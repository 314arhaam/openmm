#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------

#include "cudaKernels.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
using namespace std; 

#define SQRT sqrtf

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;
extern __global__ void kFindInteractionsWithinBlocksPeriodic_kernel(unsigned int*);

void SetCalculateAmoebaMultipoleForcesSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "SetCalculateAmoebaMultipoleForcesSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));     
    RTERROR(status, "SetCalculateAmoebaMultipoleForcesSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaMultipoleForcesSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "GetCalculateAmoebaMultipoleForcesSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));     
    RTERROR(status, "GetCalculateAmoebaMultipoleForcesSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

__device__ static float normVector3( float* vector )
{

    float norm                    = DOT3( vector, vector );
    float returnNorm              = SQRT( norm );
    norm                          = returnNorm > 0.0f ? 1.0f/returnNorm : 0.0f;

    vector[0]                    *= norm;
    vector[1]                    *= norm;
    vector[2]                    *= norm;

    return returnNorm;
}

#undef AMOEBA_DEBUG

// ZThenX     == 0
// Bisector   == 1
// ZBisect    == 2
// ThreeFold  == 3
// ZOnly      == 4
// NoAxisType == 5

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kCudaComputeCheckChiral_kernel( void )
{

    const int AD          = 0;
    const int BD          = 1;
    const int CD          = 2;
    const int C           = 3;
    float delta[4][3];
 
    float4* atomCoord            = cSim.pPosq;
    int4* multiPoleAtoms         = cAmoebaSim.pMultipoleParticlesIdsAndAxisType;
    float* molecularDipole       = cAmoebaSim.pMolecularDipole;
    float* molecularQuadrupole   = cAmoebaSim.pMolecularQuadrupole;
    float* labFrameDipole        = cAmoebaSim.pLabFrameDipole;
    float* labFrameQuadrupole    = cAmoebaSim.pLabFrameQuadrupole;
 
    // ---------------------------------------------------------------------------------------
 
    int atomIndex                = blockIdx.x;
    if( atomIndex >= cSim.atoms )return;
 
    int axisType                 = multiPoleAtoms[atomIndex].w; 
 
    float* molDipole             = &(molecularDipole[atomIndex*3]);
    float* labDipole             = &(labFrameDipole[atomIndex*3]);
    labDipole[0]                 = molDipole[0];
    labDipole[1]                 = molDipole[1];
    labDipole[2]                 = molDipole[2];
 
    float* molQuadrupole         = &(molecularQuadrupole[atomIndex*9]);
    float* labQuadrupole         = &(labFrameQuadrupole[atomIndex*9]);
    labQuadrupole[0]             = molQuadrupole[0];
    labQuadrupole[1]             = molQuadrupole[1];
    labQuadrupole[2]             = molQuadrupole[2];
    labQuadrupole[3]             = molQuadrupole[3];
    labQuadrupole[4]             = molQuadrupole[4];
    labQuadrupole[5]             = molQuadrupole[5];
    labQuadrupole[6]             = molQuadrupole[6];
    labQuadrupole[7]             = molQuadrupole[7];
    labQuadrupole[8]             = molQuadrupole[8];

    // skip z-then-x

    if( axisType == 0 || multiPoleAtoms[atomIndex].y < 0 )return;
 
    // ---------------------------------------------------------------------------------------
 
    int atomA                    = atomIndex;
    int atomB                    = multiPoleAtoms[atomIndex].z;
    int atomC                    = multiPoleAtoms[atomIndex].x;
    int atomD                    = multiPoleAtoms[atomIndex].y;

    delta[AD][0]                 = atomCoord[atomA].x - atomCoord[atomD].x;
    delta[AD][1]                 = atomCoord[atomA].y - atomCoord[atomD].y;
    delta[AD][2]                 = atomCoord[atomA].z - atomCoord[atomD].z;

    delta[BD][0]                 = atomCoord[atomB].x - atomCoord[atomD].x;
    delta[BD][1]                 = atomCoord[atomB].y - atomCoord[atomD].y;
    delta[BD][2]                 = atomCoord[atomB].z - atomCoord[atomD].z;

    delta[CD][0]                 = atomCoord[atomC].x - atomCoord[atomD].x;
    delta[CD][1]                 = atomCoord[atomC].y - atomCoord[atomD].y;
    delta[CD][2]                 = atomCoord[atomC].z - atomCoord[atomD].z;

    delta[C][0]                  = delta[BD][1]*delta[CD][2] - delta[BD][2]*delta[CD][1];
    delta[C][1]                  = delta[CD][1]*delta[AD][2] - delta[CD][2]*delta[AD][1];
    delta[C][2]                  = delta[AD][1]*delta[BD][2] - delta[AD][2]*delta[BD][1];
 
    float volume                 = delta[C][0]*delta[AD][0] + delta[C][1]*delta[BD][0] + delta[C][2]*delta[CD][0];
    if( volume < 0.0 ){
        labDipole[1]            *= -1.0f; // pole(3,i)
        labQuadrupole[1]        *= -1.0f; // pole(6,i)  && pole(8,i)
        labQuadrupole[3]        *= -1.0f; // pole(10,i) && pole(12,i)
        labQuadrupole[5]        *= -1.0f; // pole(6,i)  && pole(8,i)
        labQuadrupole[7]        *= -1.0f; // pole(10,i) && pole(12,i)
    }

}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kCudaComputeLabFrameMoments_kernel( void )
{

    float vectorX[3];
    float vectorY[3];
    float vectorZ[3];
 
    int numOfAtoms               = cSim.atoms;
    //float* rotationMatrix        = cAmoebaSim.pRotationMatrix;
    float4* atomCoord            = cSim.pPosq;
    int4* multiPoleAtoms         = cAmoebaSim.pMultipoleParticlesIdsAndAxisType;
    float* labFrameDipole        = cAmoebaSim.pLabFrameDipole;
    float* labFrameQuadrupole    = cAmoebaSim.pLabFrameQuadrupole;
 
    // ---------------------------------------------------------------------------------------
 
    int atomIndex = blockIdx.x;
 
    // ---------------------------------------------------------------------------------------
 
    // get coordinates of this atom and the z & x axis atoms
    // compute the vector between the atoms and 1/sqrt(d2), d2 is distance between
    // this atom and the axis atom
 
    // this atom is referred to as the k-atom in notes below
 
    // code common to ZThenX and Bisector
    
 /*
    vectorX                          = &(rotationMatrix[atomIndex*9]);
    vectorY                          = &(rotationMatrix[atomIndex*9+ 3]);
    vectorZ                          = &(rotationMatrix[atomIndex*9+ 6]);
 */
 
    float4 coordinatesThisAtom       = atomCoord[atomIndex];
 
    int multipoleAtomIndex           = multiPoleAtoms[atomIndex].z;
    float4 coordinatesAxisAtom       = atomCoord[multipoleAtomIndex];
 
    vectorZ[0]                       = coordinatesAxisAtom.x - coordinatesThisAtom.x;
    vectorZ[1]                       = coordinatesAxisAtom.y - coordinatesThisAtom.y;
    vectorZ[2]                       = coordinatesAxisAtom.z - coordinatesThisAtom.z;
      
    multipoleAtomIndex               = multiPoleAtoms[atomIndex].x; 
    coordinatesAxisAtom              = atomCoord[multipoleAtomIndex];
 
    vectorX[0]                       = coordinatesAxisAtom.x - coordinatesThisAtom.x;
    vectorX[1]                       = coordinatesAxisAtom.y - coordinatesThisAtom.y;
    vectorX[2]                       = coordinatesAxisAtom.z - coordinatesThisAtom.z;
 
    int axisType                     = multiPoleAtoms[atomIndex].w; 
      
    
    /*
        z-only
           (1) norm z
           (2) select random x
           (3) x = x - (x.z)z
           (4) norm x

        z-then-x
           (1) norm z
           (2) norm x (not needed)
           (3) x = x - (x.z)z
           (4) norm x

        bisector
           (1) norm z
           (2) norm x 
           (3) z = x + z
           (4) norm z
           (5) x = x - (x.z)z 
           (6) norm x 

        z-bisect
           (1) norm z
           (2) norm x 
           (3) norm y 
           (3) x = x + y
           (4) norm x
           (5) x = x - (x.z)z 
           (6) norm x 

        3-fold
           (1) norm z
           (2) norm x 
           (3) norm y 
           (4) z = x + y + z
           (5) norm z
           (6) x = x - (x.z)z 
           (7) norm x 

    */

    // branch based on axis type
     
    float sum                        = normVector3( vectorZ );

    if( axisType == 1 ){

        // bisector
        
        sum                     = normVector3( vectorX );
        
        vectorZ[0]             += vectorX[0];
        vectorZ[1]             += vectorX[1];
        vectorZ[2]             += vectorX[2];
   
        sum                     = normVector3( vectorZ );

    } else if( axisType == 2 || axisType == 3 ){ 
 
        // z-bisect

        multipoleAtomIndex      = multiPoleAtoms[atomIndex].y; 
        coordinatesAxisAtom     = atomCoord[multipoleAtomIndex];
        vectorY[0]              = coordinatesAxisAtom.x - coordinatesThisAtom.x;
        vectorY[1]              = coordinatesAxisAtom.y - coordinatesThisAtom.y;
        vectorY[2]              = coordinatesAxisAtom.z - coordinatesThisAtom.z;

        sum                     = normVector3( vectorY );
        sum                     = normVector3( vectorX );

        if( axisType == 2 ){

            vectorX[0]         += vectorY[0];
            vectorX[1]         += vectorY[1];
            vectorX[2]         += vectorY[2];
            sum                 = normVector3( vectorX );
 
        } else { 
 
            // 3-fold
    
            vectorZ[0]         += vectorX[0] + vectorY[0];
            vectorZ[1]         += vectorX[1] + vectorY[1];
            vectorZ[2]         += vectorX[2] + vectorY[2];
            sum                 = normVector3( vectorZ );
        }
 
    } else if( axisType >= 4 ){ 

        vectorX[0]             = 0.1f;
        vectorX[1]             = 0.1f;
        vectorX[2]             = 0.1f;
    }
    
    // x = x - (x.z)z

    float dot         = vectorZ[0]*vectorX[0] + vectorZ[1]*vectorX[1] + vectorZ[2]*vectorX[2];
        
    vectorX[0]       -= dot*vectorZ[0];
    vectorX[1]       -= dot*vectorZ[1];
    vectorX[2]       -= dot*vectorZ[2];
     
    sum               = normVector3( vectorX );

    vectorY[0]        = (vectorZ[1]*vectorX[2]) - (vectorZ[2]*vectorX[1]);
    vectorY[1]        = (vectorZ[2]*vectorX[0]) - (vectorZ[0]*vectorX[2]);
    vectorY[2]        = (vectorZ[0]*vectorX[1]) - (vectorZ[1]*vectorX[0]);
 
    // use identity rotation matrix for unrecognized axis types

    if( axisType < 0 || axisType > 4 ){

        vectorX[0] = 1.0f;
        vectorX[1] = 0.0f;
        vectorX[2] = 0.0f;

        vectorY[0] = 0.0f;
        vectorY[1] = 1.0f;
        vectorY[2] = 0.0f;

        vectorZ[0] = 0.0f;
        vectorZ[1] = 0.0f;
        vectorZ[2] = 1.0f;
    }

    float molDipole[3];
    float* labDipole  = &(labFrameDipole[atomIndex*3]);
    molDipole[0]      = labDipole[0];
    molDipole[1]      = labDipole[1];
    molDipole[2]      = labDipole[2];
    
    // set out-of-range elements to 0.0f
 
    labDipole[0]      = atomIndex >= numOfAtoms ? 0.0f : molDipole[0]*vectorX[0] + molDipole[1]*vectorY[0] + molDipole[2]*vectorZ[0];
    labDipole[1]      = atomIndex >= numOfAtoms ? 0.0f : molDipole[0]*vectorX[1] + molDipole[1]*vectorY[1] + molDipole[2]*vectorZ[1];
    labDipole[2]      = atomIndex >= numOfAtoms ? 0.0f : molDipole[0]*vectorX[2] + molDipole[1]*vectorY[2] + molDipole[2]*vectorZ[2];
    
    // ---------------------------------------------------------------------------------------
    
    float* rPole[3];
    float mPole[3][3];
    float* labQuadrupole       = &(labFrameQuadrupole[atomIndex*9]);
    
    for( int ii = 0; ii < 3; ii++ ){
        mPole[ii][0]   = labQuadrupole[3*ii+0];
        mPole[ii][1]   = labQuadrupole[3*ii+1];
        mPole[ii][2]   = labQuadrupole[3*ii+2];

        rPole[ii]      = labQuadrupole + ii*3;
        rPole[ii][0]   = 0.0f;
        rPole[ii][1]   = 0.0f;
        rPole[ii][2]   = 0.0f;

    }
    
    int ii = threadIdx.x;
    if( ii < 3 ){
        for( int jj = ii; jj < 3; jj++ ){
 
            rPole[ii][jj] += vectorX[ii]*vectorX[jj]*mPole[0][0];
            rPole[ii][jj] += vectorX[ii]*vectorY[jj]*mPole[0][1];
            rPole[ii][jj] += vectorX[ii]*vectorZ[jj]*mPole[0][2];
       	
            rPole[ii][jj] += vectorY[ii]*vectorX[jj]*mPole[1][0];
            rPole[ii][jj] += vectorY[ii]*vectorY[jj]*mPole[1][1];
            rPole[ii][jj] += vectorY[ii]*vectorZ[jj]*mPole[1][2];
       	
            rPole[ii][jj] += vectorZ[ii]*vectorX[jj]*mPole[2][0];
            rPole[ii][jj] += vectorZ[ii]*vectorY[jj]*mPole[2][1];
            rPole[ii][jj] += vectorZ[ii]*vectorZ[jj]*mPole[2][2];
       }
    }
 
    __syncthreads();
 
 
    rPole[1][0] = rPole[0][1];
    rPole[2][0] = rPole[0][2];
    rPole[2][1] = rPole[1][2];
 
    // set out-of-range elements to 0.0f
 
    labQuadrupole[0]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[0];
    labQuadrupole[1]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[1];
    labQuadrupole[2]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[2];
    labQuadrupole[3]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[3];
    labQuadrupole[4]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[4];
    labQuadrupole[5]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[5];
    labQuadrupole[6]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[6];
    labQuadrupole[7]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[7];
    labQuadrupole[8]   = atomIndex >= numOfAtoms ? 0.0f : labQuadrupole[8];
}

void cudaComputeAmoebaLabFrameMoments( amoebaGpuContext amoebaGpu )
{

   // ---------------------------------------------------------------------------------------

   static const char* methodName = "computeCudaAmoebaLabFrameMoments";

   // ---------------------------------------------------------------------------------------

    gpuContext gpu    = amoebaGpu->gpuContext;

    int numBlocks     =  amoebaGpu->paddedNumberOfAtoms;
    int numThreads    =  20;

//#define AMOEBA_DEBUG  
#ifdef AMOEBA_DEBUG
    if( 0 && amoebaGpu->log ){
        (void) fprintf( amoebaGpu->log, "%s: numBlocks/atoms=%d\n", methodName, numBlocks ); (void) fflush( amoebaGpu->log );
        amoebaGpu->psMultipoleParticlesIdsAndAxisType->Download();
        amoebaGpu->psMolecularDipole->Download();
        gpu->psPosq4->Download();
        for( int ii = 0; ii < gpu->natoms; ii++ ){
            int mIndex = 3*ii;
             (void) fprintf( amoebaGpu->log,"%6d [%6d %6d %6d] x[%16.9e %16.9e %16.9e] dpl[%16.9e %16.9e %16.9e]\nRot[%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e]\n\n", ii,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].x,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].y,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].w,
                             gpu->psPosq4->_pSysData[ii].x,
                             gpu->psPosq4->_pSysData[ii].y,
                             gpu->psPosq4->_pSysData[ii].z,
                             amoebaGpu->psMolecularDipole->_pSysData[mIndex],
                             amoebaGpu->psMolecularDipole->_pSysData[mIndex+1],
                             amoebaGpu->psMolecularDipole->_pSysData[mIndex+2] );
        }
    }
//    int64 kernelTime = AmoebaTiming::getTimeOfDay();
    double kernelTime = 0.0;
#endif

    kCudaComputeCheckChiral_kernel<<< numBlocks, numThreads>>> ( );
    LAUNCHERROR("kCudaComputeCheckChiral");

    kCudaComputeLabFrameMoments_kernel<<< numBlocks, numThreads>>> ( );
    LAUNCHERROR(methodName);

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){
        static int timestep = 0;
        timestep++;
        (void) fprintf( amoebaGpu->log, "Finished rotation kernel execution in %lf us\n", kernelTime ); (void) fflush( amoebaGpu->log );
        (void) fflush( amoebaGpu->log );

        amoebaGpu->psLabFrameDipole->Download();
        (void) fprintf( amoebaGpu->log, "psLabFrameDipole completed\n" );  (void) fflush( amoebaGpu->log );

        amoebaGpu->psLabFrameQuadrupole->Download();
        (void) fprintf( amoebaGpu->log, "psLabFrameQpole completed\n" );  (void) fflush( amoebaGpu->log );

        int maxPrint = 10;
        for( int ii = 0; ii < amoebaGpu->paddedNumberOfAtoms; ii++ ){

             int dipoleOffset     = 3*ii;
             int quadrupoleOffset = 9*ii;

             (void) fprintf( amoebaGpu->log,"\n%6d [%6d %6d %6d] ", ii,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].x,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].y,
                             amoebaGpu->psMultipoleParticlesIdsAndAxisType->_pSysData[ii].w );
             // coords

             (void) fprintf( amoebaGpu->log,"x[%16.9e %16.9e %16.9e]\n",
                             gpu->psPosq4->_pSysData[ii].x,
                             gpu->psPosq4->_pSysData[ii].y,
                             gpu->psPosq4->_pSysData[ii].z);
/*
             (void) fprintf( amoebaGpu->log,"   R[%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e]\n",
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+1],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+2],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+3],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+4],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+5],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+6],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+7],
                             amoebaGpu->psRotationMatrix->_pSysData[quadrupoleOffset+8] );
*/
             // dipole

             (void) fprintf( amoebaGpu->log,"   D[%16.9e %16.9e %16.9e]\n",
                             amoebaGpu->psLabFrameDipole->_pSysData[dipoleOffset],
                             amoebaGpu->psLabFrameDipole->_pSysData[dipoleOffset+1],
                             amoebaGpu->psLabFrameDipole->_pSysData[dipoleOffset+2] );
    
             // quadrupole

             (void) fprintf( amoebaGpu->log,"   Q[%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e]\n",
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+1],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+2],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+3],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+4],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+5],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+6],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+7],
                             amoebaGpu->psLabFrameQuadrupole->_pSysData[quadrupoleOffset+8] );

            if( ii == maxPrint && (ii < (gpu->natoms - maxPrint)) ){
                ii = gpu->natoms - maxPrint;
            }
        }
        int nansDetected   = checkForNansAndInfinities( amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->psLabFrameDipole );
            nansDetected  += checkForNansAndInfinities( amoebaGpu->paddedNumberOfAtoms*9, amoebaGpu->psLabFrameQuadrupole );
        if( nansDetected ){
             (void) fprintf( amoebaGpu->log,"Nans detected in dipole/quadrupoles.\n" );
             exit(0);
        }
        (void) fflush( amoebaGpu->log );
    }
#endif

    if( 0 ){
//        int particles = particles;
        int particles = amoebaGpu->paddedNumberOfAtoms;
        std::vector<int> fileId;
        //fileId.push_back( 0 );
        VectorOfDoubleVectors outputVector;
        cudaLoadCudaFloat4Array( particles, 3, gpu->psPosq4,                     outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaLoadCudaFloatArray( particles,  9, amoebaGpu->psRotationMatrix,      outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaWriteVectorOfDoubleVectorsToFile( "CudaRotationMatrices", fileId, outputVector );
    }
    if( 0 ){

        int particles = amoebaGpu->paddedNumberOfAtoms;
        std::vector<int> fileId;
        //fileId.push_back( 0 );

        VectorOfDoubleVectors outputVector;
        cudaLoadCudaFloat4Array( particles, 3, gpu->psPosq4,                     outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaLoadCudaFloatArray( particles,  3, amoebaGpu->psLabFrameDipole,      outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaLoadCudaFloatArray( particles,  9, amoebaGpu->psLabFrameQuadrupole,  outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaWriteVectorOfDoubleVectorsToFile( "CudaRotatedMoments", fileId, outputVector );
    }
  
}

void kCalculateAmoebaMultipoleForces(amoebaGpuContext amoebaGpu, bool hasAmoebaGeneralizedKirkwood ) 
{
    std::string methodName = "kCalculateAmoebaMultipoleForces";

    // compute lab frame moments

    cudaComputeAmoebaLabFrameMoments( amoebaGpu );

    if( 0 ){
        gpuContext gpu                       = amoebaGpu->gpuContext;
        std::vector<int> fileId;
        //fileId.push_back( 0 );
        VectorOfDoubleVectors outputVector;
        //cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,              outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psLabFrameDipole,     outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaLoadCudaFloatArray( gpu->natoms,  9, amoebaGpu->psLabFrameQuadrupole, outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
        cudaWriteVectorOfDoubleVectorsToFile( "CudaLabMoments", fileId, outputVector );
    }   

    // compute fixed E-field and mutual induced field 

    if( hasAmoebaGeneralizedKirkwood ){
        cudaComputeAmoebaFixedEAndGkFields( amoebaGpu );
        cudaComputeAmoebaMutualInducedAndGkField( amoebaGpu );
    } else {
        if( amoebaGpu->multipoleNonbondedMethod == AMOEBA_NO_CUTOFF ){
            cudaComputeAmoebaFixedEField( amoebaGpu );
            cudaComputeAmoebaMutualInducedField( amoebaGpu );
        } else {
            gpuContext gpu = amoebaGpu->gpuContext;
            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            //compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, amoebaGpu->psWorkUnit->_pDevData, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kFindInteractionsWithinBlocksPeriodic");
            cudaComputeAmoebaPmeFixedEField( amoebaGpu );
            cudaComputeAmoebaPmeMutualInducedField( amoebaGpu );
        }
    }

    // check if induce dipole calculation converged -- abort if it did not

    if( amoebaGpu->mutualInducedDone == 0 ){
       (void) fprintf( amoebaGpu->log, "%s induced dipole calculation did not converge -- aborting!\n", methodName.c_str() );
       (void) fflush( amoebaGpu->log );
       exit(-1);
    }

    // calculate electrostatic forces

    if( amoebaGpu->multipoleNonbondedMethod == AMOEBA_NO_CUTOFF ){

        cudaComputeAmoebaElectrostatic( amoebaGpu );

        // map torques to forces

        cudaComputeAmoebaMapTorquesAndAddTotalForce( amoebaGpu, amoebaGpu->psTorque, amoebaGpu->psForce, amoebaGpu->gpuContext->psForce4 );

        if( 0 ){
            gpuContext gpu = amoebaGpu->gpuContext;
            std::vector<int> fileId;
            //fileId.push_back( 0 );
            VectorOfDoubleVectors outputVector;
            //cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,              outputVector, gpu->psAtomIndex->_pSysData, 1.0f );
            cudaLoadCudaFloat4Array( gpu->natoms,  3, amoebaGpu->gpuContext->psForce4,     outputVector, gpu->psAtomIndex->_pSysData, 1.0f/4.184 );
            cudaWriteVectorOfDoubleVectorsToFile( "CudaMpole", fileId, outputVector );
        }   

    } else {
        cudaComputeAmoebaPmeElectrostatic( amoebaGpu );
    }
}

#undef AMOEBA_DEBUG
