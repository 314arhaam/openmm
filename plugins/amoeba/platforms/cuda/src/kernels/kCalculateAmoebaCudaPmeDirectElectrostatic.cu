#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

//#define AMOEBA_DEBUG

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaPmeDirectElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaPmeDirectElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaPmeDirectElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaPmeDirectElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaPmeDirectElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaPmeDirectElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

static int const PScaleIndex            =  0; 
static int const DScaleIndex            =  1; 
static int const UScaleIndex            =  2; 
static int const MScaleIndex            =  3;
static int const LastScalingIndex       =  4;

struct PmeDirectElectrostaticParticle {

    // coordinates charge

    float x;
    float y;
    float z;
    float q;

    // lab frame dipole

    float labFrameDipole[3];

    // lab frame quadrupole

    float labFrameQuadrupole[9];

    // induced dipole

    float inducedDipole[3];

    // polar induced dipole

    float inducedDipoleP[3];

    // scaling factors

    float thole;
    float damp;

    float force[3];

    float torque[3];
    float padding;

    float tempForce[3];
    float tempTorque[3];
};

__device__ void sumTempBuffer( PmeDirectElectrostaticParticle& atomI, PmeDirectElectrostaticParticle& atomJ ){

    atomI.tempForce[0]  += atomJ.tempForce[0];
    atomI.tempForce[1]  += atomJ.tempForce[1];
    atomI.tempForce[2]  += atomJ.tempForce[2];

    atomI.tempTorque[0] += atomJ.tempTorque[0];
    atomI.tempTorque[1] += atomJ.tempTorque[1];
    atomI.tempTorque[2] += atomJ.tempTorque[2];
}

/*
__device__ static void debugSetup( unsigned int atomI, unsigned int atomJ,
                                   float4* debugArray, float4* pullBack )
{
    unsigned int index                 = atomI + atomJ*cAmoebaSim.paddedNumberOfAtoms;
    float blockId                      = 111.0f;

    debugArray[index].x                = (float) atomI;
    debugArray[index].y                = (float) atomJ;
    debugArray[index].z                = 0.0f;
    debugArray[index].w                = blockId;

    for( int pullIndex = 0; pullIndex < 1; pullIndex++ ){
        index                             += cAmoebaSim.paddedNumberOfAtoms;
        debugArray[index].x                = pullBack[pullIndex].x;
        debugArray[index].y                = pullBack[pullIndex].y;
        debugArray[index].z                = pullBack[pullIndex].z;
        debugArray[index].w                = pullBack[pullIndex].w;
    }   
}
*/

// self-energy for PME

__device__ static void calculatePmeSelfEnergyElectrostaticPairIxn_kernel( PmeDirectElectrostaticParticle& atomI, float* energy)
{
    float term   = 2.0f*cSim.alphaEwald*cSim.alphaEwald;
    float fterm  = -(cAmoebaSim.electric/cAmoebaSim.dielec)*cSim.alphaEwald/cAmoebaSim.sqrtPi;

    float cii    = atomI.q*atomI.q;

    float dii    = atomI.labFrameDipole[0]*atomI.labFrameDipole[0] +
                   atomI.labFrameDipole[1]*atomI.labFrameDipole[1] +
                   atomI.labFrameDipole[2]*atomI.labFrameDipole[2];

    float qii    = atomI.labFrameQuadrupole[0]*atomI.labFrameQuadrupole[0] +
                   atomI.labFrameQuadrupole[4]*atomI.labFrameQuadrupole[4] +
                   atomI.labFrameQuadrupole[8]*atomI.labFrameQuadrupole[8] + 2.0f*(
                   atomI.labFrameQuadrupole[1]*atomI.labFrameQuadrupole[1] +
                   atomI.labFrameQuadrupole[2]*atomI.labFrameQuadrupole[2] +
                   atomI.labFrameQuadrupole[5]*atomI.labFrameQuadrupole[5]);

    float uii    = atomI.labFrameDipole[0]*atomI.inducedDipole[0] + atomI.labFrameDipole[1]*atomI.inducedDipole[1] + atomI.labFrameDipole[2]*atomI.inducedDipole[2];

    *energy      = (cii + term*(dii/3.0f + 2.0f*term*qii/5.0f));
    *energy     += term*uii/3.0f;
    *energy     *= fterm;
}

// self-torque for PME

__device__ static void calculatePmeSelfTorqueElectrostaticPairIxn_kernel( PmeDirectElectrostaticParticle& atomI)
{
    float term       = (4.0f/3.0f)*(cAmoebaSim.electric/cAmoebaSim.dielec)*(cSim.alphaEwald*cSim.alphaEwald*cSim.alphaEwald)/cAmoebaSim.sqrtPi;

    float uix        = 0.5f*(atomI.inducedDipole[0] + atomI.inducedDipoleP[0]);
    float uiy        = 0.5f*(atomI.inducedDipole[1] + atomI.inducedDipoleP[1]);
    float uiz        = 0.5f*(atomI.inducedDipole[2] + atomI.inducedDipoleP[2]);

    atomI.torque[0] += term*(atomI.labFrameDipole[1]*uiz - atomI.labFrameDipole[2]*uiy);
    atomI.torque[1] += term*(atomI.labFrameDipole[2]*uix - atomI.labFrameDipole[0]*uiz);
    atomI.torque[2] += term*(atomI.labFrameDipole[0]*uiy - atomI.labFrameDipole[1]*uix);
}

__device__ void calculatePmeDirectElectrostaticPairIxn_kernel( PmeDirectElectrostaticParticle& atomI,   PmeDirectElectrostaticParticle& atomJ,
                                                               float* scalingFactors, float*  outputForce, float  outputTorque[2][3], float* energy
#ifdef AMOEBA_DEBUG
                                                               ,float4* debugArray 
#endif
 ){
  

    float e,ei;
    float erl,erli;
    float di[4],qi[10];
    float dk[4],qk[10];
    float fridmp[4],findmp[4];
    float ftm2[4],ftm2i[4];
    float ftm2r[4],ftm2ri[4];
    float ttm2[4],ttm3[4];
    float ttm2i[4],ttm3i[4];
    float ttm2r[4],ttm3r[4];
    float ttm2ri[4],ttm3ri[4];
    float dixdk[4];
    float dkxui[4],dixuk[4];
    float dixukp[4],dkxuip[4];
    float uixqkr[4],ukxqir[4];
    float uixqkrp[4],ukxqirp[4];
    float qiuk[4],qkui[4];
    float qiukp[4],qkuip[4];
    float rxqiuk[4],rxqkui[4];
    float rxqiukp[4],rxqkuip[4];
    float qidk[4],qkdi[4];
    float qir[4],qkr[4];
    float qiqkr[4],qkqir[4];
    float qixqk[4],rxqir[4];
    float dixr[4],dkxr[4];
    float dixqkr[4],dkxqir[4];
    float rxqkr[4],qkrxqir[4];
    float rxqikr[4],rxqkir[4];
    float rxqidk[4],rxqkdi[4];
    float ddsc3[4],ddsc5[4];
    float ddsc7[4];
    float bn[6];
    float sc[11],gl[9];
    float sci[9],scip[9];
    float gli[8],glip[8];
    float gf[8],gfi[7];
    float gfr[8],gfri[7];
    float gti[7],gtri[7];

    float conversionFactor   = (-cAmoebaSim.electric/cAmoebaSim.dielec);

    // set the permanent multipole and induced dipole values;

    float pdi   = atomI.damp;
    float pti   = atomI.thole;
    float ci    = atomI.q;

    di[1]       = atomI.labFrameDipole[0];
    di[2]       = atomI.labFrameDipole[1];
    di[3]       = atomI.labFrameDipole[2];

    qi[1]       = atomI.labFrameQuadrupole[0];
    qi[2]       = atomI.labFrameQuadrupole[1];
    qi[3]       = atomI.labFrameQuadrupole[2];
    qi[4]       = atomI.labFrameQuadrupole[3];
    qi[5]       = atomI.labFrameQuadrupole[4];
    qi[6]       = atomI.labFrameQuadrupole[5];
    qi[7]       = atomI.labFrameQuadrupole[6];
    qi[8]       = atomI.labFrameQuadrupole[7];
    qi[9]       = atomI.labFrameQuadrupole[8];

    float xr    = atomJ.x - atomI.x;
    float yr    = atomJ.y - atomI.y;
    float zr    = atomJ.z - atomI.z;

    // periodic box 

    xr         -= floor(xr*cSim.invPeriodicBoxSizeX+0.5f)*cSim.periodicBoxSizeX;
    yr         -= floor(yr*cSim.invPeriodicBoxSizeY+0.5f)*cSim.periodicBoxSizeY;
    zr         -= floor(zr*cSim.invPeriodicBoxSizeZ+0.5f)*cSim.periodicBoxSizeZ;

    float r2    = xr*xr + yr*yr + zr*zr;
    if( r2 <= cSim.nonbondedCutoffSqr ){

        float r      = sqrt(r2);
        float ck     = atomJ.q;
      
        dk[1]  = atomJ.labFrameDipole[0];
        dk[2]  = atomJ.labFrameDipole[1];
        dk[3]  = atomJ.labFrameDipole[2];
      
        qk[1]  = atomJ.labFrameQuadrupole[0];
        qk[2]  = atomJ.labFrameQuadrupole[1];
        qk[3]  = atomJ.labFrameQuadrupole[2];
        qk[4]  = atomJ.labFrameQuadrupole[3];
        qk[5]  = atomJ.labFrameQuadrupole[4];
        qk[6]  = atomJ.labFrameQuadrupole[5];
        qk[7]  = atomJ.labFrameQuadrupole[6];
        qk[8]  = atomJ.labFrameQuadrupole[7];
        qk[9]  = atomJ.labFrameQuadrupole[8];
      
        // calculate the real space error function terms;

        float ralpha = cSim.alphaEwald*r;

               bn[0] = erfc(ralpha)/r;

        float alsq2  = 2.0f*cSim.alphaEwald*cSim.alphaEwald;
        float alsq2n = 0.0f;
        if( cSim.alphaEwald > 0.0f){
            alsq2n = 1.0f/(cAmoebaSim.sqrtPi*cSim.alphaEwald);
        }
        float exp2a  = exp(-(ralpha*ralpha));

        alsq2n      *= alsq2;
        bn[1]        = (bn[0]+alsq2n*exp2a)/r2;

        alsq2n      *= alsq2;
        bn[2]        = (3.0f*bn[1]+alsq2n*exp2a)/r2;

        alsq2n      *= alsq2;
        bn[3]        = (5.0f*bn[2]+alsq2n*exp2a)/r2;

        alsq2n      *= alsq2;
        bn[4]        = (7.0f*bn[3]+alsq2n*exp2a)/r2;

        alsq2n      *= alsq2;
        bn[5]        = (9.0f*bn[4]+alsq2n*exp2a)/r2;

        // apply Thole polarization damping to scale factors;

        float rr1    = 1.0f/r;
        float rr3    = rr1 / r2;
        float rr5    = 3.0f * rr3 / r2;
        float rr7    = 5.0f * rr5 / r2;
        float rr9    = 7.0f * rr7 / r2;
        float rr11   = 9.0f * rr9 / r2;
        float scale3 = 1.0f;
        float scale5 = 1.0f;
        float scale7 = 1.0f;

            ddsc3[1] = 0.0f;
            ddsc3[2] = 0.0f;
            ddsc3[3] = 0.0f;

            ddsc5[1] = 0.0f;
            ddsc5[2] = 0.0f;
            ddsc5[3] = 0.0f;

            ddsc7[1] = 0.0f;
            ddsc7[2] = 0.0f;
            ddsc7[3] = 0.0f;

        float pdk    = atomJ.damp;
        float ptk    = atomJ.thole;
        float damp   = pdi*pdk;
        if( damp != 0.0f ){
            float pgamma = pti < ptk ? pti : ptk;
            float ratio  = r/damp;
                damp     = -pgamma * ratio*ratio*ratio;
            if( damp > -50.0f ){
                float expdamp  = exp(damp);
                scale3         = 1.0f - expdamp;
                   scale5      = 1.0f - (1.0f-damp)*expdamp;
                   scale7      = 1.0f - (1.0f-damp+0.6f*damp*damp)*expdamp;
                float temp3    = -3.0f * damp * expdamp / r2;
                float temp5    = -damp;
                float temp7    = -0.2f - 0.6f*damp;

                ddsc3[1]       = temp3 * xr;
                ddsc3[2]       = temp3 * yr;
                ddsc3[3]       = temp3 * zr;

                ddsc5[1]       = temp5 * ddsc3[1];
                ddsc5[2]       = temp5 * ddsc3[2];
                ddsc5[3]       = temp5 * ddsc3[3];

                ddsc7[1]       = temp7 * ddsc5[1];
                ddsc7[2]       = temp7 * ddsc5[2];
                ddsc7[3]       = temp7 * ddsc5[3];
            }
        }

        float dsc3 = 1.0f - scale3*scalingFactors[DScaleIndex];
        float dsc5 = 1.0f - scale5*scalingFactors[DScaleIndex];
        float dsc7 = 1.0f - scale7*scalingFactors[DScaleIndex];

        float psc3 = 1.0f - scale3*scalingFactors[PScaleIndex];
        float psc5 = 1.0f - scale5*scalingFactors[PScaleIndex];
        float psc7 = 1.0f - scale7*scalingFactors[PScaleIndex];

        float usc3 = 1.0f - scale3*scalingFactors[UScaleIndex];
        float usc5 = 1.0f - scale5*scalingFactors[UScaleIndex];

        // construct necessary auxiliary vectors

        dixdk[1]       = di[2]*dk[3] - di[3]*dk[2];
        dixdk[2]       = di[3]*dk[1] - di[1]*dk[3];
        dixdk[3]       = di[1]*dk[2] - di[2]*dk[1];

        dixuk[1]       = di[2]*atomJ.inducedDipole[2] - di[3]*atomJ.inducedDipole[1];
        dixuk[2]       = di[3]*atomJ.inducedDipole[0] - di[1]*atomJ.inducedDipole[2];
        dixuk[3]       = di[1]*atomJ.inducedDipole[1] - di[2]*atomJ.inducedDipole[0];
        dkxui[1]       = dk[2]*atomI.inducedDipole[2] - dk[3]*atomI.inducedDipole[1];
        dkxui[2]       = dk[3]*atomI.inducedDipole[0] - dk[1]*atomI.inducedDipole[2];
        dkxui[3]       = dk[1]*atomI.inducedDipole[1] - dk[2]*atomI.inducedDipole[0];
        dixukp[1]      = di[2]*atomJ.inducedDipoleP[2] - di[3]*atomJ.inducedDipoleP[1];
        dixukp[2]      = di[3]*atomJ.inducedDipoleP[0] - di[1]*atomJ.inducedDipoleP[2];
        dixukp[3]      = di[1]*atomJ.inducedDipoleP[1] - di[2]*atomJ.inducedDipoleP[0];
        dkxuip[1]      = dk[2]*atomI.inducedDipoleP[2] - dk[3]*atomI.inducedDipoleP[1];
        dkxuip[2]      = dk[3]*atomI.inducedDipoleP[0] - dk[1]*atomI.inducedDipoleP[2];
        dkxuip[3]      = dk[1]*atomI.inducedDipoleP[1] - dk[2]*atomI.inducedDipoleP[0];
        dixr[1]        = di[2]*zr - di[3]*yr;
        dixr[2]        = di[3]*xr - di[1]*zr;
        dixr[3]        = di[1]*yr - di[2]*xr;
        dkxr[1]        = dk[2]*zr - dk[3]*yr;
        dkxr[2]        = dk[3]*xr - dk[1]*zr;
        dkxr[3]        = dk[1]*yr - dk[2]*xr;
        qir[1]         = qi[1]*xr + qi[4]*yr + qi[7]*zr;
        qir[2]         = qi[2]*xr + qi[5]*yr + qi[8]*zr;
        qir[3]         = qi[3]*xr + qi[6]*yr + qi[9]*zr;
        qkr[1]         = qk[1]*xr + qk[4]*yr + qk[7]*zr;
        qkr[2]         = qk[2]*xr + qk[5]*yr + qk[8]*zr;
        qkr[3]         = qk[3]*xr + qk[6]*yr + qk[9]*zr;
        qiqkr[1]       = qi[1]*qkr[1] + qi[4]*qkr[2] + qi[7]*qkr[3];
        qiqkr[2]       = qi[2]*qkr[1] + qi[5]*qkr[2] + qi[8]*qkr[3];
        qiqkr[3]       = qi[3]*qkr[1] + qi[6]*qkr[2] + qi[9]*qkr[3];
        qkqir[1]       = qk[1]*qir[1] + qk[4]*qir[2] + qk[7]*qir[3];
        qkqir[2]       = qk[2]*qir[1] + qk[5]*qir[2] + qk[8]*qir[3];
        qkqir[3]       = qk[3]*qir[1] + qk[6]*qir[2] + qk[9]*qir[3];
        qixqk[1]       = qi[2]*qk[3] + qi[5]*qk[6] + qi[8]*qk[9]
                       - qi[3]*qk[2] - qi[6]*qk[5] - qi[9]*qk[8];
        qixqk[2]       = qi[3]*qk[1] + qi[6]*qk[4] + qi[9]*qk[7]
                       - qi[1]*qk[3] - qi[4]*qk[6] - qi[7]*qk[9];
        qixqk[3]       = qi[1]*qk[2] + qi[4]*qk[5] + qi[7]*qk[8]
                       - qi[2]*qk[1] - qi[5]*qk[4] - qi[8]*qk[7];
        rxqir[1]       = yr*qir[3] - zr*qir[2];
        rxqir[2]       = zr*qir[1] - xr*qir[3];
        rxqir[3]       = xr*qir[2] - yr*qir[1];
        rxqkr[1]       = yr*qkr[3] - zr*qkr[2];
        rxqkr[2]       = zr*qkr[1] - xr*qkr[3];
        rxqkr[3]       = xr*qkr[2] - yr*qkr[1];
        rxqikr[1]      = yr*qiqkr[3] - zr*qiqkr[2];
        rxqikr[2]      = zr*qiqkr[1] - xr*qiqkr[3];
        rxqikr[3]      = xr*qiqkr[2] - yr*qiqkr[1];
        rxqkir[1]      = yr*qkqir[3] - zr*qkqir[2];
        rxqkir[2]      = zr*qkqir[1] - xr*qkqir[3];
        rxqkir[3]      = xr*qkqir[2] - yr*qkqir[1];
        qkrxqir[1]     = qkr[2]*qir[3] - qkr[3]*qir[2];
        qkrxqir[2]     = qkr[3]*qir[1] - qkr[1]*qir[3];
        qkrxqir[3]     = qkr[1]*qir[2] - qkr[2]*qir[1];
        qidk[1]        = qi[1]*dk[1] + qi[4]*dk[2] + qi[7]*dk[3];
        qidk[2]        = qi[2]*dk[1] + qi[5]*dk[2] + qi[8]*dk[3];
        qidk[3]        = qi[3]*dk[1] + qi[6]*dk[2] + qi[9]*dk[3];
        qkdi[1]        = qk[1]*di[1] + qk[4]*di[2] + qk[7]*di[3];
        qkdi[2]        = qk[2]*di[1] + qk[5]*di[2] + qk[8]*di[3];
        qkdi[3]        = qk[3]*di[1] + qk[6]*di[2] + qk[9]*di[3];
        qiuk[1]        = qi[1]*atomJ.inducedDipole[0] + qi[4]*atomJ.inducedDipole[1]
                       + qi[7]*atomJ.inducedDipole[2];
        qiuk[2]        = qi[2]*atomJ.inducedDipole[0] + qi[5]*atomJ.inducedDipole[1]
                       + qi[8]*atomJ.inducedDipole[2];
        qiuk[3]        = qi[3]*atomJ.inducedDipole[0] + qi[6]*atomJ.inducedDipole[1] 
                       + qi[9]*atomJ.inducedDipole[2];
        qkui[1]        = qk[1]*atomI.inducedDipole[0] + qk[4]*atomI.inducedDipole[1]
                       + qk[7]*atomI.inducedDipole[2];
        qkui[2]        = qk[2]*atomI.inducedDipole[0] + qk[5]*atomI.inducedDipole[1]
                       + qk[8]*atomI.inducedDipole[2];
        qkui[3]        = qk[3]*atomI.inducedDipole[0] + qk[6]*atomI.inducedDipole[1]
                       + qk[9]*atomI.inducedDipole[2];
        qiukp[1]       = qi[1]*atomJ.inducedDipoleP[0] + qi[4]*atomJ.inducedDipoleP[1]
                        + qi[7]*atomJ.inducedDipoleP[2];
        qiukp[2]       = qi[2]*atomJ.inducedDipoleP[0] + qi[5]*atomJ.inducedDipoleP[1]
                        + qi[8]*atomJ.inducedDipoleP[2];
        qiukp[3]       = qi[3]*atomJ.inducedDipoleP[0] + qi[6]*atomJ.inducedDipoleP[1]
                        + qi[9]*atomJ.inducedDipoleP[2];
        qkuip[1]       = qk[1]*atomI.inducedDipoleP[0] + qk[4]*atomI.inducedDipoleP[1]
                        + qk[7]*atomI.inducedDipoleP[2];
        qkuip[2]       = qk[2]*atomI.inducedDipoleP[0] + qk[5]*atomI.inducedDipoleP[1]
                        + qk[8]*atomI.inducedDipoleP[2];
        qkuip[3]       = qk[3]*atomI.inducedDipoleP[0] + qk[6]*atomI.inducedDipoleP[1]
                        + qk[9]*atomI.inducedDipoleP[2];
        dixqkr[1]      = di[2]*qkr[3] - di[3]*qkr[2];
        dixqkr[2]      = di[3]*qkr[1] - di[1]*qkr[3];
        dixqkr[3]      = di[1]*qkr[2] - di[2]*qkr[1];
        dkxqir[1]      = dk[2]*qir[3] - dk[3]*qir[2];
        dkxqir[2]      = dk[3]*qir[1] - dk[1]*qir[3];
        dkxqir[3]      = dk[1]*qir[2] - dk[2]*qir[1];
        uixqkr[1]      = atomI.inducedDipole[1]*qkr[3] - atomI.inducedDipole[2]*qkr[2];
        uixqkr[2]      = atomI.inducedDipole[2]*qkr[1] - atomI.inducedDipole[0]*qkr[3];
        uixqkr[3]      = atomI.inducedDipole[0]*qkr[2] - atomI.inducedDipole[1]*qkr[1];
        ukxqir[1]      = atomJ.inducedDipole[1]*qir[3] - atomJ.inducedDipole[2]*qir[2];
        ukxqir[2]      = atomJ.inducedDipole[2]*qir[1] - atomJ.inducedDipole[0]*qir[3];
        ukxqir[3]      = atomJ.inducedDipole[0]*qir[2] - atomJ.inducedDipole[1]*qir[1];
        uixqkrp[1]     = atomI.inducedDipoleP[1]*qkr[3] - atomI.inducedDipoleP[2]*qkr[2];
        uixqkrp[2]     = atomI.inducedDipoleP[2]*qkr[1] - atomI.inducedDipoleP[0]*qkr[3];
        uixqkrp[3]     = atomI.inducedDipoleP[0]*qkr[2] - atomI.inducedDipoleP[1]*qkr[1];
        ukxqirp[1]     = atomJ.inducedDipoleP[1]*qir[3] - atomJ.inducedDipoleP[2]*qir[2];
        ukxqirp[2]     = atomJ.inducedDipoleP[2]*qir[1] - atomJ.inducedDipoleP[0]*qir[3];
        ukxqirp[3]     = atomJ.inducedDipoleP[0]*qir[2] - atomJ.inducedDipoleP[1]*qir[1];
        rxqidk[1]      = yr*qidk[3] - zr*qidk[2];
        rxqidk[2]      = zr*qidk[1] - xr*qidk[3];
        rxqidk[3]      = xr*qidk[2] - yr*qidk[1];
        rxqkdi[1]      = yr*qkdi[3] - zr*qkdi[2];
        rxqkdi[2]      = zr*qkdi[1] - xr*qkdi[3];
        rxqkdi[3]      = xr*qkdi[2] - yr*qkdi[1];
        rxqiuk[1]      = yr*qiuk[3] - zr*qiuk[2];
        rxqiuk[2]      = zr*qiuk[1] - xr*qiuk[3];
        rxqiuk[3]      = xr*qiuk[2] - yr*qiuk[1];
        rxqkui[1]      = yr*qkui[3] - zr*qkui[2];
        rxqkui[2]      = zr*qkui[1] - xr*qkui[3];
        rxqkui[3]      = xr*qkui[2] - yr*qkui[1];
        rxqiukp[1]     = yr*qiukp[3] - zr*qiukp[2];
        rxqiukp[2]     = zr*qiukp[1] - xr*qiukp[3];
        rxqiukp[3]     = xr*qiukp[2] - yr*qiukp[1];
        rxqkuip[1]     = yr*qkuip[3] - zr*qkuip[2];
        rxqkuip[2]     = zr*qkuip[1] - xr*qkuip[3];
        rxqkuip[3]     = xr*qkuip[2] - yr*qkuip[1];

        // calculate the scalar products for permanent components

        sc[2]          = di[1]*dk[1] + di[2]*dk[2] + di[3]*dk[3];
        sc[3]          = di[1]*xr + di[2]*yr + di[3]*zr;
        sc[4]          = dk[1]*xr + dk[2]*yr + dk[3]*zr;
        sc[5]          = qir[1]*xr + qir[2]*yr + qir[3]*zr;
        sc[6]          = qkr[1]*xr + qkr[2]*yr + qkr[3]*zr;
        sc[7]          = qir[1]*dk[1] + qir[2]*dk[2] + qir[3]*dk[3];
        sc[8]          = qkr[1]*di[1] + qkr[2]*di[2] + qkr[3]*di[3];
        sc[9]          = qir[1]*qkr[1] + qir[2]*qkr[2] + qir[3]*qkr[3];
        sc[10]         = qi[1]*qk[1] + qi[2]*qk[2] + qi[3]*qk[3]
                       + qi[4]*qk[4] + qi[5]*qk[5] + qi[6]*qk[6]
                       + qi[7]*qk[7] + qi[8]*qk[8] + qi[9]*qk[9];

        // calculate the scalar products for induced components

        sci[1]          = atomI.inducedDipole[0]*dk[1] + atomI.inducedDipole[1]*dk[2]
                      + atomI.inducedDipole[2]*dk[3] + di[1]*atomJ.inducedDipole[0]
                      + di[2]*atomJ.inducedDipole[1] + di[3]*atomJ.inducedDipole[2];

        sci[2]          = atomI.inducedDipole[0]*atomJ.inducedDipole[0] + atomI.inducedDipole[1]*atomJ.inducedDipole[1]
                        + atomI.inducedDipole[2]*atomJ.inducedDipole[2];
        sci[3]          = atomI.inducedDipole[0]*xr + atomI.inducedDipole[1]*yr + atomI.inducedDipole[2]*zr;
        sci[4]          = atomJ.inducedDipole[0]*xr + atomJ.inducedDipole[1]*yr + atomJ.inducedDipole[2]*zr;
        sci[7]          = qir[1]*atomJ.inducedDipole[0] + qir[2]*atomJ.inducedDipole[1]
                        + qir[3]*atomJ.inducedDipole[2];
        sci[8]          = qkr[1]*atomI.inducedDipole[0] + qkr[2]*atomI.inducedDipole[1]
                        + qkr[3]*atomI.inducedDipole[2];
        scip[1]         = atomI.inducedDipoleP[0]*dk[1] + atomI.inducedDipoleP[1]*dk[2]
                        + atomI.inducedDipoleP[2]*dk[3] + di[1]*atomJ.inducedDipoleP[0]
                        + di[2]*atomJ.inducedDipoleP[1] + di[3]*atomJ.inducedDipoleP[2];
        scip[2]         = atomI.inducedDipole[0]*atomJ.inducedDipoleP[0]+atomI.inducedDipole[1]*atomJ.inducedDipoleP[1]
                        + atomI.inducedDipole[2]*atomJ.inducedDipoleP[2]+atomI.inducedDipoleP[0]*atomJ.inducedDipole[0]
                        + atomI.inducedDipoleP[1]*atomJ.inducedDipole[1]+atomI.inducedDipoleP[2]*atomJ.inducedDipole[2];
        scip[3]         = atomI.inducedDipoleP[0]*xr + atomI.inducedDipoleP[1]*yr + atomI.inducedDipoleP[2]*zr;
        scip[4]         = atomJ.inducedDipoleP[0]*xr + atomJ.inducedDipoleP[1]*yr + atomJ.inducedDipoleP[2]*zr;
        scip[7]         = qir[1]*atomJ.inducedDipoleP[0] + qir[2]*atomJ.inducedDipoleP[1]
                        + qir[3]*atomJ.inducedDipoleP[2];
        scip[8]         = qkr[1]*atomI.inducedDipoleP[0] + qkr[2]*atomI.inducedDipoleP[1]
                        + qkr[3]*atomI.inducedDipoleP[2];

        // calculate the gl functions for permanent components

        gl[0]           = ci*ck;
        gl[1]           = ck*sc[3] - ci*sc[4];
        gl[2]           = ci*sc[6] + ck*sc[5] - sc[3]*sc[4];
        gl[3]           = sc[3]*sc[6] - sc[4]*sc[5];
        gl[4]           = sc[5]*sc[6];
        gl[5]           = -4.0f * sc[9];
        gl[6]           = sc[2];
        gl[7]           = 2.0f * (sc[7]-sc[8]);
        gl[8]           = 2.0f * sc[10];

        // calculate the gl functions for induced components

        gli[1]          = ck*sci[3] - ci*sci[4];
        gli[2]          = -sc[3]*sci[4] - sci[3]*sc[4];
        gli[3]          = sci[3]*sc[6] - sci[4]*sc[5];
        gli[6]          = sci[1];
        gli[7]          = 2.0f * (sci[7]-sci[8]);
        glip[1]         = ck*scip[3] - ci*scip[4];
        glip[2]         = -sc[3]*scip[4] - scip[3]*sc[4];
        glip[3]         = scip[3]*sc[6] - scip[4]*sc[5];
        glip[6]         = scip[1];
        glip[7]         = 2.0f * (scip[7]-scip[8]);

        // compute the energy contributions for this interaction

        e    = bn[0]*gl[0] + bn[1]*(gl[1]+gl[6])
                 + bn[2]*(gl[2]+gl[7]+gl[8])
                 + bn[3]*(gl[3]+gl[5]) + bn[4]*gl[4];
        ei    = 0.5f * (bn[1]*(gli[1]+gli[6])
                       + bn[2]*(gli[2]+gli[7]) + bn[3]*gli[3]);

        // get the real energy without any screening function

        erl = rr1*gl[0] + rr3*(gl[1]+gl[6])
                   + rr5*(gl[2]+gl[7]+gl[8])
                   + rr7*(gl[3]+gl[5]) + rr9*gl[4];
        erli = 0.5f*(rr3*(gli[1]+gli[6])*psc3
                    + rr5*(gli[2]+gli[7])*psc5
                    + rr7*gli[3]*psc7);
        e = e - (1.0f-scalingFactors[MScaleIndex])*erl;
        ei = ei - erli;

        *energy = -conversionFactor*(e + ei);

        // increment the total intramolecular energy; assumes;
        // intramolecular distances are less than half of cell;
        // length and less than the ewald cutoff;
/*
        if (molcule(ii) .eq. molcule(kk)) {
           eintra = eintra + mscale(kk)*erl*f;
           eintra = eintra + 0.5f*pscale(kk);
&                        * (rr3*(gli[1]+gli[6])*scale3;
&                              + rr5*(gli[2]+gli[7])*scale5;
&                              + rr7*gli[3]*scale7);
        }
*/

        // intermediate variables for permanent force terms

        gf[1] = bn[1]*gl[0] + bn[2]*(gl[1]+gl[6])
                     + bn[3]*(gl[2]+gl[7]+gl[8])
                     + bn[4]*(gl[3]+gl[5]) + bn[5]*gl[4];
        gf[2] = -ck*bn[1] + sc[4]*bn[2] - sc[6]*bn[3];
        gf[3] = ci*bn[1] + sc[3]*bn[2] + sc[5]*bn[3];
        gf[4] = 2.0f * bn[2];
        gf[5] = 2.0f * (-ck*bn[2]+sc[4]*bn[3]-sc[6]*bn[4]);
        gf[6] = 2.0f * (-ci*bn[2]-sc[3]*bn[3]-sc[5]*bn[4]);
        gf[7] = 4.0f * bn[3];
        gfr[1] = rr3*gl[0] + rr5*(gl[1]+gl[6])
                      + rr7*(gl[2]+gl[7]+gl[8])
                      + rr9*(gl[3]+gl[5]) + rr11*gl[4];
        gfr[2] = -ck*rr3 + sc[4]*rr5 - sc[6]*rr7;
        gfr[3] = ci*rr3 + sc[3]*rr5 + sc[5]*rr7;
        gfr[4] = 2.0f * rr5;
        gfr[5] = 2.0f * (-ck*rr5+sc[4]*rr7-sc[6]*rr9);
        gfr[6] = 2.0f * (-ci*rr5-sc[3]*rr7-sc[5]*rr9);
        gfr[7] = 4.0f * rr7;

        // intermediate variables for induced force terms

        gfi[1] = 0.5f*bn[2]*(gli[1]+glip[1]+gli[6]+glip[6])
                      + 0.5f*bn[2]*scip[2]
                      + 0.5f*bn[3]*(gli[2]+glip[2]+gli[7]+glip[7])
                      - 0.5f*bn[3]*(sci[3]*scip[4]+scip[3]*sci[4])
                      + 0.5f*bn[4]*(gli[3]+glip[3]);
        gfi[2] = -ck*bn[1] + sc[4]*bn[2] - sc[6]*bn[3];
        gfi[3] = ci*bn[1] + sc[3]*bn[2] + sc[5]*bn[3];
        gfi[4] = 2.0f * bn[2];
        gfi[5] = bn[3] * (sci[4]+scip[4]);
        gfi[6] = -bn[3] * (sci[3]+scip[3]);
        gfri[1] = 0.5f*rr5*((gli[1]+gli[6])*psc3
                             + (glip[1]+glip[6])*dsc3
                             + scip[2]*usc3)
                  + 0.5f*rr7*((gli[7]+gli[2])*psc5
                             + (glip[7]+glip[2])*dsc5
                      - (sci[3]*scip[4]+scip[3]*sci[4])*usc5)
                  + 0.5f*rr9*(gli[3]*psc7+glip[3]*dsc7);
        gfri[2] = -rr3*ck + rr5*sc[4] - rr7*sc[6];
        gfri[3] = rr3*ci + rr5*sc[3] + rr7*sc[5];
        gfri[4] = 2.0f * rr5;
        gfri[5] = rr7 * (sci[4]*psc7+scip[4]*dsc7);
        gfri[6] = -rr7 * (sci[3]*psc7+scip[3]*dsc7);

        // get the permanent force with screening

        ftm2[1] = gf[1]*xr + gf[2]*di[1] + gf[3]*dk[1]
                       + gf[4]*(qkdi[1]-qidk[1]) + gf[5]*qir[1]
                       + gf[6]*qkr[1] + gf[7]*(qiqkr[1]+qkqir[1]);
        ftm2[2] = gf[1]*yr + gf[2]*di[2] + gf[3]*dk[2]
                       + gf[4]*(qkdi[2]-qidk[2]) + gf[5]*qir[2]
                       + gf[6]*qkr[2] + gf[7]*(qiqkr[2]+qkqir[2]);
        ftm2[3] = gf[1]*zr + gf[2]*di[3] + gf[3]*dk[3]
                       + gf[4]*(qkdi[3]-qidk[3]) + gf[5]*qir[3]
                       + gf[6]*qkr[3] + gf[7]*(qiqkr[3]+qkqir[3]);

        // get the permanent force without screening

        ftm2r[1] = gfr[1]*xr + gfr[2]*di[1] + gfr[3]*dk[1]
                       + gfr[4]*(qkdi[1]-qidk[1]) + gfr[5]*qir[1]
                       + gfr[6]*qkr[1] + gfr[7]*(qiqkr[1]+qkqir[1]);
        ftm2r[2] = gfr[1]*yr + gfr[2]*di[2] + gfr[3]*dk[2]
                       + gfr[4]*(qkdi[2]-qidk[2]) + gfr[5]*qir[2]
                       + gfr[6]*qkr[2] + gfr[7]*(qiqkr[2]+qkqir[2]);
        ftm2r[3] = gfr[1]*zr + gfr[2]*di[3] + gfr[3]*dk[3]
                       + gfr[4]*(qkdi[3]-qidk[3]) + gfr[5]*qir[3]
                       + gfr[6]*qkr[3] + gfr[7]*(qiqkr[3]+qkqir[3]);

        // get the induced force with screening

        ftm2i[1] = gfi[1]*xr + 0.5f*
              (gfi[2]*(atomI.inducedDipole[0]+atomI.inducedDipoleP[0])
             + bn[2]*(sci[4]*atomI.inducedDipoleP[0]+scip[4]*atomI.inducedDipole[0])
             + gfi[3]*(atomJ.inducedDipole[0]+atomJ.inducedDipoleP[0])
             + bn[2]*(sci[3]*atomJ.inducedDipoleP[0]+scip[3]*atomJ.inducedDipole[0])
             + (sci[4]+scip[4])*bn[2]*di[1]
             + (sci[3]+scip[3])*bn[2]*dk[1]
             + gfi[4]*(qkui[1]+qkuip[1]-qiuk[1]-qiukp[1]))
             + gfi[5]*qir[1] + gfi[6]*qkr[1];
        ftm2i[2] = gfi[1]*yr + 0.5f*
              (gfi[2]*(atomI.inducedDipole[1]+atomI.inducedDipoleP[1])
             + bn[2]*(sci[4]*atomI.inducedDipoleP[1]+scip[4]*atomI.inducedDipole[1])
             + gfi[3]*(atomJ.inducedDipole[1]+atomJ.inducedDipoleP[1])
             + bn[2]*(sci[3]*atomJ.inducedDipoleP[1]+scip[3]*atomJ.inducedDipole[1])
             + (sci[4]+scip[4])*bn[2]*di[2]
             + (sci[3]+scip[3])*bn[2]*dk[2]
             + gfi[4]*(qkui[2]+qkuip[2]-qiuk[2]-qiukp[2]))
             + gfi[5]*qir[2] + gfi[6]*qkr[2];
        ftm2i[3] = gfi[1]*zr + 0.5f*
              (gfi[2]*(atomI.inducedDipole[2]+atomI.inducedDipoleP[2])
             + bn[2]*(sci[4]*atomI.inducedDipoleP[2]+scip[4]*atomI.inducedDipole[2])
             + gfi[3]*(atomJ.inducedDipole[2]+atomJ.inducedDipoleP[2])
             + bn[2]*(sci[3]*atomJ.inducedDipoleP[2]+scip[3]*atomJ.inducedDipole[2])
             + (sci[4]+scip[4])*bn[2]*di[3]
             + (sci[3]+scip[3])*bn[2]*dk[3]
             + gfi[4]*(qkui[3]+qkuip[3]-qiuk[3]-qiukp[3]))
             + gfi[5]*qir[3] + gfi[6]*qkr[3];

        // get the induced force without screening

        ftm2ri[1] = gfri[1]*xr + 0.5f*
            (- rr3*ck*(atomI.inducedDipole[0]*psc3+atomI.inducedDipoleP[0]*dsc3)
             + rr5*sc[4]*(atomI.inducedDipole[0]*psc5+atomI.inducedDipoleP[0]*dsc5)
             - rr7*sc[6]*(atomI.inducedDipole[0]*psc7+atomI.inducedDipoleP[0]*dsc7))
             + (rr3*ci*(atomJ.inducedDipole[0]*psc3+atomJ.inducedDipoleP[0]*dsc3)
             + rr5*sc[3]*(atomJ.inducedDipole[0]*psc5+atomJ.inducedDipoleP[0]*dsc5)
             + rr7*sc[5]*(atomJ.inducedDipole[0]*psc7+atomJ.inducedDipoleP[0]*dsc7))*0.5f
             + rr5*usc5*(sci[4]*atomI.inducedDipoleP[0]+scip[4]*atomI.inducedDipole[0]
             + sci[3]*atomJ.inducedDipoleP[0]+scip[3]*atomJ.inducedDipole[0])*0.5f
             + 0.5f*(sci[4]*psc5+scip[4]*dsc5)*rr5*di[1]
             + 0.5f*(sci[3]*psc5+scip[3]*dsc5)*rr5*dk[1]
             + 0.5f*gfri[4]*((qkui[1]-qiuk[1])*psc5
             + (qkuip[1]-qiukp[1])*dsc5)
             + gfri[5]*qir[1] + gfri[6]*qkr[1];
        ftm2ri[2] = gfri[1]*yr + 0.5f*
            (- rr3*ck*(atomI.inducedDipole[1]*psc3+atomI.inducedDipoleP[1]*dsc3)
             + rr5*sc[4]*(atomI.inducedDipole[1]*psc5+atomI.inducedDipoleP[1]*dsc5)
             - rr7*sc[6]*(atomI.inducedDipole[1]*psc7+atomI.inducedDipoleP[1]*dsc7))
             + (rr3*ci*(atomJ.inducedDipole[1]*psc3+atomJ.inducedDipoleP[1]*dsc3)
             + rr5*sc[3]*(atomJ.inducedDipole[1]*psc5+atomJ.inducedDipoleP[1]*dsc5)
             + rr7*sc[5]*(atomJ.inducedDipole[1]*psc7+atomJ.inducedDipoleP[1]*dsc7))*0.5f
             + rr5*usc5*(sci[4]*atomI.inducedDipoleP[1]+scip[4]*atomI.inducedDipole[1]
             + sci[3]*atomJ.inducedDipoleP[1]+scip[3]*atomJ.inducedDipole[1])*0.5f
             + 0.5f*(sci[4]*psc5+scip[4]*dsc5)*rr5*di[2]
             + 0.5f*(sci[3]*psc5+scip[3]*dsc5)*rr5*dk[2]
             + 0.5f*gfri[4]*((qkui[2]-qiuk[2])*psc5
             + (qkuip[2]-qiukp[2])*dsc5)
             + gfri[5]*qir[2] + gfri[6]*qkr[2];
        ftm2ri[3] = gfri[1]*zr + 0.5f*
            (- rr3*ck*(atomI.inducedDipole[2]*psc3+atomI.inducedDipoleP[2]*dsc3)
             + rr5*sc[4]*(atomI.inducedDipole[2]*psc5+atomI.inducedDipoleP[2]*dsc5)
             - rr7*sc[6]*(atomI.inducedDipole[2]*psc7+atomI.inducedDipoleP[2]*dsc7))
             + (rr3*ci*(atomJ.inducedDipole[2]*psc3+atomJ.inducedDipoleP[2]*dsc3)
             + rr5*sc[3]*(atomJ.inducedDipole[2]*psc5+atomJ.inducedDipoleP[2]*dsc5)
             + rr7*sc[5]*(atomJ.inducedDipole[2]*psc7+atomJ.inducedDipoleP[2]*dsc7))*0.5f
             + rr5*usc5*(sci[4]*atomI.inducedDipoleP[2]+scip[4]*atomI.inducedDipole[2]
             + sci[3]*atomJ.inducedDipoleP[2]+scip[3]*atomJ.inducedDipole[2])*0.5f
             + 0.5f*(sci[4]*psc5+scip[4]*dsc5)*rr5*di[3]
             + 0.5f*(sci[3]*psc5+scip[3]*dsc5)*rr5*dk[3]
             + 0.5f*gfri[4]*((qkui[3]-qiuk[3])*psc5
             + (qkuip[3]-qiukp[3])*dsc5)
             + gfri[5]*qir[3] + gfri[6]*qkr[3];

        // account for partially excluded induced interactions

        float temp3 = 0.5f * rr3 * ((gli[1]+gli[6])*scalingFactors[PScaleIndex]
                                   +(glip[1]+glip[6])*scalingFactors[DScaleIndex]);
        float temp5 = 0.5f * rr5 * ((gli[2]+gli[7])*scalingFactors[PScaleIndex]
                                   +(glip[2]+glip[7])*scalingFactors[DScaleIndex]);
        float temp7 = 0.5f * rr7 * (gli[3]*scalingFactors[PScaleIndex]
                                   +glip[3]*scalingFactors[DScaleIndex]);
        fridmp[1] = temp3*ddsc3[1] + temp5*ddsc5[1] + temp7*ddsc7[1];
        fridmp[2] = temp3*ddsc3[2] + temp5*ddsc5[2] + temp7*ddsc7[2];
        fridmp[3] = temp3*ddsc3[3] + temp5*ddsc5[3] + temp7*ddsc7[3];

        // find some scaling terms for induced-induced force

        temp3 = 0.5f * rr3 * scalingFactors[UScaleIndex] * scip[2];
        temp5 = -0.5f * rr5 * scalingFactors[UScaleIndex] * (sci[3]*scip[4]+scip[3]*sci[4]);
        findmp[1] = temp3*ddsc3[1] + temp5*ddsc5[1];
        findmp[2] = temp3*ddsc3[2] + temp5*ddsc5[2];
        findmp[3] = temp3*ddsc3[3] + temp5*ddsc5[3];

        // modify the forces for partially excluded interactions

        ftm2i[1] = ftm2i[1] - fridmp[1] - findmp[1];
        ftm2i[2] = ftm2i[2] - fridmp[2] - findmp[2];
        ftm2i[3] = ftm2i[3] - fridmp[3] - findmp[3];

        // correction to convert mutual to direct polarization force

/*
        if (poltyp .eq. 'DIRECT') {
           gfd = 0.5f * (bn[2]*scip[2];
&                     - bn[3]*(scip[3]*sci[4]+sci[3]*scip[4]));
           gfdr = 0.5f * (rr5*scip[2]*usc3;
&                     - rr7*(scip[3]*sci[4];
&                           +sci[3]*scip[4])*usc5);
           ftm2i[1] = ftm2i[1] - gfd*xr - 0.5f*bn[2]*;
&                          (sci[4]*atomI.inducedDipoleP[0]+scip[4]*atomI.inducedDipole[0];
&                          +sci[3]*atomJ.inducedDipoleP[0]+scip[3]*atomJ.inducedDipole[0]);
           ftm2i[2] = ftm2i[2] - gfd*yr - 0.5f*bn[2]*;
&                          (sci[4]*atomI.inducedDipoleP[1]+scip[4]*atomI.inducedDipole[1];
&                          +sci[3]*atomJ.inducedDipoleP[1]+scip[3]*atomJ.inducedDipole[1]);
           ftm2i[3] = ftm2i[3] - gfd*zr - 0.5f*bn[2]*;
&                          (sci[4]*atomI.inducedDipoleP[2]+scip[4]*atomI.inducedDipole[2];
&                          +sci[3]*atomJ.inducedDipoleP[2]+scip[3]*atomJ.inducedDipole[2]);
           fdir[1] = gfdr*xr + 0.5f*usc5*rr5*;
&                         (sci[4]*atomI.inducedDipoleP[0]+scip[4]*atomI.inducedDipole[0];
&                        + sci[3]*atomJ.inducedDipoleP[0]+scip[3]*atomJ.inducedDipole[0]);
           fdir[2] = gfdr*yr + 0.5f*usc5*rr5*;
&                         (sci[4]*atomI.inducedDipoleP[1]+scip[4]*atomI.inducedDipole[1];
&                        + sci[3]*atomJ.inducedDipoleP[1]+scip[3]*atomJ.inducedDipole[1]);
           fdir[3] = gfdr*zr + 0.5f*usc5*rr5*;
&                         (sci[4]*atomI.inducedDipoleP[2]+scip[4]*atomI.inducedDipole[2];
&                        + sci[3]*atomJ.inducedDipoleP[2]+scip[3]*atomJ.inducedDipole[2]);
           ftm2i[1] = ftm2i[1] + fdir[1] + findmp[1];
           ftm2i[2] = ftm2i[2] + fdir[2] + findmp[2];
           ftm2i[3] = ftm2i[3] + fdir[3] + findmp[3];
        }
*/

        // intermediate variables for induced torque terms

        gti[2] = 0.5f * bn[2] * (sci[4]+scip[4]);
        gti[3] = 0.5f * bn[2] * (sci[3]+scip[3]);
        gti[4] = gfi[4];
        gti[5] = gfi[5];
        gti[6] = gfi[6];
        gtri[2] = 0.5f * rr5 * (sci[4]*psc5+scip[4]*dsc5);
        gtri[3] = 0.5f * rr5 * (sci[3]*psc5+scip[3]*dsc5);
        gtri[4] = gfri[4];
        gtri[5] = gfri[5];
        gtri[6] = gfri[6];

        // get the permanent torque with screening

        ttm2[1] = -bn[1]*dixdk[1] + gf[2]*dixr[1]
            + gf[4]*(dixqkr[1]+dkxqir[1]+rxqidk[1]-2.0f*qixqk[1])
            - gf[5]*rxqir[1] - gf[7]*(rxqikr[1]+qkrxqir[1]);
        ttm2[2] = -bn[1]*dixdk[2] + gf[2]*dixr[2]
            + gf[4]*(dixqkr[2]+dkxqir[2]+rxqidk[2]-2.0f*qixqk[2])
            - gf[5]*rxqir[2] - gf[7]*(rxqikr[2]+qkrxqir[2]);
        ttm2[3] = -bn[1]*dixdk[3] + gf[2]*dixr[3]
            + gf[4]*(dixqkr[3]+dkxqir[3]+rxqidk[3]-2.0f*qixqk[3])
            - gf[5]*rxqir[3] - gf[7]*(rxqikr[3]+qkrxqir[3]);
        ttm3[1] = bn[1]*dixdk[1] + gf[3]*dkxr[1]
            - gf[4]*(dixqkr[1]+dkxqir[1]+rxqkdi[1]-2.0f*qixqk[1])
            - gf[6]*rxqkr[1] - gf[7]*(rxqkir[1]-qkrxqir[1]);
        ttm3[2] = bn[1]*dixdk[2] + gf[3]*dkxr[2]
            - gf[4]*(dixqkr[2]+dkxqir[2]+rxqkdi[2]-2.0f*qixqk[2])
            - gf[6]*rxqkr[2] - gf[7]*(rxqkir[2]-qkrxqir[2]);
        ttm3[3] = bn[1]*dixdk[3] + gf[3]*dkxr[3]
            - gf[4]*(dixqkr[3]+dkxqir[3]+rxqkdi[3]-2.0f*qixqk[3])
            - gf[6]*rxqkr[3] - gf[7]*(rxqkir[3]-qkrxqir[3]);

        // get the permanent torque without screening

        ttm2r[1] = -rr3*dixdk[1] + gfr[2]*dixr[1]-gfr[5]*rxqir[1]
            + gfr[4]*(dixqkr[1]+dkxqir[1]+rxqidk[1]-2.0f*qixqk[1])
            - gfr[7]*(rxqikr[1]+qkrxqir[1]);
        ttm2r[2] = -rr3*dixdk[2] + gfr[2]*dixr[2]-gfr[5]*rxqir[2]
            + gfr[4]*(dixqkr[2]+dkxqir[2]+rxqidk[2]-2.0f*qixqk[2])
            - gfr[7]*(rxqikr[2]+qkrxqir[2]);
        ttm2r[3] = -rr3*dixdk[3] + gfr[2]*dixr[3]-gfr[5]*rxqir[3]
            + gfr[4]*(dixqkr[3]+dkxqir[3]+rxqidk[3]-2.0f*qixqk[3])
            - gfr[7]*(rxqikr[3]+qkrxqir[3]);
        ttm3r[1] = rr3*dixdk[1] + gfr[3]*dkxr[1] -gfr[6]*rxqkr[1]
            - gfr[4]*(dixqkr[1]+dkxqir[1]+rxqkdi[1]-2.0f*qixqk[1])
            - gfr[7]*(rxqkir[1]-qkrxqir[1]);
        ttm3r[2] = rr3*dixdk[2] + gfr[3]*dkxr[2] -gfr[6]*rxqkr[2]
            - gfr[4]*(dixqkr[2]+dkxqir[2]+rxqkdi[2]-2.0f*qixqk[2])
            - gfr[7]*(rxqkir[2]-qkrxqir[2]);
        ttm3r[3] = rr3*dixdk[3] + gfr[3]*dkxr[3] -gfr[6]*rxqkr[3]
            - gfr[4]*(dixqkr[3]+dkxqir[3]+rxqkdi[3]-2.0f*qixqk[3])
            - gfr[7]*(rxqkir[3]-qkrxqir[3]);

        // get the induced torque with screening

        ttm2i[1] = -bn[1]*(dixuk[1]+dixukp[1])*0.5f
            + gti[2]*dixr[1] + gti[4]*(ukxqir[1]+rxqiuk[1]
            + ukxqirp[1]+rxqiukp[1])*0.5f - gti[5]*rxqir[1];
        ttm2i[2] = -bn[1]*(dixuk[2]+dixukp[2])*0.5f
            + gti[2]*dixr[2] + gti[4]*(ukxqir[2]+rxqiuk[2]
            + ukxqirp[2]+rxqiukp[2])*0.5f - gti[5]*rxqir[2];
        ttm2i[3] = -bn[1]*(dixuk[3]+dixukp[3])*0.5f
            + gti[2]*dixr[3] + gti[4]*(ukxqir[3]+rxqiuk[3]
            + ukxqirp[3]+rxqiukp[3])*0.5f - gti[5]*rxqir[3];
        ttm3i[1] = -bn[1]*(dkxui[1]+dkxuip[1])*0.5f
            + gti[3]*dkxr[1] - gti[4]*(uixqkr[1]+rxqkui[1]
            + uixqkrp[1]+rxqkuip[1])*0.5f - gti[6]*rxqkr[1];
        ttm3i[2] = -bn[1]*(dkxui[2]+dkxuip[2])*0.5f
            + gti[3]*dkxr[2] - gti[4]*(uixqkr[2]+rxqkui[2]
            + uixqkrp[2]+rxqkuip[2])*0.5f - gti[6]*rxqkr[2];
        ttm3i[3] = -bn[1]*(dkxui[3]+dkxuip[3])*0.5f
            + gti[3]*dkxr[3] - gti[4]*(uixqkr[3]+rxqkui[3]
            + uixqkrp[3]+rxqkuip[3])*0.5f - gti[6]*rxqkr[3];

        // get the induced torque without screening

        ttm2ri[1] = -rr3*(dixuk[1]*psc3+dixukp[1]*dsc3)*0.5f
            + gtri[2]*dixr[1] + gtri[4]*((ukxqir[1]+rxqiuk[1])*psc5
            +(ukxqirp[1]+rxqiukp[1])*dsc5)*0.5f - gtri[5]*rxqir[1];
        ttm2ri[2] = -rr3*(dixuk[2]*psc3+dixukp[2]*dsc3)*0.5f
            + gtri[2]*dixr[2] + gtri[4]*((ukxqir[2]+rxqiuk[2])*psc5
            +(ukxqirp[2]+rxqiukp[2])*dsc5)*0.5f - gtri[5]*rxqir[2];
        ttm2ri[3] = -rr3*(dixuk[3]*psc3+dixukp[3]*dsc3)*0.5f
            + gtri[2]*dixr[3] + gtri[4]*((ukxqir[3]+rxqiuk[3])*psc5
            +(ukxqirp[3]+rxqiukp[3])*dsc5)*0.5f - gtri[5]*rxqir[3];
        ttm3ri[1] = -rr3*(dkxui[1]*psc3+dkxuip[1]*dsc3)*0.5f
            + gtri[3]*dkxr[1] - gtri[4]*((uixqkr[1]+rxqkui[1])*psc5
            +(uixqkrp[1]+rxqkuip[1])*dsc5)*0.5f - gtri[6]*rxqkr[1];
        ttm3ri[2] = -rr3*(dkxui[2]*psc3+dkxuip[2]*dsc3)*0.5f
            + gtri[3]*dkxr[2] - gtri[4]*((uixqkr[2]+rxqkui[2])*psc5
            +(uixqkrp[2]+rxqkuip[2])*dsc5)*0.5f - gtri[6]*rxqkr[2];
        ttm3ri[3] = -rr3*(dkxui[3]*psc3+dkxuip[3]*dsc3)*0.5f
            + gtri[3]*dkxr[3] - gtri[4]*((uixqkr[3]+rxqkui[3])*psc5
            +(uixqkrp[3]+rxqkuip[3])*dsc5)*0.5f - gtri[6]*rxqkr[3];

        // handle the case where scaling is used

        for( int j = 1; j <= 3; j++ ){
           ftm2[j]  = (ftm2[j]-(1.0f-scalingFactors[MScaleIndex])*ftm2r[j]);
           ftm2i[j] = (ftm2i[j]-ftm2ri[j]);
           ttm2[j]  = (ttm2[j]-(1.0f-scalingFactors[MScaleIndex])*ttm2r[j]);
           ttm2i[j] = (ttm2i[j]-ttm2ri[j]);
           ttm3[j]  = (ttm3[j]-(1.0f-scalingFactors[MScaleIndex])*ttm3r[j]);
           ttm3i[j] = (ttm3i[j]-ttm3ri[j]);
        }

        // increment gradient due to force and torque on first site;

        outputForce[0]           = conversionFactor*(ftm2[1] + ftm2i[1]);
        outputForce[1]           = conversionFactor*(ftm2[2] + ftm2i[2]);
        outputForce[2]           = conversionFactor*(ftm2[3] + ftm2i[3]);
        
        conversionFactor        *= -1.0;
        outputTorque[0][0]       =  conversionFactor*(ttm2[1] + ttm2i[1]);
        outputTorque[0][1]       =  conversionFactor*(ttm2[2] + ttm2i[2]);
        outputTorque[0][2]       =  conversionFactor*(ttm2[3] + ttm2i[3]);
    
        outputTorque[1][0]       =  conversionFactor*(ttm3[1] + ttm3i[1]);
        outputTorque[1][1]       =  conversionFactor*(ttm3[2] + ttm3i[2]);
        outputTorque[1][2]       =  conversionFactor*(ttm3[3] + ttm3i[3]);

#ifdef AMOEBA_DEBUG
    int debugIndex               = 0;
    float idTracker              = 1.0f;
/*
    debugArray[debugIndex].x = atomI.labFrameDipole[0];
    debugArray[debugIndex].y = atomI.labFrameDipole[1];
    debugArray[debugIndex].z = atomI.labFrameDipole[2];
    debugArray[debugIndex].w = r2;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = atomJ.labFrameDipole[0];
    debugArray[debugIndex].y = atomJ.labFrameDipole[1];
    debugArray[debugIndex].z = atomJ.labFrameDipole[2];
    debugArray[debugIndex].w = cSim.alphaEwald;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = atomI.inducedDipole[0];
    debugArray[debugIndex].y = atomI.inducedDipole[1];
    debugArray[debugIndex].z = atomI.inducedDipole[2];
    debugArray[debugIndex].w = idTracker;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = atomJ.inducedDipole[0];
    debugArray[debugIndex].y = atomJ.inducedDipole[1];
    debugArray[debugIndex].z = atomJ.inducedDipole[2];
    debugArray[debugIndex].w = idTracker;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = atomI.inducedDipoleP[0];
    debugArray[debugIndex].y = atomI.inducedDipoleP[1];
    debugArray[debugIndex].z = atomI.inducedDipoleP[2];
    debugArray[debugIndex].w = idTracker;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = atomJ.inducedDipoleP[0];
    debugArray[debugIndex].y = atomJ.inducedDipoleP[1];
    debugArray[debugIndex].z = atomJ.inducedDipoleP[2];
    debugArray[debugIndex].w = idTracker;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = conversionFactor*ftm2[1];
    debugArray[debugIndex].y = conversionFactor*ftm2[2];
    debugArray[debugIndex].z = conversionFactor*ftm2[3];
    debugArray[debugIndex].w = idTracker;
    debugIndex++;
*/
    idTracker               += 1.0;
    debugArray[debugIndex].x = e;
    debugArray[debugIndex].y = ei;
    debugArray[debugIndex].z = erl;
    debugArray[debugIndex].w = erli;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = r2;
    debugArray[debugIndex].y = cSim.alphaEwald;
    debugArray[debugIndex].z = conversionFactor*fridmp[3];
    debugArray[debugIndex].w = 115.0;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = conversionFactor*findmp[1];
    debugArray[debugIndex].y = conversionFactor*findmp[2];
    debugArray[debugIndex].z = conversionFactor*findmp[3];
    debugArray[debugIndex].w = cSim.alphaEwald + 1.0f;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = conversionFactor*ttm2[1];
    debugArray[debugIndex].y = conversionFactor*ttm2[2];
    debugArray[debugIndex].z = conversionFactor*ttm2[3];
    debugArray[debugIndex].w = idTracker;

    debugIndex++;
    idTracker               += 1.0;
    debugArray[debugIndex].x = conversionFactor*ttm2i[1];
    debugArray[debugIndex].y = conversionFactor*ttm2i[2];
    debugArray[debugIndex].z = conversionFactor*ttm2i[3];
    debugArray[debugIndex].w = idTracker;
#endif

    } else {

        outputForce[0]           = 0.0f;
        outputForce[1]           = 0.0f;
        outputForce[2]           = 0.0f;
        
        outputTorque[0][0]       = 0.0f;
        outputTorque[0][1]       = 0.0f;
        outputTorque[0][2]       = 0.0f;
    
        outputTorque[1][0]       = 0.0f;
        outputTorque[1][1]       = 0.0f;
        outputTorque[1][2]       = 0.0f;

        *energy                  = 0.0f;

#ifdef AMOEBA_DEBUG
for( int ii = 0; ii < 5; ii++ ){
    debugArray[ii].x = 0.0f;
    debugArray[ii].y = 0.0f;
    debugArray[ii].z = 0.0f;
    debugArray[ii].w = (float) (11*ii);
}
#endif
   
    }

    return;

}

__device__ void loadPmeDirectElectrostaticShared( struct PmeDirectElectrostaticParticle* sA, unsigned int atomI )
{
    // coordinates & charge
    sA->x                        = cSim.pPosq[atomI].x;
    sA->y                        = cSim.pPosq[atomI].y;
    sA->z                        = cSim.pPosq[atomI].z;
    sA->q                        = cSim.pPosq[atomI].w;

    // lab dipole

    sA->labFrameDipole[0]        = cAmoebaSim.pLabFrameDipole[atomI*3];
    sA->labFrameDipole[1]        = cAmoebaSim.pLabFrameDipole[atomI*3+1];
    sA->labFrameDipole[2]        = cAmoebaSim.pLabFrameDipole[atomI*3+2];


    // lab quadrupole

    sA->labFrameQuadrupole[0]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9];
    sA->labFrameQuadrupole[1]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+1];
    sA->labFrameQuadrupole[2]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+2];
    sA->labFrameQuadrupole[3]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+3];
    sA->labFrameQuadrupole[4]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+4];
    sA->labFrameQuadrupole[5]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+5];
    sA->labFrameQuadrupole[6]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+6];
    sA->labFrameQuadrupole[7]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+7];
    sA->labFrameQuadrupole[8]    = cAmoebaSim.pLabFrameQuadrupole[atomI*9+8];

    // induced dipole

    sA->inducedDipole[0]         =  cAmoebaSim.pInducedDipole[atomI*3];
    sA->inducedDipole[1]         =  cAmoebaSim.pInducedDipole[atomI*3+1];
    sA->inducedDipole[2]         =  cAmoebaSim.pInducedDipole[atomI*3+2];

    // induced dipole polar

    sA->inducedDipoleP[0]        =  cAmoebaSim.pInducedDipolePolar[atomI*3];
    sA->inducedDipoleP[1]        =  cAmoebaSim.pInducedDipolePolar[atomI*3+1];
    sA->inducedDipoleP[2]        =  cAmoebaSim.pInducedDipolePolar[atomI*3+2];

    sA->damp                     = cAmoebaSim.pDampingFactorAndThole[atomI].x;
    sA->thole                    = cAmoebaSim.pDampingFactorAndThole[atomI].y;

}

// Include versions of the kernels for N^2 calculations.

#undef USE_OUTPUT_BUFFER_PER_WARP
#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaPmeDirectElectrostatic.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaPmeDirectElectrostatic.h"

// reduce psWorkArray_3_1 -> force
// reduce psWorkArray_3_2 -> torque

static void kReduceForceTorque(amoebaGpuContext amoebaGpu )
{
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                             amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                             amoebaGpu->psWorkArray_3_1->_pDevStream[0], amoebaGpu->psForce->_pDevStream[0] );
    LAUNCHERROR("kReducePmeDirectElectrostaticForce");
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                             amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                             amoebaGpu->psWorkArray_3_2->_pDevStream[0], amoebaGpu->psTorque->_pDevStream[0] );
    LAUNCHERROR("kReducePmeDirectElectrostaticTorque");
}

//#define GET_INDUCED_DIPOLE_FROM_FILE
#ifdef GET_INDUCED_DIPOLE_FROM_FILE
#include <stdlib.h>
#endif

/**---------------------------------------------------------------------------------------

   Compute Amoeba dirrect space portion of electrostatic force & torque

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

void cudaComputeAmoebaPmeDirectElectrostatic( amoebaGpuContext amoebaGpu )
{
  
   // ---------------------------------------------------------------------------------------

    static unsigned int threadsPerBlock = 0;

#ifdef AMOEBA_DEBUG
    static const char* methodName = "cudaComputeAmoebaPmeDirectElectrostatic";
    static int timestep = 0;
    std::vector<int> fileId;
    timestep++;
    fileId.resize( 2 );
    fileId[0] = timestep;
    fileId[1] = 1;
#endif

    // ---------------------------------------------------------------------------------------

    gpuContext gpu = amoebaGpu->gpuContext;

    // apparently debug array can take up nontrivial no. registers

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){
      (void) fprintf( amoebaGpu->log, "%s %d maxCovalentDegreeSz=%d"
                      " gamma=%.3e scalingDistanceCutoff=%.3f ZZZ\n",
                      methodName, gpu->natoms,
                      amoebaGpu->maxCovalentDegreeSz, amoebaGpu->pGamma,
                      amoebaGpu->scalingDistanceCutoff );
    }   
    int paddedNumberOfAtoms                    = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
    CUDAStream<float4>* debugArray            = new CUDAStream<float4>(paddedNumberOfAtoms*paddedNumberOfAtoms, 1, "DebugArray");
    memset( debugArray->_pSysStream[0],      0, sizeof( float )*4*paddedNumberOfAtoms*paddedNumberOfAtoms);
    debugArray->Upload();
    unsigned int targetAtom                   = 10;
#endif

#ifdef GET_INDUCED_DIPOLE_FROM_FILE
    std::string fileName = "waterInducedDipole.txt";
    StringVectorVector fileContents;
    readFile( fileName, fileContents );
    unsigned int offset  = 0;
    (void) fprintf( amoebaGpu->log, "Read file: %s %u\n", fileName.c_str(), fileContents.size() ); fflush(  amoebaGpu->log );
    for( unsigned int ii = 1; ii < fileContents.size()-1; ii++ ){

        StringVector lineTokens     = fileContents[ii];
        unsigned int lineTokenIndex = 1;

        // (void) fprintf( amoebaGpu->log, "   %u %s %s\n", ii, lineTokens[0].c_str(), lineTokens[lineTokenIndex].c_str() ); fflush(  amoebaGpu->log );
        amoebaGpu->psInducedDipole->_pSysStream[0][offset++]       = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str()));
        amoebaGpu->psInducedDipole->_pSysStream[0][offset++]       = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str()));
        amoebaGpu->psInducedDipole->_pSysStream[0][offset++]       = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str())); 
        offset                                              -= 3;        
        amoebaGpu->psInducedDipolePolar->_pSysStream[0][offset++]  = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str()));
        amoebaGpu->psInducedDipolePolar->_pSysStream[0][offset++]  = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str()));
        amoebaGpu->psInducedDipolePolar->_pSysStream[0][offset++]  = static_cast<float>(atof(lineTokens[lineTokenIndex++].c_str()));
    }
    float conversion = 0.1f;
    for( int ii = 0; ii < 3*gpu->natoms; ii++ ){
        amoebaGpu->psInducedDipole->_pSysStream[0][ii]       *= conversion;
        amoebaGpu->psInducedDipolePolar->_pSysStream[0][ii]  *= conversion;
    }
    amoebaGpu->gpuContext->sim.alphaEwald = 5.4459052e+00f;
    SetCalculateAmoebaPmeDirectElectrostaticSim(amoebaGpu);
    amoebaGpu->psInducedDipole->Upload();
    amoebaGpu->psInducedDipolePolar->Upload();
#endif

    // on first pass, set threads/block

    if( threadsPerBlock == 0 ){
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 384;
        else if (gpu->sm_version >= SM_12)
            maxThreads = 128;
        else
            maxThreads = 64;
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(PmeDirectElectrostaticParticle)+sizeof(float3)), maxThreads);
    }

    kClearFields_3( amoebaGpu, 2 );

#ifdef AMOEBA_DEBUG
    (void) fprintf( amoebaGpu->log, "kCalculateAmoebaPmeDirectElectrostaticN2Forces:  threadsPerBlock=%u getThreadsPerBlock=%d sizeof=%u\n", 
                    threadsPerBlock, getThreadsPerBlock(amoebaGpu, sizeof(PmeDirectElectrostaticParticle)+sizeof(float3)),
                    (sizeof(PmeDirectElectrostaticParticle)+sizeof(float3)) );

      (void) fprintf( amoebaGpu->log, "kCalculateAmoebaPmeDirectElectrostaticN2Forces no warp:  numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%u shrd=%u Obuf=%u ixnCt=%u workUnits=%u gpu->nonbond_threads_per_block=%u\n",
                      amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                      sizeof(PmeDirectElectrostaticParticle)+sizeof(float3), (sizeof(PmeDirectElectrostaticParticle)+sizeof(float3))*threadsPerBlock, amoebaGpu->energyOutputBuffers, (*gpu->psInteractionCount)[0], gpu->sim.workUnits,
                      gpu->sim.nonbond_threads_per_block );
      (void) fflush( amoebaGpu->log );
#endif

    if (gpu->bOutputBufferPerWarp){

      kCalculateAmoebaPmeDirectElectrostaticN2ByWarpForces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, (sizeof(PmeDirectElectrostaticParticle)+sizeof(float3))*threadsPerBlock>>>(
                                                                         gpu->sim.pInteractingWorkUnit,
                                                                         amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                         amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                         debugArray->_pDevStream[0], targetAtom );
#else
                                                                         amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif

    } else {


//                                                                         gpu->sim.pInteractingWorkUnit,
//                                                                         amoebaGpu->psWorkUnit->_pDevStream[0],
      kCalculateAmoebaPmeDirectElectrostaticN2Forces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, (sizeof(PmeDirectElectrostaticParticle)+sizeof(float3))*threadsPerBlock>>>(
                                                                         gpu->sim.pInteractingWorkUnit,
                                                                         amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                         amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                         debugArray->_pDevStream[0], targetAtom );
#else
                                                                         amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif
    }
    LAUNCHERROR("kCalculateAmoebaPmeDirectElectrostaticN2Forces");
    kReduceForceTorque( amoebaGpu );

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){

        amoebaGpu->psForce->Download();
        amoebaGpu->psTorque->Download();
        debugArray->Download();
  
        (void) fprintf( amoebaGpu->log, "Finished PmeDirectElectrostatic kernel execution\n" ); (void) fflush( amoebaGpu->log );
  
        int maxPrint        = 5;
        float conversion    = 1.0f/41.84f;
        float forceSum[3]   = { 0.0f, 0.0f, 0.0f};
        for( int ii = 0; ii < gpu->natoms; ii++ ){
            (void) fprintf( amoebaGpu->log, "%5d ", ii); 
  
            int indexOffset     = ii*3;
      
            // force
  
            (void) fprintf( amoebaGpu->log,"PmeDirectElectrostaticF [%16.9e %16.9e %16.9e] ",
                            conversion*amoebaGpu->psForce->_pSysStream[0][indexOffset],
                            conversion*amoebaGpu->psForce->_pSysStream[0][indexOffset+1],
                            conversion*amoebaGpu->psForce->_pSysStream[0][indexOffset+2] );

            forceSum[0]         += amoebaGpu->psForce->_pSysStream[0][indexOffset];
            forceSum[1]         += amoebaGpu->psForce->_pSysStream[0][indexOffset+1];
            forceSum[2]         += amoebaGpu->psForce->_pSysStream[0][indexOffset+2];

            // torque
  
            (void) fprintf( amoebaGpu->log,"PmeDirectElectrostaticT [%16.9e %16.9e %16.9e] ",
                            conversion*amoebaGpu->psTorque->_pSysStream[0][indexOffset],
                            conversion*amoebaGpu->psTorque->_pSysStream[0][indexOffset+1],
                            conversion*amoebaGpu->psTorque->_pSysStream[0][indexOffset+2] );
  
            (void) fprintf( amoebaGpu->log,"\n" );
            if( ii == maxPrint && (gpu->natoms - maxPrint) > ii ){
                ii = gpu->natoms - maxPrint;
            }
        }
        (void) fflush( amoebaGpu->log );
        gpu->psEnergy->Download();
        double energy = 0.0;
        for( unsigned int ii = 0; ii < gpu->sim.energyOutputBuffers; ii++ ){
            if( (*gpu->psEnergy)[ii] !=  (*gpu->psEnergy)[ii] || (*gpu->psEnergy)[ii] == std::numeric_limits<double>::infinity() || (*gpu->psEnergy)[ii] == -std::numeric_limits<double>::infinity() ){
                (void) fprintf( amoebaGpu->log,"Energy nan at index=%d\n", ii );
            } else {
               energy += (*gpu->psEnergy)[ii];
            }
        }   
        (void) fprintf( amoebaGpu->log,"Force sums: [%16.9e %16.9e %16.9e] Energy=%16.9e\n", forceSum[0], forceSum[1], forceSum[2], energy );

        if( 0 ){
            (void) fprintf( amoebaGpu->log,"DebugElecAll\n" );
            int paddedNumberOfAtoms = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
            for( int jj = 0; jj < gpu->natoms*gpu->natoms; jj++ ){
                if( fabs( debugArray->_pSysStream[0][jj].w - 111.0 ) < 1.0e-04 ){
                    int debugIndex = jj;
                    (void) fprintf( amoebaGpu->log,"%8d [%16.9e %16.9e %16.9e %16.9e] Enr11\n", jj,
                                    debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y,
                                    debugArray->_pSysStream[0][debugIndex].z, debugArray->_pSysStream[0][debugIndex].w );
                    debugIndex += paddedNumberOfAtoms;
                    (void) fprintf( amoebaGpu->log,"%8d [%16.9e %16.9e %16.9e %16.9e] Enr12\n", jj,
                                    debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y,
                                    debugArray->_pSysStream[0][debugIndex].z, debugArray->_pSysStream[0][debugIndex].w );
                }
            }
        }
        (void) fprintf( amoebaGpu->log,"\n" );

        if( 0 ){
            (void) fprintf( amoebaGpu->log,"DebugElec\n" );
            int paddedNumberOfAtoms = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
            for( int jj = 0; jj < gpu->natoms; jj++ ){
                int debugIndex = jj;
                for( int kk = 0; kk < 6; kk++ ){
                    (void) fprintf( amoebaGpu->log,"%5d %5d [%16.9e %16.9e %16.9e %16.9e] E11\n", targetAtom, jj,
                                    debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y,
                                    debugArray->_pSysStream[0][debugIndex].z, debugArray->_pSysStream[0][debugIndex].w );
                    debugIndex += paddedNumberOfAtoms;
                }
                (void) fprintf( amoebaGpu->log,"\n" );
            }
        }
        (void) fflush( amoebaGpu->log );
  
        if( 1 ){
            std::vector<int> fileId;
            //fileId.push_back( 0 );
            VectorOfDoubleVectors outputVector;
            cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,            outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psForce,      outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psTorque,     outputVector);
            cudaWriteVectorOfDoubleVectorsToFile( "CudaPmeDirectForceTorque", fileId, outputVector );
         }

    }   
    delete debugArray;
#endif

    cudaComputeAmoebaMapTorquesAndAddTotalForce( amoebaGpu, amoebaGpu->psTorque, amoebaGpu->psForce, gpu->psForce4 );

}

/**---------------------------------------------------------------------------------------

   Compute Amoeba electrostatic force & torque using PME

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

void cudaComputeAmoebaPmeElectrostatic( amoebaGpuContext amoebaGpu )
{
    cudaComputeAmoebaPmeDirectElectrostatic( amoebaGpu );
    kCalculateAmoebaPMEInducedDipoleForces( amoebaGpu );
} 

