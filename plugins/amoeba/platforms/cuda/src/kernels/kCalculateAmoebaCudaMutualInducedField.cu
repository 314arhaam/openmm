#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

#include <stdio.h>

using namespace std;

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaCudaMutualInducedFieldSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaMutualInducedFieldSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaMutualInducedFieldSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaCudaMutualInducedFieldSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaMutualInducedFieldSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaMutualInducedFieldSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

//#define AMOEBA_DEBUG
#undef AMOEBA_DEBUG

#include "kCalculateAmoebaCudaMutualInducedParticle.h"

__device__ void calculateMutualInducedFieldPairIxn_kernel( MutualInducedParticle& atomI, MutualInducedParticle& atomJ,
                                                           float fields[4][3]

#ifdef AMOEBA_DEBUG
               , float4* debugArray
#endif

 )
{

    float deltaR[3];
    
    // ---------------------------------------------------------------------------------------
    
    // get deltaR, and r between 2 atoms
    
    deltaR[0]                                    = atomJ.x - atomI.x;
    deltaR[1]                                    = atomJ.y - atomI.y;
    deltaR[2]                                    = atomJ.z - atomI.z;

    float r                                      =  sqrtf( deltaR[0]*deltaR[0] + deltaR[1]*deltaR[1] + deltaR[2]*deltaR[2] );
    float rI                                     =  1.0f/r;
    float r2I                                    =  rI*rI;
    float rr3                                    = -rI*r2I;
    float rr5                                    = -3.0f*rr3*r2I;
    
    float dampProd                               = atomI.damp*atomJ.damp;
    float ratio                                  = (dampProd != 0.0f) ? (r/dampProd) : 1.0f;
    float pGamma                                 = atomJ.thole > atomI.thole ? atomI.thole: atomJ.thole;
    float damp                                   = ratio*ratio*ratio*pGamma;
    float dampExp                                = ( (dampProd != 0.0f) && (r < cAmoebaSim.scalingDistanceCutoff) ) ? expf( -damp ) : 0.0f; 

    rr3                                         *= (1.0f - dampExp);
    rr5                                         *= (1.0f - ( 1.0f + damp )*dampExp);
        
    float dDotDelta                              = rr5*(deltaR[0]*atomJ.inducedDipole[0]         + deltaR[1]*atomJ.inducedDipole[1]       + deltaR[2]*atomJ.inducedDipole[2] );
    fields[0][0]                                 = rr3*atomJ.inducedDipole[0] + dDotDelta*deltaR[0];
    fields[0][1]                                 = rr3*atomJ.inducedDipole[1] + dDotDelta*deltaR[1];
    fields[0][2]                                 = rr3*atomJ.inducedDipole[2] + dDotDelta*deltaR[2];
   
    dDotDelta                                    = rr5*(deltaR[0]*atomJ.inducedDipolePolar[0]    + deltaR[1]*atomJ.inducedDipolePolar[1]  + deltaR[2]*atomJ.inducedDipolePolar[2] );
    fields[1][0]                                 = rr3*atomJ.inducedDipolePolar[0] + dDotDelta*deltaR[0];
    fields[1][1]                                 = rr3*atomJ.inducedDipolePolar[1] + dDotDelta*deltaR[1];
    fields[1][2]                                 = rr3*atomJ.inducedDipolePolar[2] + dDotDelta*deltaR[2];
  
    dDotDelta                                    = rr5*(deltaR[0]*atomI.inducedDipole[0]         + deltaR[1]*atomI.inducedDipole[1]       + deltaR[2]*atomI.inducedDipole[2] );
    fields[2][0]                                 = rr3*atomI.inducedDipole[0] + dDotDelta*deltaR[0];
    fields[2][1]                                 = rr3*atomI.inducedDipole[1] + dDotDelta*deltaR[1];
    fields[2][2]                                 = rr3*atomI.inducedDipole[2] + dDotDelta*deltaR[2];
   
    dDotDelta                                    = rr5*(deltaR[0]*atomI.inducedDipolePolar[0]    + deltaR[1]*atomI.inducedDipolePolar[1]  + deltaR[2]*atomI.inducedDipolePolar[2] );
    fields[3][0]                                 = rr3*atomI.inducedDipolePolar[0] + dDotDelta*deltaR[0];
    fields[3][1]                                 = rr3*atomI.inducedDipolePolar[1] + dDotDelta*deltaR[1];
    fields[3][2]                                 = rr3*atomI.inducedDipolePolar[2] + dDotDelta*deltaR[2];
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaMutualInducedField.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaMutualInducedField.h"

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kInitializeMutualInducedField_kernel(
                   int numberOfAtoms,
                   float* fixedEField,
                   float* fixedEFieldPolar,
                   float* polarizability,
                   float* inducedDipole,
                   float* inducedDipolePolar )
{

    int threadId = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if( threadId >= 3*numberOfAtoms )return;

    fixedEField[threadId]         *= polarizability[threadId];
    inducedDipole[threadId]        = fixedEField[threadId];

    fixedEFieldPolar[threadId]    *= polarizability[threadId];
    inducedDipolePolar[threadId]   = fixedEFieldPolar[threadId];

}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kReduceMutualInducedFieldDelta_kernel(int numberOfEntries, float* arrayOfDeltas1, float* arrayOfDeltas2, float* epsilon )
{
    extern __shared__ float2 delta[];

    delta[threadIdx.x].x    = 0.0f;
    delta[threadIdx.x].y    = 0.0f;

    unsigned int pos = threadIdx.x;

    // load deltas

    while( pos < numberOfEntries )
    {   
        delta[threadIdx.x].x  += arrayOfDeltas1[pos];
        delta[threadIdx.x].y  += arrayOfDeltas2[pos];
        pos                   += blockDim.x*gridDim.x;
    }   
    __syncthreads();

    // sum the deltas

    for (int offset = 1; offset < blockDim.x; offset *= 2 )
    {   
        if (threadIdx.x + offset < blockDim.x && (threadIdx.x & (2*offset-1)) == 0)
        {
            delta[threadIdx.x].x   += delta[threadIdx.x+offset].x;
            delta[threadIdx.x].y   += delta[threadIdx.x+offset].y;
        }
        __syncthreads();
    }   

    // set epsilons

    if (threadIdx.x == 0)
    {   
        epsilon[0]  = delta[0].x > delta[0].y ? delta[0].x : delta[0].y;
        epsilon[0]  = 48.033324f*sqrtf( epsilon[0]/( (float) (numberOfEntries/3)) );
    }   
}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kSorUpdateMutualInducedField_kernel(
                   int numberOfEntries,    float* polarizability,
                   float* inducedDipole, float* inducedDipoleP,
                   float* fixedEField,   float* fixedEFieldP,
                   float* matrixProduct, float* matrixProductP )
{

    float polarSOR = 0.70f;
    int threadId   = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if( threadId  >= 3*numberOfEntries )return;

    float previousDipole                = inducedDipole[threadId];
    float previousDipoleP               = inducedDipoleP[threadId];

    inducedDipole[threadId]             = fixedEField[threadId]     + polarizability[threadId]*matrixProduct[threadId];
    inducedDipoleP[threadId]            = fixedEFieldP[threadId]    + polarizability[threadId]*matrixProductP[threadId];

    inducedDipole[threadId]             = previousDipole   + polarSOR*( inducedDipole[threadId]   - previousDipole  );   
    inducedDipoleP[threadId]            = previousDipoleP  + polarSOR*( inducedDipoleP[threadId]  - previousDipoleP );

    matrixProduct[threadId]             = ( inducedDipole[threadId]  - previousDipole  )*( inducedDipole[threadId]  - previousDipole  );
    matrixProductP[threadId]            = ( inducedDipoleP[threadId] - previousDipoleP )*( inducedDipoleP[threadId] - previousDipoleP );

}

// reduce psWorkArray_3_1
// reduce psWorkArray_3_2

static void kReduceMutualInducedFields(amoebaGpuContext amoebaGpu, CUDAStream<float>* outputArray, CUDAStream<float>* outputPolarArray )
{
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_1->_pDevData, outputArray->_pDevData );
    LAUNCHERROR("kReduceMI_Fields1");

    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_2->_pDevData, outputPolarArray->_pDevData );
    LAUNCHERROR("kReduceMI_Fields2");
}

/**---------------------------------------------------------------------------------------

   Compute mutual induce field

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

static void cudaComputeAmoebaMutualInducedFieldMatrixMultiply( amoebaGpuContext amoebaGpu,
                                                               CUDAStream<float>* outputArray, CUDAStream<float>* outputPolarArray )
{
  
   // ---------------------------------------------------------------------------------------

    static unsigned int threadsPerBlock = 0;

   // ---------------------------------------------------------------------------------------
  
    gpuContext gpu    = amoebaGpu->gpuContext;

#ifdef AMOEBA_DEBUG
    int targetAtom    = 1231;
    static const char* methodName = "cudaComputeAmoebaMutualInducedFieldMatrixMultiply";
    static int iteration = 1;
    if( 1 && amoebaGpu->log ){
        (void) fprintf( amoebaGpu->log, "%s\n", methodName );
        (void) fflush( amoebaGpu->log );
    }
    int paddedNumberOfAtoms                    = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
    CUDAStream<float4>* debugArray             = new CUDAStream<float4>(paddedNumberOfAtoms*paddedNumberOfAtoms, 1, "DebugArray");
    memset( debugArray->_pSysData,      0, sizeof( float )*4*paddedNumberOfAtoms*paddedNumberOfAtoms);
    debugArray->Upload();
#endif

    kClearFields_3( amoebaGpu, 2 );

    if( threadsPerBlock == 0 ){  
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 512; 
        else if (gpu->sm_version >= SM_12)
            maxThreads = 128; 
        else 
            maxThreads = 64; 
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(MutualInducedParticle)), maxThreads);
    }   

#ifdef AMOEBA_DEBUG
        (void) fprintf( amoebaGpu->log, "%s numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%u shrd=%u ixnCt=%u workUnits=%u\n", methodName,
                        amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                        sizeof(MutualInducedParticle), sizeof(MutualInducedParticle)*threadsPerBlock,
                        (*gpu->psInteractionCount)[0], gpu->sim.workUnits );
        (void) fflush( amoebaGpu->log );
#endif

    if (gpu->bOutputBufferPerWarp){
        kCalculateAmoebaMutualInducedFieldN2ByWarp_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(MutualInducedParticle)*threadsPerBlock>>>(
                                                                 amoebaGpu->psWorkUnit->_pDevData,
                                                                 amoebaGpu->psWorkArray_3_1->_pDevData,
#ifdef AMOEBA_DEBUG
                                                                 amoebaGpu->psWorkArray_3_2->_pDevData,
                                                                 debugArray->_pDevData, targetAtom );
#else
                                                                 amoebaGpu->psWorkArray_3_2->_pDevData );
#endif

    } else {

        kCalculateAmoebaMutualInducedFieldN2_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(MutualInducedParticle)*threadsPerBlock>>>(
                                                                 amoebaGpu->psWorkUnit->_pDevData,
                                                                 amoebaGpu->psWorkArray_3_1->_pDevData,
#ifdef AMOEBA_DEBUG
                                                                 amoebaGpu->psWorkArray_3_2->_pDevData,
                                                                 debugArray->_pDevData, targetAtom );
#else
                                                                 amoebaGpu->psWorkArray_3_2->_pDevData );
#endif


    }
    LAUNCHERROR("kCalculateAmoebaMutualInducedField");

    kReduceMutualInducedFields( amoebaGpu, outputArray, outputPolarArray );

#ifdef AMOEBA_DEBUG
    amoebaGpu->psWorkArray_3_1->Download();
    amoebaGpu->psWorkArray_3_2->Download();

    if( amoebaGpu->log && iteration == -1 ){
        (void) fprintf( amoebaGpu->log, "Finished MI kernel execution %d\n", iteration ); (void) fflush( amoebaGpu->log );
        outputArray->Download();
        outputPolarArray->Download();
        debugArray->Download();

        int maxPrint        = 1400;
        for( int ii = 0; ii < gpu->natoms; ii++ ){
           (void) fprintf( amoebaGpu->log, "%5d ", ii); 

            int indexOffset     = ii*3;
    
           // MI

           (void) fprintf( amoebaGpu->log,"Mult[%16.9e %16.9e %16.9e] ",
                           outputArray->_pSysData[indexOffset],
                           outputArray->_pSysData[indexOffset+1],
                           outputArray->_pSysData[indexOffset+2] );
    
           // MI polar

           (void) fprintf( amoebaGpu->log,"MultP[%16.9e %16.9e %16.9e] ",
                           outputPolarArray->_pSysData[indexOffset],
                           outputPolarArray->_pSysData[indexOffset+1],
                           outputPolarArray->_pSysData[indexOffset+2] );

           // coords

#if 0
            (void) fprintf( amoebaGpu->log,"x[%16.9e %16.9e %16.9e] ",
                            gpu->psPosq4->_pSysData[ii].x,
                            gpu->psPosq4->_pSysData[ii].y,
                            gpu->psPosq4->_pSysData[ii].z);


           for( int jj = 0; jj < gpu->natoms && jj < 5; jj++ ){
               int debugIndex = jj*gpu->natoms + ii;
               float xx       =  gpu->psPosq4->_pSysData[jj].x -  gpu->psPosq4->_pSysData[ii].x;
               float yy       =  gpu->psPosq4->_pSysData[jj].y -  gpu->psPosq4->_pSysData[ii].y;
               float zz       =  gpu->psPosq4->_pSysData[jj].z -  gpu->psPosq4->_pSysData[ii].z;
               (void) fprintf( amoebaGpu->log,"\n%4d %4d delta [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] ",
                               ii, jj, xx, yy, zz,
                               debugArray->_pSysData[debugIndex].x, debugArray->_pSysData[debugIndex].y, debugArray->_pSysData[debugIndex].z );

           }
#endif
           if( ii == targetAtom ){
               float sums[4][3] = { { 0.0f, 0.0f, 0.0f },
                                    { 0.0f, 0.0f, 0.0f },
                                    { 0.0f, 0.0f, 0.0f },
                                    { 0.0f, 0.0f, 0.0f } };
               (void) fprintf( amoebaGpu->log,"\n" );
               int paddedNumberOfAtoms                    = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
               unsigned int count                         = 0;
               for( int jj = 0; jj < gpu->natoms; jj++ ){
                   int debugIndex = jj;
                   (void) fprintf( amoebaGpu->log,"%4d %4d Pint [%16.9e %16.9e %16.9e %16.9e] ",
                                   ii, jj,
                                   debugArray->_pSysData[debugIndex].x, debugArray->_pSysData[debugIndex].y,
                                   debugArray->_pSysData[debugIndex].z, debugArray->_pSysData[debugIndex].w );

                   //debugIndex += gpu->natoms;
                   debugIndex += paddedNumberOfAtoms;
                   (void) fprintf( amoebaGpu->log,"[%16.9e %16.9e %16.9e] ",
                                   debugArray->_pSysData[debugIndex].x, debugArray->_pSysData[debugIndex].y, debugArray->_pSysData[debugIndex].z );

                   int index = 0;
                   sums[index][0] += debugArray->_pSysData[debugIndex].x; 
                   sums[index][1] += debugArray->_pSysData[debugIndex].y; 
                   sums[index][2] += debugArray->_pSysData[debugIndex].z; 
                   
                   if( count && ( (count % 31) == 0) ){
                      static float saveSum[3] = { 0.0f, 0.0f, 0.0f };
                      (void) fprintf( amoebaGpu->log,"Block sum [%16.9e %16.9e %16.9e] ",
                                      sums[index][0] - saveSum[0], sums[index][1] - saveSum[1], sums[index][2] - saveSum[2] );
                      saveSum[0] = sums[index][0];
                      saveSum[1] = sums[index][1];
                      saveSum[2] = sums[index][2];
                     
                   }
                   

                   debugIndex += paddedNumberOfAtoms;
                   (void) fprintf( amoebaGpu->log,"[%16.9e %16.9e %16.9e] ",
                                   debugArray->_pSysData[debugIndex].x, debugArray->_pSysData[debugIndex].y, debugArray->_pSysData[debugIndex].z );

                   index++;
                   sums[index][0] += debugArray->_pSysData[debugIndex].x; 
                   sums[index][1] += debugArray->_pSysData[debugIndex].y; 
                   sums[index][2] += debugArray->_pSysData[debugIndex].z; 

                   if( count && ( (count % 31) == 0) ){
                      static float saveSum[3] = { 0.0f, 0.0f, 0.0f };
                      (void) fprintf( amoebaGpu->log,"Block sumP [%16.9e %16.9e %16.9e] ",
                                      sums[index][0] - saveSum[0], sums[index][1] - saveSum[1], sums[index][2] - saveSum[2] );
                      saveSum[0] = sums[index][0];
                      saveSum[1] = sums[index][1];
                      saveSum[2] = sums[index][2];
                   }
                   (void) fprintf( amoebaGpu->log,"\n" );
                   count++;
               }

               (void) fprintf( amoebaGpu->log,"\n" );
               int index = 0;
               (void) fprintf( amoebaGpu->log,"Sum1 [%16.9e %16.9e %16.9e]\n", sums[index][0], sums[index][1],sums[index][2] ); index++;
               (void) fprintf( amoebaGpu->log,"Sum2 [%16.9e %16.9e %16.9e]\n", sums[index][0], sums[index][1],sums[index][2] ); index++;
               (void) fprintf( amoebaGpu->log,"Sum3 [%16.9e %16.9e %16.9e]\n", sums[index][0], sums[index][1],sums[index][2] ); index++;
               (void) fprintf( amoebaGpu->log,"Sum4 [%16.9e %16.9e %16.9e]\n", sums[index][0], sums[index][1],sums[index][2] ); index++;
           }
           (void) fprintf( amoebaGpu->log,"\n" );
           if( ii == maxPrint && (gpu->natoms - maxPrint) > ii ){
                ii = gpu->natoms - maxPrint;
           }
        }
        (void) fflush( amoebaGpu->log );
        iteration++;

     }
     delete debugArray;
#endif

}

/**---------------------------------------------------------------------------------------

   Compute mutual induce field

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

static void cudaComputeAmoebaMutualInducedFieldBySOR( amoebaGpuContext amoebaGpu )
{
  
   // ---------------------------------------------------------------------------------------

#ifdef AMOEBA_DEBUG
    static const char* methodName = "cudaComputeAmoebaMutualInducedFieldBySOR";
    static int timestep = 0;
    std::vector<int> fileId;
    timestep++;
    fileId.resize( 2 );
    fileId[0] = timestep;
    fileId[1] = 1;
#endif

   // ---------------------------------------------------------------------------------------

    int done;
    int iteration;

     gpuContext gpu    = amoebaGpu->gpuContext;
    int numOfElems     = gpu->natoms*3;
    int numThreads     = min( THREADS_PER_BLOCK, numOfElems );
    int numBlocks      = numOfElems/numThreads;

    if( (numOfElems % numThreads) != 0 )numBlocks++;

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){
        (void) fprintf( amoebaGpu->log, "%s %d numOfElems=%d numThreads=%d numBlocks=%d "
                        "maxIterations=%d targetEpsilon=%.3e\n", 
                        methodName, gpu->natoms, numOfElems, numThreads, numBlocks,
                        amoebaGpu->mutualInducedMaxIterations, amoebaGpu->mutualInducedTargetEpsilon);
        (void) fflush( amoebaGpu->log );
    }   
#endif

   // ---------------------------------------------------------------------------------------

    // set  E_Field & E_FieldPolar] to [ E_Field & E_FieldPolar]*Polarizability
    // initialize [ InducedDipole & InducedDipolePolar ] to [ E_Field & E_FieldPolar]*Polarizability

    kInitializeMutualInducedField_kernel<<< numBlocks, numThreads >>>(
         gpu->natoms,
         amoebaGpu->psE_Field->_pDevData,
         amoebaGpu->psE_FieldPolar->_pDevData,
         amoebaGpu->psPolarizability->_pDevData,
         amoebaGpu->psInducedDipole->_pDevData,
         amoebaGpu->psInducedDipolePolar->_pDevData );
    LAUNCHERROR("AmoebaMutualInducedFieldSetup");  

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){

        amoebaGpu->psE_Field->Download();
        amoebaGpu->psE_FieldPolar->Download();
        amoebaGpu->psInducedDipole->Download(),
        amoebaGpu->psInducedDipolePolar->Download();
        amoebaGpu->psPolarizability->Download();
        (void) fprintf( amoebaGpu->log, "%s Initial setup for matrix multiply\n", methodName );
        int offset   = 0;
        int maxPrint = 20000;
        for( int ii = 0; ii < gpu->natoms; ii++ ){
            (void) fprintf( amoebaGpu->log, "%4d pol=%12.4e ", ii, 
                            amoebaGpu->psPolarizability->_pSysData[offset] );
            if( amoebaGpu->psPolarizability->_pSysData[offset] != amoebaGpu->psPolarizability->_pSysData[offset+1] ||
                amoebaGpu->psPolarizability->_pSysData[offset] != amoebaGpu->psPolarizability->_pSysData[offset+2] ){
                (void) fprintf( amoebaGpu->log, "PolX!!! %12.4e %12.4e ", amoebaGpu->psPolarizability->_pSysData[offset+1], amoebaGpu->psPolarizability->_pSysData[offset+2] ); 
            }

            (void) fprintf( amoebaGpu->log," E[%14.6e %14.6e %14.6e] Mi[%14.6e %14.6e %14.6e] ",
                            amoebaGpu->psE_Field->_pSysData[offset],       amoebaGpu->psE_Field->_pSysData[offset+1],       amoebaGpu->psE_Field->_pSysData[offset+2],
                            amoebaGpu->psInducedDipole->_pSysData[offset], amoebaGpu->psInducedDipole->_pSysData[offset+1], amoebaGpu->psInducedDipole->_pSysData[offset+2] );
            (void) fprintf( amoebaGpu->log,"Ep[%14.6e %14.6e %14.6e] Mip[%14.6e %14.6e %14.6e]\n",
                            amoebaGpu->psE_FieldPolar->_pSysData[offset],       amoebaGpu->psE_FieldPolar->_pSysData[offset+1],       amoebaGpu->psE_FieldPolar->_pSysData[offset+2],
                            amoebaGpu->psInducedDipolePolar->_pSysData[offset], amoebaGpu->psInducedDipolePolar->_pSysData[offset+1], amoebaGpu->psInducedDipolePolar->_pSysData[offset+2] );
            offset += 3;
            if( ii == maxPrint && (ii < (gpu->natoms - maxPrint) ) )ii =  (gpu->natoms - maxPrint);
        }   
        (void) fflush( amoebaGpu->log );
    }   
#endif

    // ---------------------------------------------------------------------------------------
 
    done      = 0;
    iteration = 1;

    while( !done ){

        // matrix multiply

        cudaComputeAmoebaMutualInducedFieldMatrixMultiply( amoebaGpu, amoebaGpu->psWorkVector[0],  amoebaGpu->psWorkVector[1] );
        LAUNCHERROR("cudaComputeAmoebaMutualInducedFieldMatrixMultiply Loop\n");  

        // post matrix multiply

        kSorUpdateMutualInducedField_kernel<<< numBlocks, numThreads >>>(
           gpu->natoms, amoebaGpu->psPolarizability->_pDevData,
           amoebaGpu->psInducedDipole->_pDevData, amoebaGpu->psInducedDipolePolar->_pDevData,
           amoebaGpu->psE_Field->_pDevData,       amoebaGpu->psE_FieldPolar->_pDevData,
           amoebaGpu->psWorkVector[0]->_pDevData,     amoebaGpu->psWorkVector[1]->_pDevData );
        LAUNCHERROR("kSorUpdateMutualInducedField");  

        // get total epsilon -- performing sums on gpu

        kReduceMutualInducedFieldDelta_kernel<<<1, amoebaGpu->epsilonThreadsPerBlock, 2*sizeof(float)*amoebaGpu->epsilonThreadsPerBlock>>>(
           3*gpu->natoms, amoebaGpu->psWorkVector[0]->_pDevData, amoebaGpu->psWorkVector[1]->_pDevData,
           amoebaGpu->psCurrentEpsilon->_pDevData );
        LAUNCHERROR("kReduceMutualInducedFieldDelta");

        if( 0 && amoebaGpu->log ){ // trackMutualInducedIterations
            trackMutualInducedIterations( amoebaGpu, iteration);
        }

        // Debye=48.033324f
        amoebaGpu->psCurrentEpsilon->Download();
        float currentEpsilon          = amoebaGpu->psCurrentEpsilon->_pSysData[0];
        amoebaGpu->mutualInducedCurrentEpsilon   = currentEpsilon;

        if( iteration > amoebaGpu->mutualInducedMaxIterations || amoebaGpu->mutualInducedCurrentEpsilon < amoebaGpu->mutualInducedTargetEpsilon ){ 
            done = 1;
        }

#ifdef AMOEBA_DEBUG
        if( amoebaGpu->log ){
           amoebaGpu->psInducedDipole->Download();
           amoebaGpu->psInducedDipolePolar->Download();
           (void) fprintf( amoebaGpu->log, "%s iteration=%3d eps %14.6e done=%d\n",
                           methodName, iteration, amoebaGpu->mutualInducedCurrentEpsilon, done );
           (void) fflush( amoebaGpu->log );

            int offset   = 0;
            int maxPrint = 20;
            for( int ii = 0; ii < gpu->natoms; ii++ ){
                (void) fprintf( amoebaGpu->log, "%4d ", ii ); 
    
                (void) fprintf( amoebaGpu->log," Mi[%14.6e %14.6e %14.6e] ",
                                amoebaGpu->psInducedDipole->_pSysData[offset], amoebaGpu->psInducedDipole->_pSysData[offset+1], amoebaGpu->psInducedDipole->_pSysData[offset+2] );
                (void) fprintf( amoebaGpu->log,"Mip[%14.6e %14.6e %14.6e]\n",
                                amoebaGpu->psInducedDipolePolar->_pSysData[offset], amoebaGpu->psInducedDipolePolar->_pSysData[offset+1], amoebaGpu->psInducedDipolePolar->_pSysData[offset+2] );
                if( ii == maxPrint && (ii < (gpu->natoms - maxPrint) ) ){
                    ii =  (gpu->natoms - maxPrint);
                    offset = 3*(ii+1);
                } else {
                    offset += 3;
                }
            }   
            (void) fflush( amoebaGpu->log );
        }
#endif
        iteration++;
    }

    amoebaGpu->mutualInducedDone             = done;
    amoebaGpu->mutualInducedConverged        = ( !done || iteration > amoebaGpu->mutualInducedMaxIterations ) ? 0 : 1;

#ifdef AMOEBA_DEBUG
    if( 0 ){
        std::vector<int> fileId;
        //fileId.push_back( 0 );
        VectorOfDoubleVectors outputVector;
//        cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,                    outputVector, NULL, 1.0f );
        cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psInducedDipole,      outputVector, NULL, 1.0f );
        cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psInducedDipolePolar, outputVector, NULL, 1.0f );
        cudaWriteVectorOfDoubleVectorsToFile( "CudaMI", fileId, outputVector );
     }

#endif

   // ---------------------------------------------------------------------------------------
}

void cudaComputeAmoebaMutualInducedField( amoebaGpuContext amoebaGpu )
{
    if( amoebaGpu->mutualInducedIterativeMethod == 0 ){
        cudaComputeAmoebaMutualInducedFieldBySOR( amoebaGpu );
    }
}
