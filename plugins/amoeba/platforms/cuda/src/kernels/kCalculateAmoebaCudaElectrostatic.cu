#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

//#define AMOEBA_DEBUG

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

static int const PScaleIndex            =  0; 
static int const DScaleIndex            =  1; 
static int const UScaleIndex            =  2; 
static int const MScaleIndex            =  3;
static int const Scale3Index            =  4;
static int const Scale5Index            =  5;
static int const Scale7Index            =  6;
static int const Scale9Index            =  7;
static int const Ddsc30Index            =  8;
//static int const Ddsc31Index            =  9;
//static int const Ddsc32Index            = 10; 
static int const Ddsc50Index            = 11;
//static int const Ddsc51Index            = 12;
//static int const Ddsc52Index            = 13; 
static int const Ddsc70Index            = 14;
//static int const Ddsc71Index            = 15;
//static int const Ddsc72Index            = 16;
static int const LastScalingIndex       = 17;

#define DOT3_4(u,v) ((u[0])*(v[0]) + (u[1])*(v[1]) + (u[2])*(v[2]))

#define MATRIXDOT31(u,v) u[0]*v[0] + u[1]*v[1] + u[2]*v[2] + \
  u[3]*v[3] + u[4]*v[4] + u[5]*v[5] + \
  u[6]*v[6] + u[7]*v[7] + u[8]*v[8]

#define DOT31(u,v) ((u[0])*(v[0]) + (u[1])*(v[1]) + (u[2])*(v[2]))

#define i35 0.257142857f
#define one 1.0f

__device__ void acrossProductVector3(   float* vectorX, float* vectorY, float* vectorZ ){
    vectorZ[0]  = vectorX[1]*vectorY[2] - vectorX[2]*vectorY[1];
    vectorZ[1]  = vectorX[2]*vectorY[0] - vectorX[0]*vectorY[2];
    vectorZ[2]  = vectorX[0]*vectorY[1] - vectorX[1]*vectorY[0];
}

__device__ void amatrixProductVector3(   float* matrixX, float* vectorY, float* vectorZ ){
    vectorZ[0]  = matrixX[0]*vectorY[0] + matrixX[3]*vectorY[1] + matrixX[6]*vectorY[2];
    vectorZ[1]  = matrixX[1]*vectorY[0] + matrixX[4]*vectorY[1] + matrixX[7]*vectorY[2];
    vectorZ[2]  = matrixX[2]*vectorY[0] + matrixX[5]*vectorY[1] + matrixX[8]*vectorY[2];
}

__device__ void amatrixCrossProductMatrix3( float* matrixX, float* matrixY, float* vectorZ ){
  
    float* xPtr[3];
    float* yPtr[3];
        
    xPtr[0]    = matrixX;
    xPtr[1]    = matrixX + 3;
    xPtr[2]    = matrixX + 6;
    
    yPtr[0]    = matrixY;
    yPtr[1]    = matrixY + 3;
    yPtr[2]    = matrixY + 6;
          
    vectorZ[0] = DOT31( xPtr[1], yPtr[2] ) - DOT31( xPtr[2], yPtr[1] );
    vectorZ[1] = DOT31( xPtr[2], yPtr[0] ) - DOT31( xPtr[0], yPtr[2] );
    vectorZ[2] = DOT31( xPtr[0], yPtr[1] ) - DOT31( xPtr[1], yPtr[0] );
  
}

struct ElectrostaticParticle {

    // coordinates charge

    float x;
    float y;
    float z;
    float q;

    // lab frame dipole

    float labFrameDipole[3];

    // lab frame quadrupole

    float labFrameQuadrupole[9];

    // induced dipole

    float inducedDipole[3];

    // polar induced dipole

    float inducedDipoleP[3];

    // scaling factors

    float thole;
    float damp;

    float force[3];

    float torque[3];
    float padding;

};

__device__ void calculateElectrostaticPairIxn_kernel( ElectrostaticParticle& atomI,   ElectrostaticParticle& atomJ,
                                                      float scalingDistanceCutoff,   float* scalingFactors,
                                                      float*  outputForce,           float  outputTorque[2][3],
                                                      float* energy
#ifdef AMOEBA_DEBUG
                                                      ,float4* debugArray 
#endif
 ){
  
    float deltaR[3];
    
    // ---------------------------------------------------------------------------------------
    
    // ---------------------------------------------------------------------------------------

    float* ddsc3                    =  scalingFactors + Ddsc30Index;
    float* ddsc5                    =  scalingFactors + Ddsc50Index;
    float* ddsc7                    =  scalingFactors + Ddsc70Index;

    deltaR[0]                       = atomJ.x - atomI.x;
    deltaR[1]                       = atomJ.y - atomI.y;
    deltaR[2]                       = atomJ.z - atomI.z;

    float r2                        = DOT31( deltaR, deltaR );
    float r                         = sqrtf( r2 );
    float rr1                       = 1.0f/r;
    float rr2                       = rr1*rr1;
    float rr3                       = rr1*rr2;
    float rr5                       = 3.0f*rr3*rr2;
    float rr7                       = 5.0f*rr5*rr2;
    float rr9                       = 7.0f*rr7*rr2;
    float rr11                      = 9.0f*rr9*rr2;

    //-------------------------------------------

    if( atomI.damp != 0.0f && atomJ.damp != 0.0 && r < scalingDistanceCutoff ){
   
        float distanceIJ, r2I;
        distanceIJ                    = r;
        r2I                           = rr2;
        
        float ratio                   = distanceIJ/(atomI.damp*atomJ.damp);
        float pGamma                  = atomJ.thole > atomI.thole ? atomI.thole : atomJ.thole;

        float damp                          = ratio*ratio*ratio*pGamma;
        float dampExp                 = expf( -damp );
        float damp1                   = damp + one;
        float damp2                   = damp*damp;
        float damp3                   = damp2*damp;

        scalingFactors[Scale3Index]   = one - dampExp;
        scalingFactors[Scale5Index]   = one - damp1*dampExp;
        scalingFactors[Scale7Index]   = one - ( damp1 + 0.6f*damp2)*dampExp;
        scalingFactors[Scale9Index]   = one - ( damp1 + ( 2.0f*damp2 + damp3 )*i35)*dampExp;

        float factor                  = 3.0f*damp*dampExp*r2I;
        float factor7                 = -0.2f + 0.6f*damp;
        
        for( int ii = 0; ii < 3; ii++ ){
            scalingFactors[Ddsc30Index + ii] = factor*deltaR[ii];
            scalingFactors[Ddsc50Index + ii] = scalingFactors[Ddsc30Index + ii]*damp;
            scalingFactors[Ddsc70Index + ii] = scalingFactors[Ddsc50Index + ii]*factor7;
        }

    }
      
    float scaleI0 = scalingFactors[Scale3Index]*scalingFactors[UScaleIndex];
    float dsc0    = scalingFactors[Scale3Index]*scalingFactors[DScaleIndex];
    float psc0    = scalingFactors[Scale3Index]*scalingFactors[PScaleIndex];
    float scaleI1 = scalingFactors[Scale3Index+1]*scalingFactors[UScaleIndex];
    float dsc1    = scalingFactors[Scale3Index+1]*scalingFactors[DScaleIndex];
    float psc1    = scalingFactors[Scale3Index+1]*scalingFactors[PScaleIndex];
    float dsc2    = scalingFactors[Scale3Index+2]*scalingFactors[DScaleIndex];
    float psc2    = scalingFactors[Scale3Index+2]*scalingFactors[PScaleIndex];
                       
    float qIr[3], qJr[3];

    amatrixProductVector3( atomJ.labFrameQuadrupole,      deltaR,      qJr);
    amatrixProductVector3( atomI.labFrameQuadrupole,      deltaR,      qIr);

    float sc2     = DOT3_4(        atomI.labFrameDipole,  atomJ.labFrameDipole );
    float sc3     = DOT3_4(        atomI.labFrameDipole,  deltaR  );
    float sc4     = DOT3_4(        atomJ.labFrameDipole,  deltaR  );
    
    float sc5     = DOT3_4(        qIr, deltaR  );
    float sc6     = DOT3_4(        qJr, deltaR  );
    
    float sc7     = DOT3_4(        qIr, atomJ.labFrameDipole );
    float sc8     = DOT3_4(        qJr, atomI.labFrameDipole );
    
    float sc9     = DOT3_4(        qIr, qJr );
    
    float sc10    = MATRIXDOT31( atomI.labFrameQuadrupole, atomJ.labFrameQuadrupole );
    
    float sci1    = DOT3_4(        atomI.inducedDipole,  atomJ.labFrameDipole ) +
                    DOT3_4(        atomJ.inducedDipole,  atomI.labFrameDipole );
        
    float sci3    = DOT3_4(        atomI.inducedDipole,  deltaR  );
    float sci4    = DOT3_4(        atomJ.inducedDipole,  deltaR  );
    
    float sci7    = DOT3_4(        qIr, atomJ.inducedDipole );
    float sci8    = DOT3_4(        qJr, atomI.inducedDipole );
    
    float scip1   = DOT3_4(        atomI.inducedDipoleP, atomJ.labFrameDipole ) +
                    DOT3_4(        atomJ.inducedDipoleP, atomI.labFrameDipole );
    
    float scip2   = DOT3_4(        atomI.inducedDipole,  atomJ.inducedDipoleP) +
                    DOT3_4(        atomJ.inducedDipole,  atomI.inducedDipoleP);
    
    float scip3   = DOT3_4(        atomI.inducedDipoleP, deltaR );
    float scip4   = DOT3_4(        atomJ.inducedDipoleP, deltaR );
    
    float scip7   = DOT3_4(        qIr, atomJ.inducedDipoleP );
    float scip8   = DOT3_4(        qJr, atomI.inducedDipoleP );

    float findmp[3];
    float scaleF         = 0.5f*scalingFactors[UScaleIndex];
    float inducedFactor3 = scip2*rr3*scaleF;
    float inducedFactor5 = (sci3*scip4+scip3*sci4)*rr5*scaleF;
    findmp[0]            = inducedFactor3*ddsc3[0] - inducedFactor5*ddsc5[0];
    findmp[1]            = inducedFactor3*ddsc3[1] - inducedFactor5*ddsc5[1];
    findmp[2]            = inducedFactor3*ddsc3[2] - inducedFactor5*ddsc5[2];

    float gli1               = atomJ.q*sci3 - atomI.q*sci4;
    float gli2               = -sc3*sci4 - sci3*sc4;
    float gli3               = sci3*sc6 - sci4*sc5;
    float gli6               = sci1;
    float gli7               = 2.0f*(sci7-sci8);
    
    float glip1              = atomJ.q*scip3 - atomI.q*scip4;
    float glip2              = -sc3*scip4 - scip3*sc4;
    float glip3              = scip3*sc6 - scip4*sc5;
    float glip6              = scip1;
    float glip7              = 2.0f*(scip7-scip8);
    
    float fridmp[3];
    float factor3, factor5, factor7;
    
    if( scalingFactors[PScaleIndex] == 1.0f && scalingFactors[PScaleIndex] == 1.0f ){
        factor3 = rr3*( gli1  +  gli6  + glip1  + glip6 );
        factor5 = rr5*( gli2  +  gli7  + glip2  + glip7 );
        factor7 = rr7*( gli3  + glip3 );
    } else {
        factor3 = rr3*(( gli1  +  gli6)*scalingFactors[PScaleIndex] +
                       (glip1  + glip6)*scalingFactors[DScaleIndex]);
   
       factor5 = rr5*(( gli2  +  gli7)*scalingFactors[PScaleIndex] +
                      (glip2  + glip7)*scalingFactors[DScaleIndex]);
   
       factor7 = rr7*( gli3*scalingFactors[PScaleIndex] + glip3*scalingFactors[DScaleIndex]);
    }
      
    fridmp[0] = 0.5f*(factor3*ddsc3[0] + factor5*ddsc5[0] + factor7*ddsc7[0]);
    fridmp[1] = 0.5f*(factor3*ddsc3[1] + factor5*ddsc5[1] + factor7*ddsc7[1]);
    fridmp[2] = 0.5f*(factor3*ddsc3[2] + factor5*ddsc5[2] + factor7*ddsc7[2]);
      
    float gl0 = atomI.q*atomJ.q;
    float gl1 = atomJ.q*sc3 - atomI.q*sc4;
    float gl2 = atomI.q*sc6 + atomJ.q*sc5 - sc3*sc4;
    float gl3 = sc3*sc6 - sc4*sc5;
    float gl4 = sc5*sc6;
    float gl6 = sc2;
    float gl7 = 2.0f*(sc7-sc8);
    float gl8 = 2.0f*sc10;
    float gl5 = -4.0f*sc9;
    
    float gf1 = rr3*gl0 + rr5*(gl1+gl6) + rr7*(gl2+gl7+gl8) + rr9*(gl3+gl5) + rr11*gl4;
    float gf2 = -atomJ.q*rr3 + sc4*rr5 - sc6*rr7;
    float gf3 =  atomI.q*rr3 + sc3*rr5 + sc5*rr7;
    float gf4 = 2.0f*rr5;
    float gf5 = 2.0f*(-atomJ.q*rr5+sc4*rr7-sc6*rr9);
    float gf6 = 2.0f*(-atomI.q*rr5-sc3*rr7-sc5*rr9);
    float gf7 = 4.0f*rr7;

    // energy

    float conversionFactor   = (cAmoebaSim.electric/cAmoebaSim.dielec);
    float em                 = scalingFactors[MScaleIndex]*(rr1*gl0 + rr3*(gl1+gl6) + rr5*(gl2+gl7+gl8) + rr7*(gl3+gl5) + rr9*gl4);
    float ei                 = 0.5f*(rr3*(gli1+gli6)*psc0 + rr5*(gli2+gli7)*psc1 + rr7*gli3*psc2);
    *energy                  = conversionFactor*(em+ei);
    
#ifdef AMOEBA_DEBUG
#if 0
if( 1 ){
    int debugIndex           = 0;
    debugArray[debugIndex].x = conversionFactor*em;
    debugArray[debugIndex].y = conversionFactor*ei;
    debugArray[debugIndex].z = rr1;
    debugArray[debugIndex].w = rr3;

    debugIndex++;
    debugArray[debugIndex].x = gl0;
    debugArray[debugIndex].y = gl1;
    debugArray[debugIndex].z = gl6;
    debugArray[debugIndex].w = gl2;

    debugIndex++;
    debugArray[debugIndex].x = gli1;
    debugArray[debugIndex].y = gli3;
    debugArray[debugIndex].z = gli2;
    debugArray[debugIndex].w = gli7;

    debugIndex++;
    debugArray[debugIndex].x = psc0;
    debugArray[debugIndex].y = psc1;
    debugArray[debugIndex].z = psc2;
    debugArray[debugIndex].w = scalingFactors[MScaleIndex];

}
#endif
#endif

    float ftm2[3];
    float temp1[3],temp2[3],temp3[3];
    float qIqJr[3], qJqIr[3], qIdJ[3], qJdI[3];
    amatrixProductVector3( atomI.labFrameQuadrupole,      atomJ.labFrameDipole,     qIdJ );//MK
    amatrixProductVector3( atomJ.labFrameQuadrupole,      atomI.labFrameDipole,     qJdI );//MK

    amatrixProductVector3( atomI.labFrameQuadrupole,      qJr,    qIqJr );//MK
    amatrixProductVector3( atomJ.labFrameQuadrupole,      qIr,    qJqIr );//MK
    amatrixProductVector3( atomJ.labFrameQuadrupole,      qIr,    temp1 );
    amatrixProductVector3( atomJ.labFrameQuadrupole,      atomI.labFrameDipole,     temp2 );

    for( int ii = 0; ii < 3; ii++ ){
        ftm2[ii] = gf1*deltaR[ii]                             +
                   gf2*atomI.labFrameDipole[ii]     + gf3*atomJ.labFrameDipole[ii]  +
                   gf4*(temp2[ii]  - qIdJ[ii])                    +
                   gf5*qIr[ii]    + gf6*qJr[ii] +
                   gf7*(qIqJr[ii] + temp1[ii]);
    
    }

    // get the induced force;

    // intermediate variables for the induced-permanent terms;
    
    float gfi1 = rr5*0.5f*((gli1+gli6)*psc0 + (glip1+glip6)*dsc0 + scip2*scaleI0) + rr7*((gli7+gli2)*psc1 + (glip7+glip2)*dsc1 -
                                                       (sci3*scip4+scip3*sci4)*scaleI1)*0.5f + rr9*(gli3*psc2+glip3*dsc2)*0.5f;
    float gfi4 = 2.0f*rr5;
    float gfi5 = rr7* (sci4*psc2 + scip4*dsc2);
    float gfi6 = -rr7*(sci3*psc2 + scip3*dsc2);


    float ftm2i[3];
    float temp4[3];
    float temp5[3];
    float temp6[3];
    float temp7[3];
    float temp8[3];
    float temp9[3];
    float temp10[3];
    float temp11[3];
    float temp12[3];
    float temp13[3];
    float temp14[3];
    float temp15[3];
    float qIuJp[3], qJuIp[3];
    float qIuJ[3], qJuI[3];

    amatrixProductVector3(atomJ.labFrameQuadrupole,      atomI.inducedDipoleP,    temp4);

    amatrixProductVector3(atomI.labFrameQuadrupole,      atomJ.inducedDipoleP,    qIuJp);//MK
    amatrixProductVector3(atomJ.labFrameQuadrupole,      atomI.inducedDipoleP,    qJuIp);//MK
    amatrixProductVector3(atomJ.labFrameQuadrupole,      atomI.inducedDipole ,    qJuI);//MK

    amatrixProductVector3(atomJ.labFrameQuadrupole,      atomI.inducedDipole,    temp5);
    amatrixProductVector3(atomI.labFrameQuadrupole,      atomJ.inducedDipole ,     qIuJ);//MK

    float temp1_0,temp2_0,temp3_0;
    for( int ii = 0; ii < 3; ii++ ){
        temp1_0 = gfi1*deltaR[ii] +
                  0.5f*(-rr3*atomJ.q*(atomI.inducedDipole[ii]*psc0 + atomI.inducedDipoleP[ii]*dsc0) +
                  rr5*sc4*(atomI.inducedDipole[ii]*psc1 + atomI.inducedDipoleP[ii]*dsc1) -
                  rr7*sc6*(atomI.inducedDipole[ii]*psc2 + atomI.inducedDipoleP[ii]*dsc2)) ;

        temp2_0 = (rr3*atomI.q*(atomJ.inducedDipole[ii]*psc0+atomJ.inducedDipoleP[ii]*dsc0) +
                   rr5*sc3*(atomJ.inducedDipole[ii]*psc1 +atomJ.inducedDipoleP[ii]*dsc1) +
                   rr7*sc5*(atomJ.inducedDipole[ii]*psc2 +atomJ.inducedDipoleP[ii]*dsc2))*0.5f +
                   rr5*scaleI1*(sci4*atomI.inducedDipoleP[ii]+scip4*atomI.inducedDipole[ii] +
                   sci3*atomJ.inducedDipoleP[ii]+scip3*atomJ.inducedDipole[ii])*0.5f ;

        temp3_0 = 0.5f*(sci4*psc1+scip4*dsc1)*rr5*atomI.labFrameDipole[ii] +
                  0.5f*(sci3*psc1+scip3*dsc1)*rr5*atomJ.labFrameDipole[ii] +
                  0.5f*gfi4*((temp5[ii]-qIuJ[ii])*psc1 +
                  (temp4[ii]-qIuJp[ii])*dsc1) + gfi5*qIr[ii] + gfi6*qJr[ii];
        ftm2i[ii] = temp1_0 + temp2_0 + temp3_0;
    }

    // handle of scaling for partially excluded interactions;
    // correction to convert mutual to direct polarization force;
    
    ftm2i[0] -= (fridmp[0] + findmp[0]);
    ftm2i[1] -= (fridmp[1] + findmp[1]);
    ftm2i[2] -= (fridmp[2] + findmp[2]);
    
    // now perform the torque calculation;
    // intermediate terms for torque between multipoles i and j;
    
    float gti2 = 0.5f*(sci4*psc1+scip4*dsc1)*rr5;
    float gti3 = 0.5f*(sci3*psc1+scip3*dsc1)*rr5;
    float gti4 = gfi4;
    float gti5 = gfi5;
    float gti6 = gfi6;

    // get the permanent (ttm2, ttm3) and induced interaction torques (ttm2i, ttm3i)
    
    acrossProductVector3(atomI.labFrameDipole,      atomJ.labFrameDipole,      temp1);
    acrossProductVector3(atomI.labFrameDipole,      atomJ.inducedDipole ,      temp2);
    acrossProductVector3(atomI.labFrameDipole,      atomJ.inducedDipoleP,     temp3);
    acrossProductVector3(atomI.labFrameDipole,      deltaR,       temp4);
    acrossProductVector3(deltaR,       qIuJp,   temp5);
    acrossProductVector3(deltaR,       qIr,     temp6);
    acrossProductVector3(deltaR,       qIuJ,    temp7);
    acrossProductVector3(atomJ.inducedDipole ,     qIr,     temp8);
    acrossProductVector3(atomJ.inducedDipoleP,     qIr,     temp9);
    acrossProductVector3(atomI.labFrameDipole,     qJr,     temp10);
    acrossProductVector3(atomJ.labFrameDipole,     qIr,     temp11);
    acrossProductVector3(deltaR,       qIqJr,   temp12);
    acrossProductVector3(deltaR,       qIdJ,    temp13);

    amatrixCrossProductMatrix3(atomI.labFrameQuadrupole,      atomJ.labFrameQuadrupole,      temp14);
    acrossProductVector3(qJr, qIr,     temp15);

    float ttm2_0  = -rr3*temp1[0] + gf2*temp4[0]-gf5*temp6[0] + gf4*(temp10[0] + temp11[0] + temp13[0]-2.0f*temp14[0]) - gf7*(temp12[0] + temp15[0]);
    float ttm2i_0 = -rr3*(temp2[0]*psc0+temp3[0]*dsc0)*0.5f + gti2*temp4[0] + gti4*((temp8[0]+ temp7[0])*psc1 + (temp9[0] + temp5[0])*dsc1)*0.5f - gti5*temp6[0];
    float ttm2_1  = -rr3*temp1[1] + gf2*temp4[1]-gf5*temp6[1] + gf4*(temp10[1] + temp11[1] + temp13[1]-2.0f*temp14[1]) - gf7*(temp12[1] + temp15[1]);
    float ttm2i_1 = -rr3*(temp2[1]*psc0+temp3[1]*dsc0)*0.5f + gti2*temp4[1] + gti4*((temp8[1]+ temp7[1])*psc1 + (temp9[1] + temp5[1])*dsc1)*0.5f - gti5*temp6[1];
    float ttm2_2  = -rr3*temp1[2] + gf2*temp4[2]-gf5*temp6[2] + gf4*(temp10[2] + temp11[2] + temp13[2]-2.0f*temp14[2]) - gf7*(temp12[2] + temp15[2]);
    float ttm2i_2 = -rr3*(temp2[2]*psc0+temp3[2]*dsc0)*0.5f + gti2*temp4[2] + gti4*((temp8[2]+ temp7[2])*psc1 + (temp9[2] + temp5[2])*dsc1)*0.5f - gti5*temp6[2];

    acrossProductVector3(atomJ.labFrameDipole,      deltaR,       temp2  );
    acrossProductVector3(deltaR,       qJr,     temp3  );
    acrossProductVector3(atomI.labFrameDipole,      qJr,     temp4  );
    acrossProductVector3(atomJ.labFrameDipole,      qIr,     temp5  );
    acrossProductVector3(deltaR,       qJdI,    temp6  );
    acrossProductVector3(deltaR,       qJqIr,   temp7  );
    acrossProductVector3(qJr,     qIr,     temp8  ); // _qJrxqIr
    acrossProductVector3(atomJ.labFrameDipole,      atomI.inducedDipole ,      temp9  ); // _dJxuI
    acrossProductVector3(atomJ.labFrameDipole,      atomI.inducedDipoleP,     temp10 ); // _dJxuIp

    acrossProductVector3(atomI.inducedDipoleP,     qJr,     temp11 ); // _uIxqJrp
    acrossProductVector3(atomI.inducedDipole ,     qJr,     temp12 ); // _uIxqJr
    acrossProductVector3(deltaR,       qJuIp,   temp13 ); // _rxqJuIp
    acrossProductVector3(deltaR,       qJuI,    temp15 ); // _rxqJuI

    float ttm3_0 = rr3*temp1[0] + gf3*temp2[0] - gf6*temp3[0] - gf4*(temp4[0] + temp5[0] + temp6[0] - 2.0f*temp14[0]) - gf7*(temp7[0] - temp8[0]);
    float ttm3i_0 = -rr3*(temp9[0]*psc0+ temp10[0]*dsc0)*0.5f + gti3*temp2[0] - gti4*((temp12[0] + temp15[0])*psc1 + (temp11[0] + temp13[0])*dsc1)*0.5f - gti6*temp3[0];
    float ttm3_1 = rr3*temp1[1] + gf3*temp2[1] - gf6*temp3[1] - gf4*(temp4[1] + temp5[1] + temp6[1] - 2.0f*temp14[1]) - gf7*(temp7[1] - temp8[1]);
    float ttm3i_1 = -rr3*(temp9[1]*psc0+ temp10[1]*dsc0)*0.5f + gti3*temp2[1] - gti4*((temp12[1] + temp15[1])*psc1 + (temp11[1] + temp13[1])*dsc1)*0.5f - gti6*temp3[1];
    float ttm3_2 = rr3*temp1[2] + gf3*temp2[2] - gf6*temp3[2] - gf4*(temp4[2] + temp5[2] + temp6[2] - 2.0f*temp14[2]) - gf7*(temp7[2] - temp8[2]);
    float ttm3i_2 = -rr3*(temp9[2]*psc0+ temp10[2]*dsc0)*0.5f + gti3*temp2[2] - gti4*((temp12[2] + temp15[2])*psc1 + (temp11[2] + temp13[2])*dsc1)*0.5f - gti6*temp3[2];

    if( scalingFactors[MScaleIndex] < 1.0f ){
    
        ftm2[0] *= scalingFactors[MScaleIndex];
        ftm2[1] *= scalingFactors[MScaleIndex];
        ftm2[2] *= scalingFactors[MScaleIndex];
        
        ttm2_0 *= scalingFactors[MScaleIndex];
        ttm2_1 *= scalingFactors[MScaleIndex];
        ttm2_2 *= scalingFactors[MScaleIndex];
        
        ttm3_0 *= scalingFactors[MScaleIndex];
        ttm3_1 *= scalingFactors[MScaleIndex];
        ttm3_2 *= scalingFactors[MScaleIndex];
    
    }


#ifdef AMOEBA_DEBUG
#if 0
if( 0 ){
int debugIndex               = 0;
    debugArray[debugIndex].x = conversionFactor*ftm2[0];
    debugArray[debugIndex].y = conversionFactor*ftm2i[0];
    debugArray[debugIndex].z = conversionFactor*ttm3_0;
    debugArray[debugIndex].w = conversionFactor*ttm3i_0;

    debugIndex++;
    debugArray[debugIndex].x = temp1[0];
    debugArray[debugIndex].y = temp1[1];
    debugArray[debugIndex].z = temp1[2];
    debugArray[debugIndex].w = 1.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp2[0];
    debugArray[debugIndex].y = temp2[1];
    debugArray[debugIndex].z = temp2[2];
    debugArray[debugIndex].w = 2.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp3[0];
    debugArray[debugIndex].y = temp3[1];
    debugArray[debugIndex].z = temp3[2];
    debugArray[debugIndex].w = 3.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp4[0];
    debugArray[debugIndex].y = temp4[1];
    debugArray[debugIndex].z = temp4[2];
    debugArray[debugIndex].w = 4.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp5[0];
    debugArray[debugIndex].y = temp5[1];
    debugArray[debugIndex].z = temp5[2];
    debugArray[debugIndex].w = 5.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp6[0];
    debugArray[debugIndex].y = temp6[1];
    debugArray[debugIndex].z = temp6[2];
    debugArray[debugIndex].w = 6.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp14[0];
    debugArray[debugIndex].y = temp14[1];
    debugArray[debugIndex].z = temp14[2];
    debugArray[debugIndex].w = 14.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp7[0];
    debugArray[debugIndex].y = temp7[1];
    debugArray[debugIndex].z = temp7[2];
    debugArray[debugIndex].w = 7.0f;


    debugIndex++;
    debugArray[debugIndex].x = temp8[0];
    debugArray[debugIndex].y = temp8[1];
    debugArray[debugIndex].z = temp8[2];
    debugArray[debugIndex].w = 8.0f;

    debugIndex++;
    debugArray[debugIndex].x = rr3;
    debugArray[debugIndex].y = gf3;
    debugArray[debugIndex].z = gf6;
    debugArray[debugIndex].w = 20.0f;

    debugIndex++;
    debugArray[debugIndex].x = gf4;
    debugArray[debugIndex].y = gf7;
    debugArray[debugIndex].z = 0.0f;
    debugArray[debugIndex].w = 21.0f;

    debugIndex++;
    debugArray[debugIndex].x = atomJ.labFrameDipole[0];
    debugArray[debugIndex].y = atomJ.labFrameDipole[1];
    debugArray[debugIndex].z = atomJ.labFrameDipole[2];
    debugArray[debugIndex].w = 22.0f;

    debugIndex++;
    debugArray[debugIndex].x = deltaR[0];
    debugArray[debugIndex].y = deltaR[1];
    debugArray[debugIndex].z = deltaR[2];
    debugArray[debugIndex].w = 23.0f;

}
#endif
#endif

    outputForce[0]        = -conversionFactor*(ftm2[0] + ftm2i[0]);
    outputForce[1]        = -conversionFactor*(ftm2[1] + ftm2i[1]);
    outputForce[2]        = -conversionFactor*(ftm2[2] + ftm2i[2]);
    
    outputTorque[0][0]    = conversionFactor*(ttm2_0 + ttm2i_0);
    outputTorque[0][1]    = conversionFactor*(ttm2_1 + ttm2i_1);
    outputTorque[0][2]    = conversionFactor*(ttm2_2 + ttm2i_2);

    outputTorque[1][0]    = conversionFactor*(ttm3_0 + ttm3i_0);
    outputTorque[1][1]    = conversionFactor*(ttm3_1 + ttm3i_1);
    outputTorque[1][2]    = conversionFactor*(ttm3_2 + ttm3i_2);

    return;

}

__device__ void loadElectrostaticShared( struct ElectrostaticParticle* sA, unsigned int atomI,
                                         float4* atomCoord, float* labFrameDipoleJ, float* labQuadrupole,
                                         float* inducedDipole, float* inducedDipolePolar, float2* dampingFactorAndThole )
{
    // coordinates & charge

    sA->x                        = atomCoord[atomI].x;
    sA->y                        = atomCoord[atomI].y;
    sA->z                        = atomCoord[atomI].z;
    sA->q                        = atomCoord[atomI].w;

    // lab dipole

    sA->labFrameDipole[0]         = labFrameDipoleJ[atomI*3];
    sA->labFrameDipole[1]         = labFrameDipoleJ[atomI*3+1];
    sA->labFrameDipole[2]         = labFrameDipoleJ[atomI*3+2];

    // lab quadrupole

    sA->labFrameQuadrupole[0]    = labQuadrupole[atomI*9];
    sA->labFrameQuadrupole[1]    = labQuadrupole[atomI*9+1];
    sA->labFrameQuadrupole[2]    = labQuadrupole[atomI*9+2];
    sA->labFrameQuadrupole[3]    = labQuadrupole[atomI*9+3];
    sA->labFrameQuadrupole[4]    = labQuadrupole[atomI*9+4];
    sA->labFrameQuadrupole[5]    = labQuadrupole[atomI*9+5];
    sA->labFrameQuadrupole[6]    = labQuadrupole[atomI*9+6];
    sA->labFrameQuadrupole[7]    = labQuadrupole[atomI*9+7];
    sA->labFrameQuadrupole[8]    = labQuadrupole[atomI*9+8];

    // induced dipole

    sA->inducedDipole[0]          = inducedDipole[atomI*3];
    sA->inducedDipole[1]          = inducedDipole[atomI*3+1];
    sA->inducedDipole[2]          = inducedDipole[atomI*3+2];

    // induced dipole polar

    sA->inducedDipoleP[0]         = inducedDipolePolar[atomI*3];
    sA->inducedDipoleP[1]         = inducedDipolePolar[atomI*3+1];
    sA->inducedDipoleP[2]         = inducedDipolePolar[atomI*3+2];

    sA->damp                     = dampingFactorAndThole[atomI].x;
    sA->thole                    = dampingFactorAndThole[atomI].y;

}

// Include versions of the kernels for N^2 calculations.

#undef USE_OUTPUT_BUFFER_PER_WARP
#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaElectrostatic.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaElectrostatic.h"

// reduce psWorkArray_3_1 -> force
// reduce psWorkArray_3_2 -> torque

static void kReduceForceTorque(amoebaGpuContext amoebaGpu )
{
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_1->_pDevStream[0], amoebaGpu->psForce->_pDevStream[0] );
    LAUNCHERROR("kReduceElectrostaticForce");
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_2->_pDevStream[0], amoebaGpu->psTorque->_pDevStream[0] );
    LAUNCHERROR("kReduceElectrostaticTorque");
}

#ifdef AMOEBA_DEBUG
static void printElectrostaticBuffer( amoebaGpuContext amoebaGpu, unsigned int bufferIndex )
{
    (void) fprintf( amoebaGpu->log, "Electrostatic Buffer %u\n", bufferIndex );
    unsigned int start = bufferIndex*3*amoebaGpu->paddedNumberOfAtoms;
    unsigned int stop  = (bufferIndex+1)*3*amoebaGpu->paddedNumberOfAtoms;
    for( unsigned int ii = start; ii < stop; ii += 3 ){
        unsigned int ii3Index      = ii/3;
        unsigned int bufferIndex   = ii3Index/(amoebaGpu->paddedNumberOfAtoms);
        unsigned int particleIndex = ii3Index - bufferIndex*(amoebaGpu->paddedNumberOfAtoms);
        (void) fprintf( amoebaGpu->log, "   %6u %3u %6u [%14.6e %14.6e %14.6e] [%14.6e %14.6e %14.6e]\n", 
                            ii/3,  bufferIndex, particleIndex,
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii],
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii+1],
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii+2],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii+1],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii+2] );
    } 

/*
    start = 0;
    stop  = -146016;
    float maxV = -1.0e+99;
    for( unsigned int ii = start; ii < stop; ii += 3 ){
        if(  amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii] > maxV ){ 
            unsigned int ii3Index      = ii/3;
            unsigned int bufferIndex   = ii3Index/(amoebaGpu->paddedNumberOfAtoms);
            unsigned int particleIndex = ii3Index - bufferIndex*(amoebaGpu->paddedNumberOfAtoms);
            (void) fprintf( amoebaGpu->log, "MaxQ %6u %3u %6u %14.6e\n", 
                            ii/3,  bufferIndex, particleIndex,
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii] );
            maxV = amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii];
        } 
    } 
*/
}

static void printElectrostaticAtomBuffers( amoebaGpuContext amoebaGpu, unsigned int targetAtom )
{
    (void) fprintf( amoebaGpu->log, "Electrostatic atom %u\n", targetAtom );
    for( unsigned int ii = 0; ii < amoebaGpu->outputBuffers; ii++ ){
        unsigned int particleIndex = 3*(targetAtom + ii*amoebaGpu->paddedNumberOfAtoms);
        (void) fprintf( amoebaGpu->log, " %2u %6u [%14.6e %14.6e %14.6e] [%14.6e %14.6e %14.6e]\n", 
                        ii, particleIndex,
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex],
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex+1],
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex+2],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex+1],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex+2] );
    } 
}
#endif

/**---------------------------------------------------------------------------------------

   Compute Amoeba electrostatic force & torque

   @param amoebaGpu        amoebaGpu context
   @param gpu              OpenMM gpu Cuda context

   --------------------------------------------------------------------------------------- */

void cudaComputeAmoebaElectrostatic( amoebaGpuContext amoebaGpu )
{
  
   // ---------------------------------------------------------------------------------------

    static unsigned int threadsPerBlock = 0;

#ifdef AMOEBA_DEBUG
    static const char* methodName = "cudaComputeAmoebaElectrostatic";
    static int timestep = 0;
    std::vector<int> fileId;
    timestep++;
    fileId.resize( 2 );
    fileId[0] = timestep;
    fileId[1] = 1;
#endif

    // ---------------------------------------------------------------------------------------

    gpuContext gpu = amoebaGpu->gpuContext;

    // apparently debug array can take up nontrivial no. registers

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){
        (void) fprintf( amoebaGpu->log, "%s %d maxCovalentDegreeSz=%d"
                        " gamma=%.3e scalingDistanceCutoff=%.3f ZZZ\n",
                        methodName, gpu->natoms,
                        amoebaGpu->maxCovalentDegreeSz, amoebaGpu->pGamma,
                        amoebaGpu->scalingDistanceCutoff );
    }   
   int paddedNumberOfAtoms                    = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
    CUDAStream<float4>* debugArray            = new CUDAStream<float4>(paddedNumberOfAtoms*paddedNumberOfAtoms, 1, "DebugArray");
    memset( debugArray->_pSysStream[0],      0, sizeof( float )*4*paddedNumberOfAtoms*paddedNumberOfAtoms);
    debugArray->Upload();
    unsigned int targetAtom                   = 0;
#endif

    // on first pass, set threads/block

    if( threadsPerBlock == 0 ){
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 384;
        else if (gpu->sm_version >= SM_12)
            maxThreads = 128;
        else
            maxThreads = 64;
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(ElectrostaticParticle)), maxThreads);
    }

    kClearFields_3( amoebaGpu, 2 );

    if (gpu->bOutputBufferPerWarp){

        (void) fprintf( amoebaGpu->log, "kCalculateAmoebaCudaElectrostaticN2Forces warp:  numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%lu shrd=%lu Ebuf=%u ixnCt=%lu workUnits=%u\n",
                        amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                        sizeof(ElectrostaticParticle), sizeof(ElectrostaticParticle)*threadsPerBlock, amoebaGpu->energyOutputBuffers, (*gpu->psInteractionCount)[0], gpu->sim.workUnits );
        (void) fflush( amoebaGpu->log );
        kCalculateAmoebaCudaElectrostaticN2ByWarpForces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(ElectrostaticParticle)*threadsPerBlock>>>(
                                                                           amoebaGpu->psWorkUnit->_pDevStream[0],
                                                                           gpu->psPosq4->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameDipole->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameQuadrupole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipolePolar->_pDevStream[0],
                                                                           amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                           debugArray->_pDevStream[0], targetAtom );
#else
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif

    } else {

#ifdef AMOEBA_DEBUG
        (void) fprintf( amoebaGpu->log, "kCalculateAmoebaCudaElectrostaticN2Forces no warp:  numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%u shrd=%u Ebuf=%u ixnCt=%u workUnits=%u\n",
                        amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                        sizeof(ElectrostaticParticle), sizeof(ElectrostaticParticle)*threadsPerBlock, amoebaGpu->energyOutputBuffers, (*gpu->psInteractionCount)[0], gpu->sim.workUnits );
        (void) fflush( amoebaGpu->log );
#endif

        kCalculateAmoebaCudaElectrostaticN2Forces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(ElectrostaticParticle)*threadsPerBlock>>>(
                                                                           amoebaGpu->psWorkUnit->_pDevStream[0],
                                                                           gpu->psPosq4->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameDipole->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameQuadrupole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipolePolar->_pDevStream[0],
                                                                           amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                           debugArray->_pDevStream[0], targetAtom );
#else
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif
    }
    LAUNCHERROR("kCalculateAmoebaCudaElectrostaticN2Forces");

#ifdef AMOEBA_DEBUG
    if( 0 && amoebaGpu->log ){

        amoebaGpu->psWorkArray_3_1->Download();
        amoebaGpu->psWorkArray_3_2->Download();

        printElectrostaticAtomBuffers( amoebaGpu, (targetAtom + 0) );
        //printElectrostaticAtomBuffers( amoebaGpu, (targetAtom + 1231) );
        printElectrostaticBuffer( amoebaGpu, 0 );
        //printElectrostaticBuffer( amoebaGpu, 38 );
    }
#endif

    kReduceForceTorque( amoebaGpu );

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){

        amoebaGpu->psForce->Download();
        amoebaGpu->psTorque->Download();
        debugArray->Download();

        (void) fprintf( amoebaGpu->log, "Finished Electrostatic kernel execution\n" ); (void) fflush( amoebaGpu->log );

        int maxPrint        = 1400;
        for( int ii = 0; ii < gpu->natoms; ii++ ){
           (void) fprintf( amoebaGpu->log, "%5d ", ii); 

            int indexOffset     = ii*3;
    
           // force

           (void) fprintf( amoebaGpu->log,"ElectrostaticF [%16.9e %16.9e %16.9e] ",
                           amoebaGpu->psForce->_pSysStream[0][indexOffset],
                           amoebaGpu->psForce->_pSysStream[0][indexOffset+1],
                           amoebaGpu->psForce->_pSysStream[0][indexOffset+2] );
    
           // torque

           (void) fprintf( amoebaGpu->log,"ElectrostaticT [%16.9e %16.9e %16.9e] ",
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset],
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset+1],
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset+2] );

           // coords

#if 0
            (void) fprintf( amoebaGpu->log,"x[%16.9e %16.9e %16.9e] ",
                            gpu->psPosq4->_pSysStream[0][ii].x,
                            gpu->psPosq4->_pSysStream[0][ii].y,
                            gpu->psPosq4->_pSysStream[0][ii].z);


           for( int jj = 0; jj < gpu->natoms && jj < 5; jj++ ){
               int debugIndex = jj*gpu->natoms + ii;
               float xx       =  gpu->psPosq4->_pSysStream[0][jj].x -  gpu->psPosq4->_pSysStream[0][ii].x;
               float yy       =  gpu->psPosq4->_pSysStream[0][jj].y -  gpu->psPosq4->_pSysStream[0][ii].y;
               float zz       =  gpu->psPosq4->_pSysStream[0][jj].z -  gpu->psPosq4->_pSysStream[0][ii].z;
               (void) fprintf( amoebaGpu->log,"\n%4d %4d delta [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] ",
                               ii, jj, xx, yy, zz,
                               debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y, debugArray->_pSysStream[0][debugIndex].z );

           }
#endif
           (void) fprintf( amoebaGpu->log,"\n" );
           if( ii == maxPrint && (gpu->natoms - maxPrint) > ii ){
                ii = gpu->natoms - maxPrint;
           }
        }
        if( 1 ){
            (void) fprintf( amoebaGpu->log,"DebugElec\n" );
            int paddedNumberOfAtoms = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
            for( int jj = 0; jj < gpu->natoms; jj++ ){
                int debugIndex = jj;
                for( int kk = 0; kk < 5; kk++ ){
                    (void) fprintf( amoebaGpu->log,"%5d %5d [%16.9e %16.9e %16.9e %16.9e] E11\n", targetAtom, jj,
                                    debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y,
                                    debugArray->_pSysStream[0][debugIndex].z, debugArray->_pSysStream[0][debugIndex].w );
                    debugIndex += paddedNumberOfAtoms;
                }
                (void) fprintf( amoebaGpu->log,"\n" );
            }
        }
        (void) fflush( amoebaGpu->log );

        if( 0 ){
            (void) fprintf( amoebaGpu->log, "%s Tiled F & T\n", methodName ); fflush( amoebaGpu->log );
            int maxPrint = 12;
            for( int ii = 0; ii < gpu->natoms; ii++ ){
    
                // print cpu & gpu reductions
    
                int offset  = 3*ii;
    
                (void) fprintf( amoebaGpu->log,"%6d F[%16.7e %16.7e %16.7e] T[%16.7e %16.7e %16.7e]\n", ii,
                                amoebaGpu->psForce->_pSysStream[0][offset],
                                amoebaGpu->psForce->_pSysStream[0][offset+1],
                                amoebaGpu->psForce->_pSysStream[0][offset+2],
                                amoebaGpu->psTorque->_pSysStream[0][offset],
                                amoebaGpu->psTorque->_pSysStream[0][offset+1],
                                amoebaGpu->psTorque->_pSysStream[0][offset+2] );
                if( (ii == maxPrint) && (ii < (gpu->natoms - maxPrint)) )ii = gpu->natoms - maxPrint; 
            }   
        }   

        if( 1 ){
            std::vector<int> fileId;
            //fileId.push_back( 0 );
            VectorOfDoubleVectors outputVector;
            cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,            outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psForce,      outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psTorque,     outputVector);
            cudaWriteVectorOfDoubleVectorsToFile( "CudaForceTorque", fileId, outputVector );
         }

    }   
    delete debugArray;

#endif

   // ---------------------------------------------------------------------------------------
}

