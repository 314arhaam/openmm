#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

//#define AMOEBA_DEBUG

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaElectrostaticSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaElectrostaticSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaElectrostaticSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

static int const PScaleIndex            =  0; 
static int const DScaleIndex            =  1; 
static int const UScaleIndex            =  2; 
static int const MScaleIndex            =  3;
static int const Scale3Index            =  4;
static int const Scale5Index            =  5;
static int const Scale7Index            =  6;
static int const Scale9Index            =  7;
static int const Ddsc30Index            =  8;
//static int const Ddsc31Index            =  9;
//static int const Ddsc32Index            = 10; 
static int const Ddsc50Index            = 11;
//static int const Ddsc51Index            = 12;
//static int const Ddsc52Index            = 13; 
static int const Ddsc70Index            = 14;
//static int const Ddsc71Index            = 15;
//static int const Ddsc72Index            = 16;
//static int const DampIndex              = 17;
//static int const DampRatioIndex         = 18; 
//static int const DampExpIndex           = 19; 
static int const LastScalingIndex       = 20;

static int const _qI                    =  0;
static int const _qJ                    =  1;

static int const _r                     =  0; 
static int const _dI                    =  1; 
static int const _dJ                    =  2;
static int const _uI                    =  3;
static int const _uJ                    =  4;
static int const _uIp                   =  5; 
static int const _uJp                   =  6; 
static int const _qIr                   =  7; 
static int const _qJr                   =  8; 
static int const _qIqJr                 =  9;
static int const _qIdJ                  = 10; 
static int const _qIuJ                  = 11;
static int const _qIuJp                 = 12;

/*
static int const _dIxdJ                 = 13;
static int const _dIxuJ                 = 14;
static int const _dIxuJp                = 15;
static int const _dJxr                  = 16;
static int const _dJxuI                 = 17;

static int const _dJxuIp                = 18;
static int const _dIxr                  = 19;
*/

static int const _qJqIr                 = 13;
//static int const _qIxqJ                 = 21;
//static int const _rxqIr                 = 22;
//static int const _rxqJr                 = 23;
//static int const _rxqIJr                = 24;
//static int const _rxqJIr                = 25;
//static int const _qJrxqIr               = 26;
static int const _qJdI                  = 14;
//static int const _qJuI                  = 28;
static int const _qJuI                  = 15;
static int const _qJuIp                 = 16;
//static int const _dIxqJr                = 30;
//static int const _dJxqIr                = 31;
//static int const _uIxqJr                = 32;
//static int const uJxqIr                 = 33;
//static int const _uIxqJrp               = 34;
//static int const _uJxqIrp               = 35;
//static int const _rxqIdJ                = 36;
//static int const _rxqJdI                = 37;
//static int const _rxqIuJ                = 38;
//static int const _rxqJuI                = 17;
//static int const _rxqJuIp               = 18;
//static int const _rxqIuJp               = 40;
//static int const _rxqJuIp               = 41; 
static int const LastVectorFieldIndex   = 17;

#define DOT3_4(u,k,v,l) ((u[k*3+0])*(v[l*3+0]) + (u[k*3+1])*(v[l*3+1]) + (u[k*3+2])*(v[l*3+2]))

#define MATRIXDOT31(u,k,v,l) u[k*9+0]*v[l*9+0] + u[k*9+1]*v[l*9+1] + u[k*9+2]*v[l*9+2] + \
  u[k*9+3]*v[l*9+3] + u[k*9+4]*v[l*9+4] + u[k*9+5]*v[l*9+5] + \
  u[k*9+6]*v[l*9+6] + u[k*9+7]*v[l*9+7] + u[k*9+8]*v[l*9+8]

#define DOT31(u,v) ((u[0])*(v[0]) + (u[1])*(v[1]) + (u[2])*(v[2]))

#define i35 0.257142857f
#define one 1.0f

__device__ void acrossProductVector3(   float* vectorX, float* vectorY, float* vectorZ ){
    vectorZ[0]  = vectorX[1]*vectorY[2] - vectorX[2]*vectorY[1];
    vectorZ[1]  = vectorX[2]*vectorY[0] - vectorX[0]*vectorY[2];
    vectorZ[2]  = vectorX[0]*vectorY[1] - vectorX[1]*vectorY[0];
}

__device__ void amatrixProductVector3(   float* matrixX, float* vectorY, float* vectorZ ){
    vectorZ[0]  = matrixX[0]*vectorY[0] + matrixX[3]*vectorY[1] + matrixX[6]*vectorY[2];
    vectorZ[1]  = matrixX[1]*vectorY[0] + matrixX[4]*vectorY[1] + matrixX[7]*vectorY[2];
    vectorZ[2]  = matrixX[2]*vectorY[0] + matrixX[5]*vectorY[1] + matrixX[8]*vectorY[2];
}

__device__ void amatrixCrossProductMatrix3( float* matrixX, float* matrixY, float* vectorZ ){
  
    float* xPtr[3];
    float* yPtr[3];
        
    xPtr[0]    = matrixX;
    xPtr[1]    = matrixX + 3;
    xPtr[2]    = matrixX + 6;
    
    yPtr[0]    = matrixY;
    yPtr[1]    = matrixY + 3;
    yPtr[2]    = matrixY + 6;
          
    vectorZ[0] = DOT31( xPtr[1], yPtr[2] ) - DOT31( xPtr[2], yPtr[1] );
    vectorZ[1] = DOT31( xPtr[2], yPtr[0] ) - DOT31( xPtr[0], yPtr[2] );
    vectorZ[2] = DOT31( xPtr[0], yPtr[1] ) - DOT31( xPtr[1], yPtr[0] );
  
}

__device__ void calculateElectrostaticPairIxn_kernel( float4 atomCoordinatesI,       float4 atomCoordinatesJ,
                                                      float dampingFactorI,          float dampingFactorJ,
                                                      float tholeI,                  float tholeJ,
                                                      float* labFrameDipoleI,        float* labFrameDipoleJ,
                                                      float* labFrameQuadrupoleI,    float* labFrameQuadrupoleJ,
                                                      float* inducedDipoleI,         float* inducedDipoleJ,
                                                      float* inducedDipolePolarI,    float* inducedDipolePolarJ,
                                                      float scalingDistanceCutoff,   float* scalingFactors,
                                                      float*  outputForce,           float  outputTorque[2][3],
                                                      float* energy
#ifdef AMOEBA_DEBUG
                                                      ,float4* debugArray 
#endif
 ){
  
    float deltaR[5];
    
    // ---------------------------------------------------------------------------------------
    
    float vectorFields[LastVectorFieldIndex*3];
    float vectorFields1[2*9];

    float chargeI                   = atomCoordinatesI.w;
    vectorFields[_dI*3    ]         = labFrameDipoleI[0];
    vectorFields[_dI*3 + 1]         = labFrameDipoleI[1];
    vectorFields[_dI*3 + 2]         = labFrameDipoleI[2];
    
    vectorFields[_uI*3    ]         = inducedDipoleI[0];
    vectorFields[_uI*3 + 1]         = inducedDipoleI[1];
    vectorFields[_uI*3 + 2]         = inducedDipoleI[2];
    
    vectorFields[_uIp*3    ]        = inducedDipolePolarI[0];
    vectorFields[_uIp*3 + 1]        = inducedDipolePolarI[1];
    vectorFields[_uIp*3 + 2]        = inducedDipolePolarI[2];

    vectorFields1[_qI*9    ]        = labFrameQuadrupoleI[0];
    vectorFields1[_qI*9 + 1]        = labFrameQuadrupoleI[1];
    vectorFields1[_qI*9 + 2]        = labFrameQuadrupoleI[2];
    vectorFields1[_qI*9 + 3]        = labFrameQuadrupoleI[3];
    vectorFields1[_qI*9 + 4]        = labFrameQuadrupoleI[4];
    vectorFields1[_qI*9 + 5]        = labFrameQuadrupoleI[5];
    vectorFields1[_qI*9 + 6]        = labFrameQuadrupoleI[6];
    vectorFields1[_qI*9 + 7]        = labFrameQuadrupoleI[7];
    vectorFields1[_qI*9 + 8]        = labFrameQuadrupoleI[8];

    // ---------------------------------------------------------------------------------------

    float chargeJ                   = atomCoordinatesJ.w;

    vectorFields[_dJ*3 + 0]         = labFrameDipoleJ[0];
    vectorFields[_dJ*3 + 1]         = labFrameDipoleJ[1];
    vectorFields[_dJ*3 + 2]         = labFrameDipoleJ[2];

    vectorFields[_uJ*3 + 0]         = inducedDipoleJ[0];
    vectorFields[_uJ*3 + 1]         = inducedDipoleJ[1];
    vectorFields[_uJ*3 + 2]         = inducedDipoleJ[2];
    
    vectorFields[_uJp*3 + 0]        = inducedDipolePolarJ[0];
    vectorFields[_uJp*3 + 1]        = inducedDipolePolarJ[1];
    vectorFields[_uJp*3 + 2]        = inducedDipolePolarJ[2];
    
    vectorFields1[_qJ*9 + 0]        = labFrameQuadrupoleJ[0];
    vectorFields1[_qJ*9 + 1]        = labFrameQuadrupoleJ[1];
    vectorFields1[_qJ*9 + 2]        = labFrameQuadrupoleJ[2];
    vectorFields1[_qJ*9 + 3]        = labFrameQuadrupoleJ[3];
    vectorFields1[_qJ*9 + 4]        = labFrameQuadrupoleJ[4];
    vectorFields1[_qJ*9 + 5]        = labFrameQuadrupoleJ[5];
    vectorFields1[_qJ*9 + 6]        = labFrameQuadrupoleJ[6];
    vectorFields1[_qJ*9 + 7]        = labFrameQuadrupoleJ[7];
    vectorFields1[_qJ*9 + 8]        = labFrameQuadrupoleJ[8];
      
    float* ddsc3                    =  scalingFactors + Ddsc30Index;
    float* ddsc5                    =  scalingFactors + Ddsc50Index;
    float* ddsc7                    =  scalingFactors + Ddsc70Index;

    float damp                      = dampingFactorI*dampingFactorJ;      

    deltaR[0]                       = atomCoordinatesJ.x - atomCoordinatesI.x;
    deltaR[1]                       = atomCoordinatesJ.y - atomCoordinatesI.y;
    deltaR[2]                       = atomCoordinatesJ.z - atomCoordinatesI.z;

    float r2                        = DOT31( deltaR, deltaR );
    float r                         = sqrtf( r2 );
    float rr1                       = 1.0f/r;
    float rr2                       = rr1*rr1;
    float rr3                       = rr1*rr2;
    float rr5                       = 3.0f*rr3*rr2;
    float rr7                       = 5.0f*rr5*rr2;
    float rr9                       = 7.0f*rr7*rr2;
    float rr11                      = 9.0f*rr9*rr2;
    
    memcpy( &vectorFields[_r*3+0], deltaR, 3*sizeof(float) ); 

    //-------------------------------------------

    if( damp != 0.0f && r < scalingDistanceCutoff ){
   
        float distanceIJ, r2I;
        distanceIJ                    = r;
        r2I                           = rr2;
        
        float ratio                   = distanceIJ/damp;
        float pGamma                  = tholeJ > tholeI ? tholeI : tholeJ;

        damp                          = ratio*ratio*ratio*pGamma;
        float dampExp                 = expf( -damp );
        float damp1                   = damp + one;
        float damp2                   = damp*damp;
        float damp3                   = damp2*damp;

        scalingFactors[Scale3Index]   = one - dampExp;
        scalingFactors[Scale5Index]   = one - damp1*dampExp;
        scalingFactors[Scale7Index]   = one - ( damp1 + 0.6f*damp2)*dampExp;
        scalingFactors[Scale9Index]   = one - ( damp1 + ( 2.0f*damp2 + damp3 )*i35)*dampExp;

        float factor                  = 3.0f*damp*dampExp*r2I;
        float factor7                 = -0.2f + 0.6f*damp;
        
        for( int ii = 0; ii < 3; ii++ ){
            scalingFactors[Ddsc30Index + ii] = factor*deltaR[ii];
            scalingFactors[Ddsc50Index + ii] = scalingFactors[Ddsc30Index + ii]*damp;
            scalingFactors[Ddsc70Index + ii] = scalingFactors[Ddsc50Index + ii]*factor7;
        }

    }

    float scaleI[3];
    float dsc[3];
    float psc[3];
      
    for( int ii = 0; ii < 3; ii++ ){
        scaleI[ii] = scalingFactors[Scale3Index+ii]*scalingFactors[UScaleIndex];
        dsc[ii]    = scalingFactors[Scale3Index+ii]*scalingFactors[DScaleIndex];
        psc[ii]    = scalingFactors[Scale3Index+ii]*scalingFactors[PScaleIndex];
    }
                       
    float sc[11];
    float sci[9];
    float scip[9];

    amatrixProductVector3( &vectorFields1[_qJ*9],      &vectorFields[_r*3],      &vectorFields[_qJr*3]);
    amatrixProductVector3( &vectorFields1[_qI*9],      &vectorFields[_r*3],      &vectorFields[_qIr*3]);

    sc[2]     = DOT3_4(        vectorFields, _dI,  vectorFields, _dJ );
    sc[3]     = DOT3_4(        vectorFields, _dI,  vectorFields, _r  );
    sc[4]     = DOT3_4(        vectorFields, _dJ,  vectorFields, _r  );
    
    sc[5]     = DOT3_4(        vectorFields, _qIr, vectorFields, _r  );
    sc[6]     = DOT3_4(        vectorFields, _qJr, vectorFields, _r  );
    
    sc[7]     = DOT3_4(        vectorFields, _qIr, vectorFields, _dJ );
    sc[8]     = DOT3_4(        vectorFields, _qJr, vectorFields, _dI );
    
    sc[9]     = DOT3_4(        vectorFields, _qIr, vectorFields, _qJr );
    
    sc[10]    = MATRIXDOT31(   vectorFields1,_qI,  vectorFields1,_qJ );
    
    sci[1]    = DOT3_4(        vectorFields, _uI,  vectorFields, _dJ ) +
                DOT3_4(        vectorFields, _uJ,  vectorFields, _dI );
    
    sci[2]    = DOT3_4(        vectorFields, _uI,  vectorFields, _uJ );
    
    sci[3]    = DOT3_4(        vectorFields, _uI,  vectorFields, _r  );
    sci[4]    = DOT3_4(        vectorFields, _uJ,  vectorFields, _r  );
    
    sci[7]    = DOT3_4(        vectorFields, _qIr, vectorFields, _uJ );
    sci[8]    = DOT3_4(        vectorFields, _qJr, vectorFields, _uI );
    
    scip[1]   = DOT3_4(        vectorFields, _uIp, vectorFields, _dJ ) +
                DOT3_4(        vectorFields, _uJp, vectorFields, _dI );
    
    scip[2]   = DOT3_4(        vectorFields, _uI,  vectorFields, _uJp) +
                DOT3_4(        vectorFields, _uJ,  vectorFields, _uIp);
    
    scip[3]   = DOT3_4(        vectorFields, _uIp, vectorFields, _r );
    scip[4]   = DOT3_4(        vectorFields, _uJp, vectorFields, _r );
    
    scip[7]   = DOT3_4(        vectorFields, _qIr, vectorFields, _uJp );
    scip[8]   = DOT3_4(        vectorFields, _qJr, vectorFields, _uIp );

    float findmp[3];
    float scaleF         = 0.5f*scalingFactors[UScaleIndex];
    float inducedFactor3 = scip[2]*rr3*scaleF;
    float inducedFactor5 = (sci[3]*scip[4]+scip[3]*sci[4])*rr5*scaleF;
    findmp[0]            = inducedFactor3*ddsc3[0] - inducedFactor5*ddsc5[0];
    findmp[1]            = inducedFactor3*ddsc3[1] - inducedFactor5*ddsc5[1];
    findmp[2]            = inducedFactor3*ddsc3[2] - inducedFactor5*ddsc5[2];

    float gli[8];
    gli[1]               = chargeJ*sci[3] - chargeI*sci[4];
    gli[2]               = -sc[3]*sci[4] - sci[3]*sc[4];
    gli[3]               = sci[3]*sc[6] - sci[4]*sc[5];
    gli[6]               = sci[1];
    gli[7]               = 2.0f*(sci[7]-sci[8]);
    
    float glip[8];
    glip[1]              = chargeJ*scip[3] - chargeI*scip[4];
    glip[2]              = -sc[3]*scip[4] - scip[3]*sc[4];
    glip[3]              = scip[3]*sc[6] - scip[4]*sc[5];
    glip[6]              = scip[1];
    glip[7]              = 2.0f*(scip[7]-scip[8]);
    
    float fridmp[3];
    float factor3, factor5, factor7;
    
    if( scalingFactors[PScaleIndex] == 1.0f && scalingFactors[PScaleIndex] == 1.0f ){
        factor3 = rr3*( gli[1]  +  gli[6]  + glip[1]  + glip[6] );
        factor5 = rr5*( gli[2]  +  gli[7]  + glip[2]  + glip[7] );
        factor7 = rr7*( gli[3]  + glip[3] );
    } else {
        factor3 = rr3*(( gli[1]  +  gli[6])*scalingFactors[PScaleIndex] +
                       (glip[1]  + glip[6])*scalingFactors[DScaleIndex]);
   
       factor5 = rr5*(( gli[2]  +  gli[7])*scalingFactors[PScaleIndex] +
                      (glip[2]  + glip[7])*scalingFactors[DScaleIndex]);
   
       factor7 = rr7*( gli[3]*scalingFactors[PScaleIndex] + glip[3]*scalingFactors[DScaleIndex]);
    }
      
    fridmp[0] = 0.5f*(factor3*ddsc3[0] + factor5*ddsc5[0] + factor7*ddsc7[0]);
    fridmp[1] = 0.5f*(factor3*ddsc3[1] + factor5*ddsc5[1] + factor7*ddsc7[1]);
    fridmp[2] = 0.5f*(factor3*ddsc3[2] + factor5*ddsc5[2] + factor7*ddsc7[2]);
      
    float gl[9];
    
    gl[0] = chargeI*chargeJ;
    gl[1] = chargeJ*sc[3] - chargeI*sc[4];
    gl[2] = chargeI*sc[6] + chargeJ*sc[5] - sc[3]*sc[4];
    
    gl[3] = sc[3]*sc[6] - sc[4]*sc[5];
    gl[4] = sc[5]*sc[6];
    gl[6] = sc[2];
    gl[7] = 2.0f*(sc[7]-sc[8]);
    gl[8] = 2.0f*sc[10];
    gl[5] = -4.0f*sc[9];
    
    float gf[8];
    gf[1] = rr3*gl[0] + rr5*(gl[1]+gl[6]) + rr7*(gl[2]+gl[7]+gl[8]) + rr9*(gl[3]+gl[5]) + rr11*gl[4];
    gf[2] = -chargeJ*rr3 + sc[4]*rr5 - sc[6]*rr7;
    gf[3] =  chargeI*rr3 + sc[3]*rr5 + sc[5]*rr7;
    gf[4] = 2.0f*rr5;
    gf[5] = 2.0f*(-chargeJ*rr5+sc[4]*rr7-sc[6]*rr9);
    gf[6] = 2.0f*(-chargeI*rr5-sc[3]*rr7-sc[5]*rr9);
    gf[7] = 4.0f*rr7;

    // energy

    float conversionFactor   = (cAmoebaSim.electric/cAmoebaSim.dielec);
    float em                 = scalingFactors[MScaleIndex]*(rr1*gl[0] + rr3*(gl[1]+gl[6]) + rr5*(gl[2]+gl[7]+gl[8]) + rr7*(gl[3]+gl[5]) + rr9*gl[4]);
    float ei                 = 0.5f*(rr3*(gli[1]+gli[6])*psc[0] + rr5*(gli[2]+gli[7])*psc[1] + rr7*gli[3]*psc[2]);
    *energy                  = conversionFactor*(em+ei);
    
#ifdef AMOEBA_DEBUG
#if 0
if( 1 ){
    int debugIndex           = 0;
    debugArray[debugIndex].x = conversionFactor*em;
    debugArray[debugIndex].y = conversionFactor*ei;
    debugArray[debugIndex].z = rr1;
    debugArray[debugIndex].w = rr3;

    debugIndex++;
    debugArray[debugIndex].x = gl[0];
    debugArray[debugIndex].y = gl[1];
    debugArray[debugIndex].z = gl[6];
    debugArray[debugIndex].w = gl[2];

    debugIndex++;
    debugArray[debugIndex].x = gli[1];
    debugArray[debugIndex].y = gli[3];
    debugArray[debugIndex].z = gli[2];
    debugArray[debugIndex].w = gli[7];

    debugIndex++;
    debugArray[debugIndex].x = psc[0];
    debugArray[debugIndex].y = psc[1];
    debugArray[debugIndex].z = psc[2];
    debugArray[debugIndex].w = scalingFactors[MScaleIndex];

}
#endif
#endif

    float ftm2[3];
    float temp1[3],temp2[3],temp3[3];
    amatrixProductVector3( &vectorFields1[_qI*9],      &vectorFields[_dJ*3],     &vectorFields[ _qIdJ*3] );//MK
    amatrixProductVector3( &vectorFields1[_qJ*9],      &vectorFields[_dI*3],     &vectorFields[ _qJdI*3] );//MK

    amatrixProductVector3( &vectorFields1[_qI*9],      &vectorFields[_qJr*3],    &vectorFields[_qIqJr*3] );//MK
    amatrixProductVector3( &vectorFields1[_qJ*9],      &vectorFields[_qIr*3],    &vectorFields[_qJqIr*3] );//MK
    amatrixProductVector3( &vectorFields1[_qJ*9],      &vectorFields[_qIr*3],    temp1 );
    amatrixProductVector3( &vectorFields1[_qJ*9],      &vectorFields[_dI*3],     temp2 );

    for( int ii = 0; ii < 3; ii++ ){
        ftm2[ii] = gf[1]*vectorFields[_r*3+ii]                             +
                   gf[2]*vectorFields[_dI*3+ii]     + gf[3]*vectorFields[_dJ*3+ii]  +
                   gf[4]*(temp2[ii]  - vectorFields[_qIdJ*3+ii])                    +
                   gf[5]*vectorFields[_qIr*3+ii]    + gf[6]*vectorFields[_qJr*3+ii] +
                   gf[7]*(vectorFields[_qIqJr*3+ii] + temp1[ii]);
    
    }

    // get the induced force;

    // intermediate variables for the induced-permanent terms;
    
    float gfi[7];
    gfi[1] = rr5*0.5f*((gli[1]+gli[6])*psc[0] + (glip[1]+glip[6])*dsc[0] + scip[2]*scaleI[0]) + rr7*((gli[7]+gli[2])*psc[1] + (glip[7]+glip[2])*dsc[1] -
                                                       (sci[3]*scip[4]+scip[3]*sci[4])*scaleI[1])*0.5f + rr9*(gli[3]*psc[2]+glip[3]*dsc[2])*0.5f;
    gfi[2] = -rr3*chargeJ + rr5*sc[4] - rr7*sc[6];
    gfi[3] = rr3*chargeI  + rr5*sc[3] + rr7*sc[5];
    gfi[4] = 2.0f*rr5;
    gfi[5] = rr7* (sci[4]*psc[2] + scip[4]*dsc[2]);
    gfi[6] = -rr7*(sci[3]*psc[2] + scip[3]*dsc[2]);


    float ftm2i[3];
    float temp4[3];
    float temp5[3];
    float temp6[3];
    float temp7[3];
    float temp8[3];
    float temp9[3];
    float temp10[3];
    float temp11[3];
    float temp12[3];
    float temp13[3];
    float temp14[3];
    float temp15[3];

    amatrixProductVector3(&vectorFields1[_qJ*9],      &vectorFields[_uIp*3],    temp4);

    amatrixProductVector3(&vectorFields1[_qI*9],      &vectorFields[_uJp*3],    &vectorFields[ _qIuJp*3]);//MK
    amatrixProductVector3(&vectorFields1[_qJ*9],      &vectorFields[_uIp*3],    &vectorFields[ _qJuIp*3]);//MK
    amatrixProductVector3(&vectorFields1[_qJ*9],      &vectorFields[_uI*3],     &vectorFields[ _qJuI*3]);//MK

    amatrixProductVector3(&vectorFields1[_qJ*9],      &vectorFields[ _uI*3],    temp5);
    amatrixProductVector3(&vectorFields1[_qI*9],      &vectorFields[_uJ*3],     &vectorFields[ _qIuJ*3]);//MK

    float temp1_0,temp2_0,temp3_0;
    for( int ii = 0; ii < 3; ii++ ){
        temp1_0 = gfi[1]*vectorFields[_r*3+ii] +
                  0.5f*(-rr3*chargeJ*(vectorFields[_uI*3+ii]*psc[0] + vectorFields[_uIp*3+ii]*dsc[0]) +
                  rr5*sc[4]*(vectorFields[_uI*3+ii]*psc[1] + vectorFields[_uIp*3+ii]*dsc[1]) -
                  rr7*sc[6]*(vectorFields[_uI*3+ii]*psc[2] + vectorFields[_uIp*3+ii]*dsc[2])) ;

        temp2_0 = (rr3*chargeI*(vectorFields[_uJ*3+ii]*psc[0]+vectorFields[_uJp*3+ii]*dsc[0]) +
                   rr5*sc[3]*(vectorFields[_uJ*3+ii]*psc[1] +vectorFields[_uJp*3+ii]*dsc[1]) +
                   rr7*sc[5]*(vectorFields[_uJ*3+ii]*psc[2] +vectorFields[_uJp*3+ii]*dsc[2]))*0.5f +
                   rr5*scaleI[1]*(sci[4]*vectorFields[_uIp*3+ii]+scip[4]*vectorFields[_uI*3+ii] +
                   sci[3]*vectorFields[_uJp*3+ii]+scip[3]*vectorFields[_uJ*3+ii])*0.5f ;

        temp3_0 = 0.5f*(sci[4]*psc[1]+scip[4]*dsc[1])*rr5*vectorFields[_dI*3+ii] +
                  0.5f*(sci[3]*psc[1]+scip[3]*dsc[1])*rr5*vectorFields[_dJ*3+ii] +
                  0.5f*gfi[4]*((temp5[ii]-vectorFields[_qIuJ*3+ii])*psc[1] +
                  (temp4[ii]-vectorFields[_qIuJp*3+ii])*dsc[1]) + gfi[5]*vectorFields[_qIr*3+ii] + gfi[6]*vectorFields[_qJr*3+ii];
        ftm2i[ii] = temp1_0 + temp2_0 + temp3_0;
    }

    // handle of scaling for partially excluded interactions;
    // correction to convert mutual to direct polarization force;
    
    ftm2i[0] -= (fridmp[0] + findmp[0]);
    ftm2i[1] -= (fridmp[1] + findmp[1]);
    ftm2i[2] -= (fridmp[2] + findmp[2]);
    
    // now perform the torque calculation;
    // intermediate terms for torque between multipoles i and j;
    
    float gti[7];
    gti[2] = 0.5f*(sci[4]*psc[1]+scip[4]*dsc[1])*rr5;
    gti[3] = 0.5f*(sci[3]*psc[1]+scip[3]*dsc[1])*rr5;
    gti[4] = gfi[4];
    gti[5] = gfi[5];
    gti[6] = gfi[6];

    // get the permanent (ttm2, ttm3) and induced interaction torques (ttm2i, ttm3i)
    
    float ttm2[3];
    float ttm2i[3];
    float ttm3[3];
    float ttm3i[3];
    acrossProductVector3(&vectorFields[_dI*3],      &vectorFields[_dJ*3],      temp1);
    acrossProductVector3(&vectorFields[_dI*3],      &vectorFields[_uJ*3],      temp2);
    acrossProductVector3(&vectorFields[_dI*3],      &vectorFields[_uJp*3],     temp3);
    acrossProductVector3(&vectorFields[_dI*3],      &vectorFields[_r*3],       temp4);
    acrossProductVector3(&vectorFields[_r*3],       &vectorFields[_qIuJp*3],   temp5);
    acrossProductVector3(&vectorFields[_r*3],       &vectorFields[_qIr*3],     temp6);
    acrossProductVector3(&vectorFields[_r*3],       &vectorFields[_qIuJ*3],    temp7);
    acrossProductVector3(&vectorFields[_uJ*3],      &vectorFields[_qIr*3],     temp8);
    acrossProductVector3(&vectorFields[_uJp*3],     &vectorFields[_qIr*3],     temp9);
    acrossProductVector3(&vectorFields[_dI*3],      &vectorFields[_qJr*3],     temp10);
    acrossProductVector3(&vectorFields[_dJ*3],      &vectorFields[_qIr*3],     temp11);
    acrossProductVector3(&vectorFields[_r*3],       &vectorFields[_qIqJr*3],   temp12);
    acrossProductVector3(&vectorFields[_r*3],       &vectorFields[_qIdJ*3],    temp13);

    amatrixCrossProductMatrix3(&vectorFields1[_qI*9],      &vectorFields1[_qJ*9],      temp14);
    acrossProductVector3(&vectorFields[_qJr*3],     &vectorFields[_qIr*3],     temp15);

    // unroll?

    for( int ii = 0; ii < 3; ii++ ){
       ttm2[ii]  = -rr3*temp1[ii] + gf[2]*temp4[ii]-gf[5]*temp6[ii] +
                   gf[4]*(temp10[ii] + temp11[ii] + temp13[ii]-2.0f*temp14[ii]) -
                   gf[7]*(temp12[ii] + temp15[ii]);
    
       ttm2i[ii] = -rr3*(temp2[ii]*psc[0]+temp3[ii]*dsc[0])*0.5f +
                    gti[2]*temp4[ii] + gti[4]*((temp8[ii]+ temp7[ii])*psc[1] +
                    (temp9[ii] + temp5[ii])*dsc[1])*0.5f - gti[5]*temp6[ii];
    
    }

    acrossProductVector3(&vectorFields[3*_dJ],      &vectorFields[3*_r],       temp2  );
    acrossProductVector3(&vectorFields[3*_r],       &vectorFields[3*_qJr],     temp3  );
    acrossProductVector3(&vectorFields[3*_dI],      &vectorFields[3*_qJr],     temp4  );
    acrossProductVector3(&vectorFields[3*_dJ],      &vectorFields[3*_qIr],     temp5  );
    acrossProductVector3(&vectorFields[3*_r],       &vectorFields[3*_qJdI],    temp6  );
    acrossProductVector3(&vectorFields[3*_r],       &vectorFields[3*_qJqIr],   temp7  );
    acrossProductVector3(&vectorFields[3*_qJr],     &vectorFields[3*_qIr],     temp8  ); // _qJrxqIr
    acrossProductVector3(&vectorFields[3*_dJ],      &vectorFields[3*_uI],      temp9  ); // _dJxuI
    acrossProductVector3(&vectorFields[3*_dJ],      &vectorFields[3*_uIp],     temp10 ); // _dJxuIp

    acrossProductVector3(&vectorFields[3*_uIp],     &vectorFields[3*_qJr],     temp11 ); // _uIxqJrp
    acrossProductVector3(&vectorFields[3*_uI],      &vectorFields[3*_qJr],     temp12 ); // _uIxqJr
    acrossProductVector3(&vectorFields[3*_r],       &vectorFields[3*_qJuIp],   temp13 ); // _rxqJuIp
    acrossProductVector3(&vectorFields[3*_r],       &vectorFields[3*_qJuI],    temp15 ); // _rxqJuI

    // unroll?

    for( int ii = 0; ii < 3; ii++ ){
    
       ttm3[ii] = rr3*temp1[ii] +
                  gf[3]*temp2[ii] - gf[6]*temp3[ii] - gf[4]*(temp4[ii] + temp5[ii] + temp6[ii] - 2.0f*temp14[ii]) - gf[7]*(temp7[ii] - temp8[ii]);

    
       ttm3i[ii] = -rr3*(temp9[ii]*psc[0]+ temp10[ii]*dsc[0])*0.5f +
                    gti[3]*temp2[ii] - 
                    gti[4]*((temp12[ii] + temp15[ii])*psc[1] +
                    (temp11[ii] + temp13[ii])*dsc[1])*0.5f - gti[6]*temp3[ii];
    }

    if( scalingFactors[MScaleIndex] < 1.0f ){
    
        ftm2[0] *= scalingFactors[MScaleIndex];
        ftm2[1] *= scalingFactors[MScaleIndex];
        ftm2[2] *= scalingFactors[MScaleIndex];
        
        ttm2[0] *= scalingFactors[MScaleIndex];
        ttm2[1] *= scalingFactors[MScaleIndex];
        ttm2[2] *= scalingFactors[MScaleIndex];
        
        ttm3[0] *= scalingFactors[MScaleIndex];
        ttm3[1] *= scalingFactors[MScaleIndex];
        ttm3[2] *= scalingFactors[MScaleIndex];
    
    }


#ifdef AMOEBA_DEBUG
#if 0
if( 0 ){
int debugIndex               = 0;
    debugArray[debugIndex].x = conversionFactor*ftm2[0];
    debugArray[debugIndex].y = conversionFactor*ftm2i[0];
    debugArray[debugIndex].z = conversionFactor*ttm3[0];
    debugArray[debugIndex].w = conversionFactor*ttm3i[0];

    debugIndex++;
    debugArray[debugIndex].x = temp1[0];
    debugArray[debugIndex].y = temp1[1];
    debugArray[debugIndex].z = temp1[2];
    debugArray[debugIndex].w = 1.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp2[0];
    debugArray[debugIndex].y = temp2[1];
    debugArray[debugIndex].z = temp2[2];
    debugArray[debugIndex].w = 2.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp3[0];
    debugArray[debugIndex].y = temp3[1];
    debugArray[debugIndex].z = temp3[2];
    debugArray[debugIndex].w = 3.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp4[0];
    debugArray[debugIndex].y = temp4[1];
    debugArray[debugIndex].z = temp4[2];
    debugArray[debugIndex].w = 4.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp5[0];
    debugArray[debugIndex].y = temp5[1];
    debugArray[debugIndex].z = temp5[2];
    debugArray[debugIndex].w = 5.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp6[0];
    debugArray[debugIndex].y = temp6[1];
    debugArray[debugIndex].z = temp6[2];
    debugArray[debugIndex].w = 6.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp14[0];
    debugArray[debugIndex].y = temp14[1];
    debugArray[debugIndex].z = temp14[2];
    debugArray[debugIndex].w = 14.0f;

    debugIndex++;
    debugArray[debugIndex].x = temp7[0];
    debugArray[debugIndex].y = temp7[1];
    debugArray[debugIndex].z = temp7[2];
    debugArray[debugIndex].w = 7.0f;


    debugIndex++;
    debugArray[debugIndex].x = temp8[0];
    debugArray[debugIndex].y = temp8[1];
    debugArray[debugIndex].z = temp8[2];
    debugArray[debugIndex].w = 8.0f;

    debugIndex++;
    debugArray[debugIndex].x = rr3;
    debugArray[debugIndex].y = gf[3];
    debugArray[debugIndex].z = gf[6];
    debugArray[debugIndex].w = 20.0f;

    debugIndex++;
    debugArray[debugIndex].x = gf[4];
    debugArray[debugIndex].y = gf[7];
    debugArray[debugIndex].z = 0.0f;
    debugArray[debugIndex].w = 21.0f;

    debugIndex++;
    debugArray[debugIndex].x = vectorFields[3*_dJ];
    debugArray[debugIndex].y = vectorFields[3*_dJ+1];
    debugArray[debugIndex].z = vectorFields[3*_dJ+2];
    debugArray[debugIndex].w = 22.0f;

    debugIndex++;
    debugArray[debugIndex].x = vectorFields[3*_r];
    debugArray[debugIndex].y = vectorFields[3*_r+1];
    debugArray[debugIndex].z = vectorFields[3*_r+2];
    debugArray[debugIndex].w = 23.0f;

}
#endif
#endif

    outputForce[0]        = -conversionFactor*(ftm2[0] + ftm2i[0]);
    outputForce[1]        = -conversionFactor*(ftm2[1] + ftm2i[1]);
    outputForce[2]        = -conversionFactor*(ftm2[2] + ftm2i[2]);
    
    outputTorque[0][0]    = conversionFactor*(ttm2[0] + ttm2i[0]); 
    outputTorque[0][1]    = conversionFactor*(ttm2[1] + ttm2i[1]); 
    outputTorque[0][2]    = conversionFactor*(ttm2[2] + ttm2i[2]); 

    outputTorque[1][0]    = conversionFactor*(ttm3[0] + ttm3i[0]); 
    outputTorque[1][1]    = conversionFactor*(ttm3[1] + ttm3i[1]); 
    outputTorque[1][2]    = conversionFactor*(ttm3[2] + ttm3i[2]); 

    return;

}

struct ElectrostaticParticle {

    // coordinates charge

    float x;
    float y;
    float z;
    float q;

    // lab frame dipole

    float labFrameDipole_X; 
    float labFrameDipole_Y; 
    float labFrameDipole_Z; 

    // lab frame quadrupole

    float labFrameQuadrupole_XX;
    float labFrameQuadrupole_XY;
    float labFrameQuadrupole_XZ;
    float labFrameQuadrupole_YY;
    float labFrameQuadrupole_YZ;
    float labFrameQuadrupole_ZZ;

    // induced dipole

    float inducedDipole_X; 
    float inducedDipole_Y;
    float inducedDipole_Z;

    // polar induced dipole

    float inducedDipoleP_X; 
    float inducedDipoleP_Y;
    float inducedDipoleP_Z;

    // scaling factors

    float thole;
    float damp;

    float force_X;
    float force_Y;
    float force_Z;

    float torque_X;
    float torque_Y;
    float torque_Z;

};

__device__ void loadElectrostaticShared( struct ElectrostaticParticle* sA, unsigned int atomI,
                                         float4* atomCoord, float* labFrameDipoleJ, float* labQuadrupole,
                                         float* inducedDipole, float* inducedDipolePolar, float2* dampingFactorAndThole )
{
    // coordinates & charge

    sA->x                        = atomCoord[atomI].x;
    sA->y                        = atomCoord[atomI].y;
    sA->z                        = atomCoord[atomI].z;
    sA->q                        = atomCoord[atomI].w;

    // lab dipole

    sA->labFrameDipole_X         = labFrameDipoleJ[atomI*3];
    sA->labFrameDipole_Y         = labFrameDipoleJ[atomI*3+1];
    sA->labFrameDipole_Z         = labFrameDipoleJ[atomI*3+2];

    // lab quadrupole

    sA->labFrameQuadrupole_XX    = labQuadrupole[atomI*9];
    sA->labFrameQuadrupole_XY    = labQuadrupole[atomI*9+1];
    sA->labFrameQuadrupole_XZ    = labQuadrupole[atomI*9+2];
    sA->labFrameQuadrupole_YY    = labQuadrupole[atomI*9+4];
    sA->labFrameQuadrupole_YZ    = labQuadrupole[atomI*9+5];
    sA->labFrameQuadrupole_ZZ    = labQuadrupole[atomI*9+8];

    // induced dipole

    sA->inducedDipole_X          = inducedDipole[atomI*3];
    sA->inducedDipole_Y          = inducedDipole[atomI*3+1];
    sA->inducedDipole_Z          = inducedDipole[atomI*3+2];

    // induced dipole polar

    sA->inducedDipoleP_X         = inducedDipolePolar[atomI*3];
    sA->inducedDipoleP_Y         = inducedDipolePolar[atomI*3+1];
    sA->inducedDipoleP_Z         = inducedDipolePolar[atomI*3+2];

    sA->damp                     = dampingFactorAndThole[atomI].x;
    sA->thole                    = dampingFactorAndThole[atomI].y;

}

// load struct and arrays w/ shared data in sA

__device__ void loadElectrostaticData( struct ElectrostaticParticle* sA, 
                                       float4* jCoord, float* jDipole, float* jQuadrupole,
                                       float* jInducedDipole, float* jInducedDipolePolar )
{

    // load coords, charge, ...

    jCoord->x               = sA->x;
    jCoord->y               = sA->y;
    jCoord->z               = sA->z;
    jCoord->w               = sA->q;
 
    jDipole[0]              = sA->labFrameDipole_X;
    jDipole[1]              = sA->labFrameDipole_Y;
    jDipole[2]              = sA->labFrameDipole_Z;
 
    jQuadrupole[0]          = sA->labFrameQuadrupole_XX;
    jQuadrupole[1]          = sA->labFrameQuadrupole_XY;
    jQuadrupole[2]          = sA->labFrameQuadrupole_XZ;

    jQuadrupole[3]          = sA->labFrameQuadrupole_XY;
    jQuadrupole[4]          = sA->labFrameQuadrupole_YY;
    jQuadrupole[5]          = sA->labFrameQuadrupole_YZ;

    jQuadrupole[6]          = sA->labFrameQuadrupole_XZ;
    jQuadrupole[7]          = sA->labFrameQuadrupole_YZ;
    jQuadrupole[8]          = sA->labFrameQuadrupole_ZZ;
 
    jInducedDipole[0]       = sA->inducedDipole_X;
    jInducedDipole[1]       = sA->inducedDipole_Y;
    jInducedDipole[2]       = sA->inducedDipole_Z;
 
    jInducedDipolePolar[0]  = sA->inducedDipoleP_X;
    jInducedDipolePolar[1]  = sA->inducedDipoleP_Y;
    jInducedDipolePolar[2]  = sA->inducedDipoleP_Z;
 
}

// Include versions of the kernels for N^2 calculations.

#undef USE_OUTPUT_BUFFER_PER_WARP
#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaElectrostatic.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaElectrostatic.h"

// reduce psWorkArray_3_1 -> force
// reduce psWorkArray_3_2 -> torque

static void kReduceForceTorque(amoebaGpuContext amoebaGpu )
{
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_1->_pDevStream[0], amoebaGpu->psForce->_pDevStream[0] );
    LAUNCHERROR("kReduceElectrostaticForce");
    kReduceFields_kernel<<<amoebaGpu->nonbondBlocks, amoebaGpu->fieldReduceThreadsPerBlock>>>(
                               amoebaGpu->paddedNumberOfAtoms*3, amoebaGpu->outputBuffers,
                               amoebaGpu->psWorkArray_3_2->_pDevStream[0], amoebaGpu->psTorque->_pDevStream[0] );
    LAUNCHERROR("kReduceElectrostaticTorque");
}

#ifdef AMOEBA_DEBUG
static void printElectrostaticBuffer( amoebaGpuContext amoebaGpu, unsigned int bufferIndex )
{
    (void) fprintf( amoebaGpu->log, "Electrostatic Buffer %u\n", bufferIndex );
    unsigned int start = bufferIndex*3*amoebaGpu->paddedNumberOfAtoms;
    unsigned int stop  = (bufferIndex+1)*3*amoebaGpu->paddedNumberOfAtoms;
    for( unsigned int ii = start; ii < stop; ii += 3 ){
        unsigned int ii3Index      = ii/3;
        unsigned int bufferIndex   = ii3Index/(amoebaGpu->paddedNumberOfAtoms);
        unsigned int particleIndex = ii3Index - bufferIndex*(amoebaGpu->paddedNumberOfAtoms);
        (void) fprintf( amoebaGpu->log, "   %6u %3u %6u [%14.6e %14.6e %14.6e] [%14.6e %14.6e %14.6e]\n", 
                            ii/3,  bufferIndex, particleIndex,
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii],
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii+1],
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii+2],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii+1],
                            amoebaGpu->psWorkArray_3_2->_pSysStream[0][ii+2] );
    } 

/*
    start = 0;
    stop  = -146016;
    float maxV = -1.0e+99;
    for( unsigned int ii = start; ii < stop; ii += 3 ){
        if(  amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii] > maxV ){ 
            unsigned int ii3Index      = ii/3;
            unsigned int bufferIndex   = ii3Index/(amoebaGpu->paddedNumberOfAtoms);
            unsigned int particleIndex = ii3Index - bufferIndex*(amoebaGpu->paddedNumberOfAtoms);
            (void) fprintf( amoebaGpu->log, "MaxQ %6u %3u %6u %14.6e\n", 
                            ii/3,  bufferIndex, particleIndex,
                            amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii] );
            maxV = amoebaGpu->psWorkArray_3_1->_pSysStream[0][ii];
        } 
    } 
*/
}

static void printElectrostaticAtomBuffers( amoebaGpuContext amoebaGpu, unsigned int targetAtom )
{
    (void) fprintf( amoebaGpu->log, "Electrostatic atom %u\n", targetAtom );
    for( unsigned int ii = 0; ii < amoebaGpu->outputBuffers; ii++ ){
        unsigned int particleIndex = 3*(targetAtom + ii*amoebaGpu->paddedNumberOfAtoms);
        (void) fprintf( amoebaGpu->log, " %2u %6u [%14.6e %14.6e %14.6e] [%14.6e %14.6e %14.6e]\n", 
                        ii, particleIndex,
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex],
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex+1],
                        amoebaGpu->psWorkArray_3_1->_pSysStream[0][particleIndex+2],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex+1],
                        amoebaGpu->psWorkArray_3_2->_pSysStream[0][particleIndex+2] );
    } 
}
#endif

/**---------------------------------------------------------------------------------------

   Compute Amoeba electrostatic force & torque

   @param amoebaGpu        amoebaGpu context
   @param gpu              OpenMM gpu Cuda context

   --------------------------------------------------------------------------------------- */

void cudaComputeAmoebaElectrostatic( amoebaGpuContext amoebaGpu )
{
  
   // ---------------------------------------------------------------------------------------

    static unsigned int threadsPerBlock = 0;

#ifdef AMOEBA_DEBUG
    static const char* methodName = "cudaComputeAmoebaElectrostatic";
    static int timestep = 0;
    std::vector<int> fileId;
    timestep++;
    fileId.resize( 2 );
    fileId[0] = timestep;
    fileId[1] = 1;
#endif

    // ---------------------------------------------------------------------------------------

    gpuContext gpu = amoebaGpu->gpuContext;

    // apparently debug array can take up nontrivial no. registers

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){
        (void) fprintf( amoebaGpu->log, "%s %d maxCovalentDegreeSz=%d"
                        " gamma=%.3e scalingDistanceCutoff=%.3f ZZZ\n",
                        methodName, gpu->natoms,
                        amoebaGpu->maxCovalentDegreeSz, amoebaGpu->pGamma,
                        amoebaGpu->scalingDistanceCutoff );
    }   
   int paddedNumberOfAtoms                    = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
    CUDAStream<float4>* debugArray            = new CUDAStream<float4>(paddedNumberOfAtoms*paddedNumberOfAtoms, 1, "DebugArray");
    memset( debugArray->_pSysStream[0],      0, sizeof( float )*4*paddedNumberOfAtoms*paddedNumberOfAtoms);
    debugArray->Upload();
    unsigned int targetAtom                   = 0;
#endif

    // on first pass, set threads/block

    if( threadsPerBlock == 0 ){
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 256;
        else if (gpu->sm_version >= SM_12)
            maxThreads = 128;
        else
            maxThreads = 64;
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(ElectrostaticParticle)), maxThreads);
    }

    kClearFields_3( amoebaGpu, 2 );

    if (gpu->bOutputBufferPerWarp){

        (void) fprintf( amoebaGpu->log, "kCalculateAmoebaCudaElectrostaticN2Forces warp:  numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%u shrd=%u Ebuf=%u ixnCt=%u workUnits=%u\n",
                        amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                        sizeof(ElectrostaticParticle), sizeof(ElectrostaticParticle)*threadsPerBlock, amoebaGpu->energyOutputBuffers, (*gpu->psInteractionCount)[0], gpu->sim.workUnits );
        (void) fflush( amoebaGpu->log );
        kCalculateAmoebaCudaElectrostaticN2ByWarpForces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(ElectrostaticParticle)*threadsPerBlock>>>(
                                                                           amoebaGpu->psWorkUnit->_pDevStream[0],
                                                                           gpu->psPosq4->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameDipole->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameQuadrupole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipolePolar->_pDevStream[0],
                                                                           amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                           debugArray->_pDevStream[0], targetAtom );
#else
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif

    } else {

#ifdef AMOEBA_DEBUG
        (void) fprintf( amoebaGpu->log, "kCalculateAmoebaCudaElectrostaticN2Forces no warp:  numBlocks=%u numThreads=%u bufferPerWarp=%u atm=%u shrd=%u Ebuf=%u ixnCt=%u workUnits=%u\n",
                        amoebaGpu->nonbondBlocks, threadsPerBlock, amoebaGpu->bOutputBufferPerWarp,
                        sizeof(ElectrostaticParticle), sizeof(ElectrostaticParticle)*threadsPerBlock, amoebaGpu->energyOutputBuffers, (*gpu->psInteractionCount)[0], gpu->sim.workUnits );
        (void) fflush( amoebaGpu->log );
#endif

        kCalculateAmoebaCudaElectrostaticN2Forces_kernel<<<amoebaGpu->nonbondBlocks, threadsPerBlock, sizeof(ElectrostaticParticle)*threadsPerBlock>>>(
                                                                           amoebaGpu->psWorkUnit->_pDevStream[0],
                                                                           gpu->psPosq4->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameDipole->_pDevStream[0],
                                                                           amoebaGpu->psLabFrameQuadrupole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipole->_pDevStream[0],
                                                                           amoebaGpu->psInducedDipolePolar->_pDevStream[0],
                                                                           amoebaGpu->psWorkArray_3_1->_pDevStream[0],
#ifdef AMOEBA_DEBUG
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0],
                                                                           debugArray->_pDevStream[0], targetAtom );
#else
                                                                           amoebaGpu->psWorkArray_3_2->_pDevStream[0] );
#endif
    }
    LAUNCHERROR("kCalculateAmoebaCudaElectrostaticN2Forces");

#ifdef AMOEBA_DEBUG
    if( 0 && amoebaGpu->log ){

        amoebaGpu->psWorkArray_3_1->Download();
        amoebaGpu->psWorkArray_3_2->Download();

        printElectrostaticAtomBuffers( amoebaGpu, (targetAtom + 0) );
        //printElectrostaticAtomBuffers( amoebaGpu, (targetAtom + 1231) );
        printElectrostaticBuffer( amoebaGpu, 0 );
        //printElectrostaticBuffer( amoebaGpu, 38 );
    }
#endif

    kReduceForceTorque( amoebaGpu );

#ifdef AMOEBA_DEBUG
    if( amoebaGpu->log ){

        amoebaGpu->psForce->Download();
        amoebaGpu->psTorque->Download();
        debugArray->Download();

        (void) fprintf( amoebaGpu->log, "Finished Electrostatic kernel execution\n" ); (void) fflush( amoebaGpu->log );

        int maxPrint        = 1400;
        for( int ii = 0; ii < gpu->natoms; ii++ ){
           (void) fprintf( amoebaGpu->log, "%5d ", ii); 

            int indexOffset     = ii*3;
    
           // force

           (void) fprintf( amoebaGpu->log,"ElectrostaticF [%16.9e %16.9e %16.9e] ",
                           amoebaGpu->psForce->_pSysStream[0][indexOffset],
                           amoebaGpu->psForce->_pSysStream[0][indexOffset+1],
                           amoebaGpu->psForce->_pSysStream[0][indexOffset+2] );
    
           // torque

           (void) fprintf( amoebaGpu->log,"ElectrostaticT [%16.9e %16.9e %16.9e] ",
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset],
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset+1],
                           amoebaGpu->psTorque->_pSysStream[0][indexOffset+2] );

           // coords

#if 0
            (void) fprintf( amoebaGpu->log,"x[%16.9e %16.9e %16.9e] ",
                            gpu->psPosq4->_pSysStream[0][ii].x,
                            gpu->psPosq4->_pSysStream[0][ii].y,
                            gpu->psPosq4->_pSysStream[0][ii].z);


           for( int jj = 0; jj < gpu->natoms && jj < 5; jj++ ){
               int debugIndex = jj*gpu->natoms + ii;
               float xx       =  gpu->psPosq4->_pSysStream[0][jj].x -  gpu->psPosq4->_pSysStream[0][ii].x;
               float yy       =  gpu->psPosq4->_pSysStream[0][jj].y -  gpu->psPosq4->_pSysStream[0][ii].y;
               float zz       =  gpu->psPosq4->_pSysStream[0][jj].z -  gpu->psPosq4->_pSysStream[0][ii].z;
               (void) fprintf( amoebaGpu->log,"\n%4d %4d delta [%16.9e %16.9e %16.9e] [%16.9e %16.9e %16.9e] ",
                               ii, jj, xx, yy, zz,
                               debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y, debugArray->_pSysStream[0][debugIndex].z );

           }
#endif
           (void) fprintf( amoebaGpu->log,"\n" );
           if( ii == maxPrint && (gpu->natoms - maxPrint) > ii ){
                ii = gpu->natoms - maxPrint;
           }
        }
        if( 1 ){
            (void) fprintf( amoebaGpu->log,"DebugElec\n" );
            int paddedNumberOfAtoms = amoebaGpu->gpuContext->sim.paddedNumberOfAtoms;
            for( int jj = 0; jj < gpu->natoms; jj++ ){
                int debugIndex = jj;
                for( int kk = 0; kk < 5; kk++ ){
                    (void) fprintf( amoebaGpu->log,"%5d %5d [%16.9e %16.9e %16.9e %16.9e] E11\n", targetAtom, jj,
                                    debugArray->_pSysStream[0][debugIndex].x, debugArray->_pSysStream[0][debugIndex].y,
                                    debugArray->_pSysStream[0][debugIndex].z, debugArray->_pSysStream[0][debugIndex].w );
                    debugIndex += paddedNumberOfAtoms;
                }
                (void) fprintf( amoebaGpu->log,"\n" );
            }
        }
        (void) fflush( amoebaGpu->log );

        if( 0 ){
            (void) fprintf( amoebaGpu->log, "%s Tiled F & T\n", methodName ); fflush( amoebaGpu->log );
            int maxPrint = 12;
            for( int ii = 0; ii < gpu->natoms; ii++ ){
    
                // print cpu & gpu reductions
    
                int offset  = 3*ii;
    
                (void) fprintf( amoebaGpu->log,"%6d F[%16.7e %16.7e %16.7e] T[%16.7e %16.7e %16.7e]\n", ii,
                                amoebaGpu->psForce->_pSysStream[0][offset],
                                amoebaGpu->psForce->_pSysStream[0][offset+1],
                                amoebaGpu->psForce->_pSysStream[0][offset+2],
                                amoebaGpu->psTorque->_pSysStream[0][offset],
                                amoebaGpu->psTorque->_pSysStream[0][offset+1],
                                amoebaGpu->psTorque->_pSysStream[0][offset+2] );
                if( (ii == maxPrint) && (ii < (gpu->natoms - maxPrint)) )ii = gpu->natoms - maxPrint; 
            }   
        }   

        if( 1 ){
            std::vector<int> fileId;
            //fileId.push_back( 0 );
            VectorOfDoubleVectors outputVector;
            cudaLoadCudaFloat4Array( gpu->natoms, 3, gpu->psPosq4,            outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psForce,      outputVector );
            cudaLoadCudaFloatArray( gpu->natoms,  3, amoebaGpu->psTorque,     outputVector);
            cudaWriteVectorOfDoubleVectorsToFile( "CudaForceTorque", fileId, outputVector );
         }

    }   
    delete debugArray;

#endif

   // ---------------------------------------------------------------------------------------
}

