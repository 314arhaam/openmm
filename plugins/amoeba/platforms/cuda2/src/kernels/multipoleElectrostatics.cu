#include "hip/hip_runtime.h"
#define TILE_SIZE 32
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real4 posq;
    real3 force, dipole, inducedDipole, inducedDipolePolar;
    real quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ;
    float thole, damp;
} AtomData;

__device__ void computeOneInteractionF1(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real& energy, real3& outputForce);
__device__ void computeOneInteractionT1(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real3& outputForce);
__device__ void computeOneInteractionT3(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real3& outputForce);

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real* __restrict__ inducedDipole, const real* __restrict__ inducedDipolePolar, const float2* __restrict__ dampingAndThole) {
    data.posq = posq[atom];
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    data.inducedDipolePolar.x = inducedDipolePolar[atom*3];
    data.inducedDipolePolar.y = inducedDipolePolar[atom*3+1];
    data.inducedDipolePolar.z = inducedDipolePolar[atom*3+2];
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

__device__ real computeDScaleFactor(unsigned int polarizationGroup) {
    return (polarizationGroup & 1 ? 0 : 1);
}

__device__ float computeMScaleFactor(uint2 covalent) {
    bool x = (covalent.x & 1);
    bool y = (covalent.y & 1);
    return (x ? (y ? 0.0f : 0.4f) : (y ? 0.8f : 1.0f));
}

__device__ float computePScaleFactor(uint2 covalent, unsigned int polarizationGroup) {
    bool x = (covalent.x & 1);
    bool y = (covalent.y & 1);
    bool p = (polarizationGroup & 1);
    return (x && y ? 0.0f : (x && p ? 0.5f : 1.0f));
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeElectrostatics(
        unsigned long long* __restrict__ forceBuffers, real* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const unsigned int* __restrict__ exclusionIndices, const unsigned int* __restrict__ exclusionRowIndices,
        const uint2* __restrict__ covalentFlags, const unsigned int* __restrict__ polarizationGroupFlags, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const ushort2* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, unsigned int maxTiles, const unsigned int* __restrict__ interactionFlags,
#endif
        const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const real* __restrict__ inducedDipole,
        const real* __restrict__ inducedDipolePolar, const float2* __restrict__ dampingAndThole) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    unsigned int pos = (numTiles > maxTiles ? startTileIndex+warp*numTileIndices/totalWarps : warp*numTiles/totalWarps);
    unsigned int end = (numTiles > maxTiles ? startTileIndex+(warp+1)*numTileIndices/totalWarps : (warp+1)*numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    unsigned int pos = startTileIndex+warp*numTiles/totalWarps;
    unsigned int end = startTileIndex+(warp+1)*numTiles/totalWarps;
#endif
    real energy = 0;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
    __shared__ unsigned int exclusionRange[2*WARPS_PER_GROUP];
    __shared__ int exclusionIndex[WARPS_PER_GROUP];
#ifndef ENABLE_SHUFFLE
    __shared__ real tempBuffer[3*THREAD_BLOCK_SIZE];
#endif
    
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        const unsigned int localGroupIndex = threadIdx.x/TILE_SIZE;
        unsigned int x, y;
        AtomData data;
        if (pos < end) {
#ifdef USE_CUTOFF
            if (numTiles <= maxTiles) {
                ushort2 tileIndices = tiles[pos];
                x = tileIndices.x;
                y = tileIndices.y;
            }
            else
#endif
            {
                y = (unsigned int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                    y += (x < y ? -1 : 1);
                    x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                }
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
            data.force = make_real3(0);
            
            // Locate the exclusion data for this tile.

            if (tgx < 2)
                exclusionRange[2*localGroupIndex+tgx] = exclusionRowIndices[x+tgx];
            if (tgx == 0)
                exclusionIndex[localGroupIndex] = -1;
            for (unsigned int i = exclusionRange[2*localGroupIndex]+tgx; i < exclusionRange[2*localGroupIndex+1]; i += TILE_SIZE)
                if (exclusionIndices[i] == y)
                    exclusionIndex[localGroupIndex] = i*TILE_SIZE;
            bool hasExclusions = (exclusionIndex[localGroupIndex] > -1);
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                localData[threadIdx.x].posq = data.posq;
                localData[threadIdx.x].dipole = data.dipole;
                localData[threadIdx.x].quadrupoleXX = data.quadrupoleXX;
                localData[threadIdx.x].quadrupoleXY = data.quadrupoleXY;
                localData[threadIdx.x].quadrupoleXZ = data.quadrupoleXZ;
                localData[threadIdx.x].quadrupoleYY = data.quadrupoleYY;
                localData[threadIdx.x].quadrupoleYZ = data.quadrupoleYZ;
                localData[threadIdx.x].inducedDipole = data.inducedDipole;
                localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
                localData[threadIdx.x].thole = data.thole; // IS THIS CORRECT?
                localData[threadIdx.x].damp = data.damp; // IS THIS CORRECT?
                uint2 covalent = covalentFlags[exclusionIndex[localGroupIndex]+tgx];
                unsigned int polarizationGroup = polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx];
                
                // Compute forces.
                
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        real tempEnergy;
                        float d = computeDScaleFactor(polarizationGroup);
                        float p = computePScaleFactor(covalent, polarizationGroup);
                        float m = computeMScaleFactor(covalent);
                        computeOneInteractionF1(data, localData[tbx+j], d, p, m, tempEnergy, tempForce);
                        data.force += tempForce;
                        energy += 0.5f*tempEnergy;
                    }
                    covalent.x >>= 1;
                    covalent.y >>= 1;
                    polarizationGroup >>= 1;
                }
                data.force *= ENERGY_SCALE_FACTOR;
                atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0xFFFFFFFF)));
                atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0xFFFFFFFF)));
                atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0xFFFFFFFF)));
                
                // Compute torques.
                
                data.force = make_real3(0);
                covalent = covalentFlags[exclusionIndex[localGroupIndex]+tgx];
                polarizationGroup = polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx];
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        float d = computeDScaleFactor(polarizationGroup);
                        float p = computePScaleFactor(covalent, polarizationGroup);
                        float m = computeMScaleFactor(covalent);
                        computeOneInteractionT1(data, localData[tbx+j], d, p, m, tempForce);
                        data.force += tempForce;
                    }
                    covalent.x >>= 1;
                    covalent.y >>= 1;
                    polarizationGroup >>= 1;
                }
                data.force *= ENERGY_SCALE_FACTOR;
                atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0xFFFFFFFF)));
                atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0xFFFFFFFF)));
                atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0xFFFFFFFF)));
            }
            else {
                // This is an off-diagonal tile.

                unsigned int j = y*TILE_SIZE + tgx;
                loadAtomData(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
                localData[threadIdx.x].force = make_real3(0);
#ifdef USE_CUTOFF
                unsigned int flags = (numTiles <= maxTiles ? interactionFlags[pos] : 0xFFFFFFFF);
                if (!hasExclusions && flags != 0xFFFFFFFF) {
                    if (flags == 0) {
                        // No interactions in this tile.
                    }
                    else {
                        // Compute only a subset of the interactions in this tile.

                        for (j = 0; j < TILE_SIZE; j++) {
                            if ((flags&(1<<j)) != 0) {
                                int atom2 = tbx+j;
                                int bufferIndex = 3*threadIdx.x;
                                real3 dEdR1 = make_real3(0);
                                real3 dEdR2 = make_real3(0);
                                real3 delta = make_real3(localData[atom2].posq.x-data.posq.x, localData[atom2].posq.y-data.posq.y, localData[atom2].posq.z-data.posq.z);
#ifdef USE_PERIODIC
                                delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                                delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                                delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                                if (r2 < CUTOFF_SQUARED) {
#endif
                                    real invR = RSQRT(r2);
                                    real r = RECIP(invR);
                                    LOAD_ATOM2_PARAMETERS
                                    atom2 = y*TILE_SIZE+j;
                                    COMPUTE_INTERACTION
#ifdef USE_CUTOFF
                                }
#endif
#ifdef ENABLE_SHUFFLE
                                force.x -= dEdR1.x;
                                force.y -= dEdR1.y;
                                force.z -= dEdR1.z;
                                for (int i = 16; i >= 1; i /= 2) {
                                    dEdR2.x += __shfl_xor(dEdR2.x, i, 32);
                                    dEdR2.y += __shfl_xor(dEdR2.y, i, 32);
                                    dEdR2.z += __shfl_xor(dEdR2.z, i, 32);
                                }
                                if (tgx == 0) {
                                    localData[tbx+j].fx += dEdR2.x;
                                    localData[tbx+j].fy += dEdR2.y;
                                    localData[tbx+j].fz += dEdR2.z;
                                }
#else
                                force.x -= dEdR1.x;
                                force.y -= dEdR1.y;
                                force.z -= dEdR1.z;
                                tempBuffer[bufferIndex] = dEdR2.x;
                                tempBuffer[bufferIndex+1] = dEdR2.y;
                                tempBuffer[bufferIndex+2] = dEdR2.z;

                                // Sum the forces on atom2.

                                if (tgx % 4 == 0) {
                                    tempBuffer[bufferIndex] += tempBuffer[bufferIndex+3]+tempBuffer[bufferIndex+6]+tempBuffer[bufferIndex+9];
                                    tempBuffer[bufferIndex+1] += tempBuffer[bufferIndex+4]+tempBuffer[bufferIndex+7]+tempBuffer[bufferIndex+10];
                                    tempBuffer[bufferIndex+2] += tempBuffer[bufferIndex+5]+tempBuffer[bufferIndex+8]+tempBuffer[bufferIndex+11];
                                }
                                if (tgx == 0) {
                                    localData[tbx+j].fx += tempBuffer[bufferIndex]+tempBuffer[bufferIndex+12]+tempBuffer[bufferIndex+24]+tempBuffer[bufferIndex+36]+tempBuffer[bufferIndex+48]+tempBuffer[bufferIndex+60]+tempBuffer[bufferIndex+72]+tempBuffer[bufferIndex+84];
                                    localData[tbx+j].fy += tempBuffer[bufferIndex+1]+tempBuffer[bufferIndex+13]+tempBuffer[bufferIndex+25]+tempBuffer[bufferIndex+37]+tempBuffer[bufferIndex+49]+tempBuffer[bufferIndex+61]+tempBuffer[bufferIndex+73]+tempBuffer[bufferIndex+85];
                                    localData[tbx+j].fz += tempBuffer[bufferIndex+2]+tempBuffer[bufferIndex+14]+tempBuffer[bufferIndex+26]+tempBuffer[bufferIndex+38]+tempBuffer[bufferIndex+50]+tempBuffer[bufferIndex+62]+tempBuffer[bufferIndex+74]+tempBuffer[bufferIndex+86];
                                }
#endif
                            }
                        }
                    }
                }
                else
#endif
                {
                    // Compute the full set of interactions in this tile.

                    uint2 covalent = (hasExclusions ? covalentFlags[exclusionIndex[localGroupIndex]+tgx] : make_uint2(0, 0));
                    unsigned int polarizationGroup = (hasExclusions ? polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx] : 0);
                    covalent.x = (covalent.x >> tgx) | (covalent.x << (TILE_SIZE - tgx));
                    covalent.y = (covalent.y >> tgx) | (covalent.y << (TILE_SIZE - tgx));
                    polarizationGroup = (polarizationGroup >> tgx) | (polarizationGroup << (TILE_SIZE - tgx));
                    
                    // Compute forces.
                    
                    unsigned int tj = tgx;
                    for (j = 0; j < TILE_SIZE; j++) {
                        int atom2 = y*TILE_SIZE+tj;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            real3 tempForce;
                            real tempEnergy;
                            float d = computeDScaleFactor(polarizationGroup);
                            float p = computePScaleFactor(covalent, polarizationGroup);
                            float m = computeMScaleFactor(covalent);
                            computeOneInteractionF1(data, localData[tbx+tj], d, p, m, tempEnergy, tempForce);
                            data.force += tempForce;
                            localData[tbx+tj].force -= tempForce;
                            energy += tempEnergy;
                        }
                        covalent.x >>= 1;
                        covalent.y >>= 1;
                        polarizationGroup >>= 1;
                        tj = (tj + 1) & (TILE_SIZE - 1);
                    }
                    data.force *= ENERGY_SCALE_FACTOR;
                    localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
                    if (pos < end) {
                        unsigned int offset = x*TILE_SIZE + tgx;
                        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0xFFFFFFFF)));
                        offset = y*TILE_SIZE + tgx;
                        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0xFFFFFFFF)));
                    }
                    
                    // Compute torques.
                    
                    covalent = (hasExclusions ? covalentFlags[exclusionIndex[localGroupIndex]+tgx] : make_uint2(0, 0));
                    polarizationGroup = (hasExclusions ? polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx] : 0);
                    covalent.x = (covalent.x >> tgx) | (covalent.x << (TILE_SIZE - tgx));
                    covalent.y = (covalent.y >> tgx) | (covalent.y << (TILE_SIZE - tgx));
                    polarizationGroup = (polarizationGroup >> tgx) | (polarizationGroup << (TILE_SIZE - tgx));
                    for (j = 0; j < TILE_SIZE; j++) {
                        int atom2 = y*TILE_SIZE+tj;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            real3 tempForce;
                            float d = computeDScaleFactor(polarizationGroup);
                            float p = computePScaleFactor(covalent, polarizationGroup);
                            float m = computeMScaleFactor(covalent);
                            computeOneInteractionT1(data, localData[tbx+tj], d, p, m, tempForce);
                            data.force += tempForce;
                            computeOneInteractionT3(data, localData[tbx+tj], d, p, m, tempForce);
                            localData[tbx+tj].force += tempForce;
                        }
                        covalent.x >>= 1;
                        covalent.y >>= 1;
                        polarizationGroup >>= 1;
                        tj = (tj + 1) & (TILE_SIZE - 1);
                    }
                    data.force *= ENERGY_SCALE_FACTOR;
                    localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
                    if (pos < end) {
                        unsigned int offset = x*TILE_SIZE + tgx;
                        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0xFFFFFFFF)));
                        offset = y*TILE_SIZE + tgx;
                        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0xFFFFFFFF)));
                        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0xFFFFFFFF)));
                    }
                }
            }
        }
        pos++;
    } while (pos < end);
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}
