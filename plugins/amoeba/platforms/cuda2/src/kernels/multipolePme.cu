#include "hip/hip_runtime.h"
#define ARRAY(x,y) array[(x)-1+((y)-1)*PME_ORDER]

/**
 * This is called from updateBsplines().  It calculates the spline coefficients for a single atom along a single axis.
 */
__device__ void computeBSplinePoint(real4* thetai, real w, real* array) {
    // initialization to get to 2nd order recursion

    ARRAY(2,2) = w;
    ARRAY(2,1) = 1 - w;

    // perform one pass to get to 3rd order recursion

    ARRAY(3,3) = 0.5f * w * ARRAY(2,2);
    ARRAY(3,2) = 0.5f * ((1+w)*ARRAY(2,1)+(2-w)*ARRAY(2,2));
    ARRAY(3,1) = 0.5f * (1-w) * ARRAY(2,1);

    // compute standard B-spline recursion to desired order

    for (int i = 4; i <= PME_ORDER; i++)
    {
        int k = i - 1;
        real denom = RECIP(k);
        ARRAY(i,i) = denom * w * ARRAY(k,k);
        for (int j = 1; j <= i-2; j++)
            ARRAY(i,i-j) = denom * ((w+j)*ARRAY(k,i-j-1)+(i-j-w)*ARRAY(k,i-j));
        ARRAY(i,1) = denom * (1-w) * ARRAY(k,1);
    }

    // get coefficients for the B-spline first derivative

    int k = PME_ORDER - 1;
    ARRAY(k,PME_ORDER) = ARRAY(k,PME_ORDER-1);
    for (int i = PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // get coefficients for the B-spline second derivative

    k = PME_ORDER - 2;
    ARRAY(k,PME_ORDER-1) = ARRAY(k,PME_ORDER-2);
    for (int i = PME_ORDER-2; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,PME_ORDER) = ARRAY(k,PME_ORDER-1);
    for (int i = PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // get coefficients for the B-spline third derivative

    k = PME_ORDER - 3;
    ARRAY(k,PME_ORDER-2) = ARRAY(k,PME_ORDER-3);
    for (int i = PME_ORDER-3; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,PME_ORDER-1) = ARRAY(k,PME_ORDER-2);
    for (int i = PME_ORDER-2; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,PME_ORDER) = ARRAY(k,PME_ORDER-1);
    for (int i = PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // copy coefficients from temporary to permanent storage

    for (int i = 1; i <= PME_ORDER; i++)
        thetai[i-1] = make_real4(ARRAY(PME_ORDER,i), ARRAY(PME_ORDER-1,i), ARRAY(PME_ORDER-2,i), ARRAY(PME_ORDER-3,i));
}

/**
 * Compute bspline coefficients.
 */
extern "C" __global__ void updateBsplines(const real4* __restrict__ posq, int4* __restrict__ igrid, int2* __restrict__ pmeAtomGridIndex,
        real4* __restrict__ theta1, real4* __restrict__ theta2, real4* __restrict__ theta3, real4 periodicBoxSize, real4 invPeriodicBoxSize) {
    extern __shared__ real bsplines_cache[]; // size = block_size*pme_order*pme_order
    real* array = &bsplines_cache[threadIdx.x*PME_ORDER*PME_ORDER];

    //  get the B-spline coefficients for each multipole site

    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        real4 pos = posq[i];
        pos.x -= floor(pos.x*invPeriodicBoxSize.x)*periodicBoxSize.x;
        pos.y -= floor(pos.y*invPeriodicBoxSize.y)*periodicBoxSize.y;
        pos.z -= floor(pos.z*invPeriodicBoxSize.z)*periodicBoxSize.z;

        // First axis.

        real w = pos.x*invPeriodicBoxSize.x;
        real fr = GRID_SIZE_X*(w-(int)(w+0.5f)+0.5f);
        int ifr = (int) fr;
        w = fr - ifr;
        int igrid1 = ifr-PME_ORDER+1;
        computeBSplinePoint(&theta1[i*PME_ORDER], w, array);

        // Second axis.

        w = pos.y*invPeriodicBoxSize.y;
        fr = GRID_SIZE_Y*(w-(int)(w+0.5f)+0.5f);
        ifr = (int) fr;
        w = fr - ifr;
        int igrid2 = ifr-PME_ORDER+1;
        computeBSplinePoint(&theta2[i*PME_ORDER], w, array);

        // Third axis.

        w = pos.z*invPeriodicBoxSize.z;
        fr = GRID_SIZE_Z*(w-(int)(w+0.5f)+0.5f);
        ifr = (int) fr;
        w = fr - ifr;
        int igrid3 = ifr-PME_ORDER+1;
        computeBSplinePoint(&theta3[i*PME_ORDER], w, array);

        // Record the grid point.

        igrid1 += (igrid1 < 0 ? GRID_SIZE_X : 0);
        igrid2 += (igrid2 < 0 ? GRID_SIZE_Y : 0);
        igrid3 += (igrid3 < 0 ? GRID_SIZE_Z : 0);
        igrid[i] = make_int4(igrid1, igrid2, igrid3, 0);
        pmeAtomGridIndex[i] = make_int2(i, igrid1*GRID_SIZE_Y*GRID_SIZE_Z+igrid2*GRID_SIZE_Z+igrid3);
    }
}

/**
 * For each grid point, find the range of sorted atoms associated with that point.
 */
extern "C" __global__ void findAtomRangeForGrid(int2* __restrict__ pmeAtomGridIndex, int* __restrict__ pmeAtomRange,
        const real4* __restrict__ posq, real4 periodicBoxSize, real4 invPeriodicBoxSize) {
    int thread = blockIdx.x*blockDim.x+threadIdx.x;
    int start = (NUM_ATOMS*thread)/(blockDim.x*gridDim.x);
    int end = (NUM_ATOMS*(thread+1))/(blockDim.x*gridDim.x);
    int last = (start == 0 ? -1 : pmeAtomGridIndex[start-1].y);
    for (int i = start; i < end; ++i) {
        int2 atomData = pmeAtomGridIndex[i];
        int gridIndex = atomData.y;
        if (gridIndex != last) {
            for (int j = last+1; j <= gridIndex; ++j)
                pmeAtomRange[j] = i;
            last = gridIndex;
        }

        // The grid index won't be needed again.  Reuse that component to hold the z index, thus saving
        // some work in the charge spreading kernel.

        real posz = posq[atomData.x].z;
        posz -= floor(posz*invPeriodicBoxSize.z)*periodicBoxSize.z;
        real w = posz*invPeriodicBoxSize.z;
        real fr = GRID_SIZE_Z*(w-(int)(w+0.5f)+0.5f);
        int z = ((int) fr)-PME_ORDER+1;
        pmeAtomGridIndex[i].y = z;
    }

    // Fill in values beyond the last atom.

    if (thread == blockDim.x*gridDim.x-1) {
        int gridSize = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
        for (int j = last+1; j <= gridSize; ++j)
            pmeAtomRange[j] = NUM_ATOMS;
    }
}
extern "C" __global__ void gridSpreadFixedMultipoles(const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, real2* __restrict__ pmeGrid, int2* __restrict__ pmeAtomGridIndex, int* __restrict__ pmeAtomRange,
        const real4* __restrict__ theta1, const real4* __restrict__ theta2, const real4* __restrict__ theta3, real4 invPeriodicBoxSize) {
    const real xscale = GRID_SIZE_X*invPeriodicBoxSize.x;
    const real yscale = GRID_SIZE_Y*invPeriodicBoxSize.y;
    const real zscale = GRID_SIZE_Z*invPeriodicBoxSize.z;
    unsigned int numGridPoints = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
    unsigned int numThreads = gridDim.x*blockDim.x;
    for (int gridIndex = blockIdx.x*blockDim.x+threadIdx.x; gridIndex < numGridPoints; gridIndex += numThreads) {
        int3 gridPoint;
        gridPoint.x = gridIndex/(GRID_SIZE_Y*GRID_SIZE_Z);
        int remainder = gridIndex-gridPoint.x*GRID_SIZE_Y*GRID_SIZE_Z;
        gridPoint.y = remainder/GRID_SIZE_Z;
        gridPoint.z = remainder-gridPoint.y*GRID_SIZE_Z;
        real result = 0;
        for (int ix = 0; ix < PME_ORDER; ++ix) {
            int x = gridPoint.x-ix+(gridPoint.x >= ix ? 0 : GRID_SIZE_X);
            for (int iy = 0; iy < PME_ORDER; ++iy) {
                int y = gridPoint.y-iy+(gridPoint.y >= iy ? 0 : GRID_SIZE_Y);
                int z1 = gridPoint.z-PME_ORDER+1;
                z1 += (z1 >= 0 ? 0 : GRID_SIZE_Z);
                int z2 = (z1 < gridPoint.z ? gridPoint.z : GRID_SIZE_Z-1);
                int gridIndex1 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+z1;
                int gridIndex2 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+z2;
                int firstAtom = pmeAtomRange[gridIndex1];
                int lastAtom = pmeAtomRange[gridIndex2+1];
                for (int i = firstAtom; i < lastAtom; ++i) {
                    int2 atomData = pmeAtomGridIndex[i];
                    int atomIndex = atomData.x;
                    int z = atomData.y;
                    int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : GRID_SIZE_Z);
                    if (iz >= GRID_SIZE_Z)
                        iz -= GRID_SIZE_Z;
                    real atomCharge = posq[atomIndex].w;
                    real atomDipoleX = xscale*labFrameDipole[atomIndex*3];
                    real atomDipoleY = yscale*labFrameDipole[atomIndex*3+1];
                    real atomDipoleZ = zscale*labFrameDipole[atomIndex*3+2];
                    real atomQuadrupoleXX = xscale*xscale*labFrameQuadrupole[atomIndex*5];
                    real atomQuadrupoleXY = 2*xscale*yscale*labFrameQuadrupole[atomIndex*5+1];
                    real atomQuadrupoleXZ = 2*xscale*zscale*labFrameQuadrupole[atomIndex*5+2];
                    real atomQuadrupoleYY = yscale*yscale*labFrameQuadrupole[atomIndex*5+3];
                    real atomQuadrupoleYZ = 2*yscale*zscale*labFrameQuadrupole[atomIndex*5+4];
                    real atomQuadrupoleZZ = -zscale*zscale*(labFrameQuadrupole[atomIndex*5]+labFrameQuadrupole[atomIndex*5+3]);
                    real4 t = theta1[atomIndex*PME_ORDER+ix];
                    real4 u = theta2[atomIndex*PME_ORDER+iy];
                    real4 v = theta3[atomIndex*PME_ORDER+iz];
                    real term0 = atomCharge*u.x*v.x + atomDipoleY*u.y*v.x + atomDipoleZ*u.x*v.y + atomQuadrupoleYY*u.z*v.x + atomQuadrupoleZZ*u.x*v.z + atomQuadrupoleYZ*u.y*v.y;
                    real term1 = atomDipoleX*u.x*v.x + atomQuadrupoleXY*u.y*v.x + atomQuadrupoleXZ*u.x*v.y;
                    real term2 = atomQuadrupoleXX * u.x * v.x;
                    result += term0*t.x + term1*t.y + term2*t.z;
                }
                if (z1 > gridPoint.z) {
                    gridIndex1 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z;
                    gridIndex2 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+gridPoint.z;
                    firstAtom = pmeAtomRange[gridIndex1];
                    lastAtom = pmeAtomRange[gridIndex2+1];
                    for (int i = firstAtom; i < lastAtom; ++i) {
                        int2 atomData = pmeAtomGridIndex[i];
                        int atomIndex = atomData.x;
                        int z = atomData.y;
                        int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : GRID_SIZE_Z);
                        if (iz >= GRID_SIZE_Z)
                            iz -= GRID_SIZE_Z;
                        real atomCharge = posq[atomIndex].w;
                        real atomDipoleX = xscale*labFrameDipole[atomIndex*3];
                        real atomDipoleY = yscale*labFrameDipole[atomIndex*3+1];
                        real atomDipoleZ = zscale*labFrameDipole[atomIndex*3+2];
                        real atomQuadrupoleXX = xscale*xscale*labFrameQuadrupole[atomIndex*5];
                        real atomQuadrupoleXY = 2*xscale*yscale*labFrameQuadrupole[atomIndex*5+1];
                        real atomQuadrupoleXZ = 2*xscale*zscale*labFrameQuadrupole[atomIndex*5+2];
                        real atomQuadrupoleYY = yscale*yscale*labFrameQuadrupole[atomIndex*5+3];
                        real atomQuadrupoleYZ = 2*yscale*zscale*labFrameQuadrupole[atomIndex*5+4];
                        real atomQuadrupoleZZ = -zscale*zscale*(labFrameQuadrupole[atomIndex*5]+labFrameQuadrupole[atomIndex*5+3]);
                        real4 t = theta1[atomIndex*PME_ORDER+ix];
                        real4 u = theta2[atomIndex*PME_ORDER+iy];
                        real4 v = theta3[atomIndex*PME_ORDER+iz];
                        real term0 = atomCharge*u.x*v.x + atomDipoleY*u.y*v.x + atomDipoleZ*u.x*v.y + atomQuadrupoleYY*u.z*v.x + atomQuadrupoleZZ*u.x*v.z + atomQuadrupoleYZ*u.y*v.y;
                        real term1 = atomDipoleX*u.x*v.x + atomQuadrupoleXY*u.y*v.x + atomQuadrupoleXZ*u.x*v.y;
                        real term2 = atomQuadrupoleXX * u.x * v.x;
                        result += term0*t.x + term1*t.y + term2*t.z;
                    }
                }
            }
        }
        pmeGrid[gridIndex] = make_real2(result, 0);
    }
}

extern "C" __global__ void gridSpreadInducedDipoles(const real4* __restrict__ posq, const real* __restrict__ inducedDipole,
        const real* __restrict__ inducedDipolePolar, real2* __restrict__ pmeGrid, int2* __restrict__ pmeAtomGridIndex, int* __restrict__ pmeAtomRange,
        const real4* __restrict__ theta1, const real4* __restrict__ theta2, const real4* __restrict__ theta3, real4 invPeriodicBoxSize) {
    const real xscale = GRID_SIZE_X*invPeriodicBoxSize.x;
    const real yscale = GRID_SIZE_Y*invPeriodicBoxSize.y;
    const real zscale = GRID_SIZE_Z*invPeriodicBoxSize.z;
    unsigned int numGridPoints = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
    unsigned int numThreads = gridDim.x*blockDim.x;
    for (int gridIndex = blockIdx.x*blockDim.x+threadIdx.x; gridIndex < numGridPoints; gridIndex += numThreads) {
        int3 gridPoint;
        gridPoint.x = gridIndex/(GRID_SIZE_Y*GRID_SIZE_Z);
        int remainder = gridIndex-gridPoint.x*GRID_SIZE_Y*GRID_SIZE_Z;
        gridPoint.y = remainder/GRID_SIZE_Z;
        gridPoint.z = remainder-gridPoint.y*GRID_SIZE_Z;
        real2 result = make_real2(0, 0);
        for (int ix = 0; ix < PME_ORDER; ++ix) {
            int x = gridPoint.x-ix+(gridPoint.x >= ix ? 0 : GRID_SIZE_X);
            for (int iy = 0; iy < PME_ORDER; ++iy) {
                int y = gridPoint.y-iy+(gridPoint.y >= iy ? 0 : GRID_SIZE_Y);
                int z1 = gridPoint.z-PME_ORDER+1;
                z1 += (z1 >= 0 ? 0 : GRID_SIZE_Z);
                int z2 = (z1 < gridPoint.z ? gridPoint.z : GRID_SIZE_Z-1);
                int gridIndex1 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+z1;
                int gridIndex2 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+z2;
                int firstAtom = pmeAtomRange[gridIndex1];
                int lastAtom = pmeAtomRange[gridIndex2+1];
                for (int i = firstAtom; i < lastAtom; ++i) {
                    int2 atomData = pmeAtomGridIndex[i];
                    int atomIndex = atomData.x;
                    int z = atomData.y;
                    int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : GRID_SIZE_Z);
                    if (iz >= GRID_SIZE_Z)
                        iz -= GRID_SIZE_Z;
                    real inducedDipoleX = xscale*inducedDipole[atomIndex*3];
                    real inducedDipoleY = yscale*inducedDipole[atomIndex*3+1];
                    real inducedDipoleZ = zscale*inducedDipole[atomIndex*3+2];
                    real inducedDipolePolarX = xscale*inducedDipolePolar[atomIndex*3];
                    real inducedDipolePolarY = yscale*inducedDipolePolar[atomIndex*3+1];
                    real inducedDipolePolarZ = zscale*inducedDipolePolar[atomIndex*3+2];
                    real4 t = theta1[atomIndex*PME_ORDER+ix];
                    real4 u = theta2[atomIndex*PME_ORDER+iy];
                    real4 v = theta3[atomIndex*PME_ORDER+iz];
                    real term01 = inducedDipoleY*u.y*v.x + inducedDipoleZ*u.x*v.y;
                    real term11 = inducedDipoleX*u.x*v.x;
                    real term02 = inducedDipolePolarY*u.y*v.x + inducedDipolePolarZ*u.x*v.y;
                    real term12 = inducedDipolePolarX*u.x*v.x;
                    result.x += term01*t.x + term11*t.y;
                    result.y += term02*t.x + term12*t.y;
                }
                if (z1 > gridPoint.z) {
                    gridIndex1 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z;
                    gridIndex2 = x*GRID_SIZE_Y*GRID_SIZE_Z+y*GRID_SIZE_Z+gridPoint.z;
                    firstAtom = pmeAtomRange[gridIndex1];
                    lastAtom = pmeAtomRange[gridIndex2+1];
                    for (int i = firstAtom; i < lastAtom; ++i) {
                        int2 atomData = pmeAtomGridIndex[i];
                        int atomIndex = atomData.x;
                        int z = atomData.y;
                        int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : GRID_SIZE_Z);
                        if (iz >= GRID_SIZE_Z)
                            iz -= GRID_SIZE_Z;
                        real inducedDipoleX = xscale*inducedDipole[atomIndex*3];
                        real inducedDipoleY = yscale*inducedDipole[atomIndex*3+1];
                        real inducedDipoleZ = zscale*inducedDipole[atomIndex*3+2];
                        real inducedDipolePolarX = xscale*inducedDipolePolar[atomIndex*3];
                        real inducedDipolePolarY = yscale*inducedDipolePolar[atomIndex*3+1];
                        real inducedDipolePolarZ = zscale*inducedDipolePolar[atomIndex*3+2];
                        real4 t = theta1[atomIndex*PME_ORDER+ix];
                        real4 u = theta2[atomIndex*PME_ORDER+iy];
                        real4 v = theta3[atomIndex*PME_ORDER+iz];
                        real term01 = inducedDipoleY*u.y*v.x + inducedDipoleZ*u.x*v.y;
                        real term11 = inducedDipoleX*u.x*v.x;
                        real term02 = inducedDipolePolarY*u.y*v.x + inducedDipolePolarZ*u.x*v.y;
                        real term12 = inducedDipolePolarX*u.x*v.x;
                        result.x += term01*t.x + term11*t.y;
                        result.y += term02*t.x + term12*t.y;
                    }
                }
            }
        }
        pmeGrid[gridIndex] = result;
    }
}

extern "C" __global__ void reciprocalConvolution(real2* __restrict__ pmeGrid, const real* __restrict__ pmeBsplineModuliX,
        const real* __restrict__ pmeBsplineModuliY, const real* __restrict__ pmeBsplineModuliZ, real4 periodicBoxSize, real4 invPeriodicBoxSize) {
    const unsigned int gridSize = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
    real expFactor = M_PI*M_PI/(EWALD_ALPHA*EWALD_ALPHA);
    real scaleFactor = RECIP(M_PI*periodicBoxSize.x*periodicBoxSize.y*periodicBoxSize.z);
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x) {
        int kx = index/(GRID_SIZE_Y*GRID_SIZE_Z);
        int remainder = index-kx*GRID_SIZE_Y*GRID_SIZE_Z;
        int ky = remainder/GRID_SIZE_Z;
        int kz = remainder-ky*GRID_SIZE_Z;
        if (kx == 0 && ky == 0 && kz == 0) {
            pmeGrid[index] = make_real2(0, 0);
            continue;
        }
        int mx = (kx < (GRID_SIZE_X+1)/2) ? kx : (kx-GRID_SIZE_X);
        int my = (ky < (GRID_SIZE_Y+1)/2) ? ky : (ky-GRID_SIZE_Y);
        int mz = (kz < (GRID_SIZE_Z+1)/2) ? kz : (kz-GRID_SIZE_Z);
        real mhx = mx*invPeriodicBoxSize.x;
        real mhy = my*invPeriodicBoxSize.y;
        real mhz = mz*invPeriodicBoxSize.z;
        real bx = pmeBsplineModuliX[kx];
        real by = pmeBsplineModuliY[ky];
        real bz = pmeBsplineModuliZ[kz];
        real2 grid = pmeGrid[index];
        real m2 = mhx*mhx+mhy*mhy+mhz*mhz;
        real denom = m2*bx*by*bz;
        real eterm = scaleFactor*EXP(-expFactor*m2)/denom;
        pmeGrid[index] = make_real2(grid.x*eterm, grid.y*eterm);
    }
}

extern "C" __global__ void computeFixedPotentialFromGrid(const real2* __restrict__ pmeGrid, real* __restrict__ phi,
        long long* __restrict__ fieldBuffers, long long* __restrict__ fieldPolarBuffers, const int4* __restrict__ igrid, const real4* __restrict__ theta1,
        const real4* __restrict__ theta2, const real4* __restrict__ theta3, const real* __restrict__ labFrameDipole, real4 invPeriodicBoxSize) {
    // extract the permanent multipole field at each site

    for (int m = blockIdx.x*blockDim.x+threadIdx.x; m < NUM_ATOMS; m += blockDim.x*gridDim.x) {
        int4 gridPoint = igrid[m];
        real tuv000 = 0;
        real tuv001 = 0;
        real tuv010 = 0;
        real tuv100 = 0;
        real tuv200 = 0;
        real tuv020 = 0;
        real tuv002 = 0;
        real tuv110 = 0;
        real tuv101 = 0;
        real tuv011 = 0;
        real tuv300 = 0;
        real tuv030 = 0;
        real tuv003 = 0;
        real tuv210 = 0;
        real tuv201 = 0;
        real tuv120 = 0;
        real tuv021 = 0;
        real tuv102 = 0;
        real tuv012 = 0;
        real tuv111 = 0;
        for (int iz = 0; iz < PME_ORDER; iz++) {
            int k = gridPoint.z+iz-(gridPoint.z+iz >= GRID_SIZE_Z ? GRID_SIZE_Z : 0);
            real4 v = theta3[m*PME_ORDER+iz];
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int j = gridPoint.y+iy-(gridPoint.y+iy >= GRID_SIZE_Y ? GRID_SIZE_Y : 0);
                real4 u = theta2[m*PME_ORDER+iy];
                real4 t = make_real4(0, 0, 0, 0);
                for (int ix = 0; ix < PME_ORDER; ix++) {
                    int i = gridPoint.x+ix-(gridPoint.x+ix >= GRID_SIZE_X ? GRID_SIZE_X : 0);
                    int gridIndex = i*GRID_SIZE_Y*GRID_SIZE_Z + j*GRID_SIZE_Z + k;
                    real tq = pmeGrid[gridIndex].x;
                    real4 tadd = theta1[m*PME_ORDER+ix];
                    t.x += tq*tadd.x;
                    t.y += tq*tadd.y;
                    t.z += tq*tadd.z;
                    t.w += tq*tadd.w;
                }
                tu00 += t.x*u.x;
                tu10 += t.y*u.x;
                tu01 += t.x*u.y;
                tu20 += t.z*u.x;
                tu11 += t.y*u.y;
                tu02 += t.x*u.z;
                tu30 += t.w*u.x;
                tu21 += t.z*u.y;
                tu12 += t.y*u.z;
                tu03 += t.x*u.w;
            }
            tuv000 += tu00*v.x;
            tuv100 += tu10*v.x;
            tuv010 += tu01*v.x;
            tuv001 += tu00*v.y;
            tuv200 += tu20*v.x;
            tuv020 += tu02*v.x;
            tuv002 += tu00*v.z;
            tuv110 += tu11*v.x;
            tuv101 += tu10*v.y;
            tuv011 += tu01*v.y;
            tuv300 += tu30*v.x;
            tuv030 += tu03*v.x;
            tuv003 += tu00*v.w;
            tuv210 += tu21*v.x;
            tuv201 += tu20*v.y;
            tuv120 += tu12*v.x;
            tuv021 += tu02*v.y;
            tuv102 += tu10*v.z;
            tuv012 += tu01*v.z;
            tuv111 += tu11*v.y;
        }
        phi[20*m] = tuv000;
        phi[20*m+1] = tuv100;
        phi[20*m+2] = tuv010;
        phi[20*m+3] = tuv001;
        phi[20*m+4] = tuv200;
        phi[20*m+5] = tuv020;
        phi[20*m+6] = tuv002;
        phi[20*m+7] = tuv110;
        phi[20*m+8] = tuv101;
        phi[20*m+9] = tuv011;
        phi[20*m+10] = tuv300;
        phi[20*m+11] = tuv030;
        phi[20*m+12] = tuv003;
        phi[20*m+13] = tuv210;
        phi[20*m+14] = tuv201;
        phi[20*m+15] = tuv120;
        phi[20*m+16] = tuv021;
        phi[20*m+17] = tuv102;
        phi[20*m+18] = tuv012;
        phi[20*m+19] = tuv111;
        real dipoleScale = (4/(real) 3)*(EWALD_ALPHA*EWALD_ALPHA*EWALD_ALPHA)/SQRT(M_PI);
        long long fieldx = (long long) ((dipoleScale*labFrameDipole[m*3]-GRID_SIZE_X*invPeriodicBoxSize.x*tuv100)*0xFFFFFFFF);
        fieldBuffers[m] = fieldx;
        fieldPolarBuffers[m] = fieldx;
        long long fieldy = (long long) ((dipoleScale*labFrameDipole[m*3+1]-GRID_SIZE_Y*invPeriodicBoxSize.y*tuv010)*0xFFFFFFFF);
        fieldBuffers[m+PADDED_NUM_ATOMS] = fieldy;
        fieldPolarBuffers[m+PADDED_NUM_ATOMS] = fieldy;
        long long fieldz = (long long) ((dipoleScale*labFrameDipole[m*3+2]-GRID_SIZE_Z*invPeriodicBoxSize.z*tuv001)*0xFFFFFFFF);
        fieldBuffers[m+2*PADDED_NUM_ATOMS] = fieldz;
        fieldPolarBuffers[m+2*PADDED_NUM_ATOMS] = fieldz;
    }
}

extern "C" __global__ void computeInducedPotentialFromGrid(const real2* __restrict__ pmeGrid, real* __restrict__ phid,
        real* __restrict__ phip, real* __restrict__ phidp, const int4* __restrict__ igrid, const real4* __restrict__ theta1,
        const real4* __restrict__ theta2, const real4* __restrict__ theta3, real4 invPeriodicBoxSize) {
    // extract the induced dipole field at each site

    for (int m = blockIdx.x*blockDim.x+threadIdx.x; m < NUM_ATOMS; m += blockDim.x*gridDim.x) {
        int4 gridPoint = igrid[m];
        real tuv100_1 = 0;
        real tuv010_1 = 0;
        real tuv001_1 = 0;
        real tuv200_1 = 0;
        real tuv020_1 = 0;
        real tuv002_1 = 0;
        real tuv110_1 = 0;
        real tuv101_1 = 0;
        real tuv011_1 = 0;
        real tuv100_2 = 0;
        real tuv010_2 = 0;
        real tuv001_2 = 0;
        real tuv200_2 = 0;
        real tuv020_2 = 0;
        real tuv002_2 = 0;
        real tuv110_2 = 0;
        real tuv101_2 = 0;
        real tuv011_2 = 0;
        real tuv000 = 0;
        real tuv001 = 0;
        real tuv010 = 0;
        real tuv100 = 0;
        real tuv200 = 0;
        real tuv020 = 0;
        real tuv002 = 0;
        real tuv110 = 0;
        real tuv101 = 0;
        real tuv011 = 0;
        real tuv300 = 0;
        real tuv030 = 0;
        real tuv003 = 0;
        real tuv210 = 0;
        real tuv201 = 0;
        real tuv120 = 0;
        real tuv021 = 0;
        real tuv102 = 0;
        real tuv012 = 0;
        real tuv111 = 0;
        for (int iz = 0; iz < PME_ORDER; iz++) {
            int k = gridPoint.z+iz-(gridPoint.z+iz >= GRID_SIZE_Z ? GRID_SIZE_Z : 0);
            real4 v = theta3[m*PME_ORDER+iz];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int j = gridPoint.y+iy-(gridPoint.y+iy >= GRID_SIZE_Y ? GRID_SIZE_Y : 0);
                real4 u = theta2[m*PME_ORDER+iy];
                real t0_1 = 0;
                real t1_1 = 0;
                real t2_1 = 0;
                real t0_2 = 0;
                real t1_2 = 0;
                real t2_2 = 0;
                real t3 = 0;
                for (int ix = 0; ix < PME_ORDER; ix++) {
                    int i = gridPoint.x+ix-(gridPoint.x+ix >= GRID_SIZE_X ? GRID_SIZE_X : 0);
                    int gridIndex = i*GRID_SIZE_Y*GRID_SIZE_Z + j*GRID_SIZE_Z + k;
                    real2 tq = pmeGrid[gridIndex];
                    real4 tadd = theta1[m*PME_ORDER+ix];
                    t0_1 += tq.x*tadd.x;
                    t1_1 += tq.x*tadd.y;
                    t2_1 += tq.x*tadd.z;
                    t0_2 += tq.y*tadd.x;
                    t1_2 += tq.y*tadd.y;
                    t2_2 += tq.y*tadd.z;
                    t3 += (tq.x+tq.y)*tadd.w;
                }
                tu00_1 += t0_1*u.x;
                tu10_1 += t1_1*u.x;
                tu01_1 += t0_1*u.y;
                tu20_1 += t2_1*u.x;
                tu11_1 += t1_1*u.y;
                tu02_1 += t0_1*u.z;
                tu00_2 += t0_2*u.x;
                tu10_2 += t1_2*u.x;
                tu01_2 += t0_2*u.y;
                tu20_2 += t2_2*u.x;
                tu11_2 += t1_2*u.y;
                tu02_2 += t0_2*u.z;
                real t0 = t0_1 + t0_2;
                real t1 = t1_1 + t1_2;
                real t2 = t2_1 + t2_2;
                tu00 += t0*u.x;
                tu10 += t1*u.x;
                tu01 += t0*u.y;
                tu20 += t2*u.x;
                tu11 += t1*u.y;
                tu02 += t0*u.z;
                tu30 += t3*u.x;
                tu21 += t2*u.y;
                tu12 += t1*u.z;
                tu03 += t0*u.w;
            }
            tuv100_1 += tu10_1*v.x;
            tuv010_1 += tu01_1*v.x;
            tuv001_1 += tu00_1*v.y;
            tuv200_1 += tu20_1*v.x;
            tuv020_1 += tu02_1*v.x;
            tuv002_1 += tu00_1*v.z;
            tuv110_1 += tu11_1*v.x;
            tuv101_1 += tu10_1*v.y;
            tuv011_1 += tu01_1*v.y;
            tuv100_2 += tu10_2*v.x;
            tuv010_2 += tu01_2*v.x;
            tuv001_2 += tu00_2*v.y;
            tuv200_2 += tu20_2*v.x;
            tuv020_2 += tu02_2*v.x;
            tuv002_2 += tu00_2*v.z;
            tuv110_2 += tu11_2*v.x;
            tuv101_2 += tu10_2*v.y;
            tuv011_2 += tu01_2*v.y;
            tuv000 += tu00*v.x;
            tuv100 += tu10*v.x;
            tuv010 += tu01*v.x;
            tuv001 += tu00*v.y;
            tuv200 += tu20*v.x;
            tuv020 += tu02*v.x;
            tuv002 += tu00*v.z;
            tuv110 += tu11*v.x;
            tuv101 += tu10*v.y;
            tuv011 += tu01*v.y;
            tuv300 += tu30*v.x;
            tuv030 += tu03*v.x;
            tuv003 += tu00*v.w;
            tuv210 += tu21*v.x;
            tuv201 += tu20*v.y;
            tuv120 += tu12*v.x;
            tuv021 += tu02*v.y;
            tuv102 += tu10*v.z;
            tuv012 += tu01*v.z;
            tuv111 += tu11*v.y;
        }
        phid[10*m]   = 0;
        phid[10*m+1] = tuv100_1;
        phid[10*m+2] = tuv010_1;
        phid[10*m+3] = tuv001_1;
        phid[10*m+4] = tuv200_1;
        phid[10*m+5] = tuv020_1;
        phid[10*m+6] = tuv002_1;
        phid[10*m+7] = tuv110_1;
        phid[10*m+8] = tuv101_1;
        phid[10*m+9] = tuv011_1;

        phip[10*m]   = 0;
        phip[10*m+1] = tuv100_2;
        phip[10*m+2] = tuv010_2;
        phip[10*m+3] = tuv001_2;
        phip[10*m+4] = tuv200_2;
        phip[10*m+5] = tuv020_2;
        phip[10*m+6] = tuv002_2;
        phip[10*m+7] = tuv110_2;
        phip[10*m+8] = tuv101_2;
        phip[10*m+9] = tuv011_2;

        phidp[20*m] = tuv000;
        phidp[20*m+1] = tuv100;
        phidp[20*m+2] = tuv010;
        phidp[20*m+3] = tuv001;
        phidp[20*m+4] = tuv200;
        phidp[20*m+5] = tuv020;
        phidp[20*m+6] = tuv002;
        phidp[20*m+7] = tuv110;
        phidp[20*m+8] = tuv101;
        phidp[20*m+9] = tuv011;
        phidp[20*m+10] = tuv300;
        phidp[20*m+11] = tuv030;
        phidp[20*m+12] = tuv003;
        phidp[20*m+13] = tuv210;
        phidp[20*m+14] = tuv201;
        phidp[20*m+15] = tuv120;
        phidp[20*m+16] = tuv021;
        phidp[20*m+17] = tuv102;
        phidp[20*m+18] = tuv012;
        phidp[20*m+19] = tuv111;
    }
}

extern "C" __global__ void computeFixedMultipoleForceAndEnergy(real4* __restrict__ posq, unsigned long long* __restrict__ forceBuffers,
        long long* __restrict__ torqueBuffers, real* __restrict__ energyBuffer, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real* __restrict__ phi_global, real4 invPeriodicBoxSize) {
    real multipole[10];
    const int deriv1[] = {1, 4, 7, 8, 10, 15, 17, 13, 14, 19};
    const int deriv2[] = {2, 7, 5, 9, 13, 11, 18, 15, 19, 16};
    const int deriv3[] = {3, 8, 9, 6, 14, 16, 12, 19, 17, 18};
    const real xscale = GRID_SIZE_X*invPeriodicBoxSize.x;
    const real yscale = GRID_SIZE_Y*invPeriodicBoxSize.y;
    const real zscale = GRID_SIZE_Z*invPeriodicBoxSize.z;
    real energy = 0;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        // Compute the torque.

        multipole[0] = posq[i].w;
        multipole[1] = labFrameDipole[i*3];
        multipole[2] = labFrameDipole[i*3+1];
        multipole[3] = labFrameDipole[i*3+2];
        multipole[4] = labFrameQuadrupole[i*5];
        multipole[5] = labFrameQuadrupole[i*5+3];
        multipole[6] = -(multipole[4]+multipole[5]);
        multipole[7] = 2*labFrameQuadrupole[i*5+1];
        multipole[8] = 2*labFrameQuadrupole[i*5+2];
        multipole[9] = 2*labFrameQuadrupole[i*5+4];

        const real* phi = &phi_global[20*i];

        torqueBuffers[i] = (long long) (EPSILON_FACTOR*(multipole[3]*yscale*phi[2] - multipole[2]*zscale*phi[3]
                      + 2*(multipole[6]-multipole[5])*yscale*zscale*phi[9]
                      + multipole[8]*xscale*yscale*phi[7] + multipole[9]*yscale*yscale*phi[5]
                      - multipole[7]*xscale*zscale*phi[8] - multipole[9]*zscale*zscale*phi[6])*0xFFFFFFFF);

        torqueBuffers[i+PADDED_NUM_ATOMS] = (long long) (EPSILON_FACTOR*(multipole[1]*zscale*phi[3] - multipole[3]*xscale*phi[1]
                      + 2*(multipole[4]-multipole[6])*xscale*zscale*phi[8]
                      + multipole[7]*yscale*zscale*phi[9] + multipole[8]*zscale*zscale*phi[6]
                      - multipole[8]*xscale*xscale*phi[4] - multipole[9]*xscale*yscale*phi[7])*0xFFFFFFFF);

        torqueBuffers[i+PADDED_NUM_ATOMS*2] = (long long) (EPSILON_FACTOR*(multipole[2]*xscale*phi[1] - multipole[1]*yscale*phi[2]
                      + 2*(multipole[5]-multipole[4])*xscale*yscale*phi[7]
                      + multipole[7]*xscale*xscale*phi[4] + multipole[9]*xscale*zscale*phi[8]
                      - multipole[7]*yscale*yscale*phi[5] - multipole[8]*yscale*zscale*phi[9])*0xFFFFFFFF);

        // Compute the force and energy.

        multipole[1] *= xscale;
        multipole[2] *= yscale;
        multipole[3] *= zscale;
        multipole[4] *= xscale*xscale;
        multipole[5] *= yscale*yscale;
        multipole[6] *= zscale*zscale;
        multipole[7] *= xscale*yscale;
        multipole[8] *= xscale*zscale;
        multipole[9] *= yscale*zscale;

        real4 f = make_real4(0, 0, 0, 0);
        for (int k = 0; k < 10; k++) {
            energy += multipole[k]*phi[k];
            f.x += multipole[k]*phi[deriv1[k]];
            f.y += multipole[k]*phi[deriv2[k]];
            f.z += multipole[k]*phi[deriv3[k]];
        }
        f.x *= EPSILON_FACTOR*xscale;
        f.y *= EPSILON_FACTOR*yscale;
        f.z *= EPSILON_FACTOR*zscale;
        forceBuffers[i] -= static_cast<unsigned long long>((long long) (f.x*0xFFFFFFFF));
        forceBuffers[i+PADDED_NUM_ATOMS] -= static_cast<unsigned long long>((long long) (f.y*0xFFFFFFFF));
        forceBuffers[i+PADDED_NUM_ATOMS*2] -= static_cast<unsigned long long>((long long) (f.z*0xFFFFFFFF));
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*EPSILON_FACTOR*energy;
}

extern "C" __global__ void computeInducedDipoleForceAndEnergy(real4* __restrict__ posq, unsigned long long* __restrict__ forceBuffers,
        long long* __restrict__ torqueBuffers, real* __restrict__ energyBuffer, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real* __restrict__ inducedDipole_global, const real* __restrict__ inducedDipolePolar_global,
        const real* __restrict__ phi_global, const real* __restrict__ phid_global, const real* __restrict__ phip_global,
        const real* __restrict__ phidp_global, real4 invPeriodicBoxSize) {
    real multipole[10];
    real inducedDipole[3];
    real inducedDipolePolar[3];
    real scales[3];
    const int deriv1[] = {1, 4, 7, 8, 10, 15, 17, 13, 14, 19};
    const int deriv2[] = {2, 7, 5, 9, 13, 11, 18, 15, 19, 16};
    const int deriv3[] = {3, 8, 9, 6, 14, 16, 12, 19, 17, 18};
    const real xscale = GRID_SIZE_X*invPeriodicBoxSize.x;
    const real yscale = GRID_SIZE_Y*invPeriodicBoxSize.y;
    const real zscale = GRID_SIZE_Z*invPeriodicBoxSize.z;
    scales[0] = xscale;
    scales[1] = yscale;
    scales[2] = zscale;
    real energy = 0;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        // Compute the torque.

        multipole[0] = posq[i].w;
        multipole[1] = labFrameDipole[i*3];
        multipole[2] = labFrameDipole[i*3+1];
        multipole[3] = labFrameDipole[i*3+2];
        multipole[4] = labFrameQuadrupole[i*5];
        multipole[5] = labFrameQuadrupole[i*5+3];
        multipole[6] = -(multipole[4]+multipole[5]);
        multipole[7] = 2*labFrameQuadrupole[i*5+1];
        multipole[8] = 2*labFrameQuadrupole[i*5+2];
        multipole[9] = 2*labFrameQuadrupole[i*5+4];
        const real* phidp = &phidp_global[20*i];
 
        torqueBuffers[i] += (long long) (0.5f*EPSILON_FACTOR*(multipole[3]*yscale*phidp[2] - multipole[2]*zscale*phidp[3]
                      + 2*(multipole[6]-multipole[5])*yscale*zscale*phidp[9]
                      + multipole[8]*xscale*yscale*phidp[7] + multipole[9]*yscale*yscale*phidp[5]
                      - multipole[7]*xscale*zscale*phidp[8] - multipole[9]*zscale*zscale*phidp[6])*0xFFFFFFFF);

        torqueBuffers[i+PADDED_NUM_ATOMS] += (long long) (0.5f*EPSILON_FACTOR*(multipole[1]*zscale*phidp[3] - multipole[3]*xscale*phidp[1]
                      + 2*(multipole[4]-multipole[6])*xscale*zscale*phidp[8]
                      + multipole[7]*yscale*zscale*phidp[9] + multipole[8]*zscale*zscale*phidp[6]
                      - multipole[8]*xscale*xscale*phidp[4] - multipole[9]*xscale*yscale*phidp[7])*0xFFFFFFFF);

        torqueBuffers[i+PADDED_NUM_ATOMS*2] += (long long) (0.5f*EPSILON_FACTOR*(multipole[2]*xscale*phidp[1] - multipole[1]*yscale*phidp[2]
                      + 2*(multipole[5]-multipole[4])*xscale*yscale*phidp[7]
                      + multipole[7]*xscale*xscale*phidp[4] + multipole[9]*xscale*zscale*phidp[8]
                      - multipole[7]*yscale*yscale*phidp[5] - multipole[8]*yscale*zscale*phidp[9])*0xFFFFFFFF);

        // Compute the force and energy.

        multipole[1] *= xscale;
        multipole[2] *= yscale;
        multipole[3] *= zscale;
        multipole[4] *= xscale*xscale;
        multipole[5] *= yscale*yscale;
        multipole[6] *= zscale*zscale;
        multipole[7] *= xscale*yscale;
        multipole[8] *= xscale*zscale;
        multipole[9] *= yscale*zscale;

        inducedDipole[0] = inducedDipole_global[i*3];
        inducedDipole[1] = inducedDipole_global[i*3+1];
        inducedDipole[2] = inducedDipole_global[i*3+2];
        inducedDipolePolar[0] = inducedDipolePolar_global[i*3];
        inducedDipolePolar[1] = inducedDipolePolar_global[i*3+1];
        inducedDipolePolar[2] = inducedDipolePolar_global[i*3+2];
        const real* phi = &phi_global[20*i];
        const real* phip = &phip_global[10*i];
        const real* phid = &phid_global[10*i];
        real4 f = make_real4(0, 0, 0, 0);

        energy += GRID_SIZE_X*invPeriodicBoxSize.x*inducedDipole[0]*phi[1];
        energy += GRID_SIZE_Y*invPeriodicBoxSize.y*inducedDipole[1]*phi[2];
        energy += GRID_SIZE_Z*invPeriodicBoxSize.z*inducedDipole[2]*phi[3];

        for (int k = 0; k < 3; k++) {
            int j1 = deriv1[k+1];
            int j2 = deriv2[k+1];
            int j3 = deriv3[k+1];
            f.x += (inducedDipole[k]+inducedDipolePolar[k])*phi[j1]*(scales[k]/xscale);
            f.y += (inducedDipole[k]+inducedDipolePolar[k])*phi[j2]*(scales[k]/yscale);
            f.z += (inducedDipole[k]+inducedDipolePolar[k])*phi[j3]*(scales[k]/zscale);
#ifndef DIRECT_POLARIZATION
            f.x += (inducedDipole[k]*phip[j1] + inducedDipolePolar[k]*phid[j1])*(scales[k]/xscale);
            f.y += (inducedDipole[k]*phip[j2] + inducedDipolePolar[k]*phid[j2])*(scales[k]/yscale);
            f.z += (inducedDipole[k]*phip[j3] + inducedDipolePolar[k]*phid[j3])*(scales[k]/zscale);
#endif
        }

        f.x *= GRID_SIZE_X*invPeriodicBoxSize.x;
        f.y *= GRID_SIZE_Y*invPeriodicBoxSize.y;
        f.z *= GRID_SIZE_Z*invPeriodicBoxSize.z;
        for (int k = 0; k < 10; k++) {
            f.x += multipole[k]*phidp[deriv1[k]];
            f.y += multipole[k]*phidp[deriv2[k]];
            f.z += multipole[k]*phidp[deriv3[k]];
        }
        f.x *= 0.5f*EPSILON_FACTOR*xscale;
        f.y *= 0.5f*EPSILON_FACTOR*yscale;
        f.z *= 0.5f*EPSILON_FACTOR*zscale;
        forceBuffers[i] -= static_cast<unsigned long long>((long long) (f.x*0xFFFFFFFF));
        forceBuffers[i+PADDED_NUM_ATOMS] -= static_cast<unsigned long long>((long long) (f.y*0xFFFFFFFF));
        forceBuffers[i+PADDED_NUM_ATOMS*2] -= static_cast<unsigned long long>((long long) (f.z*0xFFFFFFFF));
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*EPSILON_FACTOR*energy;
}

extern "C" __global__ void recordInducedFieldDipoles(const real* __restrict__ phid, real* const __restrict__ phip,
        real* __restrict__ inducedDipole, real* __restrict__ inducedDipolePolar, real4 invPeriodicBoxSize) {
    real xscale = GRID_SIZE_X*invPeriodicBoxSize.x;
    real yscale = GRID_SIZE_Y*invPeriodicBoxSize.y;
    real zscale = GRID_SIZE_Z*invPeriodicBoxSize.z;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        inducedDipole[3*i] -= xscale*phid[10*i+1];
        inducedDipole[3*i+1] -= yscale*phid[10*i+2];
        inducedDipole[3*i+2] -= zscale*phid[10*i+3];
        inducedDipolePolar[3*i] -= xscale*phip[10*i+1];
        inducedDipolePolar[3*i+1] -= yscale*phip[10*i+2];
        inducedDipolePolar[3*i+2] -= zscale*phip[10*i+3];
    }
}
