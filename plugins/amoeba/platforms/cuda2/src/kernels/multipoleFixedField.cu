#include "hip/hip_runtime.h"
#define TILE_SIZE 32
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real4 posq;
    real3 field, fieldPolar, dipole;
    real quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ, quadrupoleZZ;
    float thole, damp;
} AtomData;

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const float2* __restrict__ dampingAndThole) {
    data.posq = posq[atom];
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.quadrupoleZZ = -(data.quadrupoleXX+data.quadrupoleYY);
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

#ifdef USE_EWALD
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float dScale, float pScale, real3* fields) {
    real r2 = dot(deltaR, deltaR);
    if (r2 <= CUTOFF_SQUARED) {
        // calculate the error function damping terms

        real r = SQRT(r2);
        real ralpha = EWALD_ALPHA*r;
        real bn0 = erfc(ralpha)/r;
        real alsq2 = 2*EWALD_ALPHA*EWALD_ALPHA;
        real alsq2n = RECIP(SQRT_PI*EWALD_ALPHA);
        real exp2a = EXP(-(ralpha*ralpha));
        alsq2n *= alsq2;
        real bn1 = (bn0+alsq2n*exp2a)/r2;
        alsq2n *= alsq2;
        real bn2 = (3*bn1+alsq2n*exp2a)/r2;
        alsq2n *= alsq2;
        real bn3 = (5*bn2+alsq2n*exp2a)/r2;

        // compute the error function scaled and unscaled terms

        real scale3 = 1;
        real scale5 = 1;
        real scale7 = 1;
        real damp = atom1.damp*atom2.damp;
        if (damp != 0) {
            real ratio = (r/damp);
            ratio = ratio*ratio*ratio;
            real pgamma = (atom1.thole < atom2.thole ? atom1.thole : atom2.thole);
            damp = -pgamma*ratio;
            if (damp > -50) {
                real expdamp = EXP(damp);
                scale3 = 1 - expdamp;
                scale5 = 1 - expdamp*(1-damp);
                scale7 = 1 - expdamp*(1-damp+(0.6f*damp*damp));
            }
        }
        real dsc3 = dScale*scale3;
        real dsc5 = dScale*scale5;
        real dsc7 = dScale*scale7;

        real psc3 = pScale*scale3;
        real psc5 = pScale*scale5;
        real psc7 = pScale*scale7;

        real r3 = r*r2;
        real r5 = r3*r2;
        real r7 = r5*r2;
        real drr3 = (1-dsc3)/r3;
        real drr5 = 3*(1-dsc5)/r5;
        real drr7 = 15*(1-dsc7)/r7;

        real prr3 = (1-psc3)/r3;
        real prr5 = 3*(1-psc5)/r5;
        real prr7 = 15*(1-psc7)/r7;

        real dir = dot(atom1.dipole, deltaR);

        real3 qi;
        qi.x = atom1.quadrupoleXX*deltaR.x + atom1.quadrupoleXY*deltaR.y + atom1.quadrupoleXZ*deltaR.z;
        qi.y = atom1.quadrupoleXY*deltaR.x + atom1.quadrupoleYY*deltaR.y + atom1.quadrupoleYZ*deltaR.z;
        qi.z = atom1.quadrupoleXZ*deltaR.x + atom1.quadrupoleYZ*deltaR.y + atom1.quadrupoleZZ*deltaR.z;
        real qir = dot(qi, deltaR);

        real dkr = dot(atom2.dipole, deltaR);

        real3 qk;
        qk.x = atom2.quadrupoleXX*deltaR.x + atom2.quadrupoleXY*deltaR.y + atom2.quadrupoleXZ*deltaR.z;
        qk.y = atom2.quadrupoleXY*deltaR.x + atom2.quadrupoleYY*deltaR.y + atom2.quadrupoleYZ*deltaR.z;
        qk.z = atom2.quadrupoleXZ*deltaR.x + atom2.quadrupoleYZ*deltaR.y + atom2.quadrupoleZZ*deltaR.z;
        real qkr = dot(qk, deltaR);

        real3 fim = -deltaR*(bn1*atom2.posq.w-bn2*dkr+bn3*qkr) - bn1*atom2.dipole + 2*bn2*qk;
        real3 fkm = deltaR*(bn1*atom1.posq.w+bn2*dir+bn3*qir) - bn1*atom1.dipole - 2*bn2*qi;
        real3 fid = -deltaR*(drr3*atom2.posq.w-drr5*dkr+drr7*qkr) - drr3*atom2.dipole + 2*drr5*qk;
        real3 fkd = deltaR*(drr3*atom1.posq.w+drr5*dir+drr7*qir) - drr3*atom1.dipole - 2*drr5*qi;
        real3 fip = -deltaR*(prr3*atom2.posq.w-prr5*dkr+prr7*qkr) - prr3*atom2.dipole + 2*prr5*qk;
        real3 fkp = deltaR*(prr3*atom1.posq.w+prr5*dir+prr7*qir) - prr3*atom1.dipole - 2*prr5*qi;

        // increment the field at each site due to this interaction

        fields[0] = fim-fid;
        fields[1] = fim-fip;
        fields[2] = fkm-fkd;
        fields[3] = fkm-fkp;
    }
    else {
        fields[0] = make_real3(0);
        fields[1] = make_real3(0);
        fields[2] = make_real3(0);
        fields[3] = make_real3(0);
    }
}
#else
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float dScale, float pScale, real3* fields) {
    real rI = RSQRT(dot(deltaR, deltaR));
    real r = RECIP(rI);
    real r2I = rI*rI;

    real rr3 = rI*r2I;
    real rr5 = 3*rr3*r2I;
    real rr7 = 5*rr5*r2I;
 
    // get scaling factors, if needed
    
    float damp = atom1.damp*atom2.damp;
    real dampExp;
    if (damp != 0) {

        // get scaling factors
      
        real ratio = r/damp;
        float pGamma = atom2.thole > atom1.thole ? atom1.thole : atom2.thole; 
        damp = ratio*ratio*ratio*pGamma;
        dampExp = EXP(-damp);
    }
    else
        dampExp = 0;
      
    rr3 *= 1 - dampExp;
    rr5 *= 1 - (1+damp)*dampExp;
    rr7 *= 1 - (1+damp+(0.6f*damp*damp))*dampExp;
      
    real rr5_2 = 2*rr5;
 
    real3 qDotDelta;
    qDotDelta.x = deltaR.x*atom2.quadrupoleXX + deltaR.y*atom2.quadrupoleXY + deltaR.z*atom2.quadrupoleXZ;
    qDotDelta.y = deltaR.x*atom2.quadrupoleXY + deltaR.y*atom2.quadrupoleYY + deltaR.z*atom2.quadrupoleYZ;
    qDotDelta.z = deltaR.x*atom2.quadrupoleXZ + deltaR.y*atom2.quadrupoleYZ + deltaR.z*atom2.quadrupoleZZ;
 
    real dotdd = dot(deltaR, atom2.dipole);
    real dotqd = dot(deltaR, qDotDelta);

    real factor = -rr3*atom2.posq.w + rr5*dotdd - rr7*dotqd;
 
    real3 field1 = deltaR*factor - rr3*atom2.dipole + rr5_2*qDotDelta;
    fields[0] = dScale*field1;
    fields[1] = pScale*field1;
 
    qDotDelta.x = deltaR.x*atom1.quadrupoleXX + deltaR.y*atom1.quadrupoleXY + deltaR.z*atom1.quadrupoleXZ;
    qDotDelta.y = deltaR.x*atom1.quadrupoleXY + deltaR.y*atom1.quadrupoleYY + deltaR.z*atom1.quadrupoleYZ;
    qDotDelta.z = deltaR.x*atom1.quadrupoleXZ + deltaR.y*atom1.quadrupoleYZ + deltaR.z*atom1.quadrupoleZZ;
 
    dotdd = dot(deltaR, atom1.dipole);
    dotqd = dot(deltaR, qDotDelta);
    factor = rr3*atom1.posq.w + rr5*dotdd + rr7*dotqd;
 
    real3 field2 = deltaR*factor - rr3*atom1.dipole - rr5_2*qDotDelta;
    fields[2] = dScale*field2;
    fields[3] = pScale*field2;
}
#endif

__device__ real computeDScaleFactor(unsigned int polarizationGroup) {
    return (polarizationGroup & 1 ? 0 : 1);
}

__device__ float computePScaleFactor(uint2 covalent, unsigned int polarizationGroup) {
    bool x = (covalent.x & 1);
    bool y = (covalent.y & 1);
    bool p = (polarizationGroup & 1);
    return (x && y ? 0.0f : (x && p ? 0.5f : 1.0f));
}

/**
 * Compute nonbonded interactions.
 */
extern "C" __global__ void computeFixedField(
        unsigned long long* __restrict__ fieldBuffers, unsigned long long* __restrict__ fieldPolarBuffers, const real4* __restrict__ posq,
        const unsigned int* __restrict__ exclusionIndices, const unsigned int* __restrict__ exclusionRowIndices,
        const uint2* __restrict__ covalentFlags, const unsigned int* __restrict__ polarizationGroupFlags, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const ushort2* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, unsigned int maxTiles, const unsigned int* __restrict__ interactionFlags,
#endif
        const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const float2* __restrict__ dampingAndThole) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    unsigned int pos = (numTiles > maxTiles ? startTileIndex+warp*numTileIndices/totalWarps : warp*numTiles/totalWarps);
    unsigned int end = (numTiles > maxTiles ? startTileIndex+(warp+1)*numTileIndices/totalWarps : (warp+1)*numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    unsigned int pos = startTileIndex+warp*numTiles/totalWarps;
    unsigned int end = startTileIndex+(warp+1)*numTiles/totalWarps;
#endif
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
    __shared__ unsigned int exclusionRange[2*WARPS_PER_GROUP];
    __shared__ int exclusionIndex[WARPS_PER_GROUP];
#ifndef ENABLE_SHUFFLE
    __shared__ real tempBuffer[3*THREAD_BLOCK_SIZE];
#endif
    
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        const unsigned int localGroupIndex = threadIdx.x/TILE_SIZE;
        unsigned int x, y;
        AtomData data;
        data.field = make_real3(0);
        data.fieldPolar = make_real3(0);
        if (pos < end) {
#ifdef USE_CUTOFF
            if (numTiles <= maxTiles) {
                ushort2 tileIndices = tiles[pos];
                x = tileIndices.x;
                y = tileIndices.y;
            }
            else
#endif
            {
                y = (unsigned int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                    y += (x < y ? -1 : 1);
                    x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                }
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, dampingAndThole);
            
            // Locate the exclusion data for this tile.

            if (tgx < 2)
                exclusionRange[2*localGroupIndex+tgx] = exclusionRowIndices[x+tgx];
            if (tgx == 0)
                exclusionIndex[localGroupIndex] = -1;
            for (unsigned int i = exclusionRange[2*localGroupIndex]+tgx; i < exclusionRange[2*localGroupIndex+1]; i += TILE_SIZE)
                if (exclusionIndices[i] == y)
                    exclusionIndex[localGroupIndex] = i*TILE_SIZE;
            bool hasExclusions = (exclusionIndex[localGroupIndex] > -1);
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                const unsigned int localAtomIndex = threadIdx.x;
                localData[localAtomIndex].posq = data.posq;
                localData[localAtomIndex].dipole = data.dipole;
                localData[localAtomIndex].quadrupoleXX = data.quadrupoleXX;
                localData[localAtomIndex].quadrupoleXY = data.quadrupoleXY;
                localData[localAtomIndex].quadrupoleXZ = data.quadrupoleXZ;
                localData[localAtomIndex].quadrupoleYY = data.quadrupoleYY;
                localData[localAtomIndex].quadrupoleYZ = data.quadrupoleYZ;
                localData[localAtomIndex].quadrupoleZZ = data.quadrupoleZZ;
                localData[localAtomIndex].thole = data.thole; // IS THIS CORRECT?
                localData[localAtomIndex].damp = data.damp; // IS THIS CORRECT?
                uint2 covalent = covalentFlags[exclusionIndex[localGroupIndex]+tgx];
                unsigned int polarizationGroup = polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx];
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+j;
                    real3 delta = make_real3(localData[atom2].posq.x-data.posq.x, localData[atom2].posq.y-data.posq.y, localData[atom2].posq.z-data.posq.z);
#ifdef USE_PERIODIC
                    delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                    delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                    delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                    real3 fields[4];
                    float d = computeDScaleFactor(polarizationGroup);
                    float p = computePScaleFactor(covalent, polarizationGroup);
                    computeOneInteraction(data, localData[atom2], delta, d, p, fields);
                    atom2 = y*TILE_SIZE+j;
                    if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        data.field += fields[0];
                        data.fieldPolar += fields[1];
                    }
                    covalent.x >>= 1;
                    covalent.y >>= 1;
                    polarizationGroup >>= 1;
                }
            }
            else {
                // This is an off-diagonal tile.

                const unsigned int localAtomIndex = threadIdx.x;
                unsigned int j = y*TILE_SIZE + tgx;
                loadAtomData(localData[localAtomIndex], j, posq, labFrameDipole, labFrameQuadrupole, dampingAndThole);
                localData[localAtomIndex].field = make_real3(0);
                localData[localAtomIndex].fieldPolar = make_real3(0);
#ifdef USE_CUTOFF
                unsigned int flags = (numTiles <= maxTiles ? interactionFlags[pos] : 0xFFFFFFFF);
                if (!hasExclusions && flags != 0xFFFFFFFF) {
                    if (flags == 0) {
                        // No interactions in this tile.
                    }
                    else {
                        // Compute only a subset of the interactions in this tile.

                        for (j = 0; j < TILE_SIZE; j++) {
                            if ((flags&(1<<j)) != 0) {
                                int atom2 = tbx+j;
                                real3 delta = make_real3(localData[atom2].posq.x-data.posq.x, localData[atom2].posq.y-data.posq.y, localData[atom2].posq.z-data.posq.z);
#ifdef USE_PERIODIC
                                delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                                delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                                delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                                real3 fields[4];
                                computeOneInteraction(data, localData[atom2], delta, 1, 1, fields);
                                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
#ifdef ENABLE_SHUFFLE
                                    for (int i = 16; i >= 1; i /= 2) {
                                        fields[2].x += __shfl_xor(fields[2].x, i, 32);
                                        fields[2].y += __shfl_xor(fields[2].y, i, 32);
                                        fields[2].z += __shfl_xor(fields[2].z, i, 32);
                                        fields[3].x += __shfl_xor(fields[3].x, i, 32);
                                        fields[3].y += __shfl_xor(fields[3].y, i, 32);
                                        fields[3].z += __shfl_xor(fields[3].z, i, 32);
                                    }
                                    if (tgx == 0) {
                                        localData[atom2].field += fields[2];
                                        localData[atom2].fieldPolar += fields[3];
                                    }
#else
                                    int bufferIndex = 3*threadIdx.x;
                                    tempBuffer[bufferIndex] = fields[2].x;
                                    tempBuffer[bufferIndex+1] = fields[2].y;
                                    tempBuffer[bufferIndex+2] = fields[2].z;
                                    if (tgx % 4 == 0) {
                                        tempBuffer[bufferIndex] += tempBuffer[bufferIndex+3]+tempBuffer[bufferIndex+6]+tempBuffer[bufferIndex+9];
                                        tempBuffer[bufferIndex+1] += tempBuffer[bufferIndex+4]+tempBuffer[bufferIndex+7]+tempBuffer[bufferIndex+10];
                                        tempBuffer[bufferIndex+2] += tempBuffer[bufferIndex+5]+tempBuffer[bufferIndex+8]+tempBuffer[bufferIndex+11];
                                    }
                                    if (tgx == 0) {
                                        localData[atom2].field.x += tempBuffer[bufferIndex]+tempBuffer[bufferIndex+12]+tempBuffer[bufferIndex+24]+tempBuffer[bufferIndex+36]+tempBuffer[bufferIndex+48]+tempBuffer[bufferIndex+60]+tempBuffer[bufferIndex+72]+tempBuffer[bufferIndex+84];
                                        localData[atom2].field.y += tempBuffer[bufferIndex+1]+tempBuffer[bufferIndex+13]+tempBuffer[bufferIndex+25]+tempBuffer[bufferIndex+37]+tempBuffer[bufferIndex+49]+tempBuffer[bufferIndex+61]+tempBuffer[bufferIndex+73]+tempBuffer[bufferIndex+85];
                                        localData[atom2].field.z += tempBuffer[bufferIndex+2]+tempBuffer[bufferIndex+14]+tempBuffer[bufferIndex+26]+tempBuffer[bufferIndex+38]+tempBuffer[bufferIndex+50]+tempBuffer[bufferIndex+62]+tempBuffer[bufferIndex+74]+tempBuffer[bufferIndex+86];
                                    }
                                    tempBuffer[bufferIndex] = fields[3].x;
                                    tempBuffer[bufferIndex+1] = fields[3].y;
                                    tempBuffer[bufferIndex+2] = fields[3].z;
                                    if (tgx % 4 == 0) {
                                        tempBuffer[bufferIndex] += tempBuffer[bufferIndex+3]+tempBuffer[bufferIndex+6]+tempBuffer[bufferIndex+9];
                                        tempBuffer[bufferIndex+1] += tempBuffer[bufferIndex+4]+tempBuffer[bufferIndex+7]+tempBuffer[bufferIndex+10];
                                        tempBuffer[bufferIndex+2] += tempBuffer[bufferIndex+5]+tempBuffer[bufferIndex+8]+tempBuffer[bufferIndex+11];
                                    }
                                    if (tgx == 0) {
                                        localData[atom2].fieldPolar.x += tempBuffer[bufferIndex]+tempBuffer[bufferIndex+12]+tempBuffer[bufferIndex+24]+tempBuffer[bufferIndex+36]+tempBuffer[bufferIndex+48]+tempBuffer[bufferIndex+60]+tempBuffer[bufferIndex+72]+tempBuffer[bufferIndex+84];
                                        localData[atom2].fieldPolar.y += tempBuffer[bufferIndex+1]+tempBuffer[bufferIndex+13]+tempBuffer[bufferIndex+25]+tempBuffer[bufferIndex+37]+tempBuffer[bufferIndex+49]+tempBuffer[bufferIndex+61]+tempBuffer[bufferIndex+73]+tempBuffer[bufferIndex+85];
                                        localData[atom2].fieldPolar.z += tempBuffer[bufferIndex+2]+tempBuffer[bufferIndex+14]+tempBuffer[bufferIndex+26]+tempBuffer[bufferIndex+38]+tempBuffer[bufferIndex+50]+tempBuffer[bufferIndex+62]+tempBuffer[bufferIndex+74]+tempBuffer[bufferIndex+86];
                                    }
#endif
                                }
                            }
                        }
                    }
                }
                else
#endif
                {
                    // Compute the full set of interactions in this tile.

                    uint2 covalent = (hasExclusions ? covalentFlags[exclusionIndex[localGroupIndex]+tgx] : make_uint2(0, 0));
                    unsigned int polarizationGroup = (hasExclusions ? polarizationGroupFlags[exclusionIndex[localGroupIndex]+tgx] : 0);
                    covalent.x = (covalent.x >> tgx) | (covalent.x << (TILE_SIZE - tgx));
                    covalent.y = (covalent.y >> tgx) | (covalent.y << (TILE_SIZE - tgx));
                    polarizationGroup = (polarizationGroup >> tgx) | (polarizationGroup << (TILE_SIZE - tgx));
                    unsigned int tj = tgx;
                    for (j = 0; j < TILE_SIZE; j++) {
                        int atom2 = tbx+tj;
                        real3 delta = make_real3(localData[atom2].posq.x-data.posq.x, localData[atom2].posq.y-data.posq.y, localData[atom2].posq.z-data.posq.z);
#ifdef USE_PERIODIC
                        delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                        delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                        delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                        real3 fields[4];
                        float d = computeDScaleFactor(polarizationGroup);
                        float p = computePScaleFactor(covalent, polarizationGroup);
                        computeOneInteraction(data, localData[atom2], delta, d, p, fields);
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            data.field += fields[0];
                            data.fieldPolar += fields[1];
                            localData[atom2].field += fields[2];
                            localData[atom2].fieldPolar += fields[3];
                        }
                        covalent.x >>= 1;
                        covalent.y >>= 1;
                        polarizationGroup >>= 1;
                        tj = (tj + 1) & (TILE_SIZE - 1);
                    }
                }
            }
        }
        
        // Write results.
        
        if (pos < end) {
            const unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (data.field.x*0xFFFFFFFF)));
            atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0xFFFFFFFF)));
            atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset], static_cast<unsigned long long>((long long) (data.fieldPolar.x*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.y*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.z*0xFFFFFFFF)));
        }
        if (pos < end && x != y) {
            const unsigned int offset = y*TILE_SIZE + tgx;
            atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.x*0xFFFFFFFF)));
            atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.y*0xFFFFFFFF)));
            atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.z*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.x*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.y*0xFFFFFFFF)));
            atomicAdd(&fieldPolarBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.z*0xFFFFFFFF)));
        }
        pos++;
    } while (pos < end);
}
