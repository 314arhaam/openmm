#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float fx;
    float fy;
    float fz;
    float fb;
};


static __constant__ cudaGmxSimulation cSim;

extern "C"
void SetCalculateObcGbsaSoftcoreForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateObcGbsaSoftcoreForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

void kCalculateObcGbsaSoftcoreForces2(gpuContext gpu)
{
    //printf("kCalculateObcGbsaSoftcoreForces2\n");
    //fprintf( stderr, "kCalculateObcGbsaSoftcoreForces2 nonbondedMethod=%d warp=%d\n", gpu->sim.nonbondedMethod, gpu->bOutputBufferPerWarp);
//fprintf( stderr, "kCalculateObcGbsaSoftcoreForces2 nonbondedMethod=%d calling kReduceForces\n", gpu->sim.nonbondedMethod);
//kReduceForces(gpu);
    switch (gpu->sim.nonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcoreN2ByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit);
            else
                kCalculateObcGbsaSoftcoreN2Forces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit);
            break;
        case CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcoreCutoffByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateObcGbsaSoftcoreCutoffForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            break;
        case PERIODIC:
            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcorePeriodicByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateObcGbsaSoftcorePeriodicForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            break;
    }
    LAUNCHERROR("kCalculateObcGbsaSoftcoreForces2");
}
