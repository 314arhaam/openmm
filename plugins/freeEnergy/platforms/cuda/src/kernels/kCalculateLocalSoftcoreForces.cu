#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "GpuLJ14Softcore.h"
#include "GpuFreeEnergyCudaKernels.h"
//#include <hip/hip_runtime.h>

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaFreeEnergySimulationNonbonded14 feSim;

/* Cuda compiler on Windows does not recognized "static const float" values */
#define LOCAL_HACK_PI 3.1415926535897932384626433832795

#define DOT3(v1, v2) (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z)

#define GETNORMEDDOTPRODUCT(v1, v2, dp) \
{ \
    dp          = DOT3(v1, v2); \
    float norm1 = DOT3(v1, v1); \
    float norm2 = DOT3(v2, v2); \
    dp /= sqrt(norm1 * norm2); \
    dp = min(dp, 1.0f); \
    dp = max(dp, -1.0f); \
}

#define CROSS_PRODUCT(v1, v2, c) \
    c.x = v1.y * v2.z - v1.z * v2.y; \
    c.y = v1.z * v2.x - v1.x * v2.z; \
    c.z = v1.x * v2.y - v1.y * v2.x;

#define GETPREFACTORSGIVENANGLECOSINE(cosine, param, dEdR) \
{ \
   float angle          = acos(cosine); \
   float deltaIdeal     = angle - (param.x * (LOCAL_HACK_PI / 180.0f)); \
   dEdR                 = param.y * deltaIdeal; \
}

#define GETENERGYGIVENANGLECOSINE(cosine, param, dEdR) \
{ \
   float angle          = acos(cosine); \
   float deltaIdeal     = angle - (param.x * (LOCAL_HACK_PI / 180.0f)); \
   dEdR                 = param.y * deltaIdeal * deltaIdeal; \
}

#define GETANGLEBETWEENTWOVECTORS(v1, v2, angle) \
{ \
    float dp; \
    GETNORMEDDOTPRODUCT(v1, v2, dp); \
    angle = acos(dp); \
}

#define GETANGLECOSINEBETWEENTWOVECTORS(v1, v2, angle, cosine) \
{ \
    GETNORMEDDOTPRODUCT(v1, v2, cosine); \
    angle = acos(cosine); \
}

#define GETDIHEDRALANGLEBETWEENTHREEVECTORS(vector1, vector2, vector3, signVector, cp0, cp1, angle) \
{ \
    CROSS_PRODUCT(vector1, vector2, cp0); \
    CROSS_PRODUCT(vector2, vector3, cp1); \
    GETANGLEBETWEENTWOVECTORS(cp0, cp1, angle); \
    float dp = DOT3(signVector, cp1); \
    angle = (dp >= 0) ? angle : -angle; \
}                                                          

#define GETDIHEDRALANGLECOSINEBETWEENTHREEVECTORS(vector1, vector2, vector3, signVector, cp0, cp1, angle, cosine) \
{ \
    CROSS_PRODUCT(vector1, vector2, cp0); \
    CROSS_PRODUCT(vector2, vector3, cp1); \
    GETANGLECOSINEBETWEENTWOVECTORS(cp0, cp1, angle, cosine); \
    float dp = DOT3(signVector, cp1); \
    angle = (dp >= 0) ? angle : -angle; \
}

void SetCalculateLocalSoftcoreGpuSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateLocalSoftcoreGpuSim copy to cSim failed");

}

void SetCalculateLocalSoftcoreSim( GpuLJ14Softcore* gpuLJ14Softcore)
{
    hipError_t status;

    status = hipMemcpyToSymbol(HIP_SYMBOL(feSim), &gpuLJ14Softcore->feSim, sizeof(cudaFreeEnergySimulationNonbonded14));     
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateLocalSoftcoreSim copy to cSim failed");
}

void GetCalculateLocalSoftcoreForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: GetCalculateLocalSoftcoreForcesSim copy from cSim failed");
}
    
#define USE_SOFTCORE_LJ
#ifdef USE_SOFTCORE_LJ
#include "kSoftcoreLJ.h"
#endif

__global__ void kCalculateLocalSoftcoreForces_kernel()
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    //Vectors* A       = &sV[threadIdx.x];

    float energy = 0.0f;

#if 0
    while (pos < cSim.bond_offset)
    {
        if (pos < cSim.bonds)
        {
            int4   atom         = cSim.pBondID[pos];
            float4 atomA        = cSim.pPosq[atom.x];
            float4 atomB        = cSim.pPosq[atom.y];
            float2 bond         = cSim.pBondParameter[pos];
            float dx            = atomB.x - atomA.x;
            float dy            = atomB.y - atomA.y;
            float dz            = atomB.z - atomA.z;
            float r2            = dx * dx + dy * dy + dz * dz;
            float r             = sqrt(r2);
            float deltaIdeal    = r - bond.x;
/* E */     energy             += 0.5f * bond.y * deltaIdeal * deltaIdeal;
            float dEdR          = bond.y * deltaIdeal;
            dEdR                = (r > 0.0f) ? (dEdR / r) : 0.0f;
//            printf("D: %11.4f %11.4f %11.4f %11.4f %11.4f %11.4f\n", dx, dy, dz, r, deltaIdeal, dEdR);
            dx                 *= dEdR;
            dy                 *= dEdR;
            dz                 *= dEdR;
            unsigned int offsetA                = atom.x + atom.z * cSim.stride;
            unsigned int offsetB                = atom.y + atom.w * cSim.stride;
            float4 forceA                       = cSim.pForce4[offsetA];
            float4 forceB                       = cSim.pForce4[offsetB];
            forceA.x                           += dx;
            forceA.y                           += dy;
            forceA.z                           += dz;
            forceB.x                           -= dx;
            forceB.y                           -= dy;
            forceB.z                           -= dz;
            cSim.pForce4[offsetA]               = forceA;
            cSim.pForce4[offsetB]               = forceB;    
        }
        pos += blockDim.x * gridDim.x;
    }

    while (pos < cSim.bond_angle_offset)
    {
        unsigned int pos1   = pos - cSim.bond_offset;
        if (pos1 < cSim.bond_angles)
        {
            int4   atom1            = cSim.pBondAngleID1[pos1];  
            float2 bond_angle       = cSim.pBondAngleParameter[pos1];
            float4 a1               = cSim.pPosq[atom1.x];
            float4 a2               = cSim.pPosq[atom1.y];
            float4 a3               = cSim.pPosq[atom1.z];
            A->v0.x                 = a2.x - a1.x;
            A->v0.y                 = a2.y - a1.y;
            A->v0.z                 = a2.z - a1.z;
            A->v1.x                 = a2.x - a3.x;
            A->v1.y                 = a2.y - a3.y;
            A->v1.z                 = a2.z - a3.z;
            float3 cp;
            CROSS_PRODUCT(A->v0, A->v1, cp);
            float rp                = DOT3(cp, cp); //cx * cx + cy * cy + cz * cz;
            rp                      = max(sqrt(rp), 1.0e-06f);
            float r21               = DOT3(A->v0, A->v0); // dx1 * dx1 + dy1 * dy1 + dz1 * dz1;
            float r23               = DOT3(A->v1, A->v1); // dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
            float dot               = DOT3(A->v0, A->v1); // dx1 * dx2 + dy1 * dy2 + dz1 * dz2;
            float cosine            = dot / sqrt(r21 * r23);

            float angle_energy;
/* E */     GETENERGYGIVENANGLECOSINE(cosine, bond_angle, angle_energy);
            energy                 += 0.5f*angle_energy;

            float dEdR;
            GETPREFACTORSGIVENANGLECOSINE(cosine, bond_angle, dEdR);
            //printf("%11.4f %11.4f\n", cosine, dEdR);
            float termA             =  dEdR / (r21 * rp);
            float termC             = -dEdR / (r23 * rp);
            float3 c21;
            float3 c23;
            CROSS_PRODUCT(A->v0, cp, c21);
            CROSS_PRODUCT(A->v1, cp, c23);
            c21.x                  *= termA;
            c21.y                  *= termA;
            c21.z                  *= termA;
            c23.x                  *= termC;
            c23.y                  *= termC;
            c23.z                  *= termC;
            int2 atom2              = cSim.pBondAngleID2[pos1];
            unsigned int offset     = atom1.x + atom1.w * cSim.stride;
            float4 force            = cSim.pForce4[offset]; 
            force.x                += c21.x;
            force.y                += c21.y;
            force.z                += c21.z;
            cSim.pForce4[offset]    = force;
            offset                  = atom1.y + atom2.x * cSim.stride;
            force                   = cSim.pForce4[offset];
            force.x                -= (c21.x + c23.x);
            force.y                -= (c21.y + c23.y);
            force.z                -= (c21.z + c23.z);
            cSim.pForce4[offset]    = force;
            offset                  = atom1.z + atom2.y * cSim.stride;
            force                   = cSim.pForce4[offset];
            force.x                += c23.x;
            force.y                += c23.y;
            force.z                += c23.z;
            cSim.pForce4[offset]    = force;
        }
        pos += blockDim.x * gridDim.x;
    }

    while (pos < cSim.dihedral_offset)
    {
        unsigned int pos1 = pos - cSim.bond_angle_offset;
        if (pos1 < cSim.dihedrals)
        {
            int4   atom1        = cSim.pDihedralID1[pos1];  
            float4 atomA        = cSim.pPosq[atom1.x];
            float4 atomB        = cSim.pPosq[atom1.y];
            float4 atomC        = cSim.pPosq[atom1.z];
            float4 atomD        = cSim.pPosq[atom1.w];            
            A->v0.x             = atomA.x - atomB.x;
            A->v0.y             = atomA.y - atomB.y;
            A->v0.z             = atomA.z - atomB.z;
            A->v1.x             = atomC.x - atomB.x;
            A->v1.y             = atomC.y - atomB.y;
            A->v1.z             = atomC.z - atomB.z;
            A->v2.x             = atomC.x - atomD.x;
            A->v2.y             = atomC.y - atomD.y;
            A->v2.z             = atomC.z - atomD.z; 
            float3 cp0, cp1;
            float dihedralAngle;
            GETDIHEDRALANGLEBETWEENTHREEVECTORS(A->v0, A->v1, A->v2, A->v0, cp0, cp1, dihedralAngle);
            float4 dihedral         = cSim.pDihedralParameter[pos1];
            float deltaAngle        = dihedral.z * dihedralAngle - (dihedral.y * PI / 180.0f);

	    // ATTENTION: This section leads to a divergent deltaAngle values wrt
	    // forces and energies. We separate the case dihedral.z = n = 0, which
	    // is treated by the calculation of energies via a harmonic potential
/* E */     if (dihedral.z) energy += dihedral.x * (1.0f + cos(deltaAngle));
/* E */     else
	    {
		float deltaAngle    = dihedralAngle - dihedral.y;
		if (deltaAngle < -PI) deltaAngle += 2.0f * PI;
		else if (deltaAngle > PI) deltaAngle -= 2.0f * PI;
                energy             += dihedral.x * deltaAngle * deltaAngle;
	    }

            float sinDeltaAngle     = sin(deltaAngle);
            float dEdAngle          = -dihedral.x * dihedral.z * sinDeltaAngle;
            float normCross1        = DOT3(cp0, cp0);
            float normBC            = sqrt(DOT3(A->v1, A->v1));
            float4 ff;
            ff.x                    = (-dEdAngle * normBC) / normCross1;
            float normCross2        = DOT3(cp1, cp1);
            ff.w                    = (dEdAngle * normBC) / normCross2;
            float dp                = 1.0f / DOT3(A->v1, A->v1);
            ff.y                    = DOT3(A->v0, A->v1) * dp;
            ff.z                    = DOT3(A->v2, A->v1) * dp;
            int4  atom2             = cSim.pDihedralID2[pos1];   
            float3 internalF0;
            float3 internalF3;
            float3 s;
            
//            printf("%4d: %9.4f %9.4f %9.4f %9.4f\n", pos1, ff.x, ff.y, ff.z, ff.w);  
            unsigned int offset                 = atom1.x + atom2.x * cSim.stride;
            float4 force                        = cSim.pForce4[offset]; 
            internalF0.x                        = ff.x * cp0.x; 
            force.x                            += internalF0.x;
            internalF0.y                        = ff.x * cp0.y;
            force.y                            += internalF0.y;
            internalF0.z                        = ff.x * cp0.z;       
            force.z                            += internalF0.z;
            cSim.pForce4[offset]                = force;
            
            //printf("%4d - 0: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.w + atom2.w * cSim.stride;
            force                               = cSim.pForce4[offset];
            internalF3.x                        = ff.w * cp1.x;
            force.x                            += internalF3.x;
            internalF3.y                        = ff.w * cp1.y;
            force.y                            += internalF3.y;
            internalF3.z                        = ff.w * cp1.z;
            force.z                            += internalF3.z;
            cSim.pForce4[offset]                = force;
            
           // printf("%4d - 3: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            s.x                                 = ff.y * internalF0.x - ff.z * internalF3.x;   
            s.y                                 = ff.y * internalF0.y - ff.z * internalF3.y;  
            s.z                                 = ff.y * internalF0.z - ff.z * internalF3.z;        
            offset                              = atom1.y + atom2.y * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF0.x + s.x;
            force.y                            += -internalF0.y + s.y;
            force.z                            += -internalF0.z + s.z;
            cSim.pForce4[offset]                = force;
            
            //printf("%4d - 1: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.z + atom2.z * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF3.x - s.x;
            force.y                            += -internalF3.y - s.y;
            force.z                            += -internalF3.z - s.z;
            cSim.pForce4[offset]                = force;
            //printf("%4d - 2: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
        }
        pos += blockDim.x * gridDim.x;
    }

    // Ryckaert Bellemans dihedrals
    while (pos < cSim.rb_dihedral_offset)
    {
        unsigned int pos1 = pos - cSim.dihedral_offset;
        if (pos1 < cSim.rb_dihedrals)
        {
            int4   atom1        = cSim.pRbDihedralID1[pos1];
            float4 atomA        = cSim.pPosq[atom1.x];
            float4 atomB        = cSim.pPosq[atom1.y];
            float4 atomC        = cSim.pPosq[atom1.z];
            float4 atomD        = cSim.pPosq[atom1.w];
            A->v0.x             = atomA.x - atomB.x;
            A->v0.y             = atomA.y - atomB.y;
            A->v0.z             = atomA.z - atomB.z;
            A->v1.x             = atomC.x - atomB.x;
            A->v1.y             = atomC.y - atomB.y;
            A->v1.z             = atomC.z - atomB.z;
            A->v2.x             = atomC.x - atomD.x;
            A->v2.y             = atomC.y - atomD.y;
            A->v2.z             = atomC.z - atomD.z;
            float3 cp0, cp1;
            float dihedralAngle, cosPhi;
      //      printf("%4d - 0 : %9.4f %9.4f %9.4f\n", pos1, A->v0.x, A->v0.y, A->v0.z);
      //      printf("%4d - 1 : %9.4f %9.4f %9.4f\n", pos1, A->v1.x, A->v1.y, A->v1.z);
      //      printf("%4d - 2 : %9.4f %9.4f %9.4f\n", pos1, A->v2.x, A->v2.y, A->v2.z);
            GETDIHEDRALANGLECOSINEBETWEENTHREEVECTORS(A->v0, A->v1, A->v2, A->v0, cp0, cp1, dihedralAngle, cosPhi);
            if (dihedralAngle < 0.0f )
            {
                dihedralAngle += PI;
            }
            else
            {
                dihedralAngle -= PI;
            }
            cosPhi                  = -cosPhi;
         //   printf("%4d: %9.4f %9.4f\n", pos1, dihedralAngle, cosPhi);
            float4 dihedral1        = cSim.pRbDihedralParameter1[pos1];
            float2 dihedral2        = cSim.pRbDihedralParameter2[pos1];
            float cosFactor         = cosPhi;
            float dEdAngle          = -dihedral1.y;

/* E */     float rb_energy         = dihedral1.x;
            rb_energy              += dihedral1.y * cosFactor;
        //    printf("%4d - 1: %9.4f %9.4f\n", pos1, dEdAngle, 1.0f);
            dEdAngle               -= 2.0f * dihedral1.z * cosFactor;
       //     printf("%4d - 2: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 3.0f * dihedral1.w * cosFactor;
            rb_energy              += dihedral1.z * cosFactor;
    //       printf("%4d - 3: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 4.0f * dihedral2.x * cosFactor;
            rb_energy              += dihedral1.w * cosFactor;
  //         printf("%4d - 4: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 5.0f * dihedral2.y * cosFactor;
            rb_energy              += dihedral2.x * cosFactor;
            rb_energy              += dihedral2.y * cosFactor * cosPhi;
/* E */     energy                 += rb_energy;
 //           printf("%4d - 5: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            dEdAngle               *= sin(dihedralAngle);
//            printf("%4d - f: %9.4f\n", pos1, dEdAngle);

            float normCross1        = DOT3(cp0, cp0);
            float normBC            = sqrt(DOT3(A->v1, A->v1));
            float4 ff;
            ff.x                    = (-dEdAngle * normBC) / normCross1;
            float normCross2        = DOT3(cp1, cp1);
            ff.w                    = (dEdAngle * normBC) / normCross2;
            float dp                = 1.0f / DOT3(A->v1, A->v1);
            ff.y                    = DOT3(A->v0, A->v1) * dp;
            ff.z                    = DOT3(A->v2, A->v1) * dp;
            int4  atom2             = cSim.pRbDihedralID2[pos1];
            float3 internalF0;
            float3 internalF3;
            float3 s;

//            printf("%4d: %9.4f %9.4f %9.4f %9.4f\n", pos1, ff.x, ff.y, ff.z, ff.w);
            unsigned int offset                 = atom1.x + atom2.x * cSim.stride;
            float4 force                        = cSim.pForce4[offset];
            internalF0.x                        = ff.x * cp0.x;
            force.x                            += internalF0.x;
            internalF0.y                        = ff.x * cp0.y;
            force.y                            += internalF0.y;
            internalF0.z                        = ff.x * cp0.z;
            force.z                            += internalF0.z;
            cSim.pForce4[offset]                = force;

 //           printf("%4d - 0: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.w + atom2.w * cSim.stride;
            force                               = cSim.pForce4[offset];
            internalF3.x                        = ff.w * cp1.x;
            force.x                            += internalF3.x;
            internalF3.y                        = ff.w * cp1.y;
            force.y                            += internalF3.y;
            internalF3.z                        = ff.w * cp1.z;
            force.z                            += internalF3.z;
            cSim.pForce4[offset]                = force;

   //         printf("%4d - 3: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            s.x                                 = ff.y * internalF0.x - ff.z * internalF3.x;
            s.y                                 = ff.y * internalF0.y - ff.z * internalF3.y;
            s.z                                 = ff.y * internalF0.z - ff.z * internalF3.z;
            offset                              = atom1.y + atom2.y * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF0.x + s.x;
            force.y                            += -internalF0.y + s.y;
            force.z                            += -internalF0.z + s.z;
            cSim.pForce4[offset]                = force;
     //       printf("%4d - 1: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.z + atom2.z * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF3.x - s.x;
            force.y                            += -internalF3.y - s.y;
            force.z                            += -internalF3.z - s.z;
            cSim.pForce4[offset]                = force;
     //       printf("%4d - 2: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
        }         
        pos += blockDim.x * gridDim.x;
    }   
#endif

    if (cSim.nonbondedMethod == NO_CUTOFF)
    {
        while (pos < feSim.LJ14_offset)
        {
            //unsigned int pos1       = pos - feSim.rb_dihedral_offset;
            unsigned int pos1       = pos;
            if (pos1 < feSim.LJ14s)
            {
                int4 atom               = feSim.pLJ14ID[pos1];
                float4 LJ14             = feSim.pLJ14Parameter[pos1];
                float4 a1               = cSim.pPosq[atom.x];
                float4 a2               = cSim.pPosq[atom.y];
                float3 d;
                d.x                     = a1.x - a2.x;
                d.y                     = a1.y - a2.y;
                d.z                     = a1.z - a2.z;
                float r2                = DOT3(d, d);
                float inverseR          = 1.0f / sqrt(r2);
#ifdef USE_SOFTCORE_LJ
                float CDLJ_energy       = 0.0f;
                float dEdR              = getSoftCoreLJ( r2, LJ14.y, LJ14.x, LJ14.w, LJ14.w, &CDLJ_energy );
                energy                 += CDLJ_energy;
#else
                float sig2              = inverseR * LJ14.y;
                sig2                   *= sig2;
                float sig6              = sig2 * sig2 * sig2;
                float dEdR              = LJ14.x * (12.0f * sig6 - 6.0f) * sig6;
                energy                 += LJ14.x * (sig6 - 1.0f) * sig6;
#endif
                energy                 += LJ14.z * inverseR;
                dEdR                   += LJ14.z * inverseR;
                dEdR                   *= inverseR * inverseR;
                unsigned int offsetA    = atom.x + atom.z * cSim.stride;
                unsigned int offsetB    = atom.y + atom.w * cSim.stride;
                float4 forceA           = cSim.pForce4[offsetA];
                float4 forceB           = cSim.pForce4[offsetB];
                d.x                    *= dEdR;
                d.y                    *= dEdR;
                d.z                    *= dEdR;
                forceA.x               += d.x;
                forceA.y               += d.y;
                forceA.z               += d.z;
                forceB.x               -= d.x;
                forceB.y               -= d.y;
                forceB.z               -= d.z;
                cSim.pForce4[offsetA]   = forceA;
                cSim.pForce4[offsetB]   = forceB;
            }
            pos                    += blockDim.x * gridDim.x;
        }
    }
    else if (cSim.nonbondedMethod == CUTOFF)
    {
        float LJ14_energy;
        while (pos < feSim.LJ14_offset)
        {
            //unsigned int pos1       = pos - feSim.rb_dihedral_offset;
            unsigned int pos1       = pos;
            if (pos1 < feSim.LJ14s)
            {
                int4 atom               = feSim.pLJ14ID[pos1];
                float4 LJ14             = feSim.pLJ14Parameter[pos1];
                float4 a1               = cSim.pPosq[atom.x];
                float4 a2               = cSim.pPosq[atom.y];
                float3 d;
                d.x                     = a1.x - a2.x;
                d.y                     = a1.y - a2.y;
                d.z                     = a1.z - a2.z;
                float r2                = DOT3(d, d);
                float inverseR          = 1.0f / sqrt(r2);
#ifdef USE_SOFTCORE_LJ
                float dEdR              = getSoftCoreLJ( r2, LJ14.y, LJ14.x, LJ14.w, LJ14.w, &LJ14_energy);
#else
                float sig2              = inverseR * LJ14.y;
                sig2                   *= sig2;
                float sig6              = sig2 * sig2 * sig2;
                float dEdR              = LJ14.x * (12.0f * sig6 - 6.0f) * sig6;                
                /* E */
                LJ14_energy             = LJ14.x * (sig6 - 1.0f) * sig6;
#endif
                LJ14_energy            += LJ14.z * (inverseR + cSim.reactionFieldK * r2 - cSim.reactionFieldC);
                dEdR                   += LJ14.z * (inverseR - 2.0f * cSim.reactionFieldK * r2);
                dEdR                   *= inverseR * inverseR;
                if (r2 > cSim.nonbondedCutoffSqr)
                {                   
                    dEdR = 0.0f;
                    /* E */
                    LJ14_energy = 0.0f;
                }
                /* E */
                energy                 += LJ14_energy;
 
                unsigned int offsetA    = atom.x + atom.z * cSim.stride;
                unsigned int offsetB    = atom.y + atom.w * cSim.stride;
                float4 forceA           = cSim.pForce4[offsetA];
                float4 forceB           = cSim.pForce4[offsetB];
                d.x                    *= dEdR;
                d.y                    *= dEdR;
                d.z                    *= dEdR;
                forceA.x               += d.x;
                forceA.y               += d.y;
                forceA.z               += d.z;
                forceB.x               -= d.x;
                forceB.y               -= d.y;
                forceB.z               -= d.z;
                cSim.pForce4[offsetA]   = forceA;
                cSim.pForce4[offsetB]   = forceB;
            }
            pos                    += blockDim.x * gridDim.x;
        }
    }
    else if (cSim.nonbondedMethod == PERIODIC)
    {
        float LJ14_energy;
        while (pos < feSim.LJ14_offset)
        {
            //unsigned int pos1       = pos - feSim.rb_dihedral_offset;
            unsigned int pos1       = pos;
            if (pos1 < feSim.LJ14s)
            {
                int4 atom               = feSim.pLJ14ID[pos1];
                float4 LJ14             = feSim.pLJ14Parameter[pos1];
                float4 a1               = cSim.pPosq[atom.x];
                float4 a2               = cSim.pPosq[atom.y];
                float3 d;
                d.x                     = a1.x - a2.x;
                d.y                     = a1.y - a2.y;
                d.z                     = a1.z - a2.z;
                d.x                     -= floor(d.x/cSim.periodicBoxSizeX+0.5f)*cSim.periodicBoxSizeX;
                d.y                     -= floor(d.y/cSim.periodicBoxSizeY+0.5f)*cSim.periodicBoxSizeY;
                d.z                     -= floor(d.z/cSim.periodicBoxSizeZ+0.5f)*cSim.periodicBoxSizeZ;
                float r2                = DOT3(d, d);
                float inverseR          = 1.0f / sqrt(r2);
#ifdef USE_SOFTCORE_LJ
                float dEdR              = getSoftCoreLJ( r2, LJ14.y, LJ14.x, LJ14.w, LJ14.w, &LJ14_energy);
#else
                float sig2              = inverseR * LJ14.y;
                sig2                   *= sig2;
                float sig6              = sig2 * sig2 * sig2;
                float dEdR              = LJ14.x * (12.0f * sig6 - 6.0f) * sig6;
                /* E */
                LJ14_energy             = LJ14.x * (sig6 - 1.0f) * sig6;
#endif
                LJ14_energy            += LJ14.z * (inverseR + cSim.reactionFieldK * r2 - cSim.reactionFieldC);

                dEdR                   += LJ14.z * (inverseR - 2.0f * cSim.reactionFieldK * r2);
                dEdR                   *= inverseR * inverseR;
                if (r2 > cSim.nonbondedCutoffSqr)
                {
                    dEdR = 0.0f;
                    /* E */
                    LJ14_energy = 0.0f;
                }
                /* E */
                energy                 += LJ14_energy;

                unsigned int offsetA    = atom.x + atom.z * cSim.stride;
                unsigned int offsetB    = atom.y + atom.w * cSim.stride;
                float4 forceA           = cSim.pForce4[offsetA];
                float4 forceB           = cSim.pForce4[offsetB];
                d.x                    *= dEdR;
                d.y                    *= dEdR;
                d.z                    *= dEdR;
                forceA.x               += d.x;
                forceA.y               += d.y;
                forceA.z               += d.z;
                forceB.x               -= d.x;
                forceB.y               -= d.y;
                forceB.z               -= d.z;
                cSim.pForce4[offsetA]   = forceA;
                cSim.pForce4[offsetB]   = forceB;
            }
            pos                    += blockDim.x * gridDim.x;
        }
    }
    cSim.pEnergy[blockIdx.x * blockDim.x + threadIdx.x] += energy;
}

extern "C"
GpuLJ14Softcore* gpuSetLJ14SoftcoreParameters(gpuContext gpu, float epsfac, float fudge, const std::vector<int>& atom1, const std::vector<int>& atom2,
                                              const std::vector<float>& c6, const std::vector<float>& c12, const std::vector<float>& q1,
                                              const std::vector<float>& q2, const std::vector<float>& softcoreLJLambdaArray)
{
    int LJ14s                                   = atom1.size();
    float scale                                 = epsfac * fudge;

    GpuLJ14Softcore* gpuLJ14Softcore            = new GpuLJ14Softcore();
    gpuLJ14Softcore->feSim.LJ14s                = LJ14s;

    CUDAStream<int4>* psLJ14ID                  = new CUDAStream<int4>(LJ14s, 1, "LJ14SoftcoreID");
    gpuLJ14Softcore->psLJ14SoftcoreID           = psLJ14ID;
    gpuLJ14Softcore->feSim.pLJ14ID              = psLJ14ID->_pDevStream[0];

    CUDAStream<float4>* psLJ14Parameter         = new CUDAStream<float4>(LJ14s, 1, "LJ14SoftcoreParameter");
    gpuLJ14Softcore->psLJ14SoftcoreParameter    = psLJ14Parameter;
    gpuLJ14Softcore->feSim.pLJ14Parameter       = psLJ14Parameter->_pDevStream[0];
    gpuLJ14Softcore->feSim.LJ14_offset          = LJ14s;

    for (int i = 0; i < LJ14s; i++)
    {
        (*psLJ14ID)[i].x          = atom1[i];
        (*psLJ14ID)[i].y          = atom2[i];
        psLJ14ID->_pSysData[i].z  = gpu->pOutputBufferCounter[psLJ14ID->_pSysData[i].x]++;
        psLJ14ID->_pSysData[i].w  = gpu->pOutputBufferCounter[psLJ14ID->_pSysData[i].y]++;
        float p0, p1, p2, p3;
        if (c12[i] == 0.0f)
        {
            p0 = 0.0f;
            p1 = 1.0f;
        }
        else
        {
            p0 = c6[i] * c6[i] / c12[i];
            p1 = pow(c12[i] / c6[i], 1.0f / 6.0f);
        }
        p2 = scale * q1[i] * q2[i];
        p3 = softcoreLJLambdaArray[i];
        (*psLJ14Parameter)[i].x = p0;
        (*psLJ14Parameter)[i].y = p1;
        (*psLJ14Parameter)[i].z = p2;
        (*psLJ14Parameter)[i].w = p3;
    }
#if (DUMP_PARAMETERS == 1)
        cout << 
            i << " " <<
            (*psLJ14ID)[i].x << " " <<
            (*psLJ14ID)[i].y << " " <<
            (*psLJ14ID)[i].z << " " <<
            (*psLJ14ID)[i].w << " " <<
            (*psLJ14Parameter)[i].x << " " <<
            (*psLJ14Parameter)[i].y << " " <<
            (*psLJ14Parameter)[i].z << " " <<
            (*psLJ14Parameter)[i].w << " " <<
            p0 << " " << 
            p1 << " " << 
            p2 << " " << 
            p3 << " " << 
            endl;
#endif
    psLJ14ID->Upload();
    psLJ14Parameter->Upload();
    SetCalculateLocalSoftcoreSim( gpuLJ14Softcore );

    return gpuLJ14Softcore;
}

void kCalculateLocalSoftcoreForces(gpuContext gpu)
{
  //  printf("kCalculateLocalForces\n");
//    fprintf( stderr, "kCalculateLocalSoftcoreForces blks=%u localForces_threads_per_block=%u szVector=%u total=%u\n", gpu->sim.blocks, gpu->sim.localForces_threads_per_block, sizeof(Vectors),
//             gpu->sim.localForces_threads_per_block * sizeof(Vectors) ); fflush( stderr );

    kCalculateLocalSoftcoreForces_kernel<<<gpu->sim.blocks, gpu->sim.localForces_threads_per_block, gpu->sim.localForces_threads_per_block * sizeof(Vectors)>>>();
    LAUNCHERROR("kCalculateLocalSoftcoreForces");
}

