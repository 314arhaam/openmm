#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"
#include "cudaKernels.h"

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float fx;
    float fy;
    float fz;
    float fb;
};


static __constant__ cudaGmxSimulation cSim;

void SetCalculateGBVIForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateGBVIForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

#include "kCalculateGBVIAux.h"

/**
 * This file contains the kernel for evalauating the second stage of GBSA.  It is included
 * several times in kCalculateGBVIForces2.cu with different #defines to generate
 * different versions of the kernels.
 */

__global__ void kCalculateGBVIForces2a_kernel()
{
    unsigned int pos  = (blockIdx.x * blockDim.x + threadIdx.x);
    if( pos >=  cSim.atoms )return;

    float4 apos                     = cSim.pPosq[pos];
    float4 ar                       = cSim.pGBVIData[pos];
    float fb                        = cSim.pBornForce[pos];
    unsigned int posJ               = 0;
    float4 force;
    force.x = force.y = force.z = force.w = 0.0f;
    while ( posJ < cSim.atoms )
    {

        float4 aposJ                = cSim.pPosq[posJ];
        float4 arJ                  = cSim.pGBVIData[posJ];
        float fbJ                   = cSim.pBornForce[posJ];

        float dx                    = aposJ.x - apos.x;
        float dy                    = aposJ.y - apos.y;
        float dz                    = aposJ.z - apos.z;

        float r2                    = dx * dx + dy * dy + dz * dz;
        float r                     = sqrt(r2);

        float dE                    = getGBVI_dE2( r, ar.x, arJ.y, fb );
        dE                          = r > 1.0e-08f ? dE : 0.0f;

//dx = dy = dz = 1.0f;
        float d                     = dx*dE;
        force.x                    -= d;
        d                           = dy*dE;
        force.y                    -= d;
        d                           = dz*dE;
        force.z                    -= d;
#if 1
        dE                          = getGBVI_dE2( r, arJ.x, ar.y, fbJ );
        dE                          = r > 1.0e-08f ? dE : 0.0f;
        d                           = dx*dE;
        force.x                    -= d;
        d                           = dy*dE;
        force.y                    -= d;
        d                           = dz*dE;
        force.z                    -= d;
#endif

        posJ                       += 1;
    }

    // Write results
    cSim.pForce4a[pos]             = force;

}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateGBVIForces2.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateGBVIForces2.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateGBVIForces2.h"

void kCalculateGBVIForces2(gpuContext gpu)
{
    //printf("kCalculateGBVIForces2\n");
    size_t numWithInteractions;

#if 0
    kClearForces(gpu);
    (void) fprintf( stderr, "\nkCalculateGBVIForces2: cleared force prior loop2\n" ); (void) fflush( stderr );
    kCalculateGBVIForces2a_kernel<<<gpu->sim.blocks, 384>>>();
    (void) fprintf( stderr, "\ncalled kCalculateGBVIForces2a\n" ); (void) fflush( stderr );
    return;
#endif

    switch (gpu->sim.nonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVIN2ByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit, gpu->sim.workUnits);
            else
                kCalculateGBVIN2Forces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit, gpu->sim.workUnits);
//(void) fprintf( stderr, "\nkCalculateGBVIForces2: Born radii/force forces warp=%u\n", gpu->bOutputBufferPerWarp ); (void) fflush( stderr );
#define GBVI_DEBUG 0
#if ( GBVI_DEBUG == 1 )
                (void) fprintf( stderr, "\nkCalculateGBVIForces2: Born radii/force forces:\n" ); (void) fflush( stderr );
                gpu->psBornForce->Download();
                gpu->psForce4->Download();
                for( int ii = 0; ii < gpu->natoms; ii++ ){
                    (void) fprintf( stderr, "%d bF=%14.6e Fa[%14.6e %14.6e %14.6e] Fb[%14.6e %14.6e %14.6e]\n",
                                    ii,
                                    gpu->psBornForce->_pSysStream[0][ii],
                                    gpu->psForce4->_pSysStream[0][ii].x,
                                    gpu->psForce4->_pSysStream[0][ii].y,
                                    gpu->psForce4->_pSysStream[0][ii].z,
                                    gpu->psForce4->_pSysStream[1][ii].x,
                                    gpu->psForce4->_pSysStream[1][ii].y,
                                    gpu->psForce4->_pSysStream[1][ii].z
                                  );  
                }   
                for( int ii = 0; ii < gpu->sim.paddedNumberOfAtoms*2; ii++ ){
                    (void) fprintf( stderr, "%d bF=%14.6e Fa[%14.6e %14.6e %14.6e %14.6e]\n",
                                    ii,
                                    gpu->psBornForce->_pSysStream[0][ii],
                                    gpu->psForce4->_pSysStream[0][ii].x,
                                    gpu->psForce4->_pSysStream[0][ii].y,
                                    gpu->psForce4->_pSysStream[0][ii].z,
                                    gpu->psForce4->_pSysStream[0][ii].w
                                  );  
                }   
#endif
#undef GBVI_DEBUG

            break;
        case CUTOFF:
            numWithInteractions = gpu->psInteractionCount->_pSysData[0];
            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVICutoffByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit, numWithInteractions);
            else
                kCalculateGBVICutoffForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit, numWithInteractions);
            break;
        case PERIODIC:
            numWithInteractions = gpu->psInteractionCount->_pSysData[0];
            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVIPeriodicByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit, numWithInteractions);
            else
                kCalculateGBVIPeriodicForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit, numWithInteractions);
            break;
    }
    LAUNCHERROR("kCalculateGBVIForces2");
}
