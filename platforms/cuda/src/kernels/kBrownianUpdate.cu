#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
//#include <fstream>
using namespace std;

#include "gputypes.h"

#define DeltaShake

static __constant__ cudaGmxSimulation cSim;

void SetBrownianUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetBrownianUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

__global__ void kBrownianUpdatePart1_kernel()
{
    unsigned int pos    = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rpos   = cSim.pRandomPosition[blockIdx.x];
    __syncthreads();
    
    while (pos < cSim.atoms)
    {
        float4 random4a         = cSim.pRandom4a[rpos + pos];
        float4 apos             = cSim.pPosq[pos];
        float4 force            = cSim.pForce4[pos];

        cSim.pOldPosq[pos]      = apos;
#ifndef DeltaShake
        apos.x                 += force.x*cSim.GDT + random4a.x;
        apos.y                 += force.y*cSim.GDT + random4a.y;
        apos.z                 += force.z*cSim.GDT + random4a.z;
#else
        apos.x                  = force.x*cSim.GDT + random4a.x;
        apos.y                  = force.y*cSim.GDT + random4a.y;
        apos.z                  = force.z*cSim.GDT + random4a.z;
#endif
        cSim.pPosqP[pos]        = apos;
        pos                    += blockDim.x * gridDim.x;
    }
}

void kBrownianUpdatePart1(gpuContext gpu)
{
//    printf("kBrownianUpdatePart1\n");
    kBrownianUpdatePart1_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kBrownianUpdatePart1");
}

__global__ void kBrownianUpdatePart2_kernel()
{
    unsigned int pos            = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rpos           = cSim.pRandomPosition[blockIdx.x];
    __syncthreads();
    
    while (pos < cSim.atoms)
    {
        float4 velocity         = cSim.pVelm4[pos];
        float4 apos             = cSim.pPosq[pos];
        float4 xPrime           = cSim.pPosqP[pos];

#ifndef DeltaShake
        velocity.x              = cSim.oneOverDeltaT*(xPrime.x-apos.x);
        velocity.y              = cSim.oneOverDeltaT*(xPrime.y-apos.y);
        velocity.z              = cSim.oneOverDeltaT*(xPrime.z-apos.z);
#else
        velocity.x              = cSim.oneOverDeltaT*(xPrime.x);
        velocity.y              = cSim.oneOverDeltaT*(xPrime.y);
        velocity.z              = cSim.oneOverDeltaT*(xPrime.z);

        xPrime.x               += apos.x;
        xPrime.y               += apos.y;
        xPrime.z               += apos.z;
#endif
        cSim.pPosq[pos]         = xPrime;
        cSim.pVelm4[pos]        = velocity;
         
        pos                    += blockDim.x * gridDim.x;    
    }

    // Update random position pointer
    if (threadIdx.x == 0)
    {
        rpos                   += cSim.paddedNumberOfAtoms;
        if (rpos > cSim.randoms)
            rpos               -= cSim.randoms;
        cSim.pRandomPosition[blockIdx.x] = rpos;
    }
}

extern void kGenerateRandoms(gpuContext gpu);
void kBrownianUpdatePart2(gpuContext gpu)
{
//    printf("kBrownianUpdatePart2\n");
    kBrownianUpdatePart2_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kBrownianUpdatePart2");
    
    // Update randoms if necessary
    gpu->iterations++;
    if (gpu->iterations == gpu->sim.randomIterations)
    {
        kGenerateRandoms(gpu);
        gpu->iterations = 0;
    }
}

