#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
using namespace std;

#include "gputypes.h"

static __constant__ cudaGmxSimulation cSim;

void SetLangevinUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetLangevinUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

// Include versions of the kernels with and with center of mass motion removal.

#include "kLangevinUpdate.h"
#define REMOVE_CM
#include "kLangevinUpdate.h"

void kLangevinUpdatePart1(gpuContext gpu)
{
//    printf("kLangevinUpdatePart1\n");
    if (gpu->bRemoveCM)
    {
        kLangevinUpdatePart1CM_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block, gpu->sim.update_threads_per_block * sizeof(float3)>>>();
        LAUNCHERROR("kLangevinUpdatePart1CM");
        gpu->bRemoveCM = false;
    }
    else
    {    
        kLangevinUpdatePart1_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
        LAUNCHERROR("kLangevinUpdatePart1");
    }
}

extern void kGenerateRandoms(gpuContext gpu);
void kLangevinUpdatePart2(gpuContext gpu)
{
//    printf("kLangevinUpdatePart2\n");
    if (gpu->bCalculateCM)
    {
        kLangevinUpdatePart2CM_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block, gpu->sim.update_threads_per_block * sizeof(float3)>>>();
        LAUNCHERROR("kLangevinUpdatePart2CM");
        gpu->bCalculateCM = false;
        gpu->bRemoveCM = true;
    }
    else
    {
        kLangevinUpdatePart2_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
        LAUNCHERROR("kLangevinUpdatePart2");
    }
    
    // Update randoms if necessary
    gpu->iterations++;
    if (gpu->iterations == gpu->sim.randomIterations)
    {
        kGenerateRandoms(gpu);
        gpu->iterations = 0;
    }
}

