#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float sr2;
    float fx;
    float fy;
    float fz;
    float fb;
//    float sum;
};


__shared__ Atom sA[GT2XX_BORNFORCE2_THREADS_PER_BLOCK];
__shared__ unsigned int sWorkUnit[GT2XX_NONBOND_WORKUNITS_PER_SM];
__shared__ unsigned int sNext[GRID];

static __constant__ cudaGmxSimulation cSim;

void SetCalculateObcGbsaForces2_12Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateObcGbsaForces2_12Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

__global__ void kCalculateObcGbsaForces2_12_kernel()
{
    // Read queue of work blocks once so the remainder of
    // kernel can run asynchronously    
    int pos = cSim.bf2WorkUnitsPerBlock * blockIdx.x + min(blockIdx.x, cSim.bf2WorkUnitsPerBlockRemainder);
    int end = cSim.bf2WorkUnitsPerBlock * (blockIdx.x + 1) + min((blockIdx.x + 1), cSim.bf2WorkUnitsPerBlockRemainder);    
    if (threadIdx.x < end - pos)
    {
        sWorkUnit[threadIdx.x]          = cSim.pWorkUnit[pos + threadIdx.x];
    }
    if (threadIdx.x < GRID)
    {
        sNext[threadIdx.x]              = (threadIdx.x + 1) & (GRID - 1);
    }
    __syncthreads();

    // Now change pos and end to reflect work queue just read
    // into shared memory
    end                                 = end - pos; 
    pos                                 = end - (threadIdx.x >> GRIDBITS) - 1;
       
    while (pos >= 0)
    {  
    
        // Extract cell coordinates from appropriate work unit
        unsigned int x                  = sWorkUnit[pos];
        unsigned int y                  = ((x >> 2) & 0x7fff) << GRIDBITS;
        x                               = (x >> 17) << GRIDBITS;
        unsigned int tgx                = threadIdx.x & (GRID - 1);
        unsigned int i                  = x + tgx;
        float4 apos                     = cSim.pPosq[i];
        float2 a                        = cSim.pObcData[i];
        float fb                        = cSim.pBornForce[i];
        unsigned int tbx                = threadIdx.x - tgx;
        int tj                          = tgx; 
        Atom* psA                       = &sA[tbx];
        if (x == y) // Handle diagonals uniquely at 50% efficiency
        { 
            // Read fixed atom data into registers and GRF
            float3 af;
            sA[threadIdx.x].fx = af.x   = 0.0f;
            sA[threadIdx.x].fy = af.y   = 0.0f;
            sA[threadIdx.x].fz = af.z   = 0.0f;
//            float sum                   = 0.0f;
            sA[threadIdx.x].x           = apos.x;
            sA[threadIdx.x].y           = apos.y;
            sA[threadIdx.x].z           = apos.z;
//            float oneOverR              = 1.0f / a.x;
            sA[threadIdx.x].r           = a.x;
            sA[threadIdx.x].sr          = a.y;
            sA[threadIdx.x].sr2         = a.y * a.y;
            sA[threadIdx.x].fb          = fb;
            
            for (unsigned int j = sNext[tgx]; j != tgx; j = sNext[j])
            {
                float dx                = psA[j].x - apos.x; 
                float dy                = psA[j].y - apos.y; 
                float dz                = psA[j].z - apos.z; 
                float r2                = dx * dx + dy * dy + dz * dz;
                float r                 = sqrt(r2);
                
                
                // Atom I Born forces and sum
                float rScaledRadiusJ    = r + psA[j].sr;
                
                float l_ij          = 1.0f / max(a.x, fabs(r - psA[j].sr));
                float u_ij          = 1.0f / rScaledRadiusJ;
                float rInverse      = 1.0f / r;
                float l_ij2         = l_ij * l_ij;
                float u_ij2         = u_ij * u_ij;
                float r2Inverse     = rInverse * rInverse;
                float t1            = log (u_ij / l_ij);
                float t2            = (l_ij2 - u_ij2);
                float t3            = t2 * rInverse;
                t1                 *= rInverse;
                    
                // Born Forces term
                float term          =  0.125f * 
                                      (1.000f + psA[j].sr2 * r2Inverse) * t3 + 
                                       0.250f * t1 * r2Inverse;
                float dE            = fb * term;
                    
                // Born sum term
//                term                =  l_ij - u_ij  +
//                                      -0.25f * r * t2 +
//                                       0.50f * t1 +
//                                      (0.25f * psA[j].sr2) * t3;
//                if (a.x < (psA[j].sr - r))
//                {
//                    term           += 2.0f * (oneOverR - l_ij);
//                }
                    
                if (a.x >= rScaledRadiusJ) 
                {
                    dE              = /*term =*/ 0.0f;
                }
                float d             = dx * dE;
                af.x               -= d;
                psA[j].fx          += d;
                d                   = dy * dE;  
                af.y               -= d;
                psA[j].fy          += d;
                d                   = dz * dE;
                af.z               -= d;
                psA[j].fz          += d;                                          
//                sum                += term;
            }
            
            // Write results
            int offset                  = x + tgx + (x >> GRIDBITS) * cSim.stride;
            float4 of;
            of.x                        = af.x + sA[threadIdx.x].fx;
            of.y                        = af.y + sA[threadIdx.x].fy;
            of.z                        = af.z + sA[threadIdx.x].fz;
            of.w                        = 0.0f;
            cSim.pForce4b[offset]       = of;
//            cSim.pBornSum[offset]       = sum;
        }         
        else 
        {        
            // Read fixed atom data into registers and GRF
            int j                       = y + tgx;
            float4 temp                 = cSim.pPosq[j];
            float2 temp1                = cSim.pObcData[j];
            sA[threadIdx.x].fb          = cSim.pBornForce[j];
            float3 af;
            sA[threadIdx.x].fx = af.x   = 0.0f;
            sA[threadIdx.x].fy = af.y   = 0.0f;
            sA[threadIdx.x].fz = af.z   = 0.0f;
//            sA[threadIdx.x].sum         = 0.0f;
//            float sum                   = 0.0f;
            float sr2                   = a.y * a.y;
//            float oneOverR              = 1.0f / a.x;
            sA[threadIdx.x].x           = temp.x;
            sA[threadIdx.x].y           = temp.y;
            sA[threadIdx.x].z           = temp.z;
            sA[threadIdx.x].r           = temp1.x;
            sA[threadIdx.x].sr          = temp1.y;
            sA[threadIdx.x].sr2         = temp1.y * temp1.y;
            for (j = 0; j < GRID; j++)
            {
                float dx                = psA[tj].x - apos.x; 
                float dy                = psA[tj].y - apos.y; 
                float dz                = psA[tj].z - apos.z; 
                float r2                = dx * dx + dy * dy + dz * dz; 
                float r                 = sqrt(r2);
                
                // Interleaved Atom I and J Born Forces and sum components
                float r2Inverse         = 1.0f / r2;
                float rScaledRadiusJ    = r + psA[tj].sr;
                float rScaledRadiusI    = r + a.y;
                float rInverse          = 1.0f / r;
                float l_ijJ             = 1.0f / max(a.x, fabs(r - psA[tj].sr));
                float l_ijI             = 1.0f / max(psA[tj].r, fabs(r - a.y));
                float u_ijJ             = 1.0f / rScaledRadiusJ;
                float u_ijI             = 1.0f / rScaledRadiusI;
                float l_ij2J            = l_ijJ * l_ijJ;
                float l_ij2I            = l_ijI * l_ijI;
                float u_ij2J            = u_ijJ * u_ijJ;
                float u_ij2I            = u_ijI * u_ijI;
                float t1J               = log (u_ijJ / l_ijJ);
                float t1I               = log (u_ijI / l_ijI);
                float t2J               = (l_ij2J - u_ij2J);
                float t2I               = (l_ij2I - u_ij2I);
                float t3J               = t2J * rInverse;
                float t3I               = t2I * rInverse;
                t1J                    *= rInverse;
                t1I                    *= rInverse;
                   
                // Born Forces term
                float term              =  0.125f * 
                                          (1.000f + psA[tj].sr2 * r2Inverse) * t3J + 
                                           0.250f * t1J * r2Inverse;
                float dE                = fb * term;
                    
                // Atom I Born sum term
//                term                    =   l_ijJ - u_ijJ +
//                                           -0.25f * r * t2J +
//                                            0.50f * t1J +
//                                           (0.25f * psA[tj].sr2) * t3J;
//                if (a.x < (psA[tj].sr - r))
//                {
//                    term               += 2.0f * (oneOverR - l_ijJ);
//                }
                
                if (a.x >= rScaledRadiusJ) 
                {
                    dE                  = /*term =*/ 0.0f;
                }
                
                float d                 = dx * dE;
                af.x                   -= d;
                psA[tj].fx             += d;
                d                       = dy * dE;  
                af.y                   -= d;
                psA[tj].fy             += d;
                d                       = dz * dE;
                af.z                   -= d;
                psA[tj].fz             += d;                                          
//                sum                    += term;
               
                // Atom J Born sum term               
                term                    =  0.125f * 
                                          (1.000f + sr2 * r2Inverse) * t3I + 
                                           0.250f * t1I * r2Inverse;
                dE                      = psA[tj].fb * term;  
                
//                term                    =  l_ijI - u_ijI +
//                                          -0.25f * r * t2I +
//                                           0.50f * t1I +
//                                          (0.25f * sr2) * t3I;
//                if (psA[tj].r < (a.y - r))
//                {
//                    term               += 2.0f * ((1.0f / psA[tj].r) - l_ijI);
//                }
                
                if (psA[tj].r >= rScaledRadiusI) 
                {           
                    dE                  = /*term =*/ 0.0f;
                }                             
                dx                     *= dE;
                dy                     *= dE;
                dz                     *= dE;
                psA[tj].fx             += dx; 
                psA[tj].fy             += dy;
                psA[tj].fz             += dz; 
                af.x                   -= dx;
                af.y                   -= dy;
                af.z                   -= dz;    
//                psA[tj].sum            += term;
                tj                      = sNext[tj]; 
            }
                
            // Write results
            int offset                  = x + tgx + (y >> GRIDBITS) * cSim.stride;
            float4 of;
            of.x                        = af.x;
            of.y                        = af.y;
            of.z                        = af.z;
            of.w                        = 0.0f;
            cSim.pForce4b[offset]       = of;
//            cSim.pBornSum[offset]       = sum;
            offset                      = y + tgx + (x >> GRIDBITS) * cSim.stride;
            of.x                        = sA[threadIdx.x].fx;
            of.y                        = sA[threadIdx.x].fy;
            of.z                        = sA[threadIdx.x].fz;
            cSim.pForce4b[offset]       = of;
//            cSim.pBornSum[offset]       = sA[threadIdx.x].sum;
        }
        pos                            -= cSim.bornForce2_workBlock;     
    }
}

void kCalculateObcGbsaForces2_12(gpuContext gpu)
{
  //  printf("kCalculateObcGbsaForces2_12\n");
    kCalculateObcGbsaForces2_12_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block>>>();
    LAUNCHERROR("kCalculateObcGbsaForces2_12");
}
