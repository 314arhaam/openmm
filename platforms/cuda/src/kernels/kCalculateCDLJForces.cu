#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"
#include "cudatypes.h"

#define UNROLLXX 0
#define UNROLLXY 0

struct Atom {
    float x;
    float y;
    float z;
    float q;
    float sig;
    float eps;
    float fx;
    float fy;
    float fz;
    float eps2;
    float sig2;
};


__shared__ Atom sA[G8X_NONBOND_THREADS_PER_BLOCK];
__shared__ unsigned int sWorkUnit[G8X_NONBOND_WORKUNITS_PER_SM];
__shared__ unsigned int sNext[GRID];

static __constant__ cudaGmxSimulation cSim;

void SetCalculateCDLJForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateCDLJForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

__global__ void kCalculateCDLJForces_kernel()
{
    // Read queue of work blocks once so the remainder of
    // kernel can run asynchronously    
    int pos = cSim.nbWorkUnitsPerBlock * blockIdx.x + min(blockIdx.x, cSim.nbWorkUnitsPerBlockRemainder);
    int end = cSim.nbWorkUnitsPerBlock * (blockIdx.x + 1) + min((blockIdx.x + 1), cSim.nbWorkUnitsPerBlockRemainder);    
    if (threadIdx.x < end - pos)
    {
        sWorkUnit[threadIdx.x] = cSim.pWorkUnit[pos + threadIdx.x];
    }
    if (threadIdx.x < GRID)
    {
        sNext[threadIdx.x] = (threadIdx.x + 1) & (GRID - 1);
    }
    __syncthreads();

    // Now change pos and end to reflect work queue just read
    // into shared memory
    end = end - pos; 
    pos = end - (threadIdx.x >> GRIDBITS) - 1;
       
    while (pos >= 0)
    {  
    
        // Extract cell coordinates from appropriate work unit
        unsigned int x = sWorkUnit[pos];
        unsigned int y = ((x >> 2) & 0x7fff) << GRIDBITS;
        bool bExclusionFlag = (x & 0x1);
        x = (x >> 17) << GRIDBITS;
        float4      apos;   // Local atom x, y, z, q
        float3      af;     // Local atom fx, fy, fz
        float dx; 
        float dy; 
        float dz; 
        float r2; 
        float invR; 
        float sig; 
        float sig2; 
        float sig6; 
        float eps; 
        float dEdR;  
        unsigned int tgx = threadIdx.x & (GRID - 1);
        unsigned int tbx = threadIdx.x - tgx;
        int tj = tgx; 
        Atom* psA = &sA[tbx];
        if (!bExclusionFlag)
        {
            if (x == y) // Handle diagonals uniquely at 50% efficiency
            { 
                // Read fixed atom data into registers and GRF
                unsigned int i      = x + tgx;
                apos                = cSim.pPosq[i];
                float2 a            = cSim.pAttr[i];
                sA[threadIdx.x].x   = apos.x;
                sA[threadIdx.x].y   = apos.y;
                sA[threadIdx.x].z   = apos.z;
                sA[threadIdx.x].q   = apos.w;
                sA[threadIdx.x].sig = a.x;
                sA[threadIdx.x].eps = a.y;
                af.x                = 0.0f;
                af.y                = 0.0f;
                af.z                = 0.0f;
                apos.w             *= cSim.epsfac;
                for (unsigned int j = 0; j < GRID; j++)
                {
                    dx              = psA[j].x - apos.x; 
                    dy              = psA[j].y - apos.y; 
                    dz              = psA[j].z - apos.z; 
                    r2              = dx * dx + dy * dy + dz * dz; 
                    invR            = 1.0f / sqrt(r2);
                    sig             = a.x + psA[j].sig; 
                    sig2            = invR * sig; 
                    sig2           *= sig2;
                    sig6            = sig2 * sig2 * sig2; 
                    eps             = a.y * psA[j].eps; 
                    dEdR            = eps * (12.0f * sig6 - 6.0f) * sig6; 
                    dEdR           += apos.w * psA[j].q * invR; 
                    dEdR           *= invR * invR; 
                    dx             *= dEdR; 
                    dy             *= dEdR; 
                    dz             *= dEdR; 
                    af.x           -= dx; 
                    af.y           -= dy; 
                    af.z           -= dz; 
                }
                
                // Write results
                float4 of;
                of.x                                = af.x;
                of.y                                = af.y;
                of.z                                = af.z;
                of.w                                = 0.0f;
                int offset                          = x + tgx + (x >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
            }         
            else        // 100% utilization
            {
                // Read fixed atom data into registers and GRF
                int j                   = y + tgx;
                unsigned int i          = x + tgx;
                float4 temp             = cSim.pPosq[j];
                float2 temp1            = cSim.pAttr[j];
                apos                    = cSim.pPosq[i];
                float2 a                = cSim.pAttr[i];
                sA[threadIdx.x].x       = temp.x;
                sA[threadIdx.x].y       = temp.y;
                sA[threadIdx.x].z       = temp.z;
                sA[threadIdx.x].q       = temp.w;
                sA[threadIdx.x].sig     = temp1.x;
                sA[threadIdx.x].eps     = temp1.y;
                sA[threadIdx.x].fx      = af.x = 0.0f;
                sA[threadIdx.x].fy      = af.y = 0.0f;
                sA[threadIdx.x].fz      = af.z = 0.0f;
                sA[threadIdx.x].sig2    = a.x;
                sA[threadIdx.x].eps2    = a.y;
                apos.w                 *= cSim.epsfac;
                
                for (j = 0; j < GRID; j++)
                {
                    dx              = psA[tj].x - apos.x; 
                    dy              = psA[tj].y - apos.y; 
                    dz              = psA[tj].z - apos.z; 
                    r2              = dx * dx + dy * dy + dz * dz; 
                    invR            = 1.0f / sqrt(r2);
                    sig             = a.x + psA[tj].sig; 
                    sig2            = invR * sig; 
                    sig2           *= sig2;
                    sig6            = sig2 * sig2 * sig2; 
                    eps             = a.y * psA[tj].eps; 
                    dEdR            = eps * (12.0f * sig6 - 6.0f) * sig6; 
                    dEdR           += apos.w * psA[tj].q * invR; 
                    dEdR           *= invR * invR; 
                    dx             *= dEdR; 
                    dy             *= dEdR; 
                    dz             *= dEdR; 
                    af.x           -= dx; 
                    af.y           -= dy; 
                    af.z           -= dz; 
                    psA[tj].fx     += dx; 
                    psA[tj].fy     += dy; 
                    psA[tj].fz     += dz;
                    tj              = sNext[tj]; 
                }
                
                // Write results
                float4 of;
                of.x                                = af.x;
                of.y                                = af.y;
                of.z                                = af.z;
                of.w                                = 0.0f;
                int offset                          = x + tgx + (y >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
                of.x                                = sA[threadIdx.x].fx;
                of.y                                = sA[threadIdx.x].fy;
                of.z                                = sA[threadIdx.x].fz;
                offset                              = y + tgx + (x >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
            }
        }
        else  // bExclusion
        {
            // Read exclusion data
            
            if (x == y) // Handle diagonals uniquely at 50% efficiency
            { 
                // Read fixed atom data into registers and GRF
                unsigned int excl       = cSim.pExclusion[x * cSim.exclusionStride + y + tgx];                          
                unsigned int i          = x + tgx;
                apos                    = cSim.pPosq[i];
                float2 a                = cSim.pAttr[i];
                sA[threadIdx.x].x       = apos.x;
                sA[threadIdx.x].y       = apos.y;
                sA[threadIdx.x].z       = apos.z;
                sA[threadIdx.x].q       = apos.w;
                sA[threadIdx.x].sig     = a.x;
                sA[threadIdx.x].eps     = a.y;
                af.x                    = 0.0f;
                af.y                    = 0.0f;
                af.z                    = 0.0f;
                sA[threadIdx.x].sig2    = a.x;
                sA[threadIdx.x].eps2    = a.y;
                apos.w                 *= cSim.epsfac;
                
                for (unsigned int j = 0; j < GRID; j++)
                {
                    dx              = psA[j].x - apos.x; 
                    dy              = psA[j].y - apos.y; 
                    dz              = psA[j].z - apos.z; 
                    r2              = dx * dx + dy * dy + dz * dz; 
                    invR            = 1.0f / sqrt(r2);
                    sig             = psA[tgx].sig2 + psA[j].sig; 
                    sig2            = invR * sig; 
                    sig2           *= sig2;
                    sig6            = sig2 * sig2 * sig2; 
                    eps             = psA[tgx].eps2 * psA[j].eps; 
                    dEdR            = eps * (12.0f * sig6 - 6.0f) * sig6; 
                    dEdR           += apos.w * psA[j].q * invR; 
                    dEdR           *= invR * invR; 
                    if (!(excl & 0x1))
                    {
                        dEdR = 0.0f;
                    }
                    dx             *= dEdR; 
                    dy             *= dEdR; 
                    dz             *= dEdR; 
                    af.x           -= dx; 
                    af.y           -= dy; 
                    af.z           -= dz;
                    excl          >>= 1;               
                }
                
                // Write results
                float4 of;
                of.x                                = af.x;
                of.y                                = af.y;
                of.z                                = af.z;
                of.w                                = 0.0f;
                int offset                          = x + tgx + (x >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
            }         
            else        // 100% utilization
            {
                // Read fixed atom data into registers and GRF        
                unsigned int excl       = cSim.pExclusion[x * cSim.exclusionStride + y + tgx];
                excl                    = (excl >> tgx) | (excl << (GRID - tgx));
                int j                   = y + tgx;
                unsigned int i          = x + tgx;
                float4 temp             = cSim.pPosq[j];
                float2 temp1            = cSim.pAttr[j];
                apos                    = cSim.pPosq[i];
                float2 a                = cSim.pAttr[i];
                sA[threadIdx.x].x       = temp.x;
                sA[threadIdx.x].y       = temp.y;
                sA[threadIdx.x].z       = temp.z;
                sA[threadIdx.x].q       = temp.w;
                sA[threadIdx.x].sig     = temp1.x;
                sA[threadIdx.x].eps     = temp1.y;
                sA[threadIdx.x].fx      = af.x = 0.0f;
                sA[threadIdx.x].fy      = af.y = 0.0f;
                sA[threadIdx.x].fz      = af.z = 0.0f;
                sA[threadIdx.x].sig2    = a.x;
                sA[threadIdx.x].eps2    = a.y;
                apos.w                 *= cSim.epsfac;
                
                for (j = 0; j < GRID; j++)
                {
                    dx              = psA[tj].x - apos.x; 
                    dy              = psA[tj].y - apos.y; 
                    dz              = psA[tj].z - apos.z; 
                    r2              = dx * dx + dy * dy + dz * dz; 
                    invR            = 1.0f / sqrt(r2);
                    sig             = psA[tgx].sig2 + psA[tj].sig; 
                    sig2            = invR * sig; 
                    sig2           *= sig2;
                    sig6            = sig2 * sig2 * sig2; 
                    eps             = psA[tgx].eps2 * psA[tj].eps; 
                    dEdR            = eps * (12.0f * sig6 - 6.0f) * sig6; 
                    dEdR           += apos.w * psA[tj].q * invR; 
                    dEdR           *= invR * invR; 
                    if (!(excl & 0x1))
                    {
                        dEdR = 0.0f;
                    }
                    dx             *= dEdR; 
                    dy             *= dEdR; 
                    dz             *= dEdR; 
                    af.x           -= dx; 
                    af.y           -= dy; 
                    af.z           -= dz; 
                    psA[tj].fx     += dx; 
                    psA[tj].fy     += dy; 
                    psA[tj].fz     += dz;
                    excl          >>= 1;
                    tj              = sNext[tj]; 
                }
                
                // Write results
                float4 of;
                of.x                                = af.x;
                of.y                                = af.y;
                of.z                                = af.z;
                of.w                                = 0.0f;
                int offset                          = x + tgx + (y >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
                of.x                                = sA[threadIdx.x].fx;
                of.y                                = sA[threadIdx.x].fy;
                of.z                                = sA[threadIdx.x].fz;
                offset                              = y + tgx + (x >> GRIDBITS) * cSim.stride;
                cSim.pForce4a[offset]               = of;
            }
        }

        pos -= cSim.nonbond_workBlock;     
    }
}

__global__ extern void kCalculateCDLJForces_12_kernel();

void kCalculateCDLJForces(gpuContext gpu)
{
//    printf("kCalculateCDLJForces\n");
    if (gpu->sm_version < SM_12)
        kCalculateCDLJForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block>>>();
    else
        kCalculateCDLJForces_12_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block>>>();
    LAUNCHERROR("kCalculateCDLJForces");
}