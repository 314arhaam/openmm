#include "hip/hip_runtime.h"
#define TILE_SIZE 32
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real x, y, z;
    real q;
    real fx, fy, fz;
    ATOM_PARAMETER_DATA
#ifndef PARAMETER_SIZE_IS_EVEN
    real padding;
#endif
} AtomData;

/**
 * Compute nonbonded interactions.
 */
extern "C" __global__ void computeNonbonded(
        unsigned long long* __restrict__ forceBuffers, real* __restrict__ energyBuffer, const real4* __restrict__ posq, const unsigned int* __restrict__ exclusions,
        const unsigned int* __restrict__ exclusionIndices, const unsigned int* __restrict__ exclusionRowIndices,
        unsigned int startTileIndex, unsigned int numTileIndices
#ifdef USE_CUTOFF
        , const ushort2* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, unsigned int maxTiles, const unsigned int* __restrict__ interactionFlags
#endif
        PARAMETER_ARGUMENTS) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    unsigned int pos = (numTiles > maxTiles ? startTileIndex+warp*numTileIndices/totalWarps : warp*numTiles/totalWarps);
    unsigned int end = (numTiles > maxTiles ? startTileIndex+(warp+1)*numTileIndices/totalWarps : (warp+1)*numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    unsigned int pos = startTileIndex+warp*numTiles/totalWarps;
    unsigned int end = startTileIndex+(warp+1)*numTiles/totalWarps;
#endif
    real energy = 0.0f;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
    __shared__ unsigned int exclusionRange[2*WARPS_PER_GROUP];
    __shared__ int exclusionIndex[WARPS_PER_GROUP];
#ifndef ENABLE_SHUFFLE
    __shared__ real tempBuffer[3*THREAD_BLOCK_SIZE];
#endif
    
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        const unsigned int localGroupIndex = threadIdx.x/TILE_SIZE;
        unsigned int x, y;
        real3 force = make_real3(0);
        if (pos < end) {
#ifdef USE_CUTOFF
            if (numTiles <= maxTiles) {
                ushort2 tileIndices = tiles[pos];
                x = tileIndices.x;
                y = tileIndices.y;
            }
            else
#endif
            {
                y = (unsigned int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                    y += (x < y ? -1 : 1);
                    x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
                }
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            real4 posq1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS

            // Locate the exclusion data for this tile.

#ifdef USE_EXCLUSIONS
            if (tgx < 2)
                exclusionRange[2*localGroupIndex+tgx] = exclusionRowIndices[x+tgx];
            if (tgx == 0)
                exclusionIndex[localGroupIndex] = -1;
            for (unsigned int i = exclusionRange[2*localGroupIndex]+tgx; i < exclusionRange[2*localGroupIndex+1]; i += TILE_SIZE)
                if (exclusionIndices[i] == y)
                    exclusionIndex[localGroupIndex] = i*TILE_SIZE;
            bool hasExclusions = (exclusionIndex[localGroupIndex] > -1);
#else
            bool hasExclusions = false;
#endif
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                const unsigned int localAtomIndex = threadIdx.x;
                localData[localAtomIndex].x = posq1.x;
                localData[localAtomIndex].y = posq1.y;
                localData[localAtomIndex].z = posq1.z;
                localData[localAtomIndex].q = posq1.w;
                LOAD_LOCAL_PARAMETERS_FROM_1
#ifdef USE_EXCLUSIONS
                unsigned int excl = exclusions[exclusionIndex[localGroupIndex]+tgx];
#endif
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
#ifdef USE_EXCLUSIONS
                    bool isExcluded = !(excl & 0x1);
#endif
                    int atom2 = tbx+j;
                    real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                    delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                    delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                    delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    real invR = RSQRT(r2);
                    real r = RECIP(invR);
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+j;
#ifdef USE_SYMMETRIC
                    real dEdR = 0.0f;
#else
                    real3 dEdR1 = make_real3(0);
                    real3 dEdR2 = make_real3(0);
#endif
                    real tempEnergy = 0.0f;
                    COMPUTE_INTERACTION
                    energy += 0.5f*tempEnergy;
#ifdef USE_SYMMETRIC
                    force.x -= delta.x*dEdR;
                    force.y -= delta.y*dEdR;
                    force.z -= delta.z*dEdR;
#else
                    force.x -= dEdR1.x;
                    force.y -= dEdR1.y;
                    force.z -= dEdR1.z;
#endif
#ifdef USE_EXCLUSIONS
                    excl >>= 1;
#endif
                }
            }
            else {
                // This is an off-diagonal tile.

                const unsigned int localAtomIndex = threadIdx.x;
                unsigned int j = y*TILE_SIZE + tgx;
                real4 tempPosq = posq[j];
                localData[localAtomIndex].x = tempPosq.x;
                localData[localAtomIndex].y = tempPosq.y;
                localData[localAtomIndex].z = tempPosq.z;
                localData[localAtomIndex].q = tempPosq.w;
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
                localData[localAtomIndex].fx = 0.0f;
                localData[localAtomIndex].fy = 0.0f;
                localData[localAtomIndex].fz = 0.0f;
#ifdef USE_CUTOFF
                unsigned int flags = (numTiles <= maxTiles ? interactionFlags[pos] : 0xFFFFFFFF);
                if (!hasExclusions && flags != 0xFFFFFFFF) {
                    if (flags == 0) {
                        // No interactions in this tile.
                    }
                    else {
                        // Compute only a subset of the interactions in this tile.

                        for (j = 0; j < TILE_SIZE; j++) {
                            if ((flags&(1<<j)) != 0) {
                                bool isExcluded = false;
                                int atom2 = tbx+j;
                                int bufferIndex = 3*threadIdx.x;
#ifdef USE_SYMMETRIC
                                real dEdR = 0;
#else
                                real3 dEdR1 = make_real3(0);
                                real3 dEdR2 = make_real3(0);
#endif
                                real tempEnergy = 0.0f;
                                real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
                                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                                delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                                delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                                delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                                if (r2 < CUTOFF_SQUARED) {
#endif
                                    real invR = RSQRT(r2);
                                    real r = RECIP(invR);
                                    LOAD_ATOM2_PARAMETERS
                                    atom2 = y*TILE_SIZE+j;
                                    COMPUTE_INTERACTION
                                    energy += tempEnergy;
#ifdef USE_CUTOFF
                                }
#endif
#ifdef ENABLE_SHUFFLE
    #ifdef USE_SYMMETRIC
                                delta *= dEdR;
                                force.x -= delta.x;
                                force.y -= delta.y;
                                force.z -= delta.z;
                                for (int i = 16; i >= 1; i /= 2) {
                                    delta.x += __shfl_xor(delta.x, i, 32);
                                    delta.y += __shfl_xor(delta.y, i, 32);
                                    delta.z += __shfl_xor(delta.z, i, 32);
                                }
                                if (tgx == 0) {
                                    localData[tbx+j].fx += delta.x;
                                    localData[tbx+j].fy += delta.y;
                                    localData[tbx+j].fz += delta.z;
                                }
    #else
                                force.x -= dEdR1.x;
                                force.y -= dEdR1.y;
                                force.z -= dEdR1.z;
                                for (int i = 16; i >= 1; i /= 2) {
                                    dEdR2.x += __shfl_xor(dEdR2.x, i, 32);
                                    dEdR2.y += __shfl_xor(dEdR2.y, i, 32);
                                    dEdR2.z += __shfl_xor(dEdR2.z, i, 32);
                                }
                                if (tgx == 0) {
                                    localData[tbx+j].fx += dEdR2.x;
                                    localData[tbx+j].fy += dEdR2.y;
                                    localData[tbx+j].fz += dEdR2.z;
                                }
    #endif
#else
    #ifdef USE_SYMMETRIC
                                delta *= dEdR;
                                force.x -= delta.x;
                                force.y -= delta.y;
                                force.z -= delta.z;
                                tempBuffer[bufferIndex] = delta.x;
                                tempBuffer[bufferIndex+1] = delta.y;
                                tempBuffer[bufferIndex+2] = delta.z;
    #else
                                force.x -= dEdR1.x;
                                force.y -= dEdR1.y;
                                force.z -= dEdR1.z;
                                tempBuffer[bufferIndex] = dEdR2.x;
                                tempBuffer[bufferIndex+1] = dEdR2.y;
                                tempBuffer[bufferIndex+2] = dEdR2.z;
    #endif

                                // Sum the forces on atom2.

                                if (tgx % 4 == 0) {
                                    tempBuffer[bufferIndex] += tempBuffer[bufferIndex+3]+tempBuffer[bufferIndex+6]+tempBuffer[bufferIndex+9];
                                    tempBuffer[bufferIndex+1] += tempBuffer[bufferIndex+4]+tempBuffer[bufferIndex+7]+tempBuffer[bufferIndex+10];
                                    tempBuffer[bufferIndex+2] += tempBuffer[bufferIndex+5]+tempBuffer[bufferIndex+8]+tempBuffer[bufferIndex+11];
                                }
                                if (tgx == 0) {
                                    localData[tbx+j].fx += tempBuffer[bufferIndex]+tempBuffer[bufferIndex+12]+tempBuffer[bufferIndex+24]+tempBuffer[bufferIndex+36]+tempBuffer[bufferIndex+48]+tempBuffer[bufferIndex+60]+tempBuffer[bufferIndex+72]+tempBuffer[bufferIndex+84];
                                    localData[tbx+j].fy += tempBuffer[bufferIndex+1]+tempBuffer[bufferIndex+13]+tempBuffer[bufferIndex+25]+tempBuffer[bufferIndex+37]+tempBuffer[bufferIndex+49]+tempBuffer[bufferIndex+61]+tempBuffer[bufferIndex+73]+tempBuffer[bufferIndex+85];
                                    localData[tbx+j].fz += tempBuffer[bufferIndex+2]+tempBuffer[bufferIndex+14]+tempBuffer[bufferIndex+26]+tempBuffer[bufferIndex+38]+tempBuffer[bufferIndex+50]+tempBuffer[bufferIndex+62]+tempBuffer[bufferIndex+74]+tempBuffer[bufferIndex+86];
                                }
#endif
                            }
                        }
                    }
                }
                else
#endif
                {
                    // Compute the full set of interactions in this tile.

#ifdef USE_EXCLUSIONS
                    unsigned int excl = (hasExclusions ? exclusions[exclusionIndex[localGroupIndex]+tgx] : 0xFFFFFFFF);
                    excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
#endif
                    unsigned int tj = tgx;
                    for (j = 0; j < TILE_SIZE; j++) {
#ifdef USE_EXCLUSIONS
                        bool isExcluded = !(excl & 0x1);
#endif
                        int atom2 = tbx+tj;
                        real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
                        real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                        delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                        delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                        delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                        real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                        if (r2 < CUTOFF_SQUARED) {
#endif
                            real invR = RSQRT(r2);
                            real r = RECIP(invR);
                            LOAD_ATOM2_PARAMETERS
                            atom2 = y*TILE_SIZE+tj;
#ifdef USE_SYMMETRIC
                            real dEdR = 0.0f;
#else
                            real3 dEdR1 = make_real3(0);
                            real3 dEdR2 = make_real3(0);
#endif
                            real tempEnergy = 0.0f;
                            COMPUTE_INTERACTION
                            energy += tempEnergy;
#ifdef USE_SYMMETRIC
                            delta *= dEdR;
                            force.x -= delta.x;
                            force.y -= delta.y;
                            force.z -= delta.z;
                            localData[tbx+tj].fx += delta.x;
                            localData[tbx+tj].fy += delta.y;
                            localData[tbx+tj].fz += delta.z;
#else
                            force.x -= dEdR1.x;
                            force.y -= dEdR1.y;
                            force.z -= dEdR1.z;
                            localData[tbx+tj].fx += dEdR2.x;
                            localData[tbx+tj].fy += dEdR2.y;
                            localData[tbx+tj].fz += dEdR2.z;
#endif
#ifdef USE_CUTOFF
                        }
#endif
#ifdef USE_EXCLUSIONS
                        excl >>= 1;
#endif
                        tj = (tj + 1) & (TILE_SIZE - 1);
                    }
                }
            }
        }
        
        // Write results.
        
        if (pos < end) {
            const unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (force.x*0xFFFFFFFF)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0xFFFFFFFF)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0xFFFFFFFF)));
        }
        if (pos < end && x != y) {
            const unsigned int offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fx*0xFFFFFFFF)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fy*0xFFFFFFFF)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fz*0xFFFFFFFF)));
        }
        pos++;
    } while (pos < end);
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}
