#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <vector>
#include "gputypes.h"

using namespace std;


static __constant__ cudaGmxSimulation cSim;

void SetCShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

/**
 * Synchronize all threads across all blocks.
 */
__device__ void kSyncAllThreads_kernel(short* syncCounter, short newCount)
{
    __syncthreads();
    if (threadIdx.x == 0)
        syncCounter[blockIdx.x] = newCount;
    if (threadIdx.x < gridDim.x)
    {
        volatile short* counter = &syncCounter[threadIdx.x];
        do
        {
        } while (*counter != newCount);
    }
    __syncthreads();
}

__global__ void kApplyCShake_kernel(float4* atomPositions, bool addOldPosition)
{
    extern __shared__ float temp[];

    // Initialize counters used for monitoring convergence and doing global thread synchronization.

    __shared__ unsigned int requiredIterations;
    if (threadIdx.x == 0)
    {
        requiredIterations = 0;
        cSim.pSyncCounter[gridDim.x+blockIdx.x] = -1;
        cSim.pSyncCounter[2*gridDim.x+blockIdx.x] = -1;
        cSim.pRequiredIterations[0] = 0;
    }

    // Calculate the direction of each constraint.

    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;
    while (pos < cSim.lincsConstraints)
    {
        int2 atoms = cSim.pLincsAtoms[pos];
        float4 dir = cSim.pLincsDistance[pos];
        float4 oldPos1 = cSim.pOldPosq[atoms.x];
        float4 oldPos2 = cSim.pOldPosq[atoms.y];
        dir.x = oldPos1.x-oldPos2.x;
        dir.y = oldPos1.y-oldPos2.y;
        dir.z = oldPos1.z-oldPos2.z;
        cSim.pLincsDistance[pos] = dir;
        pos += blockDim.x*gridDim.x;
    }
    __syncthreads();

    // Iteratively update the atom positions.

    unsigned int maxIterations = 150;
    float lowerTol = 1.0f-2.0f*cSim.shakeTolerance+cSim.shakeTolerance*cSim.shakeTolerance;
    float upperTol = 1.0f+2.0f*cSim.shakeTolerance+cSim.shakeTolerance*cSim.shakeTolerance;
    for (unsigned int iteration = 0; iteration < maxIterations && iteration == requiredIterations; iteration++)
    {
        // Calculate the constraint force for each constraint.

        pos = threadIdx.x + blockIdx.x * blockDim.x;
        while (pos < cSim.lincsConstraints)
        {
            int2 atoms = cSim.pLincsAtoms[pos];
            float4 delta1 = atomPositions[atoms.x];
            float4 delta2 = atomPositions[atoms.y];
            float4 dir = cSim.pLincsDistance[pos];
            float3 rp_ij = make_float3(delta1.x-delta2.x, delta1.y-delta2.y, delta1.z-delta2.z);
            if (addOldPosition)
            {
                rp_ij.x += dir.x;
                rp_ij.y += dir.y;
                rp_ij.z += dir.z;
            }
            float rp2 = rp_ij.x*rp_ij.x + rp_ij.y*rp_ij.y + rp_ij.z*rp_ij.z;
            float dist2 = dir.w*dir.w;
            float diff = dist2 - rp2;
            float rrpr  = rp_ij.x*dir.x + rp_ij.y*dir.y + rp_ij.z*dir.z;
            float d_ij2  = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z;
            float reducedMass = cSim.pShakeReducedMass[pos];
            cSim.pLincsSolution[pos] = (rrpr > d_ij2*1e-6f ? reducedMass*diff/rrpr : 0.0f);
            if (requiredIterations == iteration && (rp2 < lowerTol*dist2 || rp2 > upperTol*dist2))
                requiredIterations = iteration+1;
            pos += blockDim.x * gridDim.x;
        }
        kSyncAllThreads_kernel(cSim.pSyncCounter, iteration);
        if (threadIdx.x == 0 && requiredIterations > iteration)
            cSim.pRequiredIterations[0] = requiredIterations;

        // Multiply by the inverse constraint matrix for each rigid cluster.

        if (cSim.rigidClusters > 0)
        {
            pos = threadIdx.x + blockIdx.x * blockDim.x;
            unsigned int block = pos/cSim.clusterShakeBlockSize;
            unsigned int indexInBlock = pos-block*cSim.clusterShakeBlockSize;
            while (block < cSim.rigidClusters)
            {
                unsigned int firstConstraint = cSim.pRigidClusterConstraintIndex[block];
                unsigned int blockSize = cSim.pRigidClusterConstraintIndex[block+1]-firstConstraint;
                if (indexInBlock < blockSize)
                {
                    // Load the constraint forces and matrix.

                    temp[threadIdx.x] = cSim.pLincsSolution[firstConstraint+indexInBlock];
                    unsigned int firstMatrixIndex = cSim.pRigidClusterMatrixIndex[block];

                    // Multiply by the matrix.

                    float sum = 0.0f;
                    for (unsigned int i = 0; i < blockSize; i++)
                        sum += temp[threadIdx.x-indexInBlock+i]*cSim.pRigidClusterMatrix[firstMatrixIndex+i*blockSize+indexInBlock];
                    cSim.pLincsSolution[firstConstraint+indexInBlock] = sum;
                }
                block += (blockDim.x*gridDim.x)/cSim.clusterShakeBlockSize;
            }
            kSyncAllThreads_kernel(&cSim.pSyncCounter[gridDim.x], iteration);
        }

        // Update the position of each atom.

        pos = threadIdx.x + blockIdx.x * blockDim.x;
        float damping = (iteration < 2 ? 0.5f : 1.0f);
        while (pos < cSim.atoms)
        {
            float4 atomPos = atomPositions[pos];
            float invMass = cSim.pVelm4[pos].w;
            int num = cSim.pLincsNumAtomConstraints[pos];
            for (int i = 0; i < num; i++)
            {
                int index = pos+i*cSim.atoms;
                int constraint = cSim.pLincsAtomConstraints[index];
                bool forward = (constraint > 0);
                constraint = (forward ? constraint-1 : -constraint-1);
                float constraintForce = damping*invMass*cSim.pLincsSolution[constraint];
                constraintForce = (forward ? constraintForce : -constraintForce);
                float4 dir = cSim.pLincsDistance[constraint];
                atomPos.x += constraintForce*dir.x;
                atomPos.y += constraintForce*dir.y;
                atomPos.z += constraintForce*dir.z;
            }
            atomPositions[pos] = atomPos;
            pos += blockDim.x*gridDim.x;
        }
        kSyncAllThreads_kernel(&cSim.pSyncCounter[2*gridDim.x], iteration);
        requiredIterations = cSim.pRequiredIterations[0];
    }

    // Reset the initial sync counter to be ready for the next call.

    if (threadIdx.x == 0)
        cSim.pSyncCounter[blockIdx.x] = -1;
}

void kApplyFirstCShake(gpuContext gpu)
{
//    printf("kApplyFirstCShake\n");
    if (gpu->sim.lincsConstraints > 0)
    {
        kApplyCShake_kernel<<<gpu->sim.blocks, gpu->sim.lincs_threads_per_block, 4*gpu->sim.lincs_threads_per_block>>>(gpu->sim.pPosqP, true);
        LAUNCHERROR("kApplyCShake");
    }
}

void kApplySecondCShake(gpuContext gpu)
{
//    printf("kApplySecondCShake\n");
    if (gpu->sim.lincsConstraints > 0)
    {
        kApplyCShake_kernel<<<gpu->sim.blocks, gpu->sim.lincs_threads_per_block, 4*gpu->sim.lincs_threads_per_block>>>(gpu->sim.pPosq, false);
        LAUNCHERROR("kApplyCShake");
    }
}
