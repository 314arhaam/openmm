#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"
#include "cudatypes.h"

#define UNROLLXX 0
#define UNROLLXY 0

struct Atom {
    float x;
    float y;
    float z;
    float4 params;
    float fx;
    float fy;
    float fz;
};

static __constant__ cudaGmxSimulation cSim;
static __constant__ Expression<128> forceExp;
static __constant__ Expression<128> energyExp;
static __constant__ Expression<64> combiningRules[4];
static __constant__ float globalParams[8];

void SetCalculateCustomNonbondedForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateCustomNonbondedForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

void SetCustomNonbondedForceExpression(const Expression<128>& expression)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(forceExp), &expression, sizeof(forceExp));
    RTERROR(status, "SetCustomNonbondedForceExpression: hipMemcpyToSymbol failed");
}

void SetCustomNonbondedEnergyExpression(const Expression<128>& expression)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(energyExp), &expression, sizeof(energyExp));
    RTERROR(status, "SetCustomNonbondedEnergyExpression: hipMemcpyToSymbol failed");
}

void SetCustomNonbondedCombiningRules(const Expression<64>* expressions)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(combiningRules), expressions, sizeof(combiningRules));
    RTERROR(status, "SetCustomNonbondedCombiningRules: hipMemcpyToSymbol failed");
}

void SetCustomNonbondedGlobalParams(const vector<float>& paramValues)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(globalParams), &paramValues[0], sizeof(globalParams));
    RTERROR(status, "SetCustomNonbondedGlobalParams: hipMemcpyToSymbol failed");
}

#define STACK(y) stack[(y)*blockDim.x+threadIdx.x]

template<int SIZE>
__device__ float kEvaluateExpression_kernel(Expression<SIZE>* expression, float* stack, float var0, float4 vars1, float4 vars2)
{
    int stackPointer = -1;
    for (int i = 0; i < expression->length; i++)
    {
        int op = expression->op[i];
        if (op < SQRT) {
            if (op < VARIABLE8) {
                if (op < VARIABLE4) {
                    if (op == CONSTANT) {
                        STACK(++stackPointer) = expression->arg[i];
                    }
                    else if (op == VARIABLE0) {
                        STACK(++stackPointer) = var0;
                    }
                    else if (op == VARIABLE1) {
                        STACK(++stackPointer) = vars1.x;
                    }
                    else if (op == VARIABLE2) {
                        STACK(++stackPointer) = vars1.y;
                    }
                    else if (op == VARIABLE3) {
                        STACK(++stackPointer) = vars1.z;
                    }
                }
                else {
                    if (op == VARIABLE4) {
                        STACK(++stackPointer) = vars1.w;
                    }
                    else if (op == VARIABLE5) {
                        STACK(++stackPointer) = vars2.x;
                    }
                    else if (op == VARIABLE6) {
                        STACK(++stackPointer) = vars2.y;
                    }
                    else if (op == VARIABLE7) {
                        STACK(++stackPointer) = vars2.z;
                    }
                }
            }
            else {
                if (op < MULTIPLY) {
                    if (op == VARIABLE8) {
                        STACK(++stackPointer) = vars2.w;
                    }
                    else if (op == GLOBAL) {
                        STACK(++stackPointer) = globalParams[(int) expression->arg[i]];
                    }
                    else if (op == CUSTOM || op == CUSTOM_DERIV) {
                        int function = (int) expression->arg[i];
                        float x = STACK(stackPointer);
                        float4 params = cSim.pTabulatedFunctionParams[function];
                        if (x < params.x || x > params.y)
                            STACK(stackPointer) = 0.0f;
                        else
                        {
                            int index = floor((x-params.x)*params.z);
                            float4 coeff = cSim.pTabulatedFunctionCoefficients[function][index];
                            x = (x-params.x)*params.z-index;
                            if (op == CUSTOM)
                                STACK(stackPointer) = coeff.x+x*(coeff.y+x*(coeff.z+x*coeff.w));
                            else
                                STACK(stackPointer) = (coeff.y+x*(2.0f*coeff.z+x*3.0f*coeff.w))*params.z;
                        }
                    }
                    else if (op == ADD) {
                        float temp = STACK(stackPointer);
                        STACK(--stackPointer) += temp;
                    }
                    else if (op == SUBTRACT) {
                        float temp = STACK(stackPointer);
                        STACK(stackPointer) = temp-STACK(--stackPointer);
                    }
                }
                else {
                    if (op == MULTIPLY) {
                        float temp = STACK(stackPointer);
                        STACK(--stackPointer) *= temp;
                    }
                    else if (op == DIVIDE) {
                        float temp = STACK(stackPointer);
                        STACK(stackPointer) = temp/STACK(--stackPointer);
                    }
                    else if (op == POWER) {
                        float temp = STACK(stackPointer);
                        STACK(stackPointer) = pow(temp, STACK(--stackPointer));
                    }
                    else if (op == NEGATE) {
                        STACK(stackPointer) *= -1.0f;
                    }
                }
            }
        }
        else {
            if (op < ASIN) {
                if (op < SEC) {
                    if (op == SQRT) {
                        STACK(stackPointer) = sqrt(STACK(stackPointer));
                    }
                    else if (op == EXP) {
                        STACK(stackPointer) = exp(STACK(stackPointer));
                    }
                    else if (op == LOG) {
                        STACK(stackPointer) = log(STACK(stackPointer));
                    }
                    else if (op == SIN) {
                        STACK(stackPointer) = sin(STACK(stackPointer));
                    }
                    else if (op == COS) {
                        STACK(stackPointer) = cos(STACK(stackPointer));
                    }
                }
                else {
                    if (op == SEC) {
                        STACK(stackPointer) = 1.0f/cos(STACK(stackPointer));
                    }
                    else if (op == CSC) {
                        STACK(stackPointer) = 1.0f/sin(STACK(stackPointer));
                    }
                    else if (op == TAN) {
                        STACK(stackPointer) = tan(STACK(stackPointer));
                    }
                    else if (op == COT) {
                        STACK(stackPointer) = 1.0f/tan(STACK(stackPointer));
                    }
                }
            }
            else {
                if (op < RECIPROCAL) {
                    if (op == ASIN) {
                        STACK(stackPointer) = asin(STACK(stackPointer));
                    }
                    else if (op == ACOS) {
                        STACK(stackPointer) = acos(STACK(stackPointer));
                    }
                    else if (op == ATAN) {
                        STACK(stackPointer) = atan(STACK(stackPointer));
                    }
                    else if (op == SQUARE) {
                        float temp = STACK(stackPointer);
                        STACK(stackPointer) *= temp;
                    }
                    else if (op == CUBE) {
                        float temp = STACK(stackPointer);
                        STACK(stackPointer) *= temp*temp;
                    }
                }
                else {
                    if (op == RECIPROCAL) {
                        STACK(stackPointer) = 1.0f/STACK(stackPointer);
                    }
                    else if (op == ADD_CONSTANT) {
                        STACK(stackPointer) += expression->arg[i];
                    }
                    else if (op == MULTIPLY_CONSTANT) {
                        STACK(stackPointer) *= expression->arg[i];
                    }
                    else if (op == POWER_CONSTANT) {
                        STACK(stackPointer) = pow(STACK(stackPointer), expression->arg[i]);
                    }
                }
            }
        }
//        switch (expression->op[i])
//        {
//            case CONSTANT:
//                STACK(++stackPointer) = expression->arg[i];
//                break;
//            case VARIABLE0:
//                STACK(++stackPointer) = var0;
//                break;
//            case VARIABLE1:
//                STACK(++stackPointer) = vars1.x;
//                break;
//            case VARIABLE2:
//                STACK(++stackPointer) = vars1.y;
//                break;
//            case VARIABLE3:
//                STACK(++stackPointer) = vars1.z;
//                break;
//            case VARIABLE4:
//                STACK(++stackPointer) = vars1.w;
//                break;
//            case VARIABLE5:
//                STACK(++stackPointer) = vars2.x;
//                break;
//            case VARIABLE6:
//                STACK(++stackPointer) = vars2.y;
//                break;
//            case VARIABLE7:
//                STACK(++stackPointer) = vars2.z;
//                break;
//            case VARIABLE8:
//                STACK(++stackPointer) = vars2.w;
//                break;
//            case GLOBAL:
//                STACK(++stackPointer) = globalParams[(int) expression->arg[i]];
//                break;
//            case ADD:
//            {
//                float temp = STACK(stackPointer);
//                STACK(--stackPointer) += temp;
//                break;
//            }
//            case SUBTRACT:
//            {
//                float temp = STACK(stackPointer);
//                STACK(stackPointer) = temp-STACK(--stackPointer);
//                break;
//            }
//            case MULTIPLY:
//            {
//                float temp = STACK(stackPointer);
//                STACK(--stackPointer) *= temp;
//                break;
//            }
//            case DIVIDE:
//            {
//                float temp = STACK(stackPointer);
//                STACK(stackPointer) = temp/STACK(--stackPointer);
//                break;
//            }
//            case POWER:
//            {
//                float temp = STACK(stackPointer);
//                STACK(stackPointer) = pow(temp, STACK(--stackPointer));
//                break;
//            }
//            case NEGATE:
//                STACK(stackPointer) *= -1.0f;
//                break;
//            case SQRT:
//                STACK(stackPointer) = sqrt(STACK(stackPointer));
//                break;
//            case EXP:
//                STACK(stackPointer) = exp(STACK(stackPointer));
//                break;
//            case LOG:
//                STACK(stackPointer) = log(STACK(stackPointer));
//                break;
//            case SIN:
//                STACK(stackPointer) = sin(STACK(stackPointer));
//                break;
//            case COS:
//                STACK(stackPointer) = cos(STACK(stackPointer));
//                break;
//            case SEC:
//                STACK(stackPointer) = 1.0f/cos(STACK(stackPointer));
//                break;
//            case CSC:
//                STACK(stackPointer) = 1.0f/sin(STACK(stackPointer));
//                break;
//            case TAN:
//                STACK(stackPointer) = tan(STACK(stackPointer));
//                break;
//            case COT:
//                STACK(stackPointer) = 1.0f/tan(STACK(stackPointer));
//                break;
//            case ASIN:
//                STACK(stackPointer) = asin(STACK(stackPointer));
//                break;
//            case ACOS:
//                STACK(stackPointer) = acos(STACK(stackPointer));
//                break;
//            case ATAN:
//                STACK(stackPointer) = atan(STACK(stackPointer));
//                break;
//            case SQUARE:
//            {
//                float temp = STACK(stackPointer);
//                STACK(stackPointer) *= temp;
//                break;
//            }
//            case CUBE:
//            {
//                float temp = STACK(stackPointer);
//                STACK(stackPointer) *= temp*temp;
//                break;
//            }
//            case RECIPROCAL:
//                STACK(stackPointer) = 1.0f/STACK(stackPointer);
//                break;
//            case ADD_CONSTANT:
//                STACK(stackPointer) += expression->arg[i];
//                break;
//            case MULTIPLY_CONSTANT:
//                STACK(stackPointer) *= expression->arg[i];
//                break;
//            case POWER_CONSTANT:
//                STACK(stackPointer) = pow(STACK(stackPointer), expression->arg[i]);
//                break;
//        }
    }
    return STACK(stackPointer);
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateCustomNonbondedForces.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateCustomNonbondedForces.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateCustomNonbondedForces.h"

__global__ void kFindBlockBoundsCutoff_kernel();
__global__ void kFindBlocksWithInteractionsCutoff_kernel();
__global__ void kFindInteractionsWithinBlocksCutoff_kernel(unsigned int* workUnit);
__global__ void kFindBlockBoundsPeriodic_kernel();
__global__ void kFindBlocksWithInteractionsPeriodic_kernel();
__global__ void kFindInteractionsWithinBlocksPeriodic_kernel(unsigned int* workUnit);

void kCalculateCustomNonbondedForces(gpuContext gpu, bool neighborListValid)
{
//    printf("kCalculateCustomNonbondedCutoffForces\n");
    CUDPPResult result;
    int sharedPerThread = sizeof(Atom)+gpu->sim.customExpressionStackSize*sizeof(float);
    if (gpu->sim.customNonbondedMethod != NO_CUTOFF)
        sharedPerThread += sizeof(float3);
    int threads = gpu->sim.nonbond_threads_per_block;
    int maxThreads = 16380/sharedPerThread;
    if (threads > maxThreads)
        threads = (maxThreads/32)*32;
    switch (gpu->sim.customNonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedN2ByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pWorkUnit);
            else
                kCalculateCustomNonbondedN2Forces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedN2Forces");
            kCalculateCustomNonbondedN2Exceptions_kernel<<<gpu->sim.blocks, gpu->sim.custom_exception_threads_per_block,
                    gpu->sim.customExpressionStackSize*sizeof(float)*gpu->sim.custom_exception_threads_per_block>>>();
            LAUNCHERROR("kCalculateCustomNonbondedN2Exceptions");
            break;
        case CUTOFF:
            if (!neighborListValid)
            {
                kFindBlockBoundsCutoff_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
                LAUNCHERROR("kFindBlockBoundsCutoff");
                kFindBlocksWithInteractionsCutoff_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
                LAUNCHERROR("kFindBlocksWithInteractionsCutoff");
                result = cudppCompact(gpu->cudpp, gpu->sim.pInteractingWorkUnit, gpu->sim.pInteractionCount,
                        gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits);
                if (result != CUDPP_SUCCESS)
                {
                    printf("Error in cudppCompact: %d\n", result);
                    exit(-1);
                }
                kFindInteractionsWithinBlocksCutoff_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedCutoffByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCustomNonbondedCutoffForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedCutoffForces");
            kCalculateCustomNonbondedCutoffExceptions_kernel<<<gpu->sim.blocks, gpu->sim.custom_exception_threads_per_block,
                    gpu->sim.customExpressionStackSize*sizeof(float)*gpu->sim.custom_exception_threads_per_block>>>();
            LAUNCHERROR("kCalculateCustomNonbondedCutoffExceptions");
            break;
        case PERIODIC:
            if (!neighborListValid)
            {
                kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
                LAUNCHERROR("kFindBlockBoundsPeriodic");
                kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
                LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
                result = cudppCompact(gpu->cudpp, gpu->sim.pInteractingWorkUnit, gpu->sim.pInteractionCount,
                        gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits);
                if (result != CUDPP_SUCCESS)
                {
                    printf("Error in cudppCompact: %d\n", result);
                    exit(-1);
                }
                kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedPeriodicByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCustomNonbondedPeriodicForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedPeriodicForces");
            kCalculateCustomNonbondedPeriodicExceptions_kernel<<<gpu->sim.blocks, gpu->sim.custom_exception_threads_per_block,
                    gpu->sim.customExpressionStackSize*sizeof(float)*gpu->sim.custom_exception_threads_per_block>>>();
            LAUNCHERROR("kCalculateCustomNonbondedPeriodicExceptions");
            break;
    }
}
