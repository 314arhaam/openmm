#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
using namespace std;

#include "gputypes.h"
#include "cudatypes.h"
#include "cudaKernels.h"

struct Atom {
    float x;
    float y;
    float z;
    float q;
    float sig;
    float eps;
    float br;
    float fx;
    float fy;
    float fz;
    float fb;
};

static __constant__ cudaGmxSimulation cSim;

void SetCalculateCDLJObcGbsaForces1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");

}

void GetCalculateCDLJObcGbsaForces1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}
texture<float, 1, hipReadModeElementType> tabulatedErfcRef;

__device__ float fastErfc(float r)
{
    float normalized = cSim.tabulatedErfcScale*r;
    int index = (int) normalized;
    float fract2 = normalized-index;
    float fract1 = 1.0f-fract2;
    return fract1*tex1Dfetch(tabulatedErfcRef, index) + fract2*tex1Dfetch(tabulatedErfcRef, index+1);
}

// Include versions of the kernel for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateCDLJObcGbsaForces1.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateCDLJObcGbsaForces1.h"

// Include versions of the kernel with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateCDLJObcGbsaForces1.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateCDLJObcGbsaForces1.h"

// Include versions of the kernel with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateCDLJObcGbsaForces1.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateCDLJObcGbsaForces1.h"

// Include versions of the kernels for Ewald

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define USE_EWALD
#define METHOD_NAME(a, b) a##Ewald##b
#include "kCalculateCDLJObcGbsaForces1.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##EwaldByWarp##b
#include "kCalculateCDLJObcGbsaForces1.h"

extern __global__ void kFindBlockBoundsCutoff_kernel();
extern __global__ void kFindBlockBoundsPeriodic_kernel();
extern __global__ void kFindBlocksWithInteractionsCutoff_kernel();
extern __global__ void kFindBlocksWithInteractionsPeriodic_kernel();
extern __global__ void kFindInteractionsWithinBlocksCutoff_kernel(unsigned int*);
extern __global__ void kFindInteractionsWithinBlocksPeriodic_kernel(unsigned int*);
extern __global__ void kCalculateEwaldFastCosSinSums_kernel();
extern __global__ void kCalculateEwaldFastForces_kernel();
extern void kCalculatePME(gpuContext gpu);

void kCalculateCDLJObcGbsaForces1(gpuContext gpu)
{
//    printf("kCalculateCDLJObcGbsaForces1\n");
    switch (gpu->sim.nonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bRecalculateBornRadii)
            {
                if( gpu->bIncludeGBVI ){
                   kCalculateGBVIBornSum(gpu);
                   kReduceGBVIBornSum(gpu);
                } else {
                   kCalculateObcGbsaBornSum(gpu);
                   kReduceObcGbsaBornSum(gpu);
                }

            }
            if (gpu->bOutputBufferPerWarp)
                   kCalculateCDLJObcGbsaN2ByWarpForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                           sizeof(Atom)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pWorkUnit);
               else
                   kCalculateCDLJObcGbsaN2Forces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                           sizeof(Atom)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pWorkUnit);
   
            LAUNCHERROR("kCalculateCDLJObcGbsaN2Forces1");
            break;
        case CUTOFF:
            kFindBlockBoundsCutoff_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsCutoff");
            kFindBlocksWithInteractionsCutoff_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsCutoff");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksCutoff_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            if (gpu->bRecalculateBornRadii)
            {
                kCalculateObcGbsaBornSum(gpu);
                kReduceObcGbsaBornSum(gpu);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJObcGbsaCutoffByWarpForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJObcGbsaCutoffForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJObcGbsaCutoffForces1");
            break;
        case PERIODIC:
            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            if (gpu->bRecalculateBornRadii)
            {
                kCalculateObcGbsaBornSum(gpu);
                kReduceObcGbsaBornSum(gpu);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJObcGbsaPeriodicByWarpForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJObcGbsaPeriodicForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJObcGbsaPeriodicForces1");
            break;
        case EWALD:
        case PARTICLE_MESH_EWALD:
            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kFindInteractionsWithinBlocksPeriodic");
            if (gpu->bRecalculateBornRadii)
            {
                kCalculateObcGbsaBornSum(gpu);
                kReduceObcGbsaBornSum(gpu);
            }
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            hipBindTexture(NULL, &tabulatedErfcRef, gpu->psTabulatedErfc->_pDevData, &channelDesc, gpu->psTabulatedErfc->_length*sizeof(float));
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJObcGbsaEwaldByWarpForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJObcGbsaEwaldForces1_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJObcGbsaEwaldForces");
            if (gpu->sim.nonbondedMethod == EWALD)
            {
                // Ewald summation
                kCalculateEwaldFastCosSinSums_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block>>>();
                LAUNCHERROR("kCalculateEwaldFastCosSinSums");
                kCalculateEwaldFastForces_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
                LAUNCHERROR("kCalculateEwaldFastForces");
            }
            else
                kCalculatePME(gpu);
    }
}
