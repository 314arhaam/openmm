#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Erik Lindahl, Rossen Apostolov, Szilard Pall, Peter Eastman       *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "gputypes.h"
#include <hip/hip_runtime.h>

using namespace std;

static __constant__ cudaGmxSimulation cSim;

void SetCalculatePMESim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculatePMESim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

inline __host__ __device__ int fast_mod(int a, int b)
{
    return (b & (b - 1)) ? a % b : a & (b - 1);
}
inline __host__ __device__ float4 make_float4(float s)
{
    return make_float4(s, s, s, s);
}
inline __host__ __device__ float4 operator-(float4 &a)
{
    return make_float4(-a.x, -a.y, -a.z, -a.w);
}
inline __host__ __device__ float4 operator-(float4 a, float4 b)
{
    return make_float4(a.x - b.x, a.y - b.y, a.z - b.z,  a.w - b.w);
}
inline __host__ __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}
inline __host__ __device__ float4 operator+(float4 a, float b)
{
    return make_float4(a.x + b, a.y + b, a.z + b, a.w + b);
}
inline __host__ __device__ float4 operator+(float a, float4 b)
{
    return make_float4(a + b.x, a + b.y, a + b.z,  a + b.w);
}
inline __host__ __device__ float4 operator*(float s, float4 a)
{
    return make_float4(a.x * s, a.y * s, a.z * s, a.w * s);
}
inline __host__ __device__ float4 operator*(float4 a, float4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w + b.w);
}
inline __host__ __device__ float4 make_float4(int3 a)
{
    return make_float4(a.x, a.y, a.z, 0);
}

__global__ void kUpdateGridIndexAndFraction_kernel()
{
    unsigned int tnb = blockDim.x * gridDim.x;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = tid; i < cSim.atoms; i += tnb)
    {
        float4 ftmp = cSim.pPosq[i];
        float3 t = make_float3((ftmp.x/cSim.periodicBoxSizeX+1.0f)*cSim.pmeGridSize.x,
                               (ftmp.y/cSim.periodicBoxSizeY+1.0f)*cSim.pmeGridSize.y,
                               (ftmp.z/cSim.periodicBoxSizeZ+1.0f)*cSim.pmeGridSize.z);
        float3 tix;
        ftmp.x = modff(t.x, &tix.x);
        ftmp.y = modff(t.y, &tix.y);
        ftmp.z = modff(t.z, &tix.z);
        cSim.pPmeParticleFraction[i] = ftmp;
        int4 itmp = make_int4(fast_mod(__float2int_rd(tix.x), cSim.pmeGridSize.x),
                              fast_mod(__float2int_rd(tix.y), cSim.pmeGridSize.y),
                              fast_mod(__float2int_rd(tix.z), cSim.pmeGridSize.z), 0);
        cSim.pPmeParticleIndex[i] = itmp;
    }

    // Compute flags for which atoms affect which groups of grid points.

    const int3 numGroups = make_int3((cSim.pmeGridSize.x+cSim.pmeGroupSize.x-1)/cSim.pmeGroupSize.x, (cSim.pmeGridSize.y+cSim.pmeGroupSize.y-1)/cSim.pmeGroupSize.y, (cSim.pmeGridSize.z+cSim.pmeGroupSize.z-1)/cSim.pmeGroupSize.z);
    const unsigned int totalGroups = numGroups.x*numGroups.y*numGroups.z;
    const float3 gridScale = make_float3(cSim.pmeGridSize.x/cSim.periodicBoxSizeX, cSim.pmeGridSize.y/cSim.periodicBoxSizeY, cSim.pmeGridSize.z/cSim.periodicBoxSizeZ);
    for (int group = tid; group < totalGroups; group += tnb)
    {
        int3 gridBase;
        gridBase.x = group/(numGroups.y*numGroups.z);
        int remainder = group-gridBase.x*numGroups.y*numGroups.z;
        gridBase.y = remainder/numGroups.z;
        gridBase.z = remainder-gridBase.y*numGroups.z;
        gridBase.x *= cSim.pmeGroupSize.x;
        gridBase.y *= cSim.pmeGroupSize.y;
        gridBase.z *= cSim.pmeGroupSize.z;
        unsigned int flags = 0;
        unsigned int baseIndex = group*(cSim.paddedNumberOfAtoms/32);
        for (int atomBlock = 0; atomBlock < cSim.paddedNumberOfAtoms>>GRIDBITS; atomBlock++)
        {
            // Decide if this block actually needs to be processed.

            int flagIndex = atomBlock%32;
            if (flagIndex == 0)
                flags = 0;
            float4 boxSize = cSim.pGridBoundingBox[atomBlock];
            float4 center = cSim.pGridCenter[atomBlock];
            int maxx = (int) ceil((center.x+boxSize.x)*gridScale.x)+cSim.pmeGroupSize.x+PME_ORDER;
            int maxy = (int) ceil((center.y+boxSize.y)*gridScale.y)+cSim.pmeGroupSize.y+PME_ORDER;
            int maxz = (int) ceil((center.z+boxSize.z)*gridScale.z)+cSim.pmeGroupSize.z+PME_ORDER;
            int minx = (int) floor((center.x-boxSize.x)*gridScale.x);
            int miny = (int) floor((center.y-boxSize.y)*gridScale.y);
            int minz = (int) floor((center.z-boxSize.z)*gridScale.z);
            int x = minx+(gridBase.x-minx)%cSim.pmeGridSize.x;
            int y = miny+(gridBase.y-miny)%cSim.pmeGridSize.y;
            int z = minz+(gridBase.z-minz)%cSim.pmeGridSize.z;
            if (maxx < x || maxy < y || maxz < z)
                flags += 1<<flagIndex;
            if (flagIndex == 31 || atomBlock == cSim.paddedNumberOfAtoms>>GRIDBITS)
                cSim.pPmeInteractionFlags[baseIndex+atomBlock/32] = flags;
        }
    }
}

__global__ void kUpdateBsplines_kernel()
{
    unsigned int    tnb = blockDim.x * gridDim.x;
    unsigned int    tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float4 bsplines_cache[]; // size = 2 * block_size * pme_order

    const float4 div_o   = make_float4(1.0f/(PME_ORDER - 1));

    for (int i = tid; i < cSim.atoms; i += tnb)
    {

        float4* data    = &bsplines_cache[threadIdx.x*PME_ORDER];
        float4* ddata   = &bsplines_cache[threadIdx.x*PME_ORDER + blockDim.x*PME_ORDER];

        for (int j = 0; j < PME_ORDER; j++)
        {
	    data[j] = make_float4(0.0f);
            ddata[j] = make_float4(0.0f);
        }

        float4 dr = cSim.pPmeParticleFraction[i];

        data[PME_ORDER - 1] = make_float4(0.0f);
        data[1]            = dr;
        data[0]            = make_float4(1.0f) - dr;

        for (int j = 3; j < PME_ORDER; j++)
        {
            float div = 1.0f / ((float)j - 1.0f);
            data[j - 1] = div * dr * data[j - 2];

            for (int k = 1; k < (j - 1); k++)
            {
                data[j - k - 1] =
                   div * (
                           (dr + float(k))          * data[j - k - 2] +
                           (-dr + ((float)(j - k))) * data[j - k - 1]);
            }
            data[0] = div * (- dr + 1) * data[0];
        }

        ddata[0] = -data[0];

        for (int j = 1; j < PME_ORDER; j++)
            ddata[j] = data[j - 1] - data[j];

        data[PME_ORDER - 1] = div_o * dr * data[PME_ORDER - 2];

        for (int j = 1; j < (PME_ORDER - 1); j++)
        {
            data[PME_ORDER - j - 1] =
                div_o * (
                    (dr + (float)j)                 * data[PME_ORDER - j - 2] +
                    (-dr + ((float)(PME_ORDER - j))) * data[PME_ORDER - j - 1]
                );
        }
        data[0] = div_o * (-dr + 1.0f) * data[0];

        for (int j = 0; j < PME_ORDER; j++)
        {
            cSim.pPmeBsplineTheta[i + j*cSim.atoms] =  data[j];
            cSim.pPmeBsplineDtheta[i + j*cSim.atoms] = ddata[j];
        }
    }
}

__global__ void kGridSpreadCharge_kernel()
{
    extern __shared__ float atomCharge[];
    int4* atomGridIndex = (int4*) &atomCharge[blockDim.x];
    const unsigned int totalWarps = gridDim.x*blockDim.x/GRID;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/GRID;
    const int3 numGroups = make_int3((cSim.pmeGridSize.x+cSim.pmeGroupSize.x-1)/cSim.pmeGroupSize.x, (cSim.pmeGridSize.y+cSim.pmeGroupSize.y-1)/cSim.pmeGroupSize.y, (cSim.pmeGridSize.z+cSim.pmeGroupSize.z-1)/cSim.pmeGroupSize.z);
    const unsigned int totalGroups = numGroups.x*numGroups.y*numGroups.z;
    unsigned int group = warp*totalGroups/totalWarps;
    const unsigned int end = (warp+1)*totalGroups/totalWarps;
    const unsigned int index = threadIdx.x & (GRID - 1);

    while (group < end)
    {
        // Process a group of grid points of size cSim.pmeGroupSize.  First figure out the base index for the group,
        // and the index of the specific point this thread will handle.

        int3 gridBase;
        gridBase.x = group/(numGroups.y*numGroups.z);
        int remainder = group-gridBase.x*numGroups.y*numGroups.z;
        gridBase.y = remainder/numGroups.z;
        gridBase.z = remainder-gridBase.y*numGroups.z;
        gridBase.x *= cSim.pmeGroupSize.x;
        gridBase.y *= cSim.pmeGroupSize.y;
        gridBase.z *= cSim.pmeGroupSize.z;
        int3 gridPoint;
        gridPoint.x = index/(cSim.pmeGroupSize.y*cSim.pmeGroupSize.z);
        remainder = index-gridPoint.x*cSim.pmeGroupSize.y*cSim.pmeGroupSize.z;
        gridPoint.y = remainder/cSim.pmeGroupSize.z;
        gridPoint.z = remainder-gridPoint.y*cSim.pmeGroupSize.z;
        gridPoint.x += gridBase.x;
        gridPoint.y += gridBase.y;
        gridPoint.z += gridBase.z;

        // Loop over blocks of atoms.

        float result = 0.0f;
        int flags = 0;
        unsigned int baseIndex = group*(cSim.paddedNumberOfAtoms/32);
        for (int atomBlock = 0; atomBlock < cSim.paddedNumberOfAtoms>>GRIDBITS; atomBlock++)
        {
            // Decide if this block actually needs to be processed.

            int flagIndex = atomBlock%32;
            if (flagIndex == 0)
                flags = cSim.pPmeInteractionFlags[baseIndex+atomBlock/32];
            if ((flags & (1<<flagIndex)) != 0)
                continue;
            int atomIndex = (atomBlock<<GRIDBITS)+index;
            if (atomIndex < cSim.atoms)
            {
                atomCharge[threadIdx.x] = cSim.pPosq[atomIndex].w;
                atomGridIndex[threadIdx.x] = cSim.pPmeParticleIndex[atomIndex];
            }
            int maxAtoms = min(GRID, cSim.atoms-(atomBlock<<GRIDBITS));
            for (int i = 0; i < maxAtoms; i++)
            {
                int localIndex = threadIdx.x-index+i;
                int atomIndex = (atomBlock<<GRIDBITS)+i;
                int ix = gridPoint.x-atomGridIndex[localIndex].x;
                int iy = gridPoint.y-atomGridIndex[localIndex].y;
                int iz = gridPoint.z-atomGridIndex[localIndex].z;
                ix += (ix < 0 ? cSim.pmeGridSize.x : 0);
                iy += (iy < 0 ? cSim.pmeGridSize.y : 0);
                iz += (iz < 0 ? cSim.pmeGridSize.z : 0);
                if (ix < PME_ORDER && iy < PME_ORDER && iz < PME_ORDER)
                    result += atomCharge[threadIdx.x-index+i]*cSim.pPmeBsplineTheta[atomIndex+ix*cSim.atoms].x*cSim.pPmeBsplineTheta[atomIndex+iy*cSim.atoms].y*cSim.pPmeBsplineTheta[atomIndex+iz*cSim.atoms].z;
            }
        }
        unsigned int gridIndex = gridPoint.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+gridPoint.y*cSim.pmeGridSize.z+gridPoint.z;
        if (gridIndex < cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z)
            cSim.pPmeGrid[gridIndex] = make_hipComplex(result*sqrt(cSim.epsfac), 0.0f);
        group++;
    }
}

__global__ void kReciprocalConvolution_kernel()
{
    const unsigned int gridSize = cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
    float expFactor = PI*PI/(cSim.alphaEwald*cSim.alphaEwald);
    float scaleFactor = 1.0/(PI*cSim.periodicBoxSizeX*cSim.periodicBoxSizeY*cSim.periodicBoxSizeZ);
    float energy = 0.0f;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x)
    {
        int kx = index/(cSim.pmeGridSize.y*cSim.pmeGridSize.z);
        int remainder = index-kx*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
        int ky = remainder/cSim.pmeGridSize.z;
        int kz = remainder-ky*cSim.pmeGridSize.z;
        if (kx == 0 && ky == 0 && kz == 0)
            continue;
        int mx = (kx < (cSim.pmeGridSize.x+1)/2) ? kx : (kx-cSim.pmeGridSize.x);
        int my = (ky < (cSim.pmeGridSize.y+1)/2) ? ky : (ky-cSim.pmeGridSize.y);
        int mz = (kz < (cSim.pmeGridSize.z+1)/2) ? kz : (kz-cSim.pmeGridSize.z);
        float mhx = mx/cSim.periodicBoxSizeX;
        float mhy = my/cSim.periodicBoxSizeY;
        float mhz = mz/cSim.periodicBoxSizeZ;
        float bx = cSim.pPmeBsplineModuli[0][kx];
        float by = cSim.pPmeBsplineModuli[1][ky];
        float bz = cSim.pPmeBsplineModuli[2][kz];
        hipComplex grid = cSim.pPmeGrid[index];
        float m2 = mhx*mhx+mhy*mhy+mhz*mhz;
        float denom = m2*bx*by*bz;
        float eterm = scaleFactor*exp(-expFactor*m2)/denom;
        cSim.pPmeGrid[index] = make_hipComplex(grid.x*eterm, grid.y*eterm);
        energy += eterm*(grid.x*grid.x + grid.y*grid.y);
    }
    cSim.pEnergy[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*energy;
}

__global__ void kGridInterpolateForce_kernel()
{
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < cSim.atoms; atom += blockDim.x*gridDim.x)
    {
        float3 force = make_float3(0.0f, 0.0f, 0.0f);
        float4 posq = cSim.pPosq[atom];
        int4 gridIndex = cSim.pPmeParticleIndex[atom];
        for (int ix = 0; ix < PME_ORDER; ix++)
        {
            int xindex = (gridIndex.x + ix) % cSim.pmeGridSize.x;
            float tx = cSim.pPmeBsplineTheta[atom+ix*cSim.atoms].x;
            float dtx = cSim.pPmeBsplineDtheta[atom+ix*cSim.atoms].x;
            for (int iy = 0; iy < PME_ORDER; iy++)
            {
                int yindex = (gridIndex.y + iy) % cSim.pmeGridSize.y;
                float ty = cSim.pPmeBsplineTheta[atom+iy*cSim.atoms].y;
                float dty = cSim.pPmeBsplineDtheta[atom+iy*cSim.atoms].y;
                for (int iz = 0; iz < PME_ORDER; iz++)
                {
                    int zindex               = (gridIndex.z + iz) % cSim.pmeGridSize.z;
                    float tz = cSim.pPmeBsplineTheta[atom+iz*cSim.atoms].z;
                    float dtz = cSim.pPmeBsplineDtheta[atom+iz*cSim.atoms].z;
                    int index                = xindex*cSim.pmeGridSize.y*cSim.pmeGridSize.z + yindex*cSim.pmeGridSize.z + zindex;
                    float gridvalue            = cSim.pPmeGrid[index].x;
                    force.x                  += dtx*ty*tz*gridvalue;
                    force.y                  += tx*dty*tz*gridvalue;
                    force.z                  += tx*ty*dtz*gridvalue;
                }
            }
        }
        float4 totalForce = cSim.pForce4[atom];
        float q = posq.w*sqrt(cSim.epsfac);
        totalForce.x -= q*force.x*cSim.pmeGridSize.x/cSim.periodicBoxSizeX;
        totalForce.y -= q*force.y*cSim.pmeGridSize.y/cSim.periodicBoxSizeY;
        totalForce.z -= q*force.z*cSim.pmeGridSize.z/cSim.periodicBoxSizeZ;
        cSim.pForce4[atom] = totalForce;
    }
}

void kCalculatePME(gpuContext gpu)
{
//    printf("kCalculatePME\n");
    kUpdateGridIndexAndFraction_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kUpdateGridIndexAndFraction");
    unsigned int threads = 16380/(2*PME_ORDER*sizeof(float4));
    kUpdateBsplines_kernel<<<gpu->sim.blocks, threads, 2*threads*PME_ORDER*sizeof(float4)>>>();
    LAUNCHERROR("kUpdateBsplines");
    kGridSpreadCharge_kernel<<<gpu->sim.blocks, 64, 64*(sizeof(float)+sizeof(int4))>>>();
    LAUNCHERROR("kGridSpreadCharge");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_FORWARD);
    kReciprocalConvolution_kernel<<<gpu->sim.blocks, gpu->sim.nonbond_threads_per_block>>>();
    LAUNCHERROR("kReciprocalConvolution");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_BACKWARD);
    kGridInterpolateForce_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kGridInterpolateForce");
}
